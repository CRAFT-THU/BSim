/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */

#include "utils/cuda/hip/hip_runtime_api.h"
#include "GExpSynapses.h"

int cudaAllocExpSynapses(void *pCpu, void *pGpu)
{
	GExpSynapses *pGpuSynapses = (GExpSynapses*)pGpu;
	GExpSynapses *p = (GExpSynapses*)pCpu;

	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pID), sizeof(ID)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->pID, p->pID, sizeof(ID)*p->num, hipMemcpyHostToDevice));

	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pType), sizeof(Type)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->pType, p->pType, sizeof(Type)*p->num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_weight), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_weight, p->p_weight, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay_steps), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay_steps, p->p_delay_steps, sizeof(int)*p->num, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay), sizeof(real)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay, p->p_delay, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C1), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C1, p->p_C1, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C1), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C1, p->p__C1, sizeof(real)*p->num, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_tau_syn), sizeof(real)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->p_tau_syn, p->p_tau_syn, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_syn), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_syn, p->p_I_syn, sizeof(real)*p->num, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__dt), sizeof(real)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->p__dt, p->p__dt, sizeof(real)*p->num, hipMemcpyHostToDevice));

	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pSrc), sizeof(int)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->pSrc, p->pSrc, sizeof(int)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pDst), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pDst, p->pDst, sizeof(int)*p->num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeExpSynapses(void *pGpu)
{
	GExpSynapses *pGpuSynapses = (GExpSynapses*)pGpu;

	//checkCudaErrors(hipFree(pGpuSynapses->pID));

	//checkCudaErrors(hipFree(pGpuSynapses->pType));

	checkCudaErrors(hipFree(pGpuSynapses->p_weight));
	checkCudaErrors(hipFree(pGpuSynapses->p_delay_steps));
	//checkCudaErrors(hipFree(pGpuSynapses->p_delay));
	checkCudaErrors(hipFree(pGpuSynapses->p_C1));
	checkCudaErrors(hipFree(pGpuSynapses->p__C1));
	//checkCudaErrors(hipFree(pGpuSynapses->p_tau_syn));
	checkCudaErrors(hipFree(pGpuSynapses->p_I_syn));
	//checkCudaErrors(hipFree(pGpuSynapses->p__dt));

	//checkCudaErrors(hipFree(pGpuSynapses->pSrc));
	checkCudaErrors(hipFree(pGpuSynapses->pDst));

	return 0;
}
