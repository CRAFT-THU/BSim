/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */

#include "utils/cuda/hip/hip_runtime_api.h"
#include "GExpSynapses.h"

int GExpSynapses::allocGSynapses(GExpSynapses *pGpuSynapses)
{
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pID), sizeof(ID)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pID, pID, sizeof(ID)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pType), sizeof(NeuronType)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pType, pType, sizeof(NeuronType)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_weight), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_weight, p_weight, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay, p_delay, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C1, p_C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C1, p__C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_tau_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_tau_syn, p_tau_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_syn, p_I_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__dt), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__dt, p__dt, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pSrc), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pSrc, pSrc, sizeof(unsigned int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pDst), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pDst, pDst, sizeof(unsigned int)*num, hipMemcpyHostToDevice));

	return 0;
}
