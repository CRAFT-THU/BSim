/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */

#include "stdlib.h"

#include "utils/cuda/hip/hip_runtime_api.h"
#include "GExpSynapses.h"

int GExpSynapses::allocSynapses(unsigned int S)
{
	num = S;
	pID = (ID*)malloc(S*sizeof(ID));
	pType = (SpikeType*)malloc(S*sizeof(SpikeType));
	p_weight = (real*)malloc(S*sizeof(real));
	p_delay = (real*)malloc(S*sizeof(real));
	p_C1 = (real*)malloc(S*sizeof(real));
	p__C1 = (real*)malloc(S*sizeof(real));
	p_tau_syn = (real*)malloc(S*sizeof(real));
	p_I_syn = (real*)malloc(S*sizeof(real));
	p__dt = (real*)malloc(S*sizeof(real));
	pSrc = (unsigned int *)malloc(S*sizeof(unsigned int));
	pDst = (unsigned int *)malloc(S*sizeof(unsigned int));

	return 0;
}

int GExpSynapses::allocGSynapses(GExpSynapses *pGpuSynapses)
{
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pID), sizeof(ID)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pID, pID, sizeof(ID)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pType), sizeof(NeuronType)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pType, pType, sizeof(NeuronType)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_weight), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_weight, p_weight, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay, p_delay, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C1, p_C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C1, p__C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_tau_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_tau_syn, p_tau_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_syn, p_I_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__dt), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__dt, p__dt, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pSrc), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pSrc, pSrc, sizeof(unsigned int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pDst), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pDst, pDst, sizeof(unsigned int)*num, hipMemcpyHostToDevice));

	return 0;
}
