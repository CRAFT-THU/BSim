#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/runtime.h"
#include "../gpu_utils/GBuffers.h"
#include "../net/MultiNetwork.h"
#include "MultiNodeSimulator.h"

using std::cout;
using std::endl;

pthread_barrier_t nodeCycleBarrier;

CrossNodeDataGPU * gCrossDataNode;

MultiNodeSimulator::MultiNodeSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

MultiNodeSimulator::~MultiNodeSimulator()
{
}

void *run_thread_node(void *para);

int MultiNodeSimulator::run(real time, FireInfo &log)
{
	return 0;
}

