#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/FileOp.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
// #include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/GBuffers.h"
#include "../gpu_utils/runtime.h"
#include "../net/MultiNetwork.h"
// #include "../gpu_utils/gpu_func.h"

#include "SingleGPUSimulator.h"

using std::cout;
using std::endl;

CrossNodeDataGPU * gCrossDataNet;

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

int SingleGPUSimulator::run(real time, FireInfo &log)
{

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pNetCPU = network->buildNetwork();

	FILE *v_file = openFile("v.data", "w+");

	FILE *input_e_file = openFile("input_e.data", "w+");

	FILE *input_i_file = openFile("input_i.data", "w+");

	FILE *ie_file = openFile("ie.data", "w+");

	FILE *ii_file = openFile("ii.data", "w+");

	FILE *log_file = openFile("sim.log", "w+");

	//findCudaDevice(0, NULL);
	checkCudaErrors(hipSetDevice(0));
	GNetwork *c_pNetGPU = copyNetworkToGPU(pNetCPU);

	int nTypeNum = c_pNetGPU->nTypeNum;
	int sTypeNum = c_pNetGPU->sTypeNum;
	int totalNeuronNum = c_pNetGPU->pNeuronNums[nTypeNum];
	int totalSynapseNum = c_pNetGPU->pSynapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int deltaDelay = pNetCPU->pConnection->maxDelay - pNetCPU->pConnection->minDelay + 1;
	printf("maxDelay: %d minDelay: %d\n", pNetCPU->pConnection->maxDelay, pNetCPU->pConnection->minDelay);


	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, deltaDelay, dt);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(totalNeuronNum);

	int copy_idx = getIndex(c_pNetGPU->pNTypes, nTypeNum, LIF);

	GLIFNeurons *c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pNetGPU->ppNeurons[copy_idx]), 1);

	real *c_g_vm = c_g_lif->pV_m;
#ifdef DEBUG 
	real *c_g_ie = c_g_lif->pI_e;
	real *c_g_ii = c_g_lif->pI_i;
#endif
#endif

	//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
	//int exp_idx = getIndex(c_pNetGPU->pSTypes, sTypeNum, Exp);
	//GExppSynapses *c_g_exp = copyFromGPU<GExppSynapses>(static_cast<GExppSynapses*>(c_pNetGPU->ppSynapses[exp_idx]), 1);
	//real *c_g_I_syn = c_g_exp->p_I_syn;

	//for (int i=0; i<nTypeNum; i++) {
	//	cout << c_pNetGPU->pNTypes[i] << ": <<<" << updateSize[c_pNetGPU->pNTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pNTypes[i]].blockSize << ">>>" << endl;
	//}
	//for (int i=0; i<sTypeNum; i++) {
	//	cout << c_pNetGPU->pSTypes[i] << ": <<<" << updateSize[c_pNetGPU->pSTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pSTypes[i]].blockSize << ">>>" << endl;
	//}

	vector<int> firedInfo;
	printf("Start runing for %d cycles\n", sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		//printf("Cycle: %d ", time);
		//fflush(stdout);

#ifdef DEBUG
		copyFromGPU<real>(c_vm, buffers->c_gNeuronInput + c_pNetGPU->pNeuronNums[copy_idx], c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(input_e_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(input_e_file, "\n");
		copyFromGPU<real>(c_vm, buffers->c_gNeuronInput_I + c_pNetGPU->pNeuronNums[copy_idx], c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(input_i_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(input_i_file, "\n");
#endif

		update_time<<<1, 1>>>(c_pNetGPU->pConnection, time, buffers->c_gFiredTableSizes);

		for (int i=0; i<nTypeNum; i++) {
			cudaUpdateType[c_pNetGPU->pNTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppNeurons[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i], c_pNetGPU->pNeuronNums[i],time, &updateSize[c_pNetGPU->pNTypes[i]]);
		}

		for (int i=0; i<sTypeNum; i++) {
			cudaUpdateType[c_pNetGPU->pSTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppSynapses[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i], c_pNetGPU->pSynapseNums[i], time, &updateSize[c_pNetGPU->pSTypes[i]]);
		}


#ifdef LOG_DATA
		//LOG DATA
		int currentIdx = time%(deltaDelay+1);

		//printf("HERE %p %d ", buffers->c_gFiredTableSizes, currentIdx);
		//fflush(stdout);
		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		//printf("HERE1\n");
		//fflush(stdout);
		copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);
		//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pNetGPU->pSynapseNums[exp_idx+1]-c_pNetGPU->pSynapseNums[exp_idx]);

		//fprintf(dataFile, "Cycle %d: ", time);
		copyFromGPU<real>(c_vm, c_g_vm, c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");
#ifdef DEBUG
		copyFromGPU<real>(c_vm, c_g_ie, c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(ie_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(ie_file, "\n");
		copyFromGPU<real>(c_vm, c_g_ii, c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(ii_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(ii_file, "\n");
		//for (int i=0; i<c_pNetGPU->pSynapseNums[1] - c_pNetGPU->pSynapseNums[0]; i++) {
		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
		//}
#endif

		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		//LOG SYNAPSE
		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);
		//int synapseCount = 0;
		//if (time > 0) {
		//	for (int i=0; i<totalSynapseNum; i++) {
		//		if (buffers->c_synapsesFired[i] == time) {
		//			if (synapseCount ==  0) {
		//				if (copySize > 0) {
		//					fprintf(logFile, ", ");
		//				}
		//				fprintf(logFile, "%s", network->idx2sid[i].getInfo().c_str());
		//				synapseCount++;
		//			} else {
		//				fprintf(logFile, ", %s", network->idx2sid[i].getInfo().c_str());
		//			}
		//		}
		//	}
		//	fprintf(logFile, "\n");
		//}
#endif
	}
	hipDeviceSynchronize();

	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Simulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	//CALC Firing Rate
	if (log.find("count") != log.end()) {
		int *rate = (int*)malloc(sizeof(int)*totalNeuronNum);
		copyFromGPU<int>(rate, buffers->c_gFireCount, totalNeuronNum);

		log["count"].size = totalNeuronNum;
		log["count"].data = rate;

		//FILE *rateFile = fopen("GFire.log", "w+");
		//if (rateFile == NULL) {
		//	printf("ERROR: Open file Sim.log failed\n");
		//	return -1;
		//}

		//for (int i=0; i<totalNeuronNum; i++) {
		//	fprintf(rateFile, "%d \t", rate[i]);
		//}

		//fflush(rateFile);
		//fclose(rateFile);
	}

	if (log.find("Y") != log.end()) {
		real *Y = (real*)malloc(sizeof(real)*totalNeuronNum);
		copyFromGPU<real>(Y, buffers->c_gXInput, totalNeuronNum);
		log["Y"].size = totalNeuronNum;
		log["Y"].data = Y;
	}

	if (log.find("X") != log.end()) {
		int *X = (int*)malloc(sizeof(int)*totalNeuronNum);
		copyFromGPU<int>(X, buffers->c_gLayerInput, totalNeuronNum);
		log["X"].size = totalNeuronNum;
		log["X"].data = X;
	}



	closeFile(v_file);
	closeFile(input_e_file);
	closeFile(input_i_file);
	closeFile(ie_file);
	closeFile(ii_file);
	closeFile(log_file);

	free_buffers(buffers);
	freeNetworkGPU(c_pNetGPU);
	freeGNetwork(pNetCPU);

	return 0;
}


int SingleGPUSimulator::runMultiNets(real time, int parts, FireInfo &log) {
	int sim_cycle = round(time/dt);
	reset();

	checkCudaErrors(hipSetDevice(0));

	MultiNetwork multiNet(network, parts);
	DistriNetwork *subnets = multiNet.buildNetworks();
	assert(subnets != NULL);
	CrossNodeDataGPU *crossData = multiNet.arrangeCrossNodeDataGPU(parts);
	assert(crossData != NULL);

	GNetwork ** networks = (GNetwork **)malloc(sizeof(GNetwork *) * parts);
	GBuffers **buffers = (GBuffers **)malloc(sizeof(GBuffers *) * parts);
	BlockSize **updateSizes = (BlockSize **)malloc(sizeof(GBuffers *) * parts);

	for (int i=0; i<parts; i++) {
		subnets[i]._sim_cycle = sim_cycle;
		subnets[i]._node_idx = i;
		subnets[i]._node_num = parts;
		subnets[i]._dt = dt;

		GNetwork *pNetCPU = subnets[i]._network;
		networks[i] = copyNetworkToGPU(pNetCPU);
		GNetwork *c_pNetGPU = networks[i];

		int nTypeNum = c_pNetGPU->nTypeNum;
		int sTypeNum = c_pNetGPU->sTypeNum;
		int nodeNeuronNum = c_pNetGPU->pNeuronNums[nTypeNum];
		int allNeuronNum = pNetCPU->pConnection->nNum;
		int nodeSynapseNum = c_pNetGPU->pSynapseNums[sTypeNum];

		int deltaDelay = c_pNetGPU->pConnection->maxDelay - c_pNetGPU->pConnection->minDelay + 1;

		buffers[i] = alloc_buffers(allNeuronNum, nodeSynapseNum, deltaDelay, dt);
		updateSizes[i] = getBlockSize(allNeuronNum, nodeSynapseNum);

		printf("Subnet %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", subnets[i]._node_idx, nTypeNum, sTypeNum);
		printf("Subnet %d NeuronNum: %d, SynapseNum: %d\n", subnets[i]._node_idx, nodeNeuronNum, nodeSynapseNum);
	}

	for (int time=0; time<sim_cycle; time++) {

		for (int p=0; p<parts; p++) {
			GNetwork *c_pNetGPU = networks[p];
			update_time<<<1, 1>>>(c_pNetGPU->pConnection, time, buffers[p]->c_gFiredTableSizes);

			for (int i=0; i<c_pNetGPU->nTypeNum; i++) {
				assert(c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i] > 0);
				cudaUpdateType[c_pNetGPU->pNTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppNeurons[i], buffers[p]->c_gNeuronInput, buffers[p]->c_gNeuronInput_I, buffers[p]->c_gFiredTable, buffers[p]->c_gFiredTableSizes, c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i], c_pNetGPU->pNeuronNums[i], time, &updateSizes[p][c_pNetGPU->pNTypes[i]]);
			}
		}

		for (int p=0; p<parts; p++) {
			GNetwork *c_pNetGPU = networks[p];
			for (int i=0; i<c_pNetGPU->sTypeNum; i++) {
				assert(c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i] > 0);
				cudaUpdateType[c_pNetGPU->pSTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppSynapses[i], buffers[p]->c_gNeuronInput, buffers[p]->c_gNeuronInput_I, buffers[p]->c_gFiredTable, buffers[p]->c_gFiredTableSizes, c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i], c_pNetGPU->pSynapseNums[i], time, &updateSizes[p][c_pNetGPU->pSTypes[i]]);
			}
		}
	}

	for (int i=0; i<parts; i++) {
		freeNetworkGPU(networks[i]);
		free_buffers(buffers[i]);
	}

	free(networks);
	free(buffers);
	free(updateSizes);

	return 0;
}

