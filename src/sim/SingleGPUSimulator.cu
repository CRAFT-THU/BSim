#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/gpu_kernel.h"
#include "SingleGPUSimulator.h"

using std::cout;
using std::endl;

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

int SingleGPUSimulator::run(real time)
{

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pCpuNet = network->buildNetwork();

	FILE *v_file = fopen("g_v.data", "w+");
	if (v_file == NULL) {
		printf("ERROR: Open file GSim.data failed\n");
		return -1;
	}

	FILE *input_e_file = fopen("g_input_e.data", "w+");
	if (input_e_file == NULL) {
		printf("Open file input_e.data failed\n");
		return -1;
	}

	FILE *input_i_file = fopen("g_input_i.data", "w+");
	if (input_i_file == NULL) {
		printf("Open file input_i.data failed\n");
		return -1;
	}

	FILE *ie_file = fopen("g_ie.data", "w+");
	if (ie_file == NULL) {
		printf("ERROR: Open file GSim.data failed\n");
		return -1;
	}

	FILE *ii_file = fopen("g_ii.data", "w+");
	if (ii_file == NULL) {
		printf("ERROR: Open file GSim.data failed\n");
		return -1;
	}

	FILE *log_file = fopen("GSim.log", "w+");
	if (log_file == NULL) {
		printf("ERROR: Open file GSim.log failed\n");
		return -1;
	}

	findCudaDevice(0, NULL);
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	printf("MAX_DELAY: %d\n", pCpuNet->MAX_DELAY);

	init_connection<<<1, 1>>>(c_pGpuNet->pN2SConnection);

	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, MAX_DELAY);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	//BlockSize preSize = { 0, 0, 0};
	//hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_pre_synapse, 0, totalNeuronNum); 
	//preSize.gridSize = (totalSynapseNum + (preSize.blockSize) - 1) / (preSize.blockSize);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(totalNeuronNum);

	int tj_idx = getIndex(pCpuNet->nTypes, nTypeNum, TJ);
	int life_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIFE);
	int lif_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;
	real *c_g_ie = NULL;
	real *c_g_ii = NULL;

	if (life_idx >= 0) {
		GLIFENeurons *c_g_lif = copyFromGPU<GLIFENeurons>(static_cast<GLIFENeurons*>(c_pGpuNet->pNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		c_g_ie = c_g_lif->p_i_E;
		c_g_ii = c_g_lif->p_i_I;
		copy_idx = life_idx;
	} else if (lif_idx >= 0) {
		GLIFNeurons *c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGpuNet->pNeurons[lif_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		copy_idx = lif_idx;
	} else if (tj_idx >= 0) {
		GTJNeurons *c_g_tj = copyFromGPU<GTJNeurons>(static_cast<GTJNeurons*>(c_pGpuNet->pNeurons[tj_idx]), 1);
		c_g_vm = c_g_tj->p_vm;
		copy_idx = tj_idx;

	} else {
	}
#endif

	//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
	//int exp_idx = getIndex(pCpuNet->sTypes, sTypeNum, Exp);
	//GExpSynapses *c_g_exp = copyFromGPU<GExpSynapses>(static_cast<GExpSynapses*>(c_pGpuNet->pSynapses[exp_idx]), 1);
	//real *c_g_I_syn = c_g_exp->p_I_syn;

	for (int i=0; i<nTypeNum; i++) {
		cout << pCpuNet->nTypes[i] << ": <<<" << updateSize[c_pGpuNet->nTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->nTypes[i]].blockSize << ">>>" << endl;
	}
	for (int i=0; i<sTypeNum; i++) {
		cout << pCpuNet->sTypes[i] << ": <<<" << updateSize[c_pGpuNet->sTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->sTypes[i]].blockSize << ">>>" << endl;
	}

	vector<int> firedInfo;
	printf("Start runing for %d cycles\n", sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		//printf("Cycle: %d ", time);
		//fflush(stdout);

#ifdef LOG_DATA
		if (copy_idx >= 0) {
			//copyFromGPU<real>(c_vm, buffers->c_gNeuronInput + c_pGpuNet->neuronNums[copy_idx], c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			copyFromGPU<real>(c_vm, buffers->c_gNeuronInput + c_pGpuNet->neuronNums[copy_idx], c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
				fprintf(input_e_file, "%.10lf \t", c_vm[i]);
			}
			//copyFromGPU<real>(c_vm, buffers->c_gNeuronInput_I + c_pGpuNet->neuronNums[copy_idx], c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			copyFromGPU<real>(c_vm, buffers->c_gNeuronInput_I + c_pGpuNet->neuronNums[copy_idx], c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
				fprintf(input_i_file, "%.10lf \t", c_vm[i]);
			}
		}
#endif

		for (int i=0; i<nTypeNum; i++) {
			cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
		}

		//update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(c_pGpuNet->pN2SConnection);

		for (int i=0; i<sTypeNum; i++) {
			cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->sTypes[i]]);
		}


#ifdef LOG_DATA
		//LOG DATA
		int currentIdx = time%(MAX_DELAY+1);

		//printf("HERE %p %d ", buffers->c_gFiredTableSizes, currentIdx);
		//fflush(stdout);
		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		//printf("HERE1\n");
		//fflush(stdout);
		copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);

		//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGpuNet->synapseNums[exp_idx+1]-c_pGpuNet->synapseNums[exp_idx]);

		//fprintf(dataFile, "Cycle %d: ", time);
		if (copy_idx >= 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
				fprintf(v_file, "%.10lf \t", c_vm[i]);
			}


			if (life_idx >= 0) {
				copyFromGPU<real>(c_vm, c_g_ie, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
				for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
					fprintf(ie_file, "%.10lf \t", c_vm[i]);
				}
				copyFromGPU<real>(c_vm, c_g_ii, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
				for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
					fprintf(ii_file, "%.10lf \t", c_vm[i]);
				}
			}
		}
		//for (int i=0; i<c_pGpuNet->synapseNums[1] - c_pGpuNet->synapseNums[0]; i++) {
		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
		//}
		fprintf(v_file, "\n");
		fprintf(input_e_file, "\n");
		fprintf(input_i_file, "\n");
		fprintf(ie_file, "\n");
		fprintf(ii_file, "\n");

		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		//LOG SYNAPSE
		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);
		//int synapseCount = 0;
		//if (time > 0) {
		//	for (int i=0; i<totalSynapseNum; i++) {
		//		if (buffers->c_synapsesFired[i] == time) {
		//			if (synapseCount ==  0) {
		//				if (copySize > 0) {
		//					fprintf(logFile, ", ");
		//				}
		//				fprintf(logFile, "%s", network->idx2sid[i].getInfo().c_str());
		//				synapseCount++;
		//			} else {
		//				fprintf(logFile, ", %s", network->idx2sid[i].getInfo().c_str());
		//			}
		//		}
		//	}
		//	fprintf(logFile, "\n");
		//}
#endif

		update_time<<<1, 1>>>();
	}
	hipDeviceSynchronize();

	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("\nSimulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	//CALC Firing Rate
	int *rate = (int*)malloc(sizeof(int)*totalNeuronNum);
	copyFromGPU<int>(rate, buffers->c_gFireCount, totalNeuronNum);

	FILE *rateFile = fopen("GFire.log", "w+");
	if (rateFile == NULL) {
		printf("Open file Sim.log failed\n");
		return -1;
	}

	for (int i=0; i<totalNeuronNum; i++) {
		fprintf(rateFile, "%d \t", rate[i]);
	}

	fclose(rateFile);

	fclose(v_file);
	fclose(input_e_file);
	fclose(input_i_file);
	fclose(ie_file);
	fclose(ii_file);
	fclose(log_file);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);

	return 0;
}

//int SingleGPUSimulator::compare_run(real time)
//{
//
//	int sim_cycle = round(time/dt);
//
//	reset();
//
//	GNetwork *pCpuNet = network->buildNetwork();
//
//	FILE *logFile = fopen("GSim_compare.log", "w+");
//	if (logFile == NULL) {
//		printf("ERROR: Open file SimGPU.log failed\n");
//		return -1;
//	}
//	FILE *dataFile = fopen("GSim_compare.data", "w+");
//	if (dataFile == NULL) {
//		printf("ERROR: Open file SimGPU.log failed\n");
//		return -1;
//	}
//
//	findCudaDevice(0, NULL);
//	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);
//
//	int nTypeNum = pCpuNet->nTypeNum;
//	int sTypeNum = pCpuNet->sTypeNum;
//	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
//	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
//	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
//	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);
//
//	int MAX_DELAY = pCpuNet->MAX_DELAY;
//	printf("MAX_DELAY: %d\n", pCpuNet->MAX_DELAY);
//
//	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, MAX_DELAY);
//
//	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
//	//BlockSize preSize = { 0, 0, 0};
//	//hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_pre_synapse, 0, totalNeuronNum); 
//	//preSize.gridSize = (totalNeuronNum + (preSize.blockSize) - 1) / (preSize.blockSize);
//
//	real *c_vm = hostMalloc<real>(totalNeuronNum);
//	int lif_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIF);
//
//	GLIFNeurons *c_g_lif = NULL;
//	real *c_g_vm = NULL;
//	if (lif_idx >= 0) {
//		c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGpuNet->pNeurons[lif_idx]), 1);
//		c_g_vm = c_g_lif->p_vm;
//	}
//	//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
//	//int exp_idx = getIndex(pCpuNet->sTypes, sTypeNum, Exp);
//	//GExpSynapses *c_g_exp = copyFromGPU<GExpSynapses>(static_cast<GExpSynapses*>(c_pGpuNet->pSynapses[exp_idx]), 1);
//	//real *c_g_I_syn = c_g_exp->p_I_syn;
//
//	vector<int> firedInfo;
//	printf("Start runing for %d cycles\n", sim_cycle);
//	struct timeval ts, te;
//	gettimeofday(&ts, NULL);
//	for (int time=0; time<sim_cycle; time++) {
//		//printf("\rCycle: %d", time);
//		//fflush(stdout);
//
//		for (int i=0; i<nTypeNum; i++) {
//			cudaUpdateAllType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
//		}
//
//		//update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(c_pGpuNet->pN2SConnection);
//
//		for (int i=0; i<sTypeNum; i++) {
//			cudaUpdateAllType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
//		}
//
//
//		int currentIdx = time%(MAX_DELAY+1);
//
//		int copySize = 0;
//		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
//		copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);
//		if (lif_idx >= 0) {
//			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[lif_idx+1]-c_pGpuNet->neuronNums[lif_idx]);
//		}
//		//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGpuNet->synapseNums[exp_idx+1]-c_pGpuNet->synapseNums[exp_idx]);
//
//		fprintf(logFile, "Cycle %d: ", time);
//		for (int i=0; i<copySize; i++) {
//			//assert(network->idx2nid.find(buffers->c_neuronsFired[i]) != network->idx2nid.end());
//			//printf("%s ", network->idx2nid[buffers->c_neuronsFired[i]].getInfo().c_str());
//			fprintf(logFile, "%d ", buffers->c_neuronsFired[i]);
//		}
//		fprintf(logFile, "\n");
//
//		//fprintf(dataFile, "Cycle %d: ", time);
//		if (lif_idx >= 0) {
//			for (int i=0; i<c_pGpuNet->neuronNums[lif_idx+1] - c_pGpuNet->neuronNums[lif_idx]; i++) {
//				fprintf(dataFile, "%lf ", c_vm[i]);
//			}
//		}
//		//for (int i=0; i<c_pGpuNet->synapseNums[1] - c_pGpuNet->synapseNums[0]; i++) {
//		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
//		//}
//		fprintf(dataFile, "\n");
//
//		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);
//		//int synapseCount = 0;
//		//if (time > 0) {
//		//	for (int i=0; i<totalSynapseNum; i++) {
//		//		if (buffers->c_synapsesFired[i] == time) {
//		//			if (synapseCount ==  0) {
//		//				if (copySize > 0) {
//		//					fprintf(logFile, ", ");
//		//				}
//		//				fprintf(logFile, "%s", network->idx2sid[i].getInfo().c_str());
//		//				synapseCount++;
//		//			} else {
//		//				fprintf(logFile, ", %s", network->idx2sid[i].getInfo().c_str());
//		//			}
//		//		}
//		//	}
//		//	fprintf(logFile, "\n");
//		//}
//
//		update_time<<<1, 1>>>();
//	}
//	gettimeofday(&te, NULL);
//	long seconds = te.tv_sec - ts.tv_sec;
//	long hours = seconds/3600;
//	seconds = seconds%3600;
//	long minutes = seconds/60;
//	seconds = seconds%60;
//	long uSeconds = te.tv_usec - ts.tv_usec;
//	if (uSeconds < 0) {
//		uSeconds += 1000000;
//		seconds = seconds - 1;
//	}
//
//	printf("\nSimulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);
//
//	fclose(logFile);
//	fclose(dataFile);
//
//	free_buffers(buffers);
//	freeGPUNetwork(c_pGpuNet);
//
//	return 0;
//}
