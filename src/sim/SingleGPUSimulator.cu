#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/FileOp.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
// #include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/GBuffers.h"
#include "../gpu_utils/runtime.h"
#include "../net/MultiNetwork.h"
// #include "../gpu_utils/gpu_func.h"

#include "SingleGPUSimulator.h"

using std::cout;
using std::endl;

CrossNodeDataGPU * gCrossDataNet;

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

int SingleGPUSimulator::run(real time, FireInfo &log)
{

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pCPUNet = network->buildNetwork();

	FILE *v_file = openFile("v.data", "w+");

	FILE *input_e_file = openFile("input_e.data", "w+");

	FILE *input_i_file = openFile("input_i.data", "w+");

	FILE *ie_file = openFile("ie.data", "w+");

	FILE *ii_file = openFile("ii.data", "w+");

	FILE *log_file = openFile("sim.log", "w+");

	//findCudaDevice(0, NULL);
	checkCudaErrors(hipSetDevice(0));
	GNetwork *c_pGPUNet = copyNetworkToGPU(pCPUNet);

	int nTypeNum = c_pGPUNet->nTypeNum;
	int sTypeNum = c_pGPUNet->sTypeNum;
	int totalNeuronNum = c_pGPUNet->neuronNums[nTypeNum];
	int totalSynapseNum = c_pGPUNet->synapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = c_pGPUNet->MAX_DELAY;
	printf("MAX_DELAY: %d\n", c_pGPUNet->MAX_DELAY);


	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, MAX_DELAY, dt);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(totalNeuronNum);

	int copy_idx = getIndex(c_pGPUNet->nTypes, nTypeNum, LIF);

	GLIFNeurons *c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGPUNet->pNeurons[copy_idx]), 1);

	real *c_g_vm = c_g_lif->p_vm;
#ifdef DEBUG 
	real *c_g_ie = c_g_lif->p_i_E;
	real *c_g_ii = c_g_lif->p_i_I;
#endif
#endif

	//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
	//int exp_idx = getIndex(c_pGPUNet->sTypes, sTypeNum, Exp);
	//GExpSynapses *c_g_exp = copyFromGPU<GExpSynapses>(static_cast<GExpSynapses*>(c_pGPUNet->pSynapses[exp_idx]), 1);
	//real *c_g_I_syn = c_g_exp->p_I_syn;

	//for (int i=0; i<nTypeNum; i++) {
	//	cout << c_pGPUNet->nTypes[i] << ": <<<" << updateSize[c_pGPUNet->nTypes[i]].gridSize << ", " << updateSize[c_pGPUNet->nTypes[i]].blockSize << ">>>" << endl;
	//}
	//for (int i=0; i<sTypeNum; i++) {
	//	cout << c_pGPUNet->sTypes[i] << ": <<<" << updateSize[c_pGPUNet->sTypes[i]].gridSize << ", " << updateSize[c_pGPUNet->sTypes[i]].blockSize << ">>>" << endl;
	//}

	vector<int> firedInfo;
	printf("Start runing for %d cycles\n", sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		//printf("Cycle: %d ", time);
		//fflush(stdout);

#ifdef DEBUG
		copyFromGPU<real>(c_vm, buffers->c_gNeuronInput + c_pGPUNet->neuronNums[copy_idx], c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
			fprintf(input_e_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(input_e_file, "\n");
		copyFromGPU<real>(c_vm, buffers->c_gNeuronInput_I + c_pGPUNet->neuronNums[copy_idx], c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
			fprintf(input_i_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(input_i_file, "\n");
#endif

		update_time<<<1, 1>>>(time, buffers->c_gFiredTableSizes);

		for (int i=0; i<nTypeNum; i++) {
			cudaUpdateNeuron[c_pGPUNet->nTypes[i]](c_pGPUNet->pNeurons[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pGPUNet->neuronNums[i+1]-c_pGPUNet->neuronNums[i], c_pGPUNet->neuronNums[i],time, &updateSize[c_pGPUNet->nTypes[i]]);
		}

		for (int i=0; i<sTypeNum; i++) {
			cudaUpdateSynapse[c_pGPUNet->sTypes[i]](c_pGPUNet->pN2SConnection, c_pGPUNet->pSynapses[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pGPUNet->synapseNums[i+1]-c_pGPUNet->synapseNums[i], c_pGPUNet->synapseNums[i], time, &updateSize[c_pGPUNet->sTypes[i]]);
		}


#ifdef LOG_DATA
		//LOG DATA
		int currentIdx = time%(MAX_DELAY+1);

		//printf("HERE %p %d ", buffers->c_gFiredTableSizes, currentIdx);
		//fflush(stdout);
		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		//printf("HERE1\n");
		//fflush(stdout);
		copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);
		//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGPUNet->synapseNums[exp_idx+1]-c_pGPUNet->synapseNums[exp_idx]);

		//fprintf(dataFile, "Cycle %d: ", time);
		copyFromGPU<real>(c_vm, c_g_vm, c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");
#ifdef DEBUG
		copyFromGPU<real>(c_vm, c_g_ie, c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
			fprintf(ie_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(ie_file, "\n");
		copyFromGPU<real>(c_vm, c_g_ii, c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
			fprintf(ii_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(ii_file, "\n");
		//for (int i=0; i<c_pGPUNet->synapseNums[1] - c_pGPUNet->synapseNums[0]; i++) {
		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
		//}
#endif

		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		//LOG SYNAPSE
		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);
		//int synapseCount = 0;
		//if (time > 0) {
		//	for (int i=0; i<totalSynapseNum; i++) {
		//		if (buffers->c_synapsesFired[i] == time) {
		//			if (synapseCount ==  0) {
		//				if (copySize > 0) {
		//					fprintf(logFile, ", ");
		//				}
		//				fprintf(logFile, "%s", network->idx2sid[i].getInfo().c_str());
		//				synapseCount++;
		//			} else {
		//				fprintf(logFile, ", %s", network->idx2sid[i].getInfo().c_str());
		//			}
		//		}
		//	}
		//	fprintf(logFile, "\n");
		//}
#endif
	}
	hipDeviceSynchronize();

	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Simulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	//CALC Firing Rate
	if (log.find("count") != log.end()) {
		int *rate = (int*)malloc(sizeof(int)*totalNeuronNum);
		copyFromGPU<int>(rate, buffers->c_gFireCount, totalNeuronNum);

		log["count"].size = totalNeuronNum;
		log["count"].data = rate;

		//FILE *rateFile = fopen("GFire.log", "w+");
		//if (rateFile == NULL) {
		//	printf("ERROR: Open file Sim.log failed\n");
		//	return -1;
		//}

		//for (int i=0; i<totalNeuronNum; i++) {
		//	fprintf(rateFile, "%d \t", rate[i]);
		//}

		//fflush(rateFile);
		//fclose(rateFile);
	}

	if (log.find("Y") != log.end()) {
		real *Y = (real*)malloc(sizeof(real)*totalNeuronNum);
		copyFromGPU<real>(Y, buffers->c_gXInput, totalNeuronNum);
		log["Y"].size = totalNeuronNum;
		log["Y"].data = Y;
	}

	if (log.find("X") != log.end()) {
		int *X = (int*)malloc(sizeof(int)*totalNeuronNum);
		copyFromGPU<int>(X, buffers->c_gLayerInput, totalNeuronNum);
		log["X"].size = totalNeuronNum;
		log["X"].data = X;
	}



	closeFile(v_file);
	closeFile(input_e_file);
	closeFile(input_i_file);
	closeFile(ie_file);
	closeFile(ii_file);
	closeFile(log_file);

	free_buffers(buffers);
	freeGPUNetwork(c_pGPUNet);
	freeGNetwork(pCPUNet);

	return 0;
}


int SingleGPUSimulator::runMultiNets(real time, int parts, FireInfo &log) {
	int sim_cycle = round(time/dt);
	reset();

	checkCudaErrors(hipSetDevice(0));

	MultiNetwork multiNet(network, parts);
	DistriNetwork *subnets = multiNet.buildNetworks();
	assert(subnets != NULL);
	CrossNodeDataGPU *crossData = multiNet.arrangeCrossNodeDataGPU(parts);
	assert(crossData != NULL);

	GNetwork ** networks = (GNetwork **)malloc(sizeof(GNetwork *) * parts);
	GBuffers **buffers = (GBuffers **)malloc(sizeof(GBuffers *) * parts);
	BlockSize **updateSizes = (BlockSize **)malloc(sizeof(GBuffers *) * parts);

	for (int i=0; i<parts; i++) {
		subnets[i]._sim_cycle = sim_cycle;
		subnets[i]._node_idx = i;
		subnets[i]._node_num = parts;
		subnets[i]._dt = dt;

		GNetwork *pCPUNet = subnets[i]._network;
		networks[i] = copyNetworkToGPU(pCPUNet);
		GNetwork *c_pGPUNet = networks[i];

		int nTypeNum = c_pGPUNet->nTypeNum;
		int sTypeNum = c_pGPUNet->sTypeNum;
		int nodeNeuronNum = c_pGPUNet->neuronNums[nTypeNum];
		int allNeuronNum = pCPUNet->pN2SConnection->n_num;
		int nodeSynapseNum = c_pGPUNet->synapseNums[sTypeNum];

		buffers[i] = alloc_buffers(allNeuronNum, nodeSynapseNum, c_pGPUNet->MAX_DELAY, dt);
		updateSizes[i] = getBlockSize(allNeuronNum, nodeSynapseNum);

		printf("Subnet %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", subnets[i]._node_idx, nTypeNum, sTypeNum);
		printf("Subnet %d NeuronNum: %d, SynapseNum: %d\n", subnets[i]._node_idx, nodeNeuronNum, nodeSynapseNum);
	}

	for (int time=0; time<sim_cycle; time++) {

		for (int p=0; p<parts; p++) {
			update_time<<<1, 1>>>(time, buffers[p]->c_gFiredTableSizes);

			GNetwork *c_pGPUNet = networks[p];
			for (int i=0; i<c_pGPUNet->nTypeNum; i++) {
				assert(c_pGPUNet->neuronNums[i+1]-c_pGPUNet->neuronNums[i] > 0);
				cudaUpdateNeuron[c_pGPUNet->nTypes[i]](c_pGPUNet->pNeurons[i], buffers[p]->c_gNeuronInput, buffers[p]->c_gNeuronInput_I, buffers[p]->c_gFiredTable, buffers[p]->c_gFiredTableSizes, c_pGPUNet->neuronNums[i+1]-c_pGPUNet->neuronNums[i], c_pGPUNet->neuronNums[i], time, &updateSizes[p][c_pGPUNet->nTypes[i]]);
			}
		}

		for (int p=0; p<parts; p++) {
			GNetwork *c_pGPUNet = networks[p];
			for (int i=0; i<c_pGPUNet->sTypeNum; i++) {
				assert(c_pGPUNet->synapseNums[i+1]-c_pGPUNet->synapseNums[i] > 0);
				cudaUpdateSynapse[c_pGPUNet->sTypes[i]](c_pGPUNet->pN2SConnection, c_pGPUNet->pSynapses[i], buffers[p]->c_gNeuronInput, buffers[p]->c_gNeuronInput_I, buffers[p]->c_gFiredTable, buffers[p]->c_gFiredTableSizes, c_pGPUNet->synapseNums[i+1]-c_pGPUNet->synapseNums[i], c_pGPUNet->synapseNums[i], time, &updateSizes[p][c_pGPUNet->sTypes[i]]);
			}
		}
	}

	for (int i=0; i<parts; i++) {
		freeGPUNetwork(networks[i]);
		free_buffers(buffers[i]);
	}

	free(networks);
	free(buffers);
	free(updateSizes);

	return 0;
}

