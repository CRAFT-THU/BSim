#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_kernel.h"
#include "SingleGPUSimulator.h"

struct gpu_buffers {
	// Arrays
	int *gTimeTable;

	// Neuron Arrays
	real *gNeuronInput;

	// Neuron Tables
	int *gFiredTable;
	int *gFiredTableSizes;

	int *gActiveTable;
	int gActiveTableSize;

	// Synapse Tables
	int *gSynapsesActiveTable;
	int *gSynapsesActiveTableSize;

	int *gSynapsesLogTable;
}

int global_malloc()
{

}

int SingleGPUSimulator::run(real time)
{
	findCudaDevice(0, NULL);

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pCpuNet = network->buildNetwork();

	FILE *logFile = fopen("GSim.log", "w+");
	if (logFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}
	FILE *dataFile = fopen("GSim.data", "w+");
	if (dataFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}

	GNetwork *c_pGpuNet = copyDataToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	printf("MAX_DELAY: %lf %lf %d\n", network->maxDelay, dt, pCpuNet->MAX_DELAY);

	int * c_n_fired = (int*)malloc(sizeof(int)*((totalNeuronNum)));
	bool * c_s_fired = (bool*)malloc(sizeof(bool)*((totalSynapseNum)));
	real * c_n_input = (real*)malloc(sizeof(real)*((totalNeuronNum)));

	checkCudaErrors(hipMalloc((void**)&c_gTimeTable, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMemset(c_gTimeTable, 0, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMalloc((void**)&c_gFiredTable, sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1))));
	checkCudaErrors(hipMemset(c_gFiredTable, 0, sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1))));
	checkCudaErrors(hipMalloc((void**)&c_gSynapsesFiredTable, sizeof(bool)*(totalSynapseNum)));
	checkCudaErrors(hipMemset(c_gSynapsesFiredTable, 0, sizeof(bool)*(totalSynapseNum)));
	checkCudaErrors(hipMalloc((void**)&c_gNeuronInput, sizeof(real)*(totalNeuronNum)));
	checkCudaErrors(hipMemset(c_gNeuronInput, 0, sizeof(real)*(totalNeuronNum)));

	init_global<<<1, 1, 0>>>(MAX_DELAY, c_gTimeTable, c_gNeuronInput, c_gFiredTable, totalNeuronNum, c_gSynapsesFiredTable, totalSynapseNum, pGpuNet);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	BlockSize preSize = { 0, 0, 0};
	BlockSize postSize = { 0, 0, 0};
	hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_lif_neuron, 0, totalNeuronNum); 
	preSize.gridSize = (totalNeuronNum + (preSize.blockSize) - 1) / (preSize.blockSize);
	hipOccupancyMaxPotentialBlockSize(&(postSize.minGridSize), &(postSize.blockSize), update_lif_neuron, 0, totalSynapseNum); 
	postSize.gridSize = (totalSynapseNum + (postSize.blockSize) - 1) / (postSize.blockSize);

	vector<int> firedInfo;
	printf("Start runing for %d cycles\n", sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		printf("\rCycle: %d", time);
		fflush(stdout);
		//update_pre_neuron<<<1, 1, 0>>>(pGpuNet, simTime);
		//update_lif_neuron<<<3, 2, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, c_pGpuNet->neuronNum, time);

		checkCudaErrors(hipMemcpy(c_n_input, c_gNeuronInput, sizeof(real)*(totalNeuronNum), hipMemcpyDeviceToHost));
		for (int i=0; i<nTypeNum; i++) {
			updateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], time, &updateSize[c_pGpuNet->nTypes[i]]);
		}

		//update_pre_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, time);
		update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(pGpuNet, time);

		checkCudaErrors(hipMemcpy(c_s_fired, c_gSynapsesFiredTable, sizeof(bool)*(totalSynapseNum), hipMemcpyDeviceToHost));

		//update_exp_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, c_pGpuNet->synapseNum, time);
		for (int i=0; i<sTypeNum; i++) {
			updateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], time, &updateSize[pCpuNet->nTypes[i]]);
		}

		update_post_synapse<<<postSize.gridSize, postSize.blockSize>>>(pGpuNet, time);

	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("\nSimulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	fclose(logFile);
	fclose(dataFile);

	checkCudaErrors(hipFree(c_gTimeTable));
	checkCudaErrors(hipFree(c_gFiredTable));
	checkCudaErrors(hipFree(c_gSynapsesFiredTable));

	freeGPUData(c_pGpuNet);
	checkCudaErrors(hipFree(pGpuNet));

	return 0;
}

