#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/gpu_kernel.h"
#include "SingleGPUSimulator.h"

using std::cout;
using std::endl;

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

int SingleGPUSimulator::run(real time)
{

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pCpuNet = network->buildNetwork();

	FILE *dataFile = fopen("GSim.data", "w+");
	if (dataFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}

	//FILE *logFile = fopen("GSim.log", "w+");
	//if (logFile == NULL) {
	//	printf("ERROR: Open file SimGPU.log failed\n");
	//	return -1;
	//}

	findCudaDevice(0, NULL);
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	printf("MAX_DELAY: %d\n", pCpuNet->MAX_DELAY);

	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, MAX_DELAY);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	BlockSize preSize = { 0, 0, 0};
	BlockSize postSize = { 0, 0, 0};
	hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_lif_neuron, 0, totalNeuronNum); 
	preSize.gridSize = (totalNeuronNum + (preSize.blockSize) - 1) / (preSize.blockSize);
	hipOccupancyMaxPotentialBlockSize(&(postSize.minGridSize), &(postSize.blockSize), update_lif_neuron, 0, totalSynapseNum); 
	postSize.gridSize = (totalSynapseNum + (postSize.blockSize) - 1) / (postSize.blockSize);

	real *c_vm = hostMalloc<real>(totalNeuronNum);

	int life_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIFE);
	int lif_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		GLIFENeurons *c_g_lif = copyFromGPU<GLIFENeurons>(static_cast<GLIFENeurons*>(c_pGpuNet->pNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		copy_idx = life_idx;
	} else if (lif_idx >= 0) {
		GLIFNeurons *c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGpuNet->pNeurons[lif_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		copy_idx = lif_idx;
	} else {
	}

	//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
	//int exp_idx = getIndex(pCpuNet->sTypes, sTypeNum, Exp);
	//GExpSynapses *c_g_exp = copyFromGPU<GExpSynapses>(static_cast<GExpSynapses*>(c_pGpuNet->pSynapses[exp_idx]), 1);
	//real *c_g_I_syn = c_g_exp->p_I_syn;

	for (int i=0; i<nTypeNum; i++) {
		cout << pCpuNet->nTypes[i] << ": <<<" << updateSize[c_pGpuNet->nTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->nTypes[i]].blockSize << ">>>" << endl;
	}
	for (int i=0; i<sTypeNum; i++) {
		cout << pCpuNet->sTypes[i] << ": <<<" << updateSize[c_pGpuNet->sTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->sTypes[i]].blockSize << ">>>" << endl;
	}

	vector<int> firedInfo;
	printf("Start runing for %d cycles\n", sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		//printf("Cycle: %d ", time);
		//fflush(stdout);

		for (int i=0; i<nTypeNum; i++) {
			cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
		}

		//update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(c_pGpuNet->pN2SConnection);

		for (int i=0; i<sTypeNum; i++) {
			cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
		}


		int currentIdx = time%(MAX_DELAY+1);

		//printf("HERE %p %d ", buffers->c_gFiredTableSizes, currentIdx);
		//fflush(stdout);
		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		//printf("HERE1\n");
		//fflush(stdout);
		//copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);

#ifdef LOG_DATA
		//LOG DATA
		//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGpuNet->synapseNums[exp_idx+1]-c_pGpuNet->synapseNums[exp_idx]);

		//fprintf(dataFile, "Cycle %d: ", time);
		if (copy_idx >= 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
				fprintf(dataFile, "%.10lf \t", c_vm[i]);
			}
		}
		//for (int i=0; i<c_pGpuNet->synapseNums[1] - c_pGpuNet->synapseNums[0]; i++) {
		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
		//}
		fprintf(dataFile, "\n");

		//fprintf(logFile, "Cycle %d: ", time);
		//for (int i=0; i<copySize; i++) {
		//	//assert(network->idx2nid.find(buffers->c_neuronsFired[i]) != network->idx2nid.end());
		//	//printf("%s ", network->idx2nid[buffers->c_neuronsFired[i]].getInfo().c_str());
		//	fprintf(logFile, "%d ", buffers->c_neuronsFired[i]);
		//}
		//fprintf(logFile, "\n");

		//LOG SYNAPSE
		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);
		//int synapseCount = 0;
		//if (time > 0) {
		//	for (int i=0; i<totalSynapseNum; i++) {
		//		if (buffers->c_synapsesFired[i] == time) {
		//			if (synapseCount ==  0) {
		//				if (copySize > 0) {
		//					fprintf(logFile, ", ");
		//				}
		//				fprintf(logFile, "%s", network->idx2sid[i].getInfo().c_str());
		//				synapseCount++;
		//			} else {
		//				fprintf(logFile, ", %s", network->idx2sid[i].getInfo().c_str());
		//			}
		//		}
		//	}
		//	fprintf(logFile, "\n");
		//}
#endif

		update_time<<<1, 1>>>();
	}

	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("\nSimulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	//CALC Firing Rate
	int *rate = hostMalloc<int>(totalNeuronNum);
	copyFromGPU<int>(rate, buffers->c_gFireCount, totalNeuronNum);

	FILE *rateFile = fopen("GFire.log", "w+");
	if (rateFile == NULL) {
		printf("Open file Sim.log failed\n");
		return -1;
	}

	for (int i=0; i<totalNeuronNum; i++) {
		fprintf(rateFile, "%d \t", rate[i]);
	}

	fclose(rateFile);

	fclose(dataFile);
	//fclose(logFile);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);

	return 0;
}

int SingleGPUSimulator::compare_run(real time)
{

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pCpuNet = network->buildNetwork();

	FILE *logFile = fopen("GSim_compare.log", "w+");
	if (logFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}
	FILE *dataFile = fopen("GSim_compare.data", "w+");
	if (dataFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}

	findCudaDevice(0, NULL);
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	printf("MAX_DELAY: %d\n", pCpuNet->MAX_DELAY);

	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, MAX_DELAY);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	BlockSize preSize = { 0, 0, 0};
	BlockSize postSize = { 0, 0, 0};
	hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_lif_neuron, 0, totalNeuronNum); 
	preSize.gridSize = (totalNeuronNum + (preSize.blockSize) - 1) / (preSize.blockSize);
	hipOccupancyMaxPotentialBlockSize(&(postSize.minGridSize), &(postSize.blockSize), update_lif_neuron, 0, totalSynapseNum); 
	postSize.gridSize = (totalSynapseNum + (postSize.blockSize) - 1) / (postSize.blockSize);

	real *c_vm = hostMalloc<real>(totalNeuronNum);
	int lif_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIF);

	GLIFNeurons *c_g_lif = NULL;
	real *c_g_vm = NULL;
	if (lif_idx >= 0) {
		c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGpuNet->pNeurons[lif_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
	}
	//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
	//int exp_idx = getIndex(pCpuNet->sTypes, sTypeNum, Exp);
	//GExpSynapses *c_g_exp = copyFromGPU<GExpSynapses>(static_cast<GExpSynapses*>(c_pGpuNet->pSynapses[exp_idx]), 1);
	//real *c_g_I_syn = c_g_exp->p_I_syn;

	vector<int> firedInfo;
	printf("Start runing for %d cycles\n", sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		//printf("\rCycle: %d", time);
		//fflush(stdout);

		for (int i=0; i<nTypeNum; i++) {
			cudaUpdateAllType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
		}

		update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(c_pGpuNet->pN2SConnection);

		for (int i=0; i<sTypeNum; i++) {
			cudaUpdateAllType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
		}


		int currentIdx = time%(MAX_DELAY+1);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);
		if (lif_idx >= 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[lif_idx+1]-c_pGpuNet->neuronNums[lif_idx]);
		}
		//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGpuNet->synapseNums[exp_idx+1]-c_pGpuNet->synapseNums[exp_idx]);

		fprintf(logFile, "Cycle %d: ", time);
		for (int i=0; i<copySize; i++) {
			//assert(network->idx2nid.find(buffers->c_neuronsFired[i]) != network->idx2nid.end());
			//printf("%s ", network->idx2nid[buffers->c_neuronsFired[i]].getInfo().c_str());
			fprintf(logFile, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(logFile, "\n");

		//fprintf(dataFile, "Cycle %d: ", time);
		if (lif_idx >= 0) {
			for (int i=0; i<c_pGpuNet->neuronNums[lif_idx+1] - c_pGpuNet->neuronNums[lif_idx]; i++) {
				fprintf(dataFile, "%lf ", c_vm[i]);
			}
		}
		//for (int i=0; i<c_pGpuNet->synapseNums[1] - c_pGpuNet->synapseNums[0]; i++) {
		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
		//}
		fprintf(dataFile, "\n");

		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);
		//int synapseCount = 0;
		//if (time > 0) {
		//	for (int i=0; i<totalSynapseNum; i++) {
		//		if (buffers->c_synapsesFired[i] == time) {
		//			if (synapseCount ==  0) {
		//				if (copySize > 0) {
		//					fprintf(logFile, ", ");
		//				}
		//				fprintf(logFile, "%s", network->idx2sid[i].getInfo().c_str());
		//				synapseCount++;
		//			} else {
		//				fprintf(logFile, ", %s", network->idx2sid[i].getInfo().c_str());
		//			}
		//		}
		//	}
		//	fprintf(logFile, "\n");
		//}

		update_time<<<1, 1>>>();
	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("\nSimulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	fclose(logFile);
	fclose(dataFile);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);

	return 0;
}
