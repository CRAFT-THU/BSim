#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
// #include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/GBuffers.h"
#include "../gpu_utils/runtime.h"
#include "../net/MultiNetwork.h"
// #include "../gpu_utils/gpu_func.h"

#include "SingleGPUSimulator.h"

using std::cout;
using std::endl;

CrossNodeDataGPU * gCrossDataNet;

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

int SingleGPUSimulator::run(real time, FireInfo &log)
{

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pCPUNet = network->buildNetwork();

	FILE *v_file = fopen("g_v.data", "w+");
	if (v_file == NULL) {
		printf("ERROR: Open file g_v.data failed\n");
		return -1;
	}

	FILE *input_e_file = fopen("g_input_e.data", "w+");
	if (input_e_file == NULL) {
		printf("Open file g_input_e.data failed\n");
		return -1;
	}

	FILE *input_i_file = fopen("g_input_i.data", "w+");
	if (input_i_file == NULL) {
		printf("Open file g_input_i.data failed\n");
		return -1;
	}

	FILE *ie_file = fopen("g_ie.data", "w+");
	if (ie_file == NULL) {
		printf("ERROR: Open file g_ie.data failed\n");
		return -1;
	}

	FILE *ii_file = fopen("g_ii.data", "w+");
	if (ii_file == NULL) {
		printf("ERROR: Open file g_ii.data failed\n");
		return -1;
	}

	FILE *info_file = fopen("GSim.info", "w+");
	if (info_file == NULL) {
		printf("ERROR: Open file GSim.info failed\n");
		return -1;
	}

	FILE *log_file = fopen("GSim.log", "w+");
	if (log_file == NULL) {
		printf("ERROR: Open file GSim.log failed\n");
		return -1;
	}

	//findCudaDevice(0, NULL);
	checkCudaErrors(hipSetDevice(0));
	GNetwork *c_pGPUNet = copyNetworkToGPU(pCPUNet);

	int nTypeNum = c_pGPUNet->nTypeNum;
	int sTypeNum = c_pGPUNet->sTypeNum;
	int totalNeuronNum = c_pGPUNet->neuronNums[nTypeNum];
	int totalSynapseNum = c_pGPUNet->synapseNums[sTypeNum];
	fprintf(info_file, "NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	fprintf(info_file, "NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = c_pGPUNet->MAX_DELAY;
	fprintf(info_file, "MAX_DELAY: %d\n", c_pGPUNet->MAX_DELAY);

	// init_connection<<<1, 1>>>(c_pGPUNet->pN2SConnection);

	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, MAX_DELAY, dt);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	//BlockSize preSize = { 0, 0, 0};
	//hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_pre_synapse, 0, totalNeuronNum); 
	//preSize.gridSize = (totalSynapseNum + (preSize.blockSize) - 1) / (preSize.blockSize);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(totalNeuronNum);

	// int tj_idx = getIndex(c_pGPUNet->nTypes, nTypeNum, TJ);
	int life_idx = getIndex(c_pGPUNet->nTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;
	real *c_g_ie = NULL;
	real *c_g_ii = NULL;

	if (life_idx >= 0) {
		GLIFNeurons *c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGPUNet->pNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		c_g_ie = c_g_lif->p_i_E;
		c_g_ii = c_g_lif->p_i_I;
		copy_idx = life_idx;
	} /* else if (tj_idx >= 0) {
		 GTJNeurons *c_g_tj = copyFromGPU<GTJNeurons>(static_cast<GTJNeurons*>(c_pGPUNet->pNeurons[tj_idx]), 1);
		 c_g_vm = c_g_tj->p_vm;
		 copy_idx = tj_idx;
		 } */ else {
		 }
#endif

//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
//int exp_idx = getIndex(c_pGPUNet->sTypes, sTypeNum, Exp);
//GExpSynapses *c_g_exp = copyFromGPU<GExpSynapses>(static_cast<GExpSynapses*>(c_pGPUNet->pSynapses[exp_idx]), 1);
//real *c_g_I_syn = c_g_exp->p_I_syn;

//for (int i=0; i<nTypeNum; i++) {
//	cout << c_pGPUNet->nTypes[i] << ": <<<" << updateSize[c_pGPUNet->nTypes[i]].gridSize << ", " << updateSize[c_pGPUNet->nTypes[i]].blockSize << ">>>" << endl;
//}
//for (int i=0; i<sTypeNum; i++) {
//	cout << c_pGPUNet->sTypes[i] << ": <<<" << updateSize[c_pGPUNet->sTypes[i]].gridSize << ", " << updateSize[c_pGPUNet->sTypes[i]].blockSize << ">>>" << endl;
//}

vector<int> firedInfo;
fprintf(info_file, "Start runing for %d cycles\n", sim_cycle);
struct timeval ts, te;
gettimeofday(&ts, NULL);
for (int time=0; time<sim_cycle; time++) {
	//printf("Cycle: %d ", time);
	//fflush(stdout);

#ifdef LOG_DATA
	if (copy_idx >= 0) {
		//copyFromGPU<real>(c_vm, buffers->c_gNeuronInput + c_pGPUNet->neuronNums[copy_idx], c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		copyFromGPU<real>(c_vm, buffers->c_gNeuronInput + c_pGPUNet->neuronNums[copy_idx], c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
			fprintf(input_e_file, "%.10lf \t", c_vm[i]);
		}
		//copyFromGPU<real>(c_vm, buffers->c_gNeuronInput_I + c_pGPUNet->neuronNums[copy_idx], c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		copyFromGPU<real>(c_vm, buffers->c_gNeuronInput_I + c_pGPUNet->neuronNums[copy_idx], c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
			fprintf(input_i_file, "%.10lf \t", c_vm[i]);
		}
	}
#endif

	update_time<<<1, 1>>>(time, buffers->c_gFiredTableSizes);

	for (int i=0; i<nTypeNum; i++) {
		cudaUpdateNeuron[c_pGPUNet->nTypes[i]](c_pGPUNet->pNeurons[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pGPUNet->neuronNums[i+1]-c_pGPUNet->neuronNums[i], c_pGPUNet->neuronNums[i],time, &updateSize[c_pGPUNet->nTypes[i]]);
	}

	//update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(c_pGPUNet->pN2SConnection);

	for (int i=0; i<sTypeNum; i++) {
		cudaUpdateSynapse[c_pGPUNet->sTypes[i]](c_pGPUNet->pN2SConnection, c_pGPUNet->pSynapses[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pGPUNet->synapseNums[i+1]-c_pGPUNet->synapseNums[i], c_pGPUNet->synapseNums[i], time, &updateSize[c_pGPUNet->sTypes[i]]);
	}


#ifdef LOG_DATA
	//LOG DATA
	int currentIdx = time%(MAX_DELAY+1);

	//printf("HERE %p %d ", buffers->c_gFiredTableSizes, currentIdx);
	//fflush(stdout);
	int copySize = 0;
	copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
	//printf("HERE1\n");
	//fflush(stdout);
	copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);

	//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGPUNet->synapseNums[exp_idx+1]-c_pGPUNet->synapseNums[exp_idx]);

	//fprintf(dataFile, "Cycle %d: ", time);
	if (copy_idx >= 0) {
		copyFromGPU<real>(c_vm, c_g_vm, c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
		for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}


		if (life_idx >= 0) {
			copyFromGPU<real>(c_vm, c_g_ie, c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
			for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
				fprintf(ie_file, "%.10lf \t", c_vm[i]);
			}
			copyFromGPU<real>(c_vm, c_g_ii, c_pGPUNet->neuronNums[copy_idx+1]-c_pGPUNet->neuronNums[copy_idx]);
			for (int i=0; i<c_pGPUNet->neuronNums[copy_idx+1] - c_pGPUNet->neuronNums[copy_idx]; i++) {
				fprintf(ii_file, "%.10lf \t", c_vm[i]);
			}
		}
	}
	//for (int i=0; i<c_pGPUNet->synapseNums[1] - c_pGPUNet->synapseNums[0]; i++) {
	//		fprintf(dataFile, ", %lf", c_I_syn[i]);
	//}
	fprintf(v_file, "\n");
	fprintf(input_e_file, "\n");
	fprintf(input_i_file, "\n");
	fprintf(ie_file, "\n");
	fprintf(ii_file, "\n");

	for (int i=0; i<copySize; i++) {
		fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
	}
	fprintf(log_file, "\n");

	//LOG SYNAPSE
	//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);
	//int synapseCount = 0;
	//if (time > 0) {
	//	for (int i=0; i<totalSynapseNum; i++) {
	//		if (buffers->c_synapsesFired[i] == time) {
	//			if (synapseCount ==  0) {
	//				if (copySize > 0) {
	//					fprintf(logFile, ", ");
	//				}
	//				fprintf(logFile, "%s", network->idx2sid[i].getInfo().c_str());
	//				synapseCount++;
	//			} else {
	//				fprintf(logFile, ", %s", network->idx2sid[i].getInfo().c_str());
	//			}
	//		}
	//	}
	//	fprintf(logFile, "\n");
	//}
#endif
}
hipDeviceSynchronize();

gettimeofday(&te, NULL);
long seconds = te.tv_sec - ts.tv_sec;
long hours = seconds/3600;
seconds = seconds%3600;
long minutes = seconds/60;
seconds = seconds%60;
long uSeconds = te.tv_usec - ts.tv_usec;
if (uSeconds < 0) {
	uSeconds += 1000000;
	seconds = seconds - 1;
}

fprintf(info_file, "Simulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

//CALC Firing Rate
if (log.find("count") != log.end()) {
	int *rate = (int*)malloc(sizeof(int)*totalNeuronNum);
	copyFromGPU<int>(rate, buffers->c_gFireCount, totalNeuronNum);

	log["count"].size = totalNeuronNum;
	log["count"].data = rate;

	//FILE *rateFile = fopen("GFire.log", "w+");
	//if (rateFile == NULL) {
	//	printf("ERROR: Open file Sim.log failed\n");
	//	return -1;
	//}

	//for (int i=0; i<totalNeuronNum; i++) {
	//	fprintf(rateFile, "%d \t", rate[i]);
	//}

	//fflush(rateFile);
	//fclose(rateFile);
}

if (log.find("Y") != log.end()) {
	real *Y = (real*)malloc(sizeof(real)*totalNeuronNum);
	copyFromGPU<real>(Y, buffers->c_gXInput, totalNeuronNum);
	log["Y"].size = totalNeuronNum;
	log["Y"].data = Y;
}

if (log.find("X") != log.end()) {
	int *X = (int*)malloc(sizeof(int)*totalNeuronNum);
	copyFromGPU<int>(X, buffers->c_gLayerInput, totalNeuronNum);
	log["X"].size = totalNeuronNum;
	log["X"].data = X;
}



fclose(v_file);
fclose(input_e_file);
fclose(input_i_file);
fclose(ie_file);
fclose(ii_file);
fclose(info_file);
fclose(log_file);

free_buffers(buffers);
freeGPUNetwork(c_pGPUNet);
freeGNetwork(pCPUNet);

return 0;
}


int SingleGPUSimulator::runMultiNets(real time, int parts, FireInfo &log) {
	int sim_cycle = round(time/dt);
	reset();

	checkCudaErrors(hipSetDevice(0));

	MultiNetwork multiNet(network, parts);
	DistriNetwork *subnets = multiNet.buildNetworks();
	assert(subnets != NULL);
	CrossNodeDataGPU *crossData = multiNet.arrangeCrossNodeDataGPU(parts);
	assert(crossData != NULL);

	GNetwork ** networks = (GNetwork **)malloc(sizeof(GNetwork *) * parts);
	GBuffers **buffers = (GBuffers **)malloc(sizeof(GBuffers *) * parts);
	BlockSize **updateSizes = (BlockSize **)malloc(sizeof(GBuffers *) * parts);

	for (int i=0; i<parts; i++) {
		subnets[i]._sim_cycle = sim_cycle;
		subnets[i]._node_idx = i;
		subnets[i]._node_num = parts;
		subnets[i]._dt = dt;

		GNetwork *pCPUNet = subnets[i]._network;
		networks[i] = copyNetworkToGPU(pCPUNet);
		GNetwork *c_pGPUNet = networks[i];

		int nTypeNum = c_pGPUNet->nTypeNum;
		int sTypeNum = c_pGPUNet->sTypeNum;
		int nodeNeuronNum = c_pGPUNet->neuronNums[nTypeNum];
		int allNeuronNum = pCPUNet->pN2SConnection->n_num;
		int nodeSynapseNum = c_pGPUNet->synapseNums[sTypeNum];

		buffers[i] = alloc_buffers(allNeuronNum, nodeSynapseNum, c_pGPUNet->MAX_DELAY, dt);
		updateSizes[i] = getBlockSize(allNeuronNum, nodeSynapseNum);

		printf("Subnet %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", subnets[i]._node_idx, nTypeNum, sTypeNum);
		printf("Subnet %d NeuronNum: %d, SynapseNum: %d\n", subnets[i]._node_idx, nodeNeuronNum, nodeSynapseNum);
	}

	for (int time=0; time<sim_cycle; time++) {

		for (int p=0; p<parts; p++) {
			update_time<<<1, 1>>>(time, buffers[p]->c_gFiredTableSizes);

			GNetwork *c_pGPUNet = networks[p];
			for (int i=0; i<c_pGPUNet->nTypeNum; i++) {
				assert(c_pGPUNet->neuronNums[i+1]-c_pGPUNet->neuronNums[i] > 0);
				cudaUpdateNeuron[c_pGPUNet->nTypes[i]](c_pGPUNet->pNeurons[i], buffers[p]->c_gNeuronInput, buffers[p]->c_gNeuronInput_I, buffers[p]->c_gFiredTable, buffers[p]->c_gFiredTableSizes, c_pGPUNet->neuronNums[i+1]-c_pGPUNet->neuronNums[i], c_pGPUNet->neuronNums[i], time, &updateSizes[p][c_pGPUNet->nTypes[i]]);
			}
		}

		for (int p=0; p<parts; p++) {
			GNetwork *c_pGPUNet = networks[p];
			for (int i=0; i<c_pGPUNet->sTypeNum; i++) {
				assert(c_pGPUNet->synapseNums[i+1]-c_pGPUNet->synapseNums[i] > 0);
				cudaUpdateSynapse[c_pGPUNet->sTypes[i]](c_pGPUNet->pN2SConnection, c_pGPUNet->pSynapses[i], buffers[p]->c_gNeuronInput, buffers[p]->c_gNeuronInput_I, buffers[p]->c_gFiredTable, buffers[p]->c_gFiredTableSizes, c_pGPUNet->synapseNums[i+1]-c_pGPUNet->synapseNums[i], c_pGPUNet->synapseNums[i], time, &updateSizes[p][c_pGPUNet->sTypes[i]]);
			}
		}
	}

	for (int i=0; i<parts; i++) {
		freeGPUNetwork(networks[i]);
		free_buffers(buffers[i]);
	}

	free(networks);
	free(buffers);
	free(updateSizes);

	return 0;
}

