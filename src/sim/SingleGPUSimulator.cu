#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/GBuffers.h"
#include "../gpu_utils/runtime.h"
#include "../net/MultiNetwork.h"

#include "SingleGPUSimulator.h"

using std::cout;
using std::endl;

CrossNodeDataGPU * gCrossDataNet;

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

int SingleGPUSimulator::run(real time, FireInfo &log)
{

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pCpuNet = network->buildNetwork();

	FILE *v_file = fopen("g_v.data", "w+");
	if (v_file == NULL) {
		printf("ERROR: Open file g_v.data failed\n");
		return -1;
	}

	FILE *input_e_file = fopen("g_input_e.data", "w+");
	if (input_e_file == NULL) {
		printf("Open file g_input_e.data failed\n");
		return -1;
	}

	FILE *input_i_file = fopen("g_input_i.data", "w+");
	if (input_i_file == NULL) {
		printf("Open file g_input_i.data failed\n");
		return -1;
	}

	FILE *ie_file = fopen("g_ie.data", "w+");
	if (ie_file == NULL) {
		printf("ERROR: Open file g_ie.data failed\n");
		return -1;
	}

	FILE *ii_file = fopen("g_ii.data", "w+");
	if (ii_file == NULL) {
		printf("ERROR: Open file g_ii.data failed\n");
		return -1;
	}

	FILE *info_file = fopen("GSim.info", "w+");
	if (info_file == NULL) {
		printf("ERROR: Open file GSim.info failed\n");
		return -1;
	}

	FILE *log_file = fopen("GSim.log", "w+");
	if (log_file == NULL) {
		printf("ERROR: Open file GSim.log failed\n");
		return -1;
	}

	//findCudaDevice(0, NULL);
	checkCudaErrors(hipSetDevice(0));
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	fprintf(info_file, "NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	fprintf(info_file, "NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	fprintf(info_file, "MAX_DELAY: %d\n", pCpuNet->MAX_DELAY);

	init_connection<<<1, 1>>>(c_pGpuNet->pN2SConnection);

	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, MAX_DELAY, dt);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	//BlockSize preSize = { 0, 0, 0};
	//hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_pre_synapse, 0, totalNeuronNum); 
	//preSize.gridSize = (totalSynapseNum + (preSize.blockSize) - 1) / (preSize.blockSize);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(totalNeuronNum);

	int tj_idx = getIndex(pCpuNet->nTypes, nTypeNum, TJ);
	int life_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIFE);
	int copy_idx = -1;
	real *c_g_vm = NULL;
	real *c_g_ie = NULL;
	real *c_g_ii = NULL;

	if (life_idx >= 0) {
		GLIFENeurons *c_g_lif = copyFromGPU<GLIFENeurons>(static_cast<GLIFENeurons*>(c_pGpuNet->pNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		c_g_ie = c_g_lif->p_i_E;
		c_g_ii = c_g_lif->p_i_I;
		copy_idx = life_idx;
	} else if (tj_idx >= 0) {
		GTJNeurons *c_g_tj = copyFromGPU<GTJNeurons>(static_cast<GTJNeurons*>(c_pGpuNet->pNeurons[tj_idx]), 1);
		c_g_vm = c_g_tj->p_vm;
		copy_idx = tj_idx;

	} else {
	}
#endif

	//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
	//int exp_idx = getIndex(pCpuNet->sTypes, sTypeNum, Exp);
	//GExpSynapses *c_g_exp = copyFromGPU<GExpSynapses>(static_cast<GExpSynapses*>(c_pGpuNet->pSynapses[exp_idx]), 1);
	//real *c_g_I_syn = c_g_exp->p_I_syn;

	//for (int i=0; i<nTypeNum; i++) {
	//	cout << pCpuNet->nTypes[i] << ": <<<" << updateSize[c_pGpuNet->nTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->nTypes[i]].blockSize << ">>>" << endl;
	//}
	//for (int i=0; i<sTypeNum; i++) {
	//	cout << pCpuNet->sTypes[i] << ": <<<" << updateSize[c_pGpuNet->sTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->sTypes[i]].blockSize << ">>>" << endl;
	//}

	vector<int> firedInfo;
	fprintf(info_file, "Start runing for %d cycles\n", sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		//printf("Cycle: %d ", time);
		//fflush(stdout);

#ifdef LOG_DATA
		if (copy_idx >= 0) {
			//copyFromGPU<real>(c_vm, buffers->c_gNeuronInput + c_pGpuNet->neuronNums[copy_idx], c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			copyFromGPU<real>(c_vm, buffers->c_gNeuronInput + c_pGpuNet->neuronNums[copy_idx], c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
				fprintf(input_e_file, "%.10lf \t", c_vm[i]);
			}
			//copyFromGPU<real>(c_vm, buffers->c_gNeuronInput_I + c_pGpuNet->neuronNums[copy_idx], c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			copyFromGPU<real>(c_vm, buffers->c_gNeuronInput_I + c_pGpuNet->neuronNums[copy_idx], c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
				fprintf(input_i_file, "%.10lf \t", c_vm[i]);
			}
		}
#endif

		update_time<<<1, 1>>>(time);

		for (int i=0; i<nTypeNum; i++) {
			cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i],time, &updateSize[c_pGpuNet->nTypes[i]]);
		}

		//update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(c_pGpuNet->pN2SConnection);

		for (int i=0; i<sTypeNum; i++) {
			cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], time, &updateSize[pCpuNet->sTypes[i]]);
		}


#ifdef LOG_DATA
		//LOG DATA
		int currentIdx = time%(MAX_DELAY+1);

		//printf("HERE %p %d ", buffers->c_gFiredTableSizes, currentIdx);
		//fflush(stdout);
		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		//printf("HERE1\n");
		//fflush(stdout);
		copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);

		//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGpuNet->synapseNums[exp_idx+1]-c_pGpuNet->synapseNums[exp_idx]);

		//fprintf(dataFile, "Cycle %d: ", time);
		if (copy_idx >= 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
				fprintf(v_file, "%.10lf \t", c_vm[i]);
			}


			if (life_idx >= 0) {
				copyFromGPU<real>(c_vm, c_g_ie, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
				for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
					fprintf(ie_file, "%.10lf \t", c_vm[i]);
				}
				copyFromGPU<real>(c_vm, c_g_ii, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
				for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
					fprintf(ii_file, "%.10lf \t", c_vm[i]);
				}
			}
		}
		//for (int i=0; i<c_pGpuNet->synapseNums[1] - c_pGpuNet->synapseNums[0]; i++) {
		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
		//}
		fprintf(v_file, "\n");
		fprintf(input_e_file, "\n");
		fprintf(input_i_file, "\n");
		fprintf(ie_file, "\n");
		fprintf(ii_file, "\n");

		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		//LOG SYNAPSE
		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);
		//int synapseCount = 0;
		//if (time > 0) {
		//	for (int i=0; i<totalSynapseNum; i++) {
		//		if (buffers->c_synapsesFired[i] == time) {
		//			if (synapseCount ==  0) {
		//				if (copySize > 0) {
		//					fprintf(logFile, ", ");
		//				}
		//				fprintf(logFile, "%s", network->idx2sid[i].getInfo().c_str());
		//				synapseCount++;
		//			} else {
		//				fprintf(logFile, ", %s", network->idx2sid[i].getInfo().c_str());
		//			}
		//		}
		//	}
		//	fprintf(logFile, "\n");
		//}
#endif
	}
	hipDeviceSynchronize();

	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	fprintf(info_file, "Simulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	//CALC Firing Rate
	if (log.find("count") != log.end()) {
		int *rate = (int*)malloc(sizeof(int)*totalNeuronNum);
		copyFromGPU<int>(rate, buffers->c_gFireCount, totalNeuronNum);

		log["count"].size = totalNeuronNum;
		log["count"].data = rate;

		//FILE *rateFile = fopen("GFire.log", "w+");
		//if (rateFile == NULL) {
		//	printf("ERROR: Open file Sim.log failed\n");
		//	return -1;
		//}

		//for (int i=0; i<totalNeuronNum; i++) {
		//	fprintf(rateFile, "%d \t", rate[i]);
		//}

		//fflush(rateFile);
		//fclose(rateFile);
	}

	if (log.find("Y") != log.end()) {
		real *Y = (real*)malloc(sizeof(real)*totalNeuronNum);
		copyFromGPU<real>(Y, buffers->c_gXInput, totalNeuronNum);
		log["Y"].size = totalNeuronNum;
		log["Y"].data = Y;
	}

	if (log.find("X") != log.end()) {
		int *X = (int*)malloc(sizeof(int)*totalNeuronNum);
		copyFromGPU<int>(X, buffers->c_gLayerInput, totalNeuronNum);
		log["X"].size = totalNeuronNum;
		log["X"].data = X;
	}



	fclose(v_file);
	fclose(input_e_file);
	fclose(input_i_file);
	fclose(ie_file);
	fclose(ii_file);
	fclose(info_file);
	fclose(log_file);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);
	freeGNetwork(pCpuNet);

	return 0;
}


int SingleGPUSimulator::runMultiNets(real time, int parts, FireInfo &log) {
	// int sim_cycle = round(time/dt);
	// reset();

	// checkCudaErrors(hipSetDevice(0));

	// MultiNetwork multiNet(network, parts);
	// DistriNetwork *node_nets = multiNet.buildNetworks();
	// assert(node_nets != NULL);
	// CrossNodeDataGPU *CrossData = multiNet.arrangeCrossNodeDataGPU(parts);
	// assert(gCrossDataNet != NULL);

	// GNetwork ** networks = (GNetwork **)malloc(sizeof(GNetwork *) * parts);

	// for (int i=0; i<parts; i++) {
	// 	node_nets[i]._sim_cycle = sim_cycle;
	// 	node_nets[i]._node_idx = i;
	// 	node_nets[i]._node_num = parts;
	// 	node_nets[i]._dt = dt;

	// 	DistriNetwork *network = &(node_nets[i]);
	// 	networks[i] = copyNetworkToGPU(network->_network);
	// 	int nTypeNum = network->_network->nTypeNum;
	// 	int sTypeNum = network->_network->sTypeNum;
	// 	printf("Subnet %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_node_idx, nTypeNum, sTypeNum);
	// 	printf("Subnet %d NeuronNum: %d, SynapseNum: %d\n", network->_node_idx, network->_network->neuronNums[nTypeNum], network->_network->synapseNums[sTypeNum]);
	// }

	// for (int time=0; time<sim_cycle; time++) {
	// 	update_time<<<1, 1>>>(time);

	// 	for (int p=0; p<parts; p++) {
	// 		GNetwork * c_pGpuNet = networks[p]; 
	// 		for (int i=0; i<c_pGpuNet->nTypeNum; i++) {
	// 			assert(c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i] > 0);
	// 			cudaUpdateType[c_pGpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], time, &updateSize[c_pGpuNet->nTypes[i]]);
	// 		}
	// 	}

	// 	hipMemset(c_g_fired_n_num, 0, sizeof(int)*network->_node_num);
	// 	cudaDeliverNeurons(c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_node_num, allNeuronNum, time);
	// 	checkCudaErrors(hipMemcpy(gCrossDataNet->_fired_num + network->_node_idx * network->_node_num, c_g_fired_n_num, sizeof(int)*network->_node_num, hipMemcpyDeviceToHost));

	// 	for (int i=0; i< network->_node_num; i++) {
	// 		int idx2i = network->_node_idx * network->_node_num + i;
	// 		assert(gCrossDataNet->_fired_num[idx2i] <= gCrossDataNet->_max_num[idx2i]);
	// 		if (gCrossDataNet->_fired_num[idx2i] > 0) {
	// 			checkCudaErrors(hipMemcpyPeer(gCrossDataNet->_fired_arrays[idx2i], i, c_g_global_cross_data + allNeuronNum * i, network->_node_idx, gCrossDataNet->_fired_num[idx2i] * sizeof(int)));
	// 		}
	// 	}

// #ifde// f LOG_DATA
// 	// 	int currentIdx = time%(MAX_DELAY+1);
// 
// 	// 	int copySize = 0;
// 	// 	copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
// 	// 	if (copySize > 0) {
// 	// 		copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (allNeuronNum*currentIdx), copySize);
// 	// 	}
// 
// 	// 	if (copy_idx >= 0 && (c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]) > 0) {
// 	// 		copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
// 	// 	}
// #endi// f

	// 	for (int i=0; i<sTypeNum; i++) {
	// 		assert(c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i] > 0);
	// 		cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], time, &updateSize[pCpuNet->sTypes[i]]);
	// 	}

	// 	for (int i=0; i< network->_node_num; i++) {
	// 		int i2idx = network->_node_idx + network->_node_num * i;
	// 		if (gCrossDataNet->_fired_num[i2idx] > 0) {
	// 			addCrossNeurons(gCrossDataNet->_fired_arrays[i2idx], gCrossDataNet->_fired_num[i2idx], time);
	// 		}
	// 	}
	// 	
// #ifde// f LOG_DATA
// 	// 	for (int i=0; i<copySize; i++) {
// 	// 		fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
// 	// 	}
// 	// 	fprintf(log_file, "\n");
// 
// 	// 	for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
// 	// 		fprintf(v_file, "%.10lf \t", c_vm[i]);
// 	// 	}
// 	// 	fprintf(v_file, "\n");
// #endi// f

	// }
	// gettimeofday(&te, NULL);
	// long seconds = te.tv_sec - ts.tv_sec;
	// long hours = seconds/3600;
	// seconds = seconds%3600;
	// long minutes = seconds/60;
	// seconds = seconds%60;
	// long uSeconds = te.tv_usec - ts.tv_usec;
	// if (uSeconds < 0) {
	// 	uSeconds += 1000000;
	// 	seconds = seconds - 1;
	// }

	// printf("Simulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	// int *rate = (int*)malloc(sizeof(int)*nodeNeuronNum);
	// copyFromGPU<int>(rate, buffers->c_gFireCount, nodeNeuronNum);

	// char fire_filename[512];
	// sprintf(fire_filename, "GFire_%d.log", network->_node_idx); 
	// FILE *rate_file = fopen(fire_filename, "w+");
	// if (rate_file == NULL) {
	// 	printf("Open file Sim.log failed\n");
	// 	return NULL;
	// }

	// for (int i=0; i<nodeNeuronNum; i++) {
	// 	fprintf(rate_file, "%d \t", rate[i]);
	// }

	// free(rate);
	// fclose(rate_file);

	// fclose(log_file);
	// fclose(v_file);

	// free_buffers(buffers);
	// freeGPUNetwork(c_pGpuNet);

	// return NULL;







	return 0;
}

