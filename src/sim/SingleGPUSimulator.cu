#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_kernel.h"
#include "SingleGPUSimulator.h"

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

int SingleGPUSimulator::run(real time)
{
	findCudaDevice(0, NULL);

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pCpuNet = network->buildNetwork();

	FILE *logFile = fopen("GSim.log", "w+");
	if (logFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}
	FILE *dataFile = fopen("GSim.data", "w+");
	if (dataFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}

	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	printf("MAX_DELAY: %lf %lf %d\n", network->maxDelay, dt, pCpuNet->MAX_DELAY);

	// Arrays
	//int *c_gTimeTable;
	// Neuron Arrays
	real *c_gNeuronInput;
	// Neuron Tables
	int *c_gFiredTable;
	int *c_gFiredTableSizes;
	int *c_gActiveTable;
	// Synapse Tables
	int *c_gSynapsesActiveTable;
	int *c_gSynapsesLogTable;

	//checkCudaErrors(hipMalloc((void**)&c_gTimeTable, sizeof(int)*(MAX_DELAY+1)));
	//checkCudaErrors(hipMemset(c_gTimeTable, 0, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMalloc((void**)&c_gNeuronInput, sizeof(real)*(totalNeuronNum)));
	checkCudaErrors(hipMemset(c_gNeuronInput, 0, sizeof(real)*(totalNeuronNum)));

	checkCudaErrors(hipMalloc((void**)&c_gFiredTable, sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1))));
	checkCudaErrors(hipMemset(c_gFiredTable, 0, sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1))));

	checkCudaErrors(hipMalloc((void**)&c_gFiredTableSizes, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMemset(c_gFiredTableSizes, 0, sizeof(int)*(MAX_DELAY+1)));

	checkCudaErrors(hipMalloc((void**)&c_gActiveTable, sizeof(int)*(totalNeuronNum)));
	checkCudaErrors(hipMemset(c_gActiveTable, 0, sizeof(int)*(totalNeuronNum)));

	checkCudaErrors(hipMalloc((void**)&c_gSynapsesActiveTable, sizeof(int)*(totalSynapseNum)));
	checkCudaErrors(hipMemset(c_gSynapsesActiveTable, 0, sizeof(int)*(totalSynapseNum)));
	
	checkCudaErrors(hipMalloc((void**)&c_gSynapsesLogTable, sizeof(int)*(totalSynapseNum)));
	checkCudaErrors(hipMemset(c_gSynapsesLogTable, 0, sizeof(int)*(totalSynapseNum)));

	int timeTableCap = MAX_DELAY+1;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&MAX_DELAY), &MAX_DELAY, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&gTimeTableCap), &timeTableCap, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&gFiredTableCap), &totalNeuronNum, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&gSynapsesTableCap), &totalSynapseNum, sizeof(int)));

	init_buffers<<<1, 1, 0>>>(/*c_gTimeTable,*/ c_gNeuronInput, c_gFiredTable, c_gFiredTableSizes, c_gActiveTable, c_gSynapsesActiveTable, c_gSynapsesLogTable);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	BlockSize preSize = { 0, 0, 0};
	BlockSize postSize = { 0, 0, 0};
	hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_lif_neuron, 0, totalNeuronNum); 
	preSize.gridSize = (totalNeuronNum + (preSize.blockSize) - 1) / (preSize.blockSize);
	hipOccupancyMaxPotentialBlockSize(&(postSize.minGridSize), &(postSize.blockSize), update_lif_neuron, 0, totalSynapseNum); 
	postSize.gridSize = (totalSynapseNum + (postSize.blockSize) - 1) / (postSize.blockSize);

	int * c_n_fired = (int*)malloc(sizeof(int)*((totalNeuronNum)));
	int * c_s_fired = (int*)malloc(sizeof(int)*((totalSynapseNum)));
	//real * c_n_input = (real*)malloc(sizeof(real)*((totalNeuronNum)));

	vector<int> firedInfo;
	printf("Start runing for %d cycles\n", sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		printf("\rCycle: %d", time);
		fflush(stdout);

		//checkCudaErrors(hipMemcpy(c_n_input, c_gNeuronInput, sizeof(real)*(totalNeuronNum), hipMemcpyDeviceToHost));
		for (int i=0; i<nTypeNum; i++) {
			updateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
		}

		update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(c_pGpuNet->pN2SConnection);

		for (int i=0; i<sTypeNum; i++) {
			updateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
		}

		update_time<<<1, 1>>>();

		int currentIdx = time%(MAX_DELAY+1);
		checkCudaErrors(hipMemcpy(c_n_fired, c_gFiredTable + totalNeuronNum*currentIdx, sizeof(int)*(totalNeuronNum), hipMemcpyDeviceToHost));

		fprintf(logFile, "Cycle %d: ", time);
		firedInfo.clear();
		for (int i=0; i<pCpuNet->neuronNums[nTypeNum]; i++) {
			if (c_n_fired[i] > 0) {
				firedInfo.push_back(i);
			}
		}
		int size = firedInfo.size();
		if (size > 0) {
			fprintf(logFile, "%d_%d", network->idx2nid[firedInfo[0]].groupId, network->idx2nid[firedInfo[0]].id);
			for (int i=1; i<size; i++) {
				fprintf(logFile, ", %d_%d", network->idx2nid[firedInfo[i]].groupId, network->idx2nid[firedInfo[i]].id);
			}
		}
		firedInfo.clear();
		for (int i=0; i<pCpuNet->synapseNums[sTypeNum]; i++) {
			if (c_s_fired[i]) {
				firedInfo.push_back(i);
			}
		}
		int size2 = firedInfo.size();
		if (size2 > 0) {
			if (size > 0) {
				fprintf(logFile, ", ");
			}
			fprintf(logFile, "%d_%d", network->idx2sid[firedInfo[0]].groupId, network->idx2sid[firedInfo[0]].id);
			for (int i=1; i<size2; i++) {
				fprintf(logFile, ", %d_%d", network->idx2sid[firedInfo[i]].groupId, network->idx2sid[firedInfo[i]].id);
			}
		}
		fprintf(logFile, "\n");
	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("\nSimulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	fclose(logFile);
	fclose(dataFile);

	//checkCudaErrors(hipFree(c_gTimeTable));
	checkCudaErrors(hipFree(c_gNeuronInput));
	checkCudaErrors(hipFree(c_gFiredTable));
	checkCudaErrors(hipFree(c_gActiveTable));
	checkCudaErrors(hipFree(c_gSynapsesActiveTable));
	checkCudaErrors(hipFree(c_gSynapsesLogTable));

	freeGPUNetwork(c_pGpuNet);

	return 0;
}

