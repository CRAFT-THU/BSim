#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
// #include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/runtime.h"
#include "../gpu_utils/GBuffers.h"
#include "../net/MultiNetwork.h"

#include "MultiGPUSimulator.h"

using std::cout;
using std::endl;

pthread_barrier_t gpuCycleBarrier;

CrossNodeDataGPU * gCrossDataGPU;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread_gpu(void *para);

int MultiGPUSimulator::run(real time, FireInfo &log)
{
	int sim_cycle = round(time/dt);
	reset();

	int device_count = 4;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);
	for (int i=0; i<device_count; i++) {
		for (int j=0; j<device_count; j++) {
			if (i!=j) {
				int access = 0;
				checkCudaErrors(hipDeviceCanAccessPeer(&access, i, j));
				if (access == 1) {
					checkCudaErrors(hipSetDevice(i));
					checkCudaErrors(hipDeviceEnablePeerAccess(j, 0));
				}
			}
		}
	}
	checkCudaErrors(hipSetDevice(0));

	pthread_barrier_init(&gpuCycleBarrier, NULL, device_count);

	MultiNetwork multiNet(network, device_count);
	DistriNetwork *node_nets = multiNet.buildNetworks();
	assert(node_nets != NULL);
	gCrossDataGPU = multiNet.arrangeCrossNodeDataGPU(device_count);
	assert(gCrossDataGPU != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);


	for (int i=0; i<device_count; i++) {
		node_nets[i]._sim_cycle = sim_cycle;
		node_nets[i]._node_idx = i;
		node_nets[i]._node_num = device_count;
		node_nets[i]._dt = dt;


		int ret = pthread_create(&(thread_ids[i]), NULL, &run_thread_gpu, (void*)&(node_nets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<device_count; i++) {
		pthread_join(thread_ids[i], NULL);
	}

	pthread_barrier_destroy(&gpuCycleBarrier);

	return 0;
}

void * run_thread_gpu(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char log_filename[512];
	sprintf(log_filename, "GSim_%d.log", network->_node_idx); 
	FILE *log_file = fopen(log_filename, "w+");
	assert(log_file != NULL);

	char v_filename[512];
	sprintf(v_filename, "g_v_%d.data", network->_node_idx); 
	FILE *v_file = fopen(v_filename, "w+");
	assert(v_file != NULL);

	checkCudaErrors(hipSetDevice(network->_node_idx));

	GNetwork *pNetCPU = network->_network;
	GNetwork *c_pNetGPU = copyNetworkToGPU(pNetCPU);

	int nTypeNum = c_pNetGPU->nTypeNum;
	int sTypeNum = c_pNetGPU->sTypeNum;
	int nodeNeuronNum = c_pNetGPU->pNeuronNums[nTypeNum];
	int allNeuronNum = pNetCPU->pConnection->nNum;
	int nodeSynapseNum = c_pNetGPU->pSynapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_node_idx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_node_idx, nodeNeuronNum, nodeSynapseNum);

	//int dataOffset = network->_node_idx * network->_node_num;
	//int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;

	int deltaDelay = pNetCPU->pConnection->maxDelay - pNetCPU->pConnection->minDelay;
	printf("Thread %d MaxDelay: %d MinDelay: %d\n", network->_node_idx, pNetCPU->pConnection->maxDelay, pNetCPU->pConnection->minDelay);

	// init_connection<<<1, 1>>>(c_pNetGPU->pConnection);

	GBuffers *buffers = alloc_buffers(allNeuronNum, nodeSynapseNum, pNetCPU->pConnection->maxDelay, network->_dt);

	BlockSize *updateSize = getBlockSize(allNeuronNum, nodeSynapseNum);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(nodeNeuronNum);
	int life_idx = getIndex(c_pNetGPU->pNTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		GLIFNeurons *c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pNetGPU->ppNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->pV_m;
		copy_idx = life_idx;
	} else {
	}
#endif

	for (int i=0; i<nTypeNum; i++) {
		cout << "Thread " << network->_node_idx << " " << c_pNetGPU->pNTypes[i] << ": <<<" << updateSize[c_pNetGPU->pNTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pNTypes[i]].blockSize << ">>>" << endl;
	}
	for (int i=0; i<sTypeNum; i++) {
		cout << "Thread " << network->_node_idx << " " << c_pNetGPU->pSTypes[i] << ": <<<" << updateSize[c_pNetGPU->pSTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pSTypes[i]].blockSize << ">>>" << endl;
	}

	//int * c_g_cross_id = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 

	int * c_g_idx2index = copyToGPU<int>(network->_crossnode_map->_idx2index, allNeuronNum);
	int * c_g_cross_index2idx = copyToGPU<int>(network->_crossnode_map->_crossnode_index2idx, network->_crossnode_map->_cross_size);
	int * c_g_global_cross_data = gpuMalloc<int>(allNeuronNum * network->_node_num);
	int * c_g_fired_n_num = gpuMalloc<int>(network->_node_num);

	vector<int> firedInfo;
	struct timeval ts, te;
	//struct timeval t0, t1, t2, t3, t4, t5,/* t6,*/ t7, t8, t9;
	//double barrier1_time = 0, gpu_cpy_time = 0, peer_cpy_time = 0, barrier2_time=0, copy_time = 0;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_sim_cycle; time++) {
		update_time<<<1, 1>>>(c_pNetGPU->pConnection, time, buffers->c_gFiredTableSizes);

		for (int i=0; i<nTypeNum; i++) {
			assert(c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pNTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppNeurons[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i], c_pNetGPU->pNeuronNums[i], time, &updateSize[c_pNetGPU->pNTypes[i]]);
		}

		//gettimeofday(&t0, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t1, NULL);
		//barrier1_time += (t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec)/1000000.0;
		hipMemset(c_g_fired_n_num, 0, sizeof(int)*network->_node_num);
		//hipDeviceSynchronize();
		//gettimeofday(&t2, NULL);
		//cudaDeliverNeurons(c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_node_num, allNeuronNum);
		//for (int i=0; i<network->_node_num; i++) {
		//	int offset = i * network->_node_num + network->_node_idx; 
		//	copyFromGPU<int>(&(global_cross_data[offset]._fired_n_num), c_g_fired_n_num + i, 1);
		//	if (global_cross_data[offset]._fired_n_num > 0) {
		//		copyFromGPU<int>(global_cross_data[offset]._fired_n_idxs, c_g_global_cross_data + allNeuronNum * i, global_cross_data[offset]._fired_n_num);
		//	}
		//}

		cudaDeliverNeurons<<<(allNeuronNum+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGPU->pConnection, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_node_num, time);

		checkCudaErrors(hipMemcpy(gCrossDataGPU->_fired_num + network->_node_idx * network->_node_num, c_g_fired_n_num, sizeof(int)*network->_node_num, hipMemcpyDeviceToHost));
		//gettimeofday(&t3, NULL);

		for (int i=0; i< network->_node_num; i++) {
			int idx2i = network->_node_idx * network->_node_num + i;
			assert(gCrossDataGPU->_fired_num[idx2i] <= gCrossDataGPU->_max_num[idx2i]);
			if (gCrossDataGPU->_fired_num[idx2i] > 0) {
				checkCudaErrors(hipMemcpyPeer(gCrossDataGPU->_fired_arrays[idx2i], i, c_g_global_cross_data + allNeuronNum * i, network->_node_idx, gCrossDataGPU->_fired_num[idx2i] * sizeof(int)));
			}
		}
		//gettimeofday(&t7, NULL);

		//gpu_cpy_time += (t3.tv_sec - t2.tv_sec) + (t3.tv_usec - t2.tv_usec)/1000000.0;
		//peer_cpy_time += (t7.tv_sec - t3.tv_sec) + (t7.tv_usec - t3.tv_usec)/1000000.0;

#ifdef LOG_DATA
		int currentIdx = time%(deltaDelay+1);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		if (copySize > 0) {
			copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (allNeuronNum*currentIdx), copySize);
		}

		if (copy_idx >= 0 && (c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]) > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		}
#endif

		for (int i=0; i<sTypeNum; i++) {
			assert(c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pSTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppSynapses[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i], c_pNetGPU->pSynapseNums[i], time, &updateSize[c_pNetGPU->pSTypes[i]]);
		}
		//hipDeviceSynchronize();

		//gettimeofday(&t4, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t5, NULL);
		//barrier2_time += (t5.tv_sec - t4.tv_sec) + (t5.tv_usec - t4.tv_usec)/1000000.0;

		//gettimeofday(&t6, NULL);
		//collectNeurons();
		//gettimeofday(&t7, NULL);
		//cpu_cpy_time += (t7.tv_sec - t6.tv_sec) + (t7.tv_usec - t6.tv_usec)/1000000.0;
		
		//gettimeofday(&t8, NULL);
		//if (global_cross_data[dataIdx]._fired_n_num > 0) {
		//	copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
		//	addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
		//}
		for (int i=0; i< network->_node_num; i++) {
			int i2idx = network->_node_idx + network->_node_num * i;
			if (gCrossDataGPU->_fired_num[i2idx] > 0) {
				int num = gCrossDataGPU->_fired_num[i2idx];
				cudaAddCrossNeurons<<<(num+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGPU->pConnection, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, gCrossDataGPU->_fired_arrays[i2idx], gCrossDataGPU->_fired_num[i2idx], time);
			}
		}
		
		//gettimeofday(&t9, NULL);
		//copy_time += (t9.tv_sec - t8.tv_sec) + (t9.tv_usec - t8.tv_usec)/1000000.0;

#ifdef LOG_DATA
		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");
#endif

		//pthread_barrier_wait(&gpuCycleBarrier);
	}
	pthread_barrier_wait(&gpuCycleBarrier);
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_node_idx, hours, minutes, seconds, uSeconds);
	//printf("Thread %d cost : barrier1 %lf, DtoH %lf, DtoD %lf, barrier2 %lf, HtoD %lf\n", network->_node_idx, barrier1_time, gpu_cpy_time, peer_cpy_time, barrier2_time, copy_time);

	int *rate = (int*)malloc(sizeof(int)*nodeNeuronNum);
	copyFromGPU<int>(rate, buffers->c_gFireCount, nodeNeuronNum);

	char fire_filename[512];
	sprintf(fire_filename, "GFire_%d.log", network->_node_idx); 
	FILE *rate_file = fopen(fire_filename, "w+");
	if (rate_file == NULL) {
		printf("Open file Sim.log failed\n");
		return NULL;
	}

	for (int i=0; i<nodeNeuronNum; i++) {
		fprintf(rate_file, "%d \t", rate[i]);
	}

	free(rate);
	fclose(rate_file);

	fclose(log_file);
	fclose(v_file);

	free_buffers(buffers);
	freeNetworkGPU(c_pNetGPU);

	return NULL;
}

