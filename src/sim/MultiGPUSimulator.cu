#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
// #include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/runtime.h"
#include "../gpu_utils/GBuffers.h"
#include "../net/MultiNetwork.h"

#include "MultiGPUSimulator.h"

using std::cout;
using std::endl;

pthread_barrier_t gpuCycleBarrier;

CrossNodeDataGPU * gCrossDataGPU;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread_gpu(void *para);

int MultiGPUSimulator::run(real time, FireInfo &log)
{
	int sim_cycle = round(time/dt);
	reset();

	int device_count = 4;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);
	for (int i=0; i<device_count; i++) {
		for (int j=0; j<device_count; j++) {
			if (i!=j) {
				int access = 0;
				checkCudaErrors(hipDeviceCanAccessPeer(&access, i, j));
				if (access == 1) {
					checkCudaErrors(hipSetDevice(i));
					checkCudaErrors(hipDeviceEnablePeerAccess(j, 0));
				}
			}
		}
	}
	checkCudaErrors(hipSetDevice(0));

	pthread_barrier_init(&gpuCycleBarrier, NULL, device_count);

	MultiNetwork multiNet(network, device_count);
	DistriNetwork *node_nets = multiNet.buildNetworks();
	assert(node_nets != NULL);
	gCrossDataGPU = multiNet.arrangeCrossNodeDataGPU(device_count);
	assert(gCrossDataGPU != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);


	for (int i=0; i<device_count; i++) {
		node_nets[i]._sim_cycle = sim_cycle;
		node_nets[i]._node_idx = i;
		node_nets[i]._node_num = device_count;
		node_nets[i]._dt = dt;


		int ret = pthread_create(&(thread_ids[i]), NULL, &run_thread_gpu, (void*)&(node_nets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<device_count; i++) {
		pthread_join(thread_ids[i], NULL);
	}

	pthread_barrier_destroy(&gpuCycleBarrier);

	return 0;
}

void * run_thread_gpu(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char log_filename[512];
	sprintf(log_filename, "GSim_%d.log", network->_node_idx); 
	FILE *log_file = fopen(log_filename, "w+");
	assert(log_file != NULL);

	char v_filename[512];
	sprintf(v_filename, "g_v_%d.data", network->_node_idx); 
	FILE *v_file = fopen(v_filename, "w+");
	assert(v_file != NULL);

	checkCudaErrors(hipSetDevice(network->_node_idx));

	GNetwork *pCpuNet = network->_network;
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = c_pGpuNet->nTypeNum;
	int sTypeNum = c_pGpuNet->sTypeNum;
	int nodeNeuronNum = c_pGpuNet->neuronNums[nTypeNum];
	int allNeuronNum = pCpuNet->pN2SConnection->n_num;
	int nodeSynapseNum = c_pGpuNet->synapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_node_idx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_node_idx, nodeNeuronNum, nodeSynapseNum);

	//int dataOffset = network->_node_idx * network->_node_num;
	//int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;

	int MAX_DELAY = c_pGpuNet->MAX_DELAY;
	printf("Thread %d MAX_DELAY: %d\n", network->_node_idx, c_pGpuNet->MAX_DELAY);

	// init_connection<<<1, 1>>>(c_pGpuNet->pN2SConnection);

	GBuffers *buffers = alloc_buffers(allNeuronNum, nodeSynapseNum, MAX_DELAY, network->_dt);

	BlockSize *updateSize = getBlockSize(allNeuronNum, nodeSynapseNum);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(nodeNeuronNum);
	int life_idx = getIndex(c_pGpuNet->nTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		GLIFNeurons *c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGpuNet->pNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		copy_idx = life_idx;
	} else {
	}
#endif

	for (int i=0; i<nTypeNum; i++) {
		cout << "Thread " << network->_node_idx << " " << c_pGpuNet->nTypes[i] << ": <<<" << updateSize[c_pGpuNet->nTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->nTypes[i]].blockSize << ">>>" << endl;
	}
	for (int i=0; i<sTypeNum; i++) {
		cout << "Thread " << network->_node_idx << " " << c_pGpuNet->sTypes[i] << ": <<<" << updateSize[c_pGpuNet->sTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->sTypes[i]].blockSize << ">>>" << endl;
	}

	//int * c_g_cross_id = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 

	int * c_g_idx2index = copyToGPU<int>(network->_crossnode_map->_idx2index, allNeuronNum);
	int * c_g_cross_index2idx = copyToGPU<int>(network->_crossnode_map->_crossnode_index2idx, network->_crossnode_map->_cross_size);
	int * c_g_global_cross_data = gpuMalloc<int>(allNeuronNum * network->_node_num);
	int * c_g_fired_n_num = gpuMalloc<int>(network->_node_num);

	vector<int> firedInfo;
	struct timeval ts, te;
	//struct timeval t0, t1, t2, t3, t4, t5,/* t6,*/ t7, t8, t9;
	//double barrier1_time = 0, gpu_cpy_time = 0, peer_cpy_time = 0, barrier2_time=0, copy_time = 0;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_sim_cycle; time++) {
		update_time<<<1, 1>>>(time, buffers->c_gFiredTableSizes);

		for (int i=0; i<nTypeNum; i++) {
			assert(c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i] > 0);
			cudaUpdateNeuron[c_pGpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], time, &updateSize[c_pGpuNet->nTypes[i]]);
		}

		//gettimeofday(&t0, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t1, NULL);
		//barrier1_time += (t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec)/1000000.0;
		hipMemset(c_g_fired_n_num, 0, sizeof(int)*network->_node_num);
		//hipDeviceSynchronize();
		//gettimeofday(&t2, NULL);
		//cudaDeliverNeurons(c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_node_num, allNeuronNum);
		//for (int i=0; i<network->_node_num; i++) {
		//	int offset = i * network->_node_num + network->_node_idx; 
		//	copyFromGPU<int>(&(global_cross_data[offset]._fired_n_num), c_g_fired_n_num + i, 1);
		//	if (global_cross_data[offset]._fired_n_num > 0) {
		//		copyFromGPU<int>(global_cross_data[offset]._fired_n_idxs, c_g_global_cross_data + allNeuronNum * i, global_cross_data[offset]._fired_n_num);
		//	}
		//}

		cudaDeliverNeurons<<<(allNeuronNum+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_node_num, time);

		checkCudaErrors(hipMemcpy(gCrossDataGPU->_fired_num + network->_node_idx * network->_node_num, c_g_fired_n_num, sizeof(int)*network->_node_num, hipMemcpyDeviceToHost));
		//gettimeofday(&t3, NULL);

		for (int i=0; i< network->_node_num; i++) {
			int idx2i = network->_node_idx * network->_node_num + i;
			assert(gCrossDataGPU->_fired_num[idx2i] <= gCrossDataGPU->_max_num[idx2i]);
			if (gCrossDataGPU->_fired_num[idx2i] > 0) {
				checkCudaErrors(hipMemcpyPeer(gCrossDataGPU->_fired_arrays[idx2i], i, c_g_global_cross_data + allNeuronNum * i, network->_node_idx, gCrossDataGPU->_fired_num[idx2i] * sizeof(int)));
			}
		}
		//gettimeofday(&t7, NULL);

		//gpu_cpy_time += (t3.tv_sec - t2.tv_sec) + (t3.tv_usec - t2.tv_usec)/1000000.0;
		//peer_cpy_time += (t7.tv_sec - t3.tv_sec) + (t7.tv_usec - t3.tv_usec)/1000000.0;

#ifdef LOG_DATA
		int currentIdx = time%(MAX_DELAY+1);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		if (copySize > 0) {
			copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (allNeuronNum*currentIdx), copySize);
		}

		if (copy_idx >= 0 && (c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]) > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
		}
#endif

		for (int i=0; i<sTypeNum; i++) {
			assert(c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i] > 0);
			cudaUpdateSynapse[c_pGpuNet->sTypes[i]](c_pGpuNet->pN2SConnection, c_pGpuNet->pSynapses[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], time, &updateSize[c_pGpuNet->sTypes[i]]);
		}
		//hipDeviceSynchronize();

		//gettimeofday(&t4, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t5, NULL);
		//barrier2_time += (t5.tv_sec - t4.tv_sec) + (t5.tv_usec - t4.tv_usec)/1000000.0;

		//gettimeofday(&t6, NULL);
		//collectNeurons();
		//gettimeofday(&t7, NULL);
		//cpu_cpy_time += (t7.tv_sec - t6.tv_sec) + (t7.tv_usec - t6.tv_usec)/1000000.0;
		
		//gettimeofday(&t8, NULL);
		//if (global_cross_data[dataIdx]._fired_n_num > 0) {
		//	copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
		//	addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
		//}
		for (int i=0; i< network->_node_num; i++) {
			int i2idx = network->_node_idx + network->_node_num * i;
			if (gCrossDataGPU->_fired_num[i2idx] > 0) {
				int num = gCrossDataGPU->_fired_num[i2idx];
				cudaAddCrossNeurons<<<(num+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(buffers->c_gFiredTable, buffers->c_gFiredTableSizes, gCrossDataGPU->_fired_arrays[i2idx], gCrossDataGPU->_fired_num[i2idx], time);
			}
		}
		
		//gettimeofday(&t9, NULL);
		//copy_time += (t9.tv_sec - t8.tv_sec) + (t9.tv_usec - t8.tv_usec)/1000000.0;

#ifdef LOG_DATA
		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");
#endif

		//pthread_barrier_wait(&gpuCycleBarrier);
	}
	pthread_barrier_wait(&gpuCycleBarrier);
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_node_idx, hours, minutes, seconds, uSeconds);
	//printf("Thread %d cost : barrier1 %lf, DtoH %lf, DtoD %lf, barrier2 %lf, HtoD %lf\n", network->_node_idx, barrier1_time, gpu_cpy_time, peer_cpy_time, barrier2_time, copy_time);

	int *rate = (int*)malloc(sizeof(int)*nodeNeuronNum);
	copyFromGPU<int>(rate, buffers->c_gFireCount, nodeNeuronNum);

	char fire_filename[512];
	sprintf(fire_filename, "GFire_%d.log", network->_node_idx); 
	FILE *rate_file = fopen(fire_filename, "w+");
	if (rate_file == NULL) {
		printf("Open file Sim.log failed\n");
		return NULL;
	}

	for (int i=0; i<nodeNeuronNum; i++) {
		fprintf(rate_file, "%d \t", rate[i]);
	}

	free(rate);
	fclose(rate_file);

	fclose(log_file);
	fclose(v_file);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);

	return NULL;
}

