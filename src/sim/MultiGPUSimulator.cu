#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/runtime.h"
#include "../gpu_utils/GBuffers.h"
#include "../net/Network.h"
#include "../neuron/lif/LIFData.h"

#include "MultiGPUSimulator.h"

using std::cout;
using std::endl;

pthread_barrier_t gpuCycleBarrier;

CrossNodeDataGPU * gCrossDataGPU;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : Simulator(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread_gpu(void *para);

int MultiGPUSimulator::run(real time, FireInfo &log)
{
	int sim_cycle = round(time/_dt);
	reset();

	int device_count = 4;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);
	for (int i=0; i<device_count; i++) {
		for (int j=0; j<device_count; j++) {
			if (i!=j) {
				int access = 0;
				checkCudaErrors(hipDeviceCanAccessPeer(&access, i, j));
				if (access == 1) {
					checkCudaErrors(hipSetDevice(i));
					checkCudaErrors(hipDeviceEnablePeerAccess(j, 0));
				}
			}
		}
	}
	checkCudaErrors(hipSetDevice(0));

	pthread_barrier_init(&gpuCycleBarrier, NULL, device_count);

	// MultiNetwork multiNet(_network, device_count);
	_network->setNodeNum(device_count);

	SimInfo info(_dt);
	DistriNetwork *node_nets = _network->buildNetworks(info);
	assert(node_nets != NULL);
	gCrossDataGPU = _network->arrangeCrossNodeDataGPU(device_count);
	assert(gCrossDataGPU != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);


	for (int i=0; i<device_count; i++) {
		node_nets[i]._simCycle = sim_cycle;
		node_nets[i]._nodeIdx = i;
		node_nets[i]._nodeNum = device_count;
		node_nets[i]._dt = _dt;


		int ret = pthread_create(&(thread_ids[i]), NULL, &run_thread_gpu, (void*)&(node_nets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<device_count; i++) {
		pthread_join(thread_ids[i], NULL);
	}

	pthread_barrier_destroy(&gpuCycleBarrier);

	return 0;
}

void * run_thread_gpu(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char log_filename[512];
	sprintf(log_filename, "sim_%d.log", network->_nodeIdx); 
	FILE *log_file = fopen(log_filename, "w+");
	assert(log_file != NULL);

	char v_filename[512];
	sprintf(v_filename, "v_%d.data", network->_nodeIdx); 
	FILE *v_file = fopen(v_filename, "w+");
	assert(v_file != NULL);

	checkCudaErrors(hipSetDevice(network->_nodeIdx));

	GNetwork *pNetCPU = network->_network;
	GNetwork *c_pNetGPU = copyNetworkToGPU(pNetCPU);

	int nTypeNum = c_pNetGPU->nTypeNum;
	int sTypeNum = c_pNetGPU->sTypeNum;
	int nodeNeuronNum = c_pNetGPU->pNeuronNums[nTypeNum];
	int allNeuronNum = pNetCPU->pConnection->nNum;
	int nodeSynapseNum = c_pNetGPU->pSynapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_nodeIdx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_nodeIdx, nodeNeuronNum, nodeSynapseNum);

	//int dataOffset = network->_nodeIdx * network->_nodeNum;
	//int dataIdx = network->_nodeIdx * network->_nodeNum + network->_nodeIdx;

	int maxDelay = pNetCPU->pConnection->maxDelay;
	int minDelay = pNetCPU->pConnection->minDelay;
	// int deltaDelay = maxDelay - minDelay;
	// int deltaDelay = pNetCPU->pConnection->maxDelay - pNetCPU->pConnection->minDelay;
	printf("Thread %d MaxDelay: %d MinDelay: %d\n", network->_nodeIdx, maxDelay,  minDelay);

	// init_connection<<<1, 1>>>(c_pNetGPU->pConnection);

	GBuffers *buffers = alloc_buffers(allNeuronNum, nodeSynapseNum, pNetCPU->pConnection->maxDelay, network->_dt);

	BlockSize *updateSize = getBlockSize(allNeuronNum, nodeSynapseNum);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(nodeNeuronNum);
	int life_idx = getIndex(c_pNetGPU->pNTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		LIFData *c_g_lif = copyFromGPU<LIFData>(static_cast<LIFData *>(c_pNetGPU->ppNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->pV_m;
		copy_idx = life_idx;
	} else {
	}
#endif

	for (int i=0; i<nTypeNum; i++) {
		cout << "Thread " << network->_nodeIdx << " " << c_pNetGPU->pNTypes[i] << ": <<<" << updateSize[c_pNetGPU->pNTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pNTypes[i]].blockSize << ">>>" << endl;
	}
	for (int i=0; i<sTypeNum; i++) {
		cout << "Thread " << network->_nodeIdx << " " << c_pNetGPU->pSTypes[i] << ": <<<" << updateSize[c_pNetGPU->pSTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pSTypes[i]].blockSize << ">>>" << endl;
	}

	//int * c_g_cross_id = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 

	int * c_g_idx2index = copyToGPU<int>(network->_crossnodeMap->_idx2index, allNeuronNum);
	int * c_g_cross_index2idx = copyToGPU<int>(network->_crossnodeMap->_crossnodeIndex2idx, network->_crossnodeMap->_crossSize);
	int * c_g_global_cross_data = gpuMalloc<int>(allNeuronNum * network->_nodeNum);
	int * c_g_fired_n_num = gpuMalloc<int>(network->_nodeNum);

	vector<int> firedInfo;
	struct timeval ts, te;
	//struct timeval t0, t1, t2, t3, t4, t5,/* t6,*/ t7, t8, t9;
	//double barrier1_time = 0, gpu_cpy_time = 0, peer_cpy_time = 0, barrier2_time=0, copy_time = 0;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_simCycle; time++) {
		update_time<<<1, 1>>>(c_pNetGPU->pConnection, time, buffers->c_gFiredTableSizes);

		for (int i=0; i<nTypeNum; i++) {
			assert(c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pNTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppNeurons[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i], c_pNetGPU->pNeuronNums[i], time, &updateSize[c_pNetGPU->pNTypes[i]]);
		}

		//gettimeofday(&t0, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t1, NULL);
		//barrier1_time += (t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec)/1000000.0;
		hipMemset(c_g_fired_n_num, 0, sizeof(int)*network->_nodeNum);
		//hipDeviceSynchronize();
		//gettimeofday(&t2, NULL);
		//cudaDeliverNeurons(c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_nodeNum, allNeuronNum);
		//for (int i=0; i<network->_nodeNum; i++) {
		//	int offset = i * network->_nodeNum + network->_nodeIdx; 
		//	copyFromGPU<int>(&(global_cross_data[offset]._fired_n_num), c_g_fired_n_num + i, 1);
		//	if (global_cross_data[offset]._fired_n_num > 0) {
		//		copyFromGPU<int>(global_cross_data[offset]._fired_n_idxs, c_g_global_cross_data + allNeuronNum * i, global_cross_data[offset]._fired_n_num);
		//	}
		//}

		cudaDeliverNeurons<<<(allNeuronNum+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGPU->pConnection, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_nodeNum, time);

		checkCudaErrors(hipMemcpy(gCrossDataGPU->_firedNum + network->_nodeIdx * network->_nodeNum, c_g_fired_n_num, sizeof(int)*network->_nodeNum, hipMemcpyDeviceToHost));
		//gettimeofday(&t3, NULL);

		for (int i=0; i< network->_nodeNum; i++) {
			int idx2i = network->_nodeIdx * network->_nodeNum + i;
			assert(gCrossDataGPU->_firedNum[idx2i] <= gCrossDataGPU->_maxNum[idx2i]);
			if (gCrossDataGPU->_firedNum[idx2i] > 0) {
				checkCudaErrors(hipMemcpyPeer(gCrossDataGPU->_firedArrays[idx2i], i, c_g_global_cross_data + allNeuronNum * i, network->_nodeIdx, gCrossDataGPU->_firedNum[idx2i] * sizeof(int)));
			}
		}
		//gettimeofday(&t7, NULL);

		//gpu_cpy_time += (t3.tv_sec - t2.tv_sec) + (t3.tv_usec - t2.tv_usec)/1000000.0;
		//peer_cpy_time += (t7.tv_sec - t3.tv_sec) + (t7.tv_usec - t3.tv_usec)/1000000.0;

#ifdef LOG_DATA
		int currentIdx = time%(maxDelay+1);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		if (copySize > 0) {
			copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (allNeuronNum*currentIdx), copySize);
		}

		if (copy_idx >= 0 && (c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]) > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		}
#endif

		for (int i=0; i<sTypeNum; i++) {
			assert(c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pSTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppSynapses[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i], c_pNetGPU->pSynapseNums[i], time, &updateSize[c_pNetGPU->pSTypes[i]]);
		}
		//hipDeviceSynchronize();

		//gettimeofday(&t4, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t5, NULL);
		//barrier2_time += (t5.tv_sec - t4.tv_sec) + (t5.tv_usec - t4.tv_usec)/1000000.0;

		//gettimeofday(&t6, NULL);
		//collectNeurons();
		//gettimeofday(&t7, NULL);
		//cpu_cpy_time += (t7.tv_sec - t6.tv_sec) + (t7.tv_usec - t6.tv_usec)/1000000.0;
		
		//gettimeofday(&t8, NULL);
		//if (global_cross_data[dataIdx]._fired_n_num > 0) {
		//	copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
		//	addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
		//}
		for (int i=0; i< network->_nodeNum; i++) {
			int i2idx = network->_nodeIdx + network->_nodeNum * i;
			if (gCrossDataGPU->_firedNum[i2idx] > 0) {
				int num = gCrossDataGPU->_firedNum[i2idx];
				cudaAddCrossNeurons<<<(num+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGPU->pConnection, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, gCrossDataGPU->_firedArrays[i2idx], gCrossDataGPU->_firedNum[i2idx], time);
			}
		}
		
		//gettimeofday(&t9, NULL);
		//copy_time += (t9.tv_sec - t8.tv_sec) + (t9.tv_usec - t8.tv_usec)/1000000.0;

#ifdef LOG_DATA
		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");
#endif

		//pthread_barrier_wait(&gpuCycleBarrier);
	}
	pthread_barrier_wait(&gpuCycleBarrier);
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_nodeIdx, hours, minutes, seconds, uSeconds);
	//printf("Thread %d cost : barrier1 %lf, DtoH %lf, DtoD %lf, barrier2 %lf, HtoD %lf\n", network->_nodeIdx, barrier1_time, gpu_cpy_time, peer_cpy_time, barrier2_time, copy_time);

	int *rate = (int*)malloc(sizeof(int)*nodeNeuronNum);
	copyFromGPU<int>(rate, buffers->c_gFireCount, nodeNeuronNum);

	char fire_filename[512];
	sprintf(fire_filename, "fire_%d.count", network->_nodeIdx); 
	FILE *rate_file = fopen(fire_filename, "w+");
	if (rate_file == NULL) {
		printf("Open file Sim.log failed\n");
		return NULL;
	}

	for (int i=0; i<nodeNeuronNum; i++) {
		fprintf(rate_file, "%d \t", rate[i]);
	}

	free(rate);
	fclose(rate_file);

	fclose(log_file);
	fclose(v_file);

	free_buffers(buffers);
	freeNetworkGPU(c_pNetGPU);

	return NULL;
}

