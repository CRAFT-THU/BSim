#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/runtime.h"
#include "../gpu_utils/GBuffers.h"
#include "../net/MultiNetwork.h"
#include "MultiGPUSimulator.h"

using std::cout;
using std::endl;

pthread_barrier_t cycle_barrier;

CrossNodeDataGPU * global_cross_data_gpu;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread(void *para);

int MultiGPUSimulator::run(real time, FireInfo &log)
{
	int sim_cycle = round(time/dt);
	reset();

	int device_count = 4;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);
	for (int i=0; i<device_count; i++) {
		for (int j=0; j<device_count; j++) {
			if (i!=j) {
				int access = 0;
				checkCudaErrors(hipDeviceCanAccessPeer(&access, i, j));
				if (access == 1) {
					checkCudaErrors(hipSetDevice(i));
					checkCudaErrors(hipDeviceEnablePeerAccess(j, 0));
				}
			}
		}
	}
	checkCudaErrors(hipSetDevice(0));

	pthread_barrier_init(&cycle_barrier, NULL, device_count);

	MultiNetwork multiNet(network, device_count);
	DistriNetwork *node_nets = multiNet.buildNetworks();
	assert(node_nets != NULL);
	global_cross_data_gpu = multiNet.arrangeCrossNodeDataGPU(device_count);
	assert(global_cross_data_gpu != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);


	for (int i=0; i<device_count; i++) {
		node_nets[i]._sim_cycle = sim_cycle;
		node_nets[i]._node_idx = i;
		node_nets[i]._node_num = device_count;
		node_nets[i]._dt = dt;


		int ret = pthread_create(&(thread_ids[i]), NULL, &run_thread, (void*)&(node_nets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<device_count; i++) {
		pthread_join(thread_ids[i], NULL);
	}

	pthread_barrier_destroy(&cycle_barrier);

	return 0;
}

void * run_thread(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char log_filename[512];
	sprintf(log_filename, "GSim_%d.log", network->_node_idx); 
	FILE *log_file = fopen(log_filename, "w+");
	assert(log_file != NULL);

	char v_filename[512];
	sprintf(v_filename, "g_v_%d.data", network->_node_idx); 
	FILE *v_file = fopen(v_filename, "w+");
	assert(v_file != NULL);

	checkCudaErrors(hipSetDevice(network->_node_idx));

	GNetwork *pCpuNet = network->_network;
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int nodeNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int allNeuronNum = pCpuNet->pN2SConnection->n_num;
	int nodeSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_node_idx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_node_idx, nodeNeuronNum, nodeSynapseNum);

	//int dataOffset = network->_node_idx * network->_node_num;
	//int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	printf("Thread %d MAX_DELAY: %d\n", network->_node_idx, pCpuNet->MAX_DELAY);

	init_connection<<<1, 1>>>(c_pGpuNet->pN2SConnection);

	GBuffers *buffers = alloc_buffers(allNeuronNum, nodeSynapseNum, MAX_DELAY, network->_dt);

	BlockSize *updateSize = getBlockSize(allNeuronNum, nodeSynapseNum);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(nodeNeuronNum);
	int life_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIFE);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		GLIFENeurons *c_g_lif = copyFromGPU<GLIFENeurons>(static_cast<GLIFENeurons*>(c_pGpuNet->pNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		copy_idx = life_idx;
	} else {
	}
#endif

	for (int i=0; i<nTypeNum; i++) {
		cout << "Thread " << network->_node_idx << " " << pCpuNet->nTypes[i] << ": <<<" << updateSize[c_pGpuNet->nTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->nTypes[i]].blockSize << ">>>" << endl;
	}
	for (int i=0; i<sTypeNum; i++) {
		cout << "Thread " << network->_node_idx << " " << pCpuNet->sTypes[i] << ": <<<" << updateSize[c_pGpuNet->sTypes[i]].gridSize << ", " << updateSize[c_pGpuNet->sTypes[i]].blockSize << ">>>" << endl;
	}

	//int * c_g_cross_id = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 

	int * c_g_idx2index = copyToGPU<int>(network->_crossnode_map->_idx2index, allNeuronNum);
	int * c_g_cross_index2idx = copyToGPU<int>(network->_crossnode_map->_crossnode_index2idx, network->_crossnode_map->_cross_size);
	int * c_g_global_cross_data = gpuMalloc<int>(allNeuronNum * network->_node_num);
	int * c_g_fired_n_num = gpuMalloc<int>(network->_node_num);

	vector<int> firedInfo;
	struct timeval ts, te;
	//struct timeval t0, t1, t2, t3, t4, t5,/* t6,*/ t7, t8, t9;
	//double barrier1_time = 0, gpu_cpy_time = 0, peer_cpy_time = 0, barrier2_time=0, copy_time = 0;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_sim_cycle; time++) {
		for (int i=0; i<nTypeNum; i++) {
			assert(c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i] > 0);
			cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], time, &updateSize[c_pGpuNet->nTypes[i]]);
		}

		//gettimeofday(&t0, NULL);
		pthread_barrier_wait(&cycle_barrier);
		//gettimeofday(&t1, NULL);
		//barrier1_time += (t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec)/1000000.0;
		hipMemset(c_g_fired_n_num, 0, sizeof(int)*network->_node_num);
		//hipDeviceSynchronize();
		//gettimeofday(&t2, NULL);
		//cudaDeliverNeurons(c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_node_num, allNeuronNum);
		//for (int i=0; i<network->_node_num; i++) {
		//	int offset = i * network->_node_num + network->_node_idx; 
		//	copyFromGPU<int>(&(global_cross_data[offset]._fired_n_num), c_g_fired_n_num + i, 1);
		//	if (global_cross_data[offset]._fired_n_num > 0) {
		//		copyFromGPU<int>(global_cross_data[offset]._fired_n_idxs, c_g_global_cross_data + allNeuronNum * i, global_cross_data[offset]._fired_n_num);
		//	}
		//}
		cudaDeliverNeurons(c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_node_num, allNeuronNum);
		checkCudaErrors(hipMemcpy(global_cross_data_gpu->_fired_num + network->_node_idx * network->_node_num, c_g_fired_n_num, sizeof(int)*network->_node_num, hipMemcpyDeviceToHost));
		//gettimeofday(&t3, NULL);

		for (int i=0; i< network->_node_num; i++) {
			int idx2i = network->_node_idx * network->_node_num + i;
			assert(global_cross_data_gpu->_fired_num[idx2i] <= global_cross_data_gpu->_max_num[idx2i]);
			if (global_cross_data_gpu->_fired_num[idx2i] > 0) {
				checkCudaErrors(hipMemcpyPeer(global_cross_data_gpu->_fired_arrays[idx2i], i, c_g_global_cross_data + allNeuronNum * i, network->_node_idx, global_cross_data_gpu->_fired_num[idx2i] * sizeof(int)));
			}
		}
		//gettimeofday(&t7, NULL);

		//gpu_cpy_time += (t3.tv_sec - t2.tv_sec) + (t3.tv_usec - t2.tv_usec)/1000000.0;
		//peer_cpy_time += (t7.tv_sec - t3.tv_sec) + (t7.tv_usec - t3.tv_usec)/1000000.0;

#ifdef LOG_DATA
		int currentIdx = time%(MAX_DELAY+1);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		if (copySize > 0) {
			copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (allNeuronNum*currentIdx), copySize);
		}

		if (copy_idx >= 0 && (c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]) > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
		}
#endif

		for (int i=0; i<sTypeNum; i++) {
			assert(c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i] > 0);
			cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], time, &updateSize[pCpuNet->sTypes[i]]);
		}
		//hipDeviceSynchronize();

		//gettimeofday(&t4, NULL);
		pthread_barrier_wait(&cycle_barrier);
		//gettimeofday(&t5, NULL);
		//barrier2_time += (t5.tv_sec - t4.tv_sec) + (t5.tv_usec - t4.tv_usec)/1000000.0;

		//gettimeofday(&t6, NULL);
		//collectNeurons();
		//gettimeofday(&t7, NULL);
		//cpu_cpy_time += (t7.tv_sec - t6.tv_sec) + (t7.tv_usec - t6.tv_usec)/1000000.0;
		
		//gettimeofday(&t8, NULL);
		//if (global_cross_data[dataIdx]._fired_n_num > 0) {
		//	copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
		//	addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
		//}
		for (int i=0; i< network->_node_num; i++) {
			int i2idx = network->_node_idx + network->_node_num * i;
			if (global_cross_data_gpu->_fired_num[i2idx] > 0) {
				addCrossNeurons(global_cross_data_gpu->_fired_arrays[i2idx], global_cross_data_gpu->_fired_num[i2idx]);
			}
		}
		
		//gettimeofday(&t9, NULL);
		//copy_time += (t9.tv_sec - t8.tv_sec) + (t9.tv_usec - t8.tv_usec)/1000000.0;

#ifdef LOG_DATA
		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");
#endif

		//pthread_barrier_wait(&cycle_barrier);
	}
	pthread_barrier_wait(&cycle_barrier);
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_node_idx, hours, minutes, seconds, uSeconds);
	//printf("Thread %d cost : barrier1 %lf, DtoH %lf, DtoD %lf, barrier2 %lf, HtoD %lf\n", network->_node_idx, barrier1_time, gpu_cpy_time, peer_cpy_time, barrier2_time, copy_time);

	int *rate = (int*)malloc(sizeof(int)*nodeNeuronNum);
	copyFromGPU<int>(rate, buffers->c_gFireCount, nodeNeuronNum);

	char fire_filename[512];
	sprintf(fire_filename, "GFire_%d.log", network->_node_idx); 
	FILE *rate_file = fopen(fire_filename, "w+");
	if (rate_file == NULL) {
		printf("Open file Sim.log failed\n");
		return NULL;
	}

	for (int i=0; i<nodeNeuronNum; i++) {
		fprintf(rate_file, "%d \t", rate[i]);
	}

	free(rate);
	fclose(rate_file);

	fclose(log_file);
	fclose(v_file);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);

	return NULL;
}

