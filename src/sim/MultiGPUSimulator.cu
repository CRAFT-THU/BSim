#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/gpu_kernel.h"
#include "../net/MultiNetwork.h"
#include "MultiGPUSimulator.h"

pthread_barrier_t cycle_barrier;

CrossNodeData * global_cross_data;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread(void *para);
void *run_single(void *para);

int MultiGPUSimulator::run(real time)
{
	int sim_cycle = round(time/dt);
	reset();

	int device_count = 1;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);

	pthread_barrier_init(&cycle_barrier, NULL, device_count);

	MultiNetwork multiNet(network, device_count);
	DistriNetwork *node_nets = multiNet.buildNetworks();
	assert(node_nets != NULL);
	global_cross_data = multiNet.arrangeCrossNodeData(device_count);
	assert(global_cross_data != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);


	for (int i=0; i<device_count; i++) {
		node_nets[i]._sim_cycle = sim_cycle;
		node_nets[i]._node_idx = i;
		node_nets[i]._node_num = device_count;

		int ret = pthread_create(&(thread_ids[i]), NULL, &run_thread, (void*)&(node_nets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<device_count; i++) {
		pthread_join(thread_ids[i], NULL);
	}

	pthread_barrier_destroy(&cycle_barrier);

	return 0;
}

void * run_thread(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char log_filename[512];
	sprintf(log_filename, "GSim_%d.log", network->_node_idx); 
	FILE *log_file = fopen(log_filename, "w+");
	assert(log_file != NULL);

	char v_filename[512];
	sprintf(v_filename, "g_v_%d.data", network->_node_idx); 
	FILE *v_file = fopen(v_filename, "w+");
	assert(v_file != NULL);

	checkCudaErrors(hipSetDevice(network->_node_idx));

	GNetwork *pCpuNet = network->_network;
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_node_idx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_node_idx, totalNeuronNum, totalSynapseNum);

	int dataOffset = network->_node_idx * network->_node_num;
	int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	//printf("MAX_DELAY: %d\n", pCpuNet->MAX_DELAY);

	GBuffers *buffers = alloc_buffers(pCpuNet->pN2SConnection->n_num, totalSynapseNum, MAX_DELAY);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	BlockSize preSize = { 0, 0, 0};
	hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_pre_synapse, 0, totalNeuronNum); 
	preSize.gridSize = (upzero_else_set_one(totalNeuronNum) + (preSize.blockSize) - 1) / (preSize.blockSize);

	real *c_vm = hostMalloc<real>(totalNeuronNum);
	int life_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIFE);
	int lif_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		GLIFENeurons *c_g_lif = copyFromGPU<GLIFENeurons>(static_cast<GLIFENeurons*>(c_pGpuNet->pNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		copy_idx = life_idx;
	} else if (lif_idx >= 0) {
		GLIFNeurons *c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGpuNet->pNeurons[lif_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		copy_idx = lif_idx;
	} else {
	}

	int * c_g_cross_id = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 


	vector<int> firedInfo;
	//printf("Thread %d: Start runing for %d cycles\n", network->_node_idx, network->_sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_sim_cycle; time++) {
		//printf("Thread %d Cycle: %d\n", network->_node_idx, time);
		//fflush(stdout);

		for (int i=0; i<nTypeNum; i++) {
			//printf("Thread %d update neuron size %d %d\n", network->_node_idx, updateSize[c_pGpuNet->nTypes[i]].gridSize, updateSize[c_pGpuNet->nTypes[i]].blockSize);
			assert(c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i] > 0);
			cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
		}
		//printf("Thread %d before copy size %p\n", network->_node_idx, buffers->c_gFiredTableSizes);

		int currentIdx = time%(MAX_DELAY+1);
		//printf("Thread %d current idx %d\n", network->_node_idx, currentIdx);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		//printf("Thread %d copy size: %d\n", network->_node_idx, copySize);
		if (copySize > 0) {
			copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (pCpuNet->pN2SConnection->n_num*currentIdx), copySize);
		}

		if (copy_idx >= 0 && (c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]) > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGpuNet->synapseNums[exp_idx+1]-c_pGpuNet->synapseNums[exp_idx]);
		}

		//printf("Thread %d before update connect\n", network->_node_idx);
		//printf("Thread %d update connect size %d %d\n", network->_node_idx, preSize.gridSize, preSize.blockSize);
		//printf("Thread %d after update connect\n", network->_node_idx);

		for (int i=0; i<sTypeNum; i++) {
			assert(c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i] > 0);
			cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
		}

		for (int i=0; i<network->_node_num; i++) {
			int offset = i * network->_node_num + network->_node_idx; 
			global_cross_data[offset]._fired_n_num = 0;
		}

		for (int i=0; i<copySize; i++) {
			int nid = buffers->c_neuronsFired[i];
			int tmp = network->_crossnode_map->_idx2index[nid];
			if (tmp >= 0) {
				for (int j=0; j<network->_node_num; j++) {
					int tmp2 = tmp * network->_node_num + j;
					int map_nid = network->_crossnode_map->_crossnode_index2idx[tmp2];
					if (map_nid >= 0) {
						//_node_idx to j 
						int offset = j * network->_node_num + network->_node_idx; 
						global_cross_data[offset]._fired_n_idxs[global_cross_data[offset]._fired_n_num] = map_nid; 
						global_cross_data[offset]._fired_n_num++;
					}
				}
			}

		}

		pthread_barrier_wait(&cycle_barrier);

		for (int i=0; i<network->_node_num; i++) {
			if (i != network->_node_idx) {
				memcpy(global_cross_data[dataIdx]._fired_n_idxs + global_cross_data[dataIdx]._fired_n_num, global_cross_data[dataOffset+i]._fired_n_idxs, global_cross_data[dataOffset+i]._fired_n_num * sizeof(int));
				global_cross_data[dataIdx]._fired_n_num += global_cross_data[dataOffset+i]._fired_n_num;
			}
		}

		
		if (global_cross_data[dataIdx]._fired_n_num > 0) {
			copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
			addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
		}

		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");

		pthread_barrier_wait(&cycle_barrier);
		update_time<<<1, 1>>>();
	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_node_idx, hours, minutes, seconds, uSeconds);

	fclose(log_file);
	fclose(v_file);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);

	return NULL;
}

int MultiGPUSimulator::single_run(real time)
{
	int sim_cycle = round(time/dt);
	reset();

	int device_count = 1;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);

	MultiNetwork multiNet(network, device_count);
	DistriNetwork *node_nets = multiNet.buildNetworks();
	assert(node_nets != NULL);
	global_cross_data = multiNet.arrangeCrossNodeData(device_count);
	assert(global_cross_data != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);

	FILE **log_files = (FILE**)malloc(sizeof(FILE*)*device_count);
	FILE **v_files = (FILE**)malloc(sizeof(FILE*)*device_count);
	GBuffers **node_buffers = (GBuffers **)malloc(sizeof(GBuffers*)*device_count);
	BlockSize **node_update_sizes = (BlockSize **)malloc(sizeof(BlockSize*)*device_count);
	int **c_g_cross_ids = (int **)malloc(sizeof(int*)*device_count); 
	int *copy_idxs = (int *)malloc(sizeof(int)*device_count); 
	real **c_g_vms = (real**)malloc(sizeof(real*)*device_count);
	real **c_vms = (real**)malloc(sizeof(real*)*device_count);

	for (int i=0; i<device_count; i++) {
		node_nets[i]._sim_cycle = sim_cycle;
		node_nets[i]._node_idx = i;
		node_nets[i]._node_num = device_count;

		DistriNetwork *network = &(node_nets[i]);
		GNetwork *pCpuNet = network->_network;
		GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);
		int nTypeNum = pCpuNet->nTypeNum;
		int sTypeNum = pCpuNet->sTypeNum;
		int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
		int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
		int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;
		int MAX_DELAY = pCpuNet->MAX_DELAY;

		char log_filename[512];
		sprintf(log_filename, "GSim_single_%d.log", node_nets[i]._node_idx); 
		FILE *log_file = fopen(log_filename, "w+");
		assert(log_file != NULL);
		log_files[i] = log_file;

		char v_filename[512];
		sprintf(v_filename, "g_v_single_%d.data", node_nets[i]._node_idx); 
		FILE *v_file = fopen(v_filename, "w+");
		assert(v_file != NULL);
		v_files[i] = v_file;

		node_buffers[i] = alloc_buffers(pCpuNet->pN2SConnection->n_num, totalSynapseNum, MAX_DELAY);
		node_update_sizes[i] = getBlockSize(totalNeuronNum, totalSynapseNum);
		c_g_cross_ids[i] = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 
		c_vms[i] = hostMalloc<real>(totalNeuronNum);
		copy_idxs[i] = getIndex(pCpuNet->nTypes, nTypeNum, LIFE);
		GLIFENeurons *c_g_lif = copyFromGPU<GLIFENeurons>(static_cast<GLIFENeurons*>(c_pGpuNet->pNeurons[copy_idxs[i]]), 1);
		c_g_vms[i] = c_g_lif->p_vm;

	}

	checkCudaErrors(hipSetDevice(0));

	for (int time=0; time<sim_cycle; time++) {
		for (int node=0; node<device_count; node++) {
			DistriNetwork *network = &(node_nets[node]);
			GNetwork *pCpuNet = network->_network;
			GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);
			int nTypeNum = pCpuNet->nTypeNum;
			int sTypeNum = pCpuNet->sTypeNum;
			//int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
			//int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
			//int dataOffset = network->_node_idx * network->_node_num;
			int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;
			int MAX_DELAY = pCpuNet->MAX_DELAY;
			int *c_g_cross_id = c_g_cross_ids[node];

			GBuffers *buffers = node_buffers[node];
			BlockSize *updateSize = node_update_sizes[node];

			int copy_idx = copy_idxs[node];
			real *c_vm = c_vms[node];
			real *c_g_vm = c_g_vms[node];
			FILE *v_file = v_files[node];

			if (time > 0) {
				if (global_cross_data[dataIdx]._fired_n_num > 0) {
					copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
					addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
				}

				update_time<<<1, 1>>>();
			}
			
			for (int i=0; i<nTypeNum; i++) {
				assert(c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i] > 0);
				cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
			}

			int currentIdx = time%(MAX_DELAY+1);

			int copySize = 0;
			copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
			if (copySize > 0) {
				copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (pCpuNet->pN2SConnection->n_num*currentIdx), copySize);
			}

			if (copy_idx >= 0 && (c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]) > 0) {
				copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
			}

			for (int i=0; i<sTypeNum; i++) {
				assert(c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i] > 0);
				cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
			}

			for (int i=0; i<network->_node_num; i++) {
				int offset = i * network->_node_num + network->_node_idx; 
				global_cross_data[offset]._fired_n_num = 0;
			}

			for (int i=0; i<copySize; i++) {
				int nid = buffers->c_neuronsFired[i];
				int tmp = network->_crossnode_map->_idx2index[nid];
				if (tmp >= 0) {
					for (int j=0; j<network->_node_num; j++) {
						int tmp2 = tmp * network->_node_num + j;
						int map_nid = network->_crossnode_map->_crossnode_index2idx[tmp2];
						if (map_nid >= 0) {
							//_node_idx to j 
							int offset = j * network->_node_num + network->_node_idx; 
							global_cross_data[offset]._fired_n_idxs[global_cross_data[offset]._fired_n_num] = map_nid; 
							global_cross_data[offset]._fired_n_num++;
						}
					}
				}

			}

			for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
				fprintf(v_file, "%.10lf \t", c_vm[i]);
			}
			fprintf(v_file, "\n");

			freeGPUNetwork(c_pGpuNet);
		}

		for (int node=0; node<device_count; node++) {
			DistriNetwork *network = &(node_nets[node]);
			int dataOffset = network->_node_idx * network->_node_num;
			int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;
			for (int i=0; i<network->_node_num; i++) {
				if (i != network->_node_idx) {
					memcpy(global_cross_data[dataIdx]._fired_n_idxs + global_cross_data[dataIdx]._fired_n_num, global_cross_data[dataOffset+i]._fired_n_idxs, global_cross_data[dataOffset+i]._fired_n_num * sizeof(int));
					global_cross_data[dataIdx]._fired_n_num += global_cross_data[dataOffset+i]._fired_n_num;
				}
			}
		}

	}

	for (int i=0; i<device_count; i++) {
		fclose(log_files[i]);
		fclose(v_files[i]);
		free_buffers(node_buffers[i]);
	}



	return 0;
}
