#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/gpu_kernel.h"
#include "../net/MultiNetwork.h"
#include "MultiGPUSimulator.h"

pthread_barrier_t cycle_barrier;

CrossNodeData * global_cross_data;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread(void *para);

int MultiGPUSimulator::run(real time)
{
	int sim_cycle = round(time/dt);
	reset();

	int device_count = 1;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);

	pthread_barrier_init(&cycle_barrier, NULL, device_count);

	MultiNetwork multiNet(network, device_count);
	DistriNetwork *node_nets = multiNet.buildNetworks();
	assert(node_nets != NULL);
	global_cross_data = multiNet.arrangeCrossNodeData(device_count);
	assert(global_cross_data != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);


	for (int i=0; i<device_count; i++) {
		node_nets[i]._sim_cycle = sim_cycle;
		node_nets[i]._node_idx = i;
		node_nets[i]._node_num = device_count;

		int ret = pthread_create(&(thread_ids[i]), NULL, &run_thread, (void*)&(node_nets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<device_count; i++) {
		pthread_join(thread_ids[i], NULL);
	}

	pthread_barrier_destroy(&cycle_barrier);

	return 0;
}

void * run_thread(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char logFilename[512];
	sprintf(logFilename, "GSim_%d.log", network->_node_idx); 
	FILE *logFile = fopen(logFilename, "w+");
	assert(logFile != NULL);

	char dataFilename[512];
	sprintf(dataFilename, "g_v_%d.data", network->_node_idx); 
	FILE *dataFile = fopen(dataFilename, "w+");
	assert(dataFile != NULL);

	checkCudaErrors(hipSetDevice(network->_node_idx));

	GNetwork *pCpuNet = network->_network;
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_node_idx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_node_idx, totalNeuronNum, totalSynapseNum);

	int dataOffset = network->_node_idx * network->_node_num;
	int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	//printf("MAX_DELAY: %d\n", pCpuNet->MAX_DELAY);

	GBuffers *buffers = alloc_buffers(pCpuNet->pN2SConnection->n_num, totalSynapseNum, MAX_DELAY);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	BlockSize preSize = { 0, 0, 0};
	hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_pre_synapse, 0, totalNeuronNum); 
	preSize.gridSize = (upzero_else_set_one(totalNeuronNum) + (preSize.blockSize) - 1) / (preSize.blockSize);

	real *c_vm = hostMalloc<real>(totalNeuronNum);
	int life_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIFE);
	int lif_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		GLIFENeurons *c_g_lif = copyFromGPU<GLIFENeurons>(static_cast<GLIFENeurons*>(c_pGpuNet->pNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		copy_idx = life_idx;
	} else if (lif_idx >= 0) {
		GLIFNeurons *c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGpuNet->pNeurons[lif_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
		copy_idx = lif_idx;
	} else {
	}

	int * c_g_cross_id = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 


	vector<int> firedInfo;
	//printf("Thread %d: Start runing for %d cycles\n", network->_node_idx, network->_sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_sim_cycle; time++) {
		//printf("Thread %d Cycle: %d\n", network->_node_idx, time);
		//fflush(stdout);

		for (int i=0; i<nTypeNum; i++) {
			//printf("Thread %d update neuron size %d %d\n", network->_node_idx, updateSize[c_pGpuNet->nTypes[i]].gridSize, updateSize[c_pGpuNet->nTypes[i]].blockSize);
			assert(c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i] > 0);
			cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
		}
		//printf("Thread %d before copy size %p\n", network->_node_idx, buffers->c_gFiredTableSizes);

		int currentIdx = time%(MAX_DELAY+1);
		//printf("Thread %d current idx %d\n", network->_node_idx, currentIdx);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		//printf("Thread %d copy size: %d\n", network->_node_idx, copySize);
		if (copySize > 0) {
			copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (pCpuNet->pN2SConnection->n_num*currentIdx), copySize);
		}

		if (lif_idx >= 0 && (c_pGpuNet->neuronNums[lif_idx+1]-c_pGpuNet->neuronNums[lif_idx]) > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[lif_idx+1]-c_pGpuNet->neuronNums[lif_idx]);
			//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGpuNet->synapseNums[exp_idx+1]-c_pGpuNet->synapseNums[exp_idx]);
		}

		//printf("Thread %d before update connect\n", network->_node_idx);
		//printf("Thread %d update connect size %d %d\n", network->_node_idx, preSize.gridSize, preSize.blockSize);
		//printf("Thread %d after update connect\n", network->_node_idx);

		for (int i=0; i<sTypeNum; i++) {
			assert(c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i] > 0);
			cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
		}

		for (int i=0; i<network->_node_num; i++) {
			int offset = i * network->_node_num + network->_node_idx; 
			global_cross_data[offset]._fired_n_num = 0;
		}

		for (int i=0; i<copySize; i++) {
			int nid = buffers->c_neuronsFired[i];
			int tmp = network->_crossnode_map->_idx2index[nid];
			if (tmp >= 0) {
				for (int j=0; j<network->_node_num; j++) {
					int tmp2 = tmp * network->_node_num + j;
					int map_nid = network->_crossnode_map->_crossnode_index2idx[tmp2];
					if (map_nid >= 0) {
						//_node_idx to j 
						int offset = j * network->_node_num + network->_node_idx; 
						global_cross_data[offset]._fired_n_idxs[global_cross_data[offset]._fired_n_num] = map_nid; 
						global_cross_data[offset]._fired_n_num++;
					}
				}
			}

		}

		pthread_barrier_wait(&cycle_barrier);

		for (int i=0; i<network->_node_num; i++) {
			if (i != network->_node_idx) {
				memcpy(global_cross_data[dataIdx]._fired_n_idxs + global_cross_data[dataIdx]._fired_n_num, global_cross_data[dataOffset+i]._fired_n_idxs, global_cross_data[dataOffset+i]._fired_n_num * sizeof(int));
				global_cross_data[dataIdx]._fired_n_num += global_cross_data[dataOffset+i]._fired_n_num;
			}
		}

		
		if (global_cross_data[dataIdx]._fired_n_num > 0) {
			copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
			addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
		}

		fprintf(logFile, "Cycle %d: ", time);
		for (int i=0; i<copySize; i++) {
			//if (network->_node_idx == 1) printf("hehe\n");
			fprintf(logFile, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(logFile, "\n");

		//fprintf(dataFile, "Cycle %d: ", time);
		for (int i=0; i<c_pGpuNet->neuronNums[lif_idx+1] - c_pGpuNet->neuronNums[lif_idx]; i++) {
			fprintf(dataFile, "%.10lf \t", c_vm[i]);
		}
		//for (int i=0; i<c_pGpuNet->synapseNums[1] - c_pGpuNet->synapseNums[0]; i++) {
		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
		//}
		fprintf(dataFile, "\n");

		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);

		//int synapseCount = 0;
		//if (time > 0) {
		//	for (int i=0; i<totalSynapseNum; i++) {
		//		if (buffers->c_synapsesFired[i] == time) {
		//			if (synapseCount ==  0) {
		//				if (copySize > 0) {
		//					fprintf(logFile, ", ");
		//				}
		//				fprintf(logFile, "%d", i);
		//				synapseCount++;
		//			} else {
		//				fprintf(logFile, ", %d", i);
		//			}
		//		}
		//	}
		//	fprintf(logFile, "\n");
		//}

		pthread_barrier_wait(&cycle_barrier);
		update_time<<<1, 1>>>();
	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_node_idx, hours, minutes, seconds, uSeconds);

	fclose(logFile);
	fclose(dataFile);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);

	return NULL;
}
