#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/gpu_kernel.h"
#include "../net/MultiNetwork.h"
#include "MultiGPUSimulator.h"

struct DistriNetwork {
	int simCycle;
	int nodeIdx;
	int nodeNum;
	GNetwork * network;
	CrossNodeMap *crossNodeMap;
	CrossNodeData *crossNodeData;
};

pthread_barrier_t cycle_barrier;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread(void *para);

int MultiGPUSimulator::run(real time)
{
	int simCycle = round(time/dt);
	reset();

	int deviceCount = 1;
	checkCudaErrors(hipGetDeviceCount(&deviceCount));
	assert(deviceCount != 0);

	pthread_barrier_init(&cycle_barrier, NULL, deviceCount);

	MultiNetwork multiNet(network);
	GNetwork *pCpuNets = multiNet.buildNetworks(deviceCount);

	pthread_t *threadIds = (pthread_t *)malloc(sizeof(pthread_t) * deviceCount);
	DistriNetwork *nodeNets = (DistriNetwork *)malloc(sizeof(DistriNetwork) * deviceCount);

	for (int i=0; i<deviceCount; i++) {
		nodeNets[i].simCycle = simCycle;
		nodeNets[i].nodeIdx = i;
		nodeNets[i].nodeNum = deviceCount;
		nodeNets[i].network = &(pCpuNets[i]);
		nodeNets[i].crossNodeMap = &(multiNet.crossNodeMap[i]); 
		nodeNets[i].crossNodeData = multiNet.crossNodeData; 

		int ret = pthread_create(&(threadIds[i]), NULL, &run_thread, (void*)&(nodeNets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<deviceCount; i++) {
		pthread_join(threadIds[i], NULL);
	}

	pthread_barrier_destroy(&cycle_barrier);

	return 0;
}

void * run_thread(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char logFilename[512];
	sprintf(logFilename, "GSim_%d.log", network->nodeIdx); 
	FILE *logFile = fopen(logFilename, "w+");
	assert(logFile != NULL);

	char dataFilename[512];
	sprintf(dataFilename, "GSim_%d.data", network->nodeIdx); 
	FILE *dataFile = fopen(dataFilename, "w+");
	assert(dataFile != NULL);

	GNetwork *pCpuNet = network->network;
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->nodeIdx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->nodeIdx, totalNeuronNum, totalSynapseNum);

	int dataOffset = network->nodeIdx * network->nodeNum;
	int dataIdx = network->nodeIdx * network->nodeNum + network->nodeIdx;

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	//printf("MAX_DELAY: %d\n", pCpuNet->MAX_DELAY);

	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, MAX_DELAY);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	BlockSize preSize = { 0, 0, 0};
	BlockSize postSize = { 0, 0, 0};
	hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_lif_neuron, 0, totalNeuronNum); 
	preSize.gridSize = (totalNeuronNum + (preSize.blockSize) - 1) / (preSize.blockSize);
	hipOccupancyMaxPotentialBlockSize(&(postSize.minGridSize), &(postSize.blockSize), update_lif_neuron, 0, totalSynapseNum); 
	postSize.gridSize = (totalSynapseNum + (postSize.blockSize) - 1) / (postSize.blockSize);

	real *c_vm = hostMalloc<real>(totalNeuronNum);
	int lif_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIF);
	GLIFNeurons *c_g_lif;
	real *c_g_vm;
	if (lif_idx > 0) {
		c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGpuNet->pNeurons[lif_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
	}
	int * c_g_cross_id = gpuMalloc<int>(network->crossNodeData[dataIdx].maxNeuronNum);
	//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
	//int exp_idx = getIndex(pCpuNet->sTypes, sTypeNum, Exp);
	//GExpSynapses *c_g_exp = copyFromGPU<GExpSynapses>(static_cast<GExpSynapses*>(c_pGpuNet->pSynapses[exp_idx]), 1);
	//real *c_g_I_syn = c_g_exp->p_I_syn;


	vector<int> firedInfo;
	//printf("Start runing for %d cycles\n", network->simCycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->simCycle; time++) {
		//printf("\rCycle: %d", time);
		//fflush(stdout);

		for (int i=0; i<nTypeNum; i++) {
			cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
		}

		int currentIdx = time%(MAX_DELAY+1);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);
		if (lif_idx > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[lif_idx+1]-c_pGpuNet->neuronNums[lif_idx]);
			//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGpuNet->synapseNums[exp_idx+1]-c_pGpuNet->synapseNums[exp_idx]);
		}

		update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(c_pGpuNet->pN2SConnection);

		for (int i=0; i<sTypeNum; i++) {
			cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
		}

		for (int i=0; i<network->nodeNum; i++) {
			int offset = i * network->nodeNum + network->nodeIdx; 
			network->crossNodeData[offset].firedNeuronNum = 0;
		}

		for (int i=0; i<copySize; i++) {
			int nid = buffers->c_neuronsFired[i];
			int tmp = network->crossNodeMap->idx2index[nid];
			if (tmp >= 0) {
				for (int j=0; j<network->nodeNum; j++) {
					int tmp2 = tmp * network->nodeIdx + j;
					int map_nid = network->crossNodeMap->crossNodeMap[tmp2];
					if (map_nid >= 0) {
						//nodeIdx to j 
						int offset = j * network->nodeNum + network->nodeIdx; 
						network->crossNodeData[offset].firedNeuronIdx[network->crossNodeData[offset].firedNeuronNum] = map_nid; 
						network->crossNodeData[offset].firedNeuronNum++;
					}
				}
			}

		}

		pthread_barrier_wait(&cycle_barrier);

		for (int i=0; i<network->nodeNum; i++) {
			if (i != network->nodeIdx) {
				memcpy(network->crossNodeData[dataIdx].firedNeuronIdx + network->crossNodeData[dataIdx].firedNeuronNum, network->crossNodeData[dataOffset+i].firedNeuronIdx, network->crossNodeData[dataOffset+i].firedNeuronNum * sizeof(int));
				network->crossNodeData[dataIdx].firedNeuronNum += network->crossNodeData[dataOffset+i].firedNeuronNum;
			}
		}

		
		copyToGPU(c_g_cross_id, network->crossNodeData[dataIdx].firedNeuronIdx, network->crossNodeData[dataIdx].firedNeuronNum);
		addCrossNeurons(c_g_cross_id, network->crossNodeData[dataIdx].firedNeuronNum);

		fprintf(logFile, "Cycle %d: ", time);
		for (int i=0; i<copySize; i++) {
			fprintf(logFile, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(logFile, "\n");

		fprintf(dataFile, "Cycle %d: ", time);
		for (int i=0; i<c_pGpuNet->neuronNums[2] - c_pGpuNet->neuronNums[1]; i++) {
			if (i ==  0) {
				fprintf(dataFile, "%lf", c_vm[i]);
			} else {
				fprintf(dataFile, ", %lf", c_vm[i]);
			}
		}
		//for (int i=0; i<c_pGpuNet->synapseNums[1] - c_pGpuNet->synapseNums[0]; i++) {
		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
		//}
		fprintf(dataFile, "\n");

		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);

		//int synapseCount = 0;
		//if (time > 0) {
		//	for (int i=0; i<totalSynapseNum; i++) {
		//		if (buffers->c_synapsesFired[i] == time) {
		//			if (synapseCount ==  0) {
		//				if (copySize > 0) {
		//					fprintf(logFile, ", ");
		//				}
		//				fprintf(logFile, "%d", i);
		//				synapseCount++;
		//			} else {
		//				fprintf(logFile, ", %d", i);
		//			}
		//		}
		//	}
		//	fprintf(logFile, "\n");
		//}

		pthread_barrier_wait(&cycle_barrier);
		update_time<<<1, 1>>>();
	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->nodeIdx, hours, minutes, seconds, uSeconds);

	fclose(logFile);
	fclose(dataFile);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);

	return NULL;
}
