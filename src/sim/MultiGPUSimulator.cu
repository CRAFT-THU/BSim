#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/gpu_kernel.h"
#include "../net/MultiNetwork.h"
#include "MultiGPUSimulator.h"

pthread_barrier_t cycle_barrier;

CrossNodeData * global_cross_data;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread(void *para);

int MultiGPUSimulator::run(real time)
{
	int sim_cycle = round(time/dt);
	reset();

	int device_count = 1;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);

	pthread_barrier_init(&cycle_barrier, NULL, device_count);

	MultiNetwork multiNet(network, device_count);
	DistriNetwork *node_nets = multiNet.buildNetworks();
	assert(node_nets != NULL);
	global_cross_data = multiNet.arrangeCrossNodeData(device_count);
	assert(global_cross_data != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);


	for (int i=0; i<device_count; i++) {
		node_nets[i]._sim_cycle = sim_cycle;
		node_nets[i]._node_idx = i;
		node_nets[i]._node_num = device_count;

		int ret = pthread_create(&(thread_ids[i]), NULL, &run_thread, (void*)&(node_nets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<device_count; i++) {
		pthread_join(thread_ids[i], NULL);
	}

	pthread_barrier_destroy(&cycle_barrier);

	return 0;
}

void * run_thread(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char logFilename[512];
	sprintf(logFilename, "GSim_%d.log", network->_node_idx); 
	FILE *logFile = fopen(logFilename, "w+");
	assert(logFile != NULL);

	char dataFilename[512];
	sprintf(dataFilename, "GSim_%d.data", network->_node_idx); 
	FILE *dataFile = fopen(dataFilename, "w+");
	assert(dataFile != NULL);

	checkCudaErrors(hipSetDevice(network->_node_idx));

	GNetwork *pCpuNet = network->_network;
	GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_node_idx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_node_idx, totalNeuronNum, totalSynapseNum);

	int dataOffset = network->_node_idx * network->_node_num;
	int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;

	int MAX_DELAY = pCpuNet->MAX_DELAY;
	//printf("MAX_DELAY: %d\n", pCpuNet->MAX_DELAY);

	GBuffers *buffers = alloc_buffers(totalNeuronNum, totalSynapseNum, MAX_DELAY);

	BlockSize *updateSize = getBlockSize(totalNeuronNum, totalSynapseNum);
	BlockSize preSize = { 0, 0, 0};
	BlockSize postSize = { 0, 0, 0};
	hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_lif_neuron, 0, totalNeuronNum); 
	preSize.gridSize = (totalNeuronNum + (preSize.blockSize) - 1) / (preSize.blockSize);
	hipOccupancyMaxPotentialBlockSize(&(postSize.minGridSize), &(postSize.blockSize), update_lif_neuron, 0, totalSynapseNum); 
	postSize.gridSize = (totalSynapseNum + (postSize.blockSize) - 1) / (postSize.blockSize);

	real *c_vm = hostMalloc<real>(totalNeuronNum);
	int lif_idx = getIndex(pCpuNet->nTypes, nTypeNum, LIF);
	GLIFNeurons *c_g_lif;
	real *c_g_vm;
	if (lif_idx > 0) {
		c_g_lif = copyFromGPU<GLIFNeurons>(static_cast<GLIFNeurons*>(c_pGpuNet->pNeurons[lif_idx]), 1);
		c_g_vm = c_g_lif->p_vm;
	}
	int * c_g_cross_id = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num);
	//real *c_I_syn = hostMalloc<real>(totalSynapseNum);
	//int exp_idx = getIndex(pCpuNet->sTypes, sTypeNum, Exp);
	//GExpSynapses *c_g_exp = copyFromGPU<GExpSynapses>(static_cast<GExpSynapses*>(c_pGpuNet->pSynapses[exp_idx]), 1);
	//real *c_g_I_syn = c_g_exp->p_I_syn;


	vector<int> firedInfo;
	//printf("Start runing for %d cycles\n", network->simCycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_sim_cycle; time++) {
		//printf("\rCycle: %d", time);
		//fflush(stdout);

		for (int i=0; i<nTypeNum; i++) {
			cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
		}

		int currentIdx = time%(MAX_DELAY+1);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (totalNeuronNum*currentIdx), copySize);
		if (lif_idx > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pGpuNet->neuronNums[lif_idx+1]-c_pGpuNet->neuronNums[lif_idx]);
			//copyFromGPU<real>(c_I_syn, c_g_I_syn, c_pGpuNet->synapseNums[exp_idx+1]-c_pGpuNet->synapseNums[exp_idx]);
		}

		update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(c_pGpuNet->pN2SConnection);

		for (int i=0; i<sTypeNum; i++) {
			cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
		}

		for (int i=0; i<network->_node_num; i++) {
			int offset = i * network->_node_num + network->_node_idx; 
			global_cross_data[offset]._fired_n_num = 0;
		}

		for (int i=0; i<copySize; i++) {
			int nid = buffers->c_neuronsFired[i];
			int tmp = network->_crossnode_map->_idx2index[nid];
			if (tmp >= 0) {
				for (int j=0; j<network->_node_num; j++) {
					int tmp2 = tmp * network->_node_idx + j;
					int map_nid = network->_crossnode_map->_crossnode_index2idx[tmp2];
					if (map_nid >= 0) {
						//_node_idx to j 
						int offset = j * network->_node_num + network->_node_idx; 
						global_cross_data[offset]._fired_n_idxs[global_cross_data[offset]._fired_n_num] = map_nid; 
						global_cross_data[offset]._fired_n_num++;
					}
				}
			}

		}

		pthread_barrier_wait(&cycle_barrier);

		for (int i=0; i<network->_node_num; i++) {
			if (i != network->_node_idx) {
				memcpy(global_cross_data[dataIdx]._fired_n_idxs + global_cross_data[dataIdx]._fired_n_num, global_cross_data[dataOffset+i]._fired_n_idxs, global_cross_data[dataOffset+i]._fired_n_num * sizeof(int));
				global_cross_data[dataIdx]._fired_n_num += global_cross_data[dataOffset+i]._fired_n_num;
			}
		}

		
		copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
		addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);

		fprintf(logFile, "Cycle %d: ", time);
		for (int i=0; i<copySize; i++) {
			fprintf(logFile, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(logFile, "\n");

		fprintf(dataFile, "Cycle %d: ", time);
		for (int i=0; i<c_pGpuNet->neuronNums[2] - c_pGpuNet->neuronNums[1]; i++) {
			if (i ==  0) {
				fprintf(dataFile, "%lf", c_vm[i]);
			} else {
				fprintf(dataFile, ", %lf", c_vm[i]);
			}
		}
		//for (int i=0; i<c_pGpuNet->synapseNums[1] - c_pGpuNet->synapseNums[0]; i++) {
		//		fprintf(dataFile, ", %lf", c_I_syn[i]);
		//}
		fprintf(dataFile, "\n");

		//copyFromGPU<int>(buffers->c_synapsesFired, buffers->c_gSynapsesLogTable, totalSynapseNum);

		//int synapseCount = 0;
		//if (time > 0) {
		//	for (int i=0; i<totalSynapseNum; i++) {
		//		if (buffers->c_synapsesFired[i] == time) {
		//			if (synapseCount ==  0) {
		//				if (copySize > 0) {
		//					fprintf(logFile, ", ");
		//				}
		//				fprintf(logFile, "%d", i);
		//				synapseCount++;
		//			} else {
		//				fprintf(logFile, ", %d", i);
		//			}
		//		}
		//	}
		//	fprintf(logFile, "\n");
		//}

		pthread_barrier_wait(&cycle_barrier);
		update_time<<<1, 1>>>();
	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_node_idx, hours, minutes, seconds, uSeconds);

	fclose(logFile);
	fclose(dataFile);

	free_buffers(buffers);
	freeGPUNetwork(c_pGpuNet);

	return NULL;
}
