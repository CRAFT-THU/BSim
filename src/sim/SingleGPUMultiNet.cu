#include "hip/hip_runtime.h"

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/gpu_kernel.h"
#include "../net/MultiNetwork.h"
#include "MultiGPUSimulator.h"

CrossNodeData * global_cross_data;

void deliverNeurons(DistriNetwork *network, GBuffers *buffers, int copySize)
{
	for (int i=0; i<network->_node_num; i++) {
		int offset = i * network->_node_num + network->_node_idx; 
		global_cross_data[offset]._fired_n_num = 0;
	}

	for (int i=0; i<copySize; i++) {
		int nid = buffers->c_neuronsFired[i];
		int tmp = network->_crossnode_map->_idx2index[nid];
		if (tmp >= 0) {
			for (int j=0; j<network->_node_num; j++) {
				int tmp2 = tmp * network->_node_num + j;
				int map_nid = network->_crossnode_map->_crossnode_index2idx[tmp2];
				if (map_nid >= 0) {
					//_node_idx to j 
					int offset = j * network->_node_num + network->_node_idx; 
					global_cross_data[offset]._fired_n_idxs[global_cross_data[offset]._fired_n_num] = map_nid; 
					global_cross_data[offset]._fired_n_num++;
				}
			}
		}

	}
}

void collectNeurons(DistriNetwork *network)
{
	int dataOffset = network->_node_idx * network->_node_num;
	int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;
	for (int i=0; i<network->_node_num; i++) {
		if ((i != network->_node_idx) && (global_cross_data[dataOffset+i]._fired_n_num > 0)) {
			memcpy(global_cross_data[dataIdx]._fired_n_idxs + global_cross_data[dataIdx]._fired_n_num, global_cross_data[dataOffset+i]._fired_n_idxs, global_cross_data[dataOffset+i]._fired_n_num * sizeof(int));
			global_cross_data[dataIdx]._fired_n_num += global_cross_data[dataOffset+i]._fired_n_num;
		}
	}
}

int MultiGPUSimulator::single_run(real time)
{
	int sim_cycle = round(time/dt);
	reset();

	int device_count = 1;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);

	MultiNetwork multiNet(network, device_count);
	DistriNetwork *node_nets = multiNet.buildNetworks();
	assert(node_nets != NULL);
	global_cross_data = multiNet.arrangeCrossNodeData(device_count);
	assert(global_cross_data != NULL);

	FILE **log_files = (FILE**)malloc(sizeof(FILE*)*device_count);
	FILE **v_files = (FILE**)malloc(sizeof(FILE*)*device_count);
	GBuffers **node_buffers = (GBuffers **)malloc(sizeof(GBuffers*)*device_count);
	BlockSize **node_update_sizes = (BlockSize **)malloc(sizeof(BlockSize*)*device_count);
	int **c_g_cross_ids = (int **)malloc(sizeof(int*)*device_count); 
	int *copy_idxs = (int *)malloc(sizeof(int)*device_count); 

	for (int i=0; i<device_count; i++) {
		node_nets[i]._sim_cycle = sim_cycle;
		node_nets[i]._node_idx = i;
		node_nets[i]._node_num = device_count;

		DistriNetwork *network = &(node_nets[i]);
		GNetwork *pCpuNet = network->_network;
		int nTypeNum = pCpuNet->nTypeNum;
		int sTypeNum = pCpuNet->sTypeNum;
		//int nodeNeuronNum = pCpuNet->neuronNums[nTypeNum];
		int allNeuronNum = pCpuNet->pN2SConnection->n_num;
		int nodeSynapseNum = pCpuNet->synapseNums[sTypeNum];
		int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;
		int MAX_DELAY = pCpuNet->MAX_DELAY;

		char log_filename[512];
		sprintf(log_filename, "GSim_single_%d.log", node_nets[i]._node_idx); 
		FILE *log_file = fopen(log_filename, "w+");
		assert(log_file != NULL);
		log_files[i] = log_file;

		char v_filename[512];
		sprintf(v_filename, "g_v_single_%d.data", node_nets[i]._node_idx); 
		FILE *v_file = fopen(v_filename, "w+");
		assert(v_file != NULL);
		v_files[i] = v_file;

		node_buffers[i] = alloc_buffers(allNeuronNum, nodeSynapseNum, MAX_DELAY);
		node_update_sizes[i] = getBlockSize(allNeuronNum, nodeSynapseNum);
		c_g_cross_ids[i] = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 
		copy_idxs[i] = getIndex(pCpuNet->nTypes, nTypeNum, LIFE);
	}

	checkCudaErrors(hipSetDevice(0));

	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		for (int node=0; node<device_count; node++) {
			DistriNetwork *network = &(node_nets[node]);
			GNetwork *pCpuNet = network->_network;
			GNetwork *c_pGpuNet = copyNetworkToGPU(pCpuNet);
			int nTypeNum = pCpuNet->nTypeNum;
			int sTypeNum = pCpuNet->sTypeNum;
			int allNeuronNum = pCpuNet->pN2SConnection->n_num;
			int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;
			int MAX_DELAY = pCpuNet->MAX_DELAY;
			int *c_g_cross_id = c_g_cross_ids[node];

			
			GBuffers *buffers = node_buffers[node];
			init_buffers(buffers);
			init_connection<<<1, 1>>>(c_pGpuNet->pN2SConnection);
			BlockSize *updateSize = node_update_sizes[node];

			int copy_idx = copy_idxs[node];
			FILE *v_file = v_files[node];
			FILE *log_file = log_files[node];

			if (time > 0) {
				init_time<<<1, 1>>>(time - 1);
				if (global_cross_data[dataIdx]._fired_n_num > 0) {
					copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
					addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
				}
				update_time<<<1, 1>>>();

			}
			
			for (int i=0; i<nTypeNum; i++) {
				assert(c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i] > 0);
				cudaUpdateType[pCpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], c_pGpuNet->neuronNums[i+1]-c_pGpuNet->neuronNums[i], c_pGpuNet->neuronNums[i], &updateSize[c_pGpuNet->nTypes[i]]);
			}

			int currentIdx = time%(MAX_DELAY+1);

			int copySize = 0;
			copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
			if (copySize > 0) {
				copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (allNeuronNum*currentIdx), copySize);
			}

			for (int i=0; i<sTypeNum; i++) {
				assert(c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i] > 0);
				cudaUpdateType[pCpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], c_pGpuNet->synapseNums[i+1]-c_pGpuNet->synapseNums[i], c_pGpuNet->synapseNums[i], &updateSize[pCpuNet->nTypes[i]]);
			}

			for (int i=0; i<network->_node_num; i++) {
				int offset = i * network->_node_num + network->_node_idx; 
				global_cross_data[offset]._fired_n_num = 0;
			}

			for (int i=0; i<copySize; i++) {
				int nid = buffers->c_neuronsFired[i];
				int tmp = network->_crossnode_map->_idx2index[nid];
				if (tmp >= 0) {
					for (int j=0; j<network->_node_num; j++) {
						int tmp2 = tmp * network->_node_num + j;
						int map_nid = network->_crossnode_map->_crossnode_index2idx[tmp2];
						if (map_nid >= 0) {
							//_node_idx to j 
							int offset = j * network->_node_num + network->_node_idx; 
							global_cross_data[offset]._fired_n_idxs[global_cross_data[offset]._fired_n_num] = map_nid; 
							global_cross_data[offset]._fired_n_num++;
						}
					}
				}

			}

			if (copy_idx >= 0 && (c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]) > 0) {
				GLIFENeurons *c_g_lif = copyFromGPU<GLIFENeurons>(static_cast<GLIFENeurons*>(c_pGpuNet->pNeurons[copy_idx]), 1);
				real *c_vm = copyFromGPU<real>(c_g_lif->p_vm, c_pGpuNet->neuronNums[copy_idx+1]-c_pGpuNet->neuronNums[copy_idx]);
				for (int i=0; i<c_pGpuNet->neuronNums[copy_idx+1] - c_pGpuNet->neuronNums[copy_idx]; i++) {
					fprintf(v_file, "%.10lf \t", c_vm[i]);
				}
				fprintf(v_file, "\n");
				free(c_g_lif);
				free(c_vm);
			}

			for (int i=0; i<copySize; i++) {
				fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
			}
			fprintf(log_file, "\n");

			fetchNetworkFromGPU(pCpuNet, c_pGpuNet);

			freeGPUNetwork(c_pGpuNet);
		}

		for (int node=0; node<device_count; node++) {
			DistriNetwork *network = &(node_nets[node]);
			int dataOffset = network->_node_idx * network->_node_num;
			int dataIdx = network->_node_idx * network->_node_num + network->_node_idx;
			for (int i=0; i<network->_node_num; i++) {
				if (i != network->_node_idx) {
					memcpy(global_cross_data[dataIdx]._fired_n_idxs + global_cross_data[dataIdx]._fired_n_num, global_cross_data[dataOffset+i]._fired_n_idxs, global_cross_data[dataOffset+i]._fired_n_num * sizeof(int));
					global_cross_data[dataIdx]._fired_n_num += global_cross_data[dataOffset+i]._fired_n_num;
				}
			}
		}

	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Simulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);


	for (int i=0; i<device_count; i++) {
		DistriNetwork *network = &(node_nets[i]);
		GNetwork *pCpuNet = network->_network;
		int nTypeNum = pCpuNet->nTypeNum;
		int nodeNeuronNum = pCpuNet->neuronNums[nTypeNum];
		GBuffers *buffers = node_buffers[i];

		int *rate = (int*)malloc(sizeof(int)*nodeNeuronNum);
		copyFromGPU<int>(rate, buffers->c_gFireCount, nodeNeuronNum);

		char fire_filename[512];
		sprintf(fire_filename, "GFire_single_%d.log", i); 
		FILE *rate_file = fopen(fire_filename, "w+");
		if (rate_file == NULL) {
			printf("Open file Sim.log failed\n");
			return -1;
		}

		for (int i=0; i<nodeNeuronNum; i++) {
			fprintf(rate_file, "%d \t", rate[i]);
		}

		free(rate);
		fclose(rate_file);
	}

	for (int i=0; i<device_count; i++) {
		fclose(log_files[i]);
		fclose(v_files[i]);
		free_buffers(node_buffers[i]);
	}



	return 0;
}
