#include "hip/hip_runtime.h"

#include "gpu_kernel.h"

#define MAX_FIRED 512

__device__ int MAX_DELAY;
__device__ int gCurrentTime;
__device__ int *gTimeTable;
__device__ int gTimeTableSize;
__device__ bool *gFiredTable;
__device__ int gFiredTableSize;
__device__ int gFiredCnt;
__device__ int gFiredCntTest;
__device__ bool *gSynapsesFiredTable;
__device__ int gSynapsesFiredTableSize;

__device__ double atomicAdd(double *address, double val)
{
	unsigned long long int *address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed = 0;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);

	return __longlong_as_double(old);
}

__device__ int updateTimeTable(int simTime)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gTimeTable[gCurrentTime] = simTime;
		gCurrentTime = (gCurrentTime +1)%(MAX_DELAY + 1);
	}
	__syncthreads();
	return 0;
}

__device__ int updateFiredTable(int firedID, bool fired, int simTime)
{
	gFiredTable[gCurrentTime*gFiredTableSize + firedID] = fired;

	return 0;
}

__global__ void init_global(int max_delay, int *c_gTimeTable, int c_gTimeTableSize, bool *c_gFiredTable, int c_gFiredTableSize, bool *c_gSynapsesFiredTable, int c_gSynapsesFiredTableSize) 
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		MAX_DELAY = max_delay;
		gCurrentTime = 0;
		gTimeTable = c_gTimeTable;
		gTimeTableSize = c_gTimeTableSize;
		gFiredTable = c_gFiredTable;
		gFiredTableSize = c_gFiredTableSize;
		gSynapsesFiredTable = c_gSynapsesFiredTable;
		gSynapsesFiredTableSize = c_gSynapsesFiredTableSize;
		gFiredCnt = 0;
		gFiredCntTest = 0;
	}
}

__global__ void update_pre_synapse(GLIFNeurons *d_neurons, GExpSynapses* d_synapses, int simTime)
{
	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int time = 0; time<MAX_DELAY+1; time++) {
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		int start_t = gTimeTable[time];
		for (int idx = tid; idx < gFiredTableSize; idx += blockDim.x*gridDim.x) {
			int nid = idx;
			if (gFiredTable[time*gFiredTableSize + nid]) {
				for (int i=0; i<d_neurons->pSynapsesNum[nid]; i++) {
					int loc = d_neurons->pSynapsesLoc[nid];
					int sid = d_neurons->pSynapsesIdx[i+loc];
					if (simTime == start_t +(int)(d_synapses->p_delay[sid]/d_synapses->p__dt[sid]))
						gSynapsesFiredTable[d_neurons->pSynapsesIdx[i+loc]] = true;
				}
			}
		}
	}
	__syncthreads();
}

__global__ void update_lif_neuron(GLIFNeurons *d_neurons, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int nid = 0;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		nid = idx;
		if (nid < num) {
			bool fired = false;
			real I = d_neurons->p_i_syn[nid] + d_neurons->p_i_tmp[nid];
			d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
			d_neurons->p_i_syn[nid] = 0;

			if (d_neurons->p_refrac_step[nid] > 0) {
				d_neurons->p_refrac_step[nid] --;
				d_neurons->p_vm[nid] = 0;
			}

			if (d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid]) {
				fired = true;
				d_neurons->p_refrac_step[nid] = (int)(d_neurons->p_tau_refrac[nid]/d_neurons->p__dt[nid]) - 1;
				d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
			}
			updateFiredTable(nid, fired, simTime);
		}
	}
	__syncthreads();

	updateTimeTable(simTime);
	__syncthreads();
}

__global__ void update_alpha_synapse(GLIFNeurons *d_neurons, GAlphaSynapses *d_synapses, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
		int sid = tid;
		if (sid < num) {
			d_synapses->p_I_syn[sid] = d_synapses->p_C1[sid] * d_synapses->p_I_syn[sid] + d_synapses->p_C2[sid] * d_synapses->p_I_tmp[sid];
			d_synapses->p_I_tmp[sid] *= d_synapses->p_C1[sid];

			if (gSynapsesFiredTable[sid]) {
				real I_t = d_synapses->p_C2[sid] * d_synapses->p_I_syn[sid] + d_synapses->p__C2[sid] * d_synapses->p_I_tmp[sid];
				d_synapses->p_I_tmp[sid] += d_synapses->p_weight[sid]/d_synapses->p__C1[sid];
				d_synapses->p_I_syn[sid] = (I_t - d_synapses->p__C2[sid] * d_synapses->p_I_tmp[sid])/d_synapses->p__C1[sid];
				atomicAdd(&(d_neurons->p_i_syn[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
				gSynapsesFiredTable[sid] = false;
			}
		}
	}
	__syncthreads();
}

__global__ void update_exp_synapse(GLIFNeurons *d_neurons, GExpSynapses *d_synapses, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
		int sid = tid;
		if (sid < num) {
			d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
		}

		if (gSynapsesFiredTable[sid]) {
			d_synapses->p_I_syn[sid] += (d_synapses->p_weight[sid]/d_synapses->p__C1[sid]);
			atomicAdd(&(d_neurons->p_i_syn[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
			gSynapsesFiredTable[sid] = false;
		}
	}
	__syncthreads();
}
