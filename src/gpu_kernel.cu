#include "hip/hip_runtime.h"

#include "gpu_kernel.h"

#define MAX_FIRED 512

__device__ unsigned int MAX_DELAY;
__device__ unsigned int *gTimeTable;
__device__ unsigned int gTimeTableSize;
__device__ unsigned int *gFiredTable;
__device__ unsigned int gFiredTableSize;
__device__ unsigned int gFiredCnt;
__device__ unsigned int gFiredCntTest;
__device__ bool *gSynapsesFiredTable;
__device__ unsigned int gSynapsesFiredTableSize;

__device__ double atomicAdd(double *address, double val)
{
	unsigned long long int *address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed = 0;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);

	return __longlong_as_double(old);
}

__device__ int updateTimeTable(unsigned int simTime)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gTimeTable[simTime + MAX_DELAY + 1] = gFiredCnt;
	}
	__syncthreads();
	return 0;
}

__device__ int updateFiredTable(unsigned int *fireTable, unsigned int fireCnt, unsigned int simTime)
{
	__shared__ volatile unsigned int cnt;
	unsigned int idx = 0;
	if (threadIdx.x == 0) {
		cnt = atomicAdd(&gFiredCntTest, fireCnt);
		//TODO: check over add items
		cnt = atomicAdd(&gFiredCnt, fireCnt);
	}
	__syncthreads();

	for (int i=threadIdx.x; i<fireCnt; i+=blockDim.x) {
		idx = atomicAdd((unsigned int*)&cnt, 1);
		gFiredTable[idx] = fireTable[i];
	}
	return 0;
}

__global__ void init_global(unsigned int max_delay, unsigned int *c_gTimeTable, unsigned int c_gTimeTableSize, unsigned int *c_gFiredTable, unsigned int c_gFiredTableSize, bool *c_gSynapsesFiredTable, unsigned int c_gSynapsesFiredTableSize) 
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		MAX_DELAY = max_delay;
		gTimeTable = c_gTimeTable;
		gTimeTableSize = c_gTimeTableSize;
		gFiredTable = c_gFiredTable;
		gFiredTableSize = c_gFiredTableSize;
		gSynapsesFiredTable = c_gSynapsesFiredTable;
		gSynapsesFiredTableSize = c_gSynapsesFiredTableSize;
		gFiredCnt = 0;
		gFiredCntTest = 0;
	}
}

__global__ void update_pre_synapse(GLIFNeurons *d_neurons, GExpSynapses* d_synapses, unsigned int simTime)
{
	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idx_s = gTimeTable[simTime];	
	unsigned int idx_e = gTimeTable[simTime+MAX_DELAY];
	for (unsigned int idx = idx_s + threadIdx.x; idx <= idx_e; idx += blockDim.x) {
		unsigned int nid = gFiredTable[idx];
		unsigned int t = 0;
		for (t=max(0, simTime-MAX_DELAY-1); t<simTime+MAX_DELAY+1; t++) {
			if (gTimeTable[t+MAX_DELAY+1] > idx) {
				break;
			}
		}
		if (simTime == t) {
			d_neurons->p_refrac_step[nid]= (int)(d_neurons->p_tau_refrac[nid]/d_neurons->p__dt[nid]) - 1;
			d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];

		}
		for (unsigned int i=0; i<d_neurons->pSynapsesNum[nid]; i++) {
			unsigned int loc = d_neurons->pSynapsesLoc[nid];
			unsigned int sid = d_neurons->pSynapsesIdx[i+loc];
			if (simTime == t+(unsigned int)(d_synapses->p_delay[sid]/d_synapses->p__dt[sid]))
					gSynapsesFiredTable[d_neurons->pSynapsesIdx[i+loc]] = true;
		}

	}
	__syncthreads();
}

__global__ void update_lif_neuron(GLIFNeurons *d_neurons, int num, unsigned int simTime)
{
	__shared__ volatile unsigned int fireCnt;
	__shared__ volatile unsigned int fireCntTest;
	__shared__ unsigned int fireTable[MAX_FIRED];

	int idx = 0;
	if (threadIdx.x == 0) {
		fireCnt = 0;
		fireCntTest = 0;
	}

	__syncthreads();

	bool fired = false;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int nid = tid;
	if (nid < num) {
		real I = d_neurons->p_i_syn[nid] + d_neurons->p_i_tmp[nid];
		d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
		d_neurons->p_i_syn[nid] = 0;

		if (d_neurons->p_refrac_step[nid] > 0) {
			d_neurons->p_refrac_step[nid] --;
			d_neurons->p_vm[nid] = 0;
		}

		if (d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid]) {
			fired = true;
			//d_neurons[nid].refrac_step = (int)(d_neurons[nid].tau_refrac/d_neurons[nid]._dt) - 1;
			//d_neurons[nid].vm = d_neurons[nid].v_reset;
		}


	}
	__syncthreads();

	if (fired) {
		idx = atomicAdd((unsigned int *)&fireCntTest, 1);
	}
	if (fired && idx < MAX_FIRED) {
		idx = atomicAdd((unsigned int *)&fireCnt, 1);
		fired = false;
		fireTable[idx] = nid;
	}
	__syncthreads();

	if (fireCnt > 0) {
		updateFiredTable(fireTable, fireCnt, simTime);
	}
	__syncthreads();

	updateTimeTable(simTime);
	__syncthreads();
}

__global__ void update_alpha_synapse(GLIFNeurons *d_neurons, GAlphaSynapses *d_synapses, unsigned int num, unsigned int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int sid = tid;
	if (sid < num) {
		d_synapses->p_I_syn[sid] = d_synapses->p_C1[sid] * d_synapses->p_I_syn[sid] + d_synapses->p_C2[sid] * d_synapses->p_I_tmp[sid];
		d_synapses->p_I_tmp[sid] *= d_synapses->p_C1[sid];
	}
	__syncthreads();
	if (sid < num) {
		if (gSynapsesFiredTable[sid]) {
			real I_t = d_synapses->p_C2[sid] * d_synapses->p_I_syn[sid] + d_synapses->p__C2[sid] * d_synapses->p_I_tmp[sid];
			d_synapses->p_I_tmp[sid] += d_synapses->p_weight[sid]/d_synapses->p__C1[sid];
			d_synapses->p_I_syn[sid] = (I_t - d_synapses->p__C2[sid] * d_synapses->p_I_tmp[sid])/d_synapses->p__C1[sid];
			atomicAdd(&(d_neurons->p_i_syn[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
			gSynapsesFiredTable[sid] = false;
		}
	}
	__syncthreads();
}

__global__ void update_exp_synapse(GLIFNeurons *d_neurons, GExpSynapses *d_synapses, unsigned int num, unsigned int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int sid = tid;
	if (sid < num) {
		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
	}
	__syncthreads();

	if (sid < num) {
		if (gSynapsesFiredTable[sid]) {
			d_synapses->p_I_syn[sid] += (d_synapses->p_weight[sid]/d_synapses->p__C1[sid]);
			atomicAdd(&(d_neurons->p_i_syn[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
			gSynapsesFiredTable[sid] = false;
		}
	}
	__syncthreads();
}
