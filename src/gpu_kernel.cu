#include "hip/hip_runtime.h"

#include "gpu_kernel.h"

#define MAX_FIRED 512

__device__ PlainNetwork * pGpuNet;
__device__ unsigned int *gTimeTable;
__device__ bool *gSynapsesFiredTable;
__device__ unsigned int *gFiredTable;
__device__ unsigned int gFiredCnt;
__device__ unsigned int gFiredCntTest;

__device__ double atomicAdd(double *address, double val)
{
	unsigned long long int *address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed = 0;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);

	return __longlong_as_double(old);
}

__device__ int updateTimeTable(unsigned int simTime)
{
	if ((threadIdx.x == 0) && (blockDim.x == 0)) {
		gTimeTable[simTime + pGpuNet->MAX_DELAY + 1] = gFiredCnt;
	}
	__syncthreads();
	return 0;
}

__device__ int updateFiredTable(int *fireTable, unsigned int fireCnt, unsigned int simTime)
{
	__shared__ volatile unsigned int cnt;
	unsigned int idx = 0;
	if (threadIdx.x == 0) {
		cnt = atomicAdd(&gFiredCntTest, fireCnt);
		//TODO: check over add items
		cnt = atomicAdd(&gFiredCnt, fireCnt);
	}
	__syncthreads();

	for (int i=threadIdx.x; i<fireCnt; i+=blockDim.x) {
		idx = atomicAdd((unsigned int*)&cnt, 1);
		gFiredTable[idx] = fireTable[i];
	}
	return 0;
}

__global__ void init_global(PlainNetwork * c_pGpuNet, unsigned int *c_gTimeTable, unsigned int *c_gFiredTable, bool *c_gSynapsesFiredTable) {
	pGpuNet = c_pGpuNet;
	gTimeTable = c_gTimeTable;
	gFiredTable = c_gFiredTable;
	gSynapsesFiredTable = c_gSynapsesFiredTable;
	gFiredCnt = 0;
	gFiredCntTest = 0;
}

__global__ void update_postSynapse(GLIFNeurons *d_neurons, GExpSynapses* d_synapses, unsigned int simTime)
{
	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idx_s = gTimeTable[simTime];	
	unsigned int idx_e = gTimeTable[simTime+pGpuNet->MAX_DELAY];
	for (unsigned int idx = idx_s + threadIdx.x; idx <= idx_e; idx += blockDim.x) {
		unsigned int nid = gFiredTable[idx];
		for (unsigned int i=0; i<d_neurons->pSynapsesNum[nid]; i++) {
			unsigned int loc = d_neurons->pSynapsesLoc[nid];
			gSynapsesFiredTable[d_neurons->pSynapsesIdx[i+loc]] = true;
		}
	}
	__syncthreads();
}

__global__ void update_lif_neuron(GLIFNeurons *d_neurons, int num, unsigned int simTime)
{
	__shared__ volatile unsigned int fireCnt;
	__shared__ volatile unsigned int fireCntTest;
	__shared__ int fireTable[MAX_FIRED];

	int idx = 0;
	if (threadIdx.x == 0) {
		fireCnt = 0;
		fireCntTest = 0;
	}

	__syncthreads();

	bool fired = false;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int nid = tid;
	if (nid < num) {
		real I = d_neurons->p_i_syn[nid] + d_neurons->p_i_tmp[nid];
		d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
		d_neurons->p_i_syn[nid] = 0;

		if (d_neurons->p_refrac_step[nid] > 0) {
			d_neurons->p_refrac_step[nid] --;
			d_neurons->p_vm[nid] = 0;
		}

		if (d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid]) {
			fired = true;
			//d_neurons[nid].refrac_step = (int)(d_neurons[nid].tau_refrac/d_neurons[nid]._dt) - 1;
			//d_neurons[nid].vm = d_neurons[nid].v_reset;
		}


	}
	__syncthreads();

	if (fired) {
		idx = atomicAdd((unsigned int *)&fireCntTest, 1);
	}
	if (fired && idx < MAX_FIRED) {
		idx = atomicAdd((unsigned int *)&fireCnt, 1);
		fired = false;
		fireTable[idx] = nid;
	}
	__syncthreads();

	if (fireCnt) {
		updateFiredTable(fireTable, fireCnt, simTime);
	}
	__syncthreads();
}

__global__ void update_alpha_synapse(GAlphaSynapses *d_synapses, unsigned int num, unsigned int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int sid = tid;
	if (sid < num) {
		d_synapses->p_I_syn[sid] = d_synapses->p_C1[sid] * d_synapses->p_I_syn[sid] + d_synapses->p_C2[sid] * d_synapses->p_I_tmp[sid];
		d_synapses->p_I_tmp[sid] *= d_synapses->p_C1[sid];
	}
	__syncthreads();
}

__global__ void update_exp_synapse(GLIFNeurons *d_neurons, GExpSynapses *d_synapses, unsigned int num, unsigned int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int sid = tid;
	if (sid < num) {
		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
	}
	__syncthreads();

	if (sid < num) {
		if (gSynapsesFiredTable[sid]) {
			atomicAdd(&(d_neurons->p_vm[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
		}
	}
	__syncthreads();
}
