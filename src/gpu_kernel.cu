#include "hip/hip_runtime.h"
#include "Neuron.h"
#include "Synapse.h"

#define MAX_DELAY 10
#define MAX_FIRED 1000
#define ONESECOND 1000

__device__ unsigned int gTimeTable[ONESECOND];
__device__ unsigned int *gFiredTable;
__device__ unsigned int gFiredCnt;
__device__ unsigned int gFiredCntTest;

__device__ double atomicAdd(double *address, double val)
{
	unsigned long long int *address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed = 0;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);

	return __longlong_as_double(old);
}

__device__ int updateTimeTable(unsigned int simTime)
{
	if ((threadIdx.x == 0) && (blockDim.x == 0)) {
		gTimeTable[simTime + MAX_DELAY + 1] = gFiredCnt;
	}
	__syncthreads();
	return 0;
}

__device__ int updateFiredTable(int *fireTable, unsigned int fireCnt, unsigned int simTime)
{
	__shared__ volatile unsigned int cnt;
	unsigned int idx = 0;
	if (threadIdx.x == 0) {
		cnt = atomicAdd(&gFiredCntTest, fireCnt);
		//check over add items
		cnt = atomicAdd(&gFiredCnt, fireCnt);
	}
	__syncthreads();

	for (int i=threadIdx.x; i<fireCnt; i+=blockDim.x) {
		idx = atomicAdd((unsigned int*)&cnt, 1);
		gFiredTable[idx] = fireTable[i];
	}
	return 0;
}

__global__ void postSynapse()
{

}

__global__ void update_lif_neuron(GLIFNeuron *d_neurons, int num, unsigned int simTime)
{
	__shared__ volatile unsigned int fireCnt;
	__shared__ volatile unsigned int fireCntTest;
	__shared__ int fireTable[MAX_FIRED];

	int idx = 0;
	if (threadIdx.x == 0) {
		fireCnt = 0;
	}

	__syncthreads();

	bool fired = false;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int nid = tid;
	if (nid < num) {
		real I = d_neurons[nid].i_syn + d_neurons[nid].i_tmp;
		d_neurons[nid].vm = d_neurons[nid].vm * d_neurons[nid].C1 + d_neurons[nid].C2 * I; 
		d_neurons[nid].i_syn = 0;

		if (d_neurons[nid].refrac_step > 0) {
			d_neurons[nid].refrac_step --;
			d_neurons[nid].vm = 0;
		}

		if (d_neurons[nid].vm >= d_neurons[nid].v_thresh) {
			fired = true;
			//d_neurons[nid].refrac_step = (int)(d_neurons[nid].tau_refrac/d_neurons[nid]._dt) - 1;
			//d_neurons[nid].vm = d_neurons[nid].v_reset;
		}


	}
	__syncthreads();

	if (fired) {
		idx = atomicAdd((int*)&fireCntTest, 1);
	}
	if (fired && idx < MAX_FIRED) {
		idx = atomicAdd((int*)&fireCnt, 1);
		fired = false;
		fireTable[idx] = nid;
	}
	__syncthreads();
	if (fireCnt) {
		updateFiredTable(fireTable, fireCnt, simTime);
	}
}

__global__ void update_alpha_synapse(GAlphaSynapse *d_synapses, int num)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int sid = tid;
	if (sid < num) {
		d_synapses[sid].I_syn = d_synapses[sid].C1 * d_synapses[sid].I_syn + d_synapses[sid].C2 * d_synapses[sid].I_tmp;
		d_synapses[sid].I_tmp *= d_synapses[sid].C1;
	}
}

__global__ void update_exp_synapse(GExpSynapse *d_synapses, int num)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int sid = tid;
	if (tid < num) {
		d_synapses[sid].I_syn *= d_synapses[sid].C1;
	}
}
