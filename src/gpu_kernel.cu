#include "hip/hip_runtime.h"

#include "gpu_macros.h"
#include "gpu_kernel.h"

#define MAX_FIRED 512

__device__ int MAX_DELAY;
__device__ int gCurrentTime;
__device__ int *gTimeTable;
__device__ int gTimeTableSize;
__device__ real *gNeuronInput;
__device__ int gNeuronNum;
__device__ int *gFiredTable;
__device__ int gFiredTableSize;
__device__ int gFiredCnt;
__device__ int gFiredCntTest;
__device__ bool *gSynapsesFiredTable;
__device__ int gSynapsesFiredTableSize;
__device__ GNetwork *gGpuNet;

__device__ int get_type(int *array, int num, int value, int *offset)
{
	for (int i=0; i<num; i++) {
		if (array[i+1] > value) {
			if (offset != NULL) {
				*offset = value - array[i];
			}

			return i;
		}
	}
	return 0;
}

//int gnid = get_gid(gGpuNet->neuronNums, gGpuNet->nTypes, gGpuNet->nTypeNum, LIF, nid);
//__device__ int get_gid(int *array, Type *types, int num, int type, int offset)
__device__ int type2gnid(GNetwork *net, Type type, int offset)
{
	for (int i=0; i<net->nTypeNum; i++) {
		if (net->nTypes[i] == type) {
			return (net->gNeuronNums[i] + net->nOffsets[i] + offset);
		}
	}

	return 0;
}

__device__ int get_gnid(GNetwork *net, int type, int offset)
{
	return (net->gNeuronNums[type] + net->nOffsets[type] + offset);
}

__device__ int get_gsid(GNetwork *net, int type, int offset)
{
	return (net->gSynapseNums[type] + net->sOffsets[type] + offset);
	//for (int i=0; i<net->sTypeNum; i++) {
	//	if (net->sTypes[i] == type) {
	//		return (net->gSynapseNums[i] + net->sOffsets[i] + offset);
	//	}
	//}

	//return 0;
}

//__device__ double atomicAdd(double *address, double val)
//{
//	unsigned long long int *address_as_ull = (unsigned long long int*)address;
//	unsigned long long int old = *address_as_ull, assumed = 0;
//	do {
//		assumed = old;
//		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
//	} while (assumed != old);
//
//	return __longlong_as_double(old);
//}

__device__ int updateTimeTable(int simTime)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gTimeTable[gCurrentTime] = simTime;
		gCurrentTime = (gCurrentTime +1)%(MAX_DELAY + 1);
	}
	__syncthreads();
	return 0;
}

__device__ int updateFiredTable(int firedID, int fired, int simTime)
{
	gFiredTable[gCurrentTime*gFiredTableSize + firedID] = fired;

	return 0;
}

__global__ void init_global(int max_delay, int *c_gTimeTable, real *c_gNeuronInput, int *c_gFiredTable, int c_gFiredTableSize, bool *c_gSynapsesFiredTable, int c_gSynapsesFiredTableSize, GNetwork* network) 
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		MAX_DELAY = max_delay;
		gCurrentTime = 0;
		gTimeTable = c_gTimeTable;
		gTimeTableSize = MAX_DELAY + 1;
		gNeuronInput = c_gNeuronInput;
		gNeuronNum = c_gFiredTableSize;
		gFiredTable = c_gFiredTable;
		gFiredTableSize = c_gFiredTableSize;
		gSynapsesFiredTable = c_gSynapsesFiredTable;
		gSynapsesFiredTableSize = c_gSynapsesFiredTableSize;
		gFiredCnt = 0;
		gFiredCntTest = 0;
		gGpuNet = network;
	}
}


//__device__ int get_alpha_delay(void *data, int num, int sid)
//{
//	GAlphaSynapses *d_synapses = (GAlphaSynapses*)data;
//	return (int)(d_synapses->p_delay_steps[sid]);
//}
//
//__device__ int get_exp_delay(void *data, int num, int sid)
//{
//	GExpSynapses *d_synapses = (GExpSynapses*)data;
//	return (int)(d_synapses->p_delay_steps[sid]);
//}

GET_DELAY(basic, Basic)
GET_DELAY(alpha, Alpha)
GET_DELAY(exp, Exp)

__device__ int (*get_delay[])(void *, int, int) = { NULL, NULL, get_basic_delay, get_alpha_delay, get_exp_delay };

__device__ int update_constant_spike(void *data, int num, int nid, int start_t, int simTime)
{
	if (nid >= num) {
		return -1;
	}

	GConstantNeurons *d_neurons = (GConstantNeurons*)data;
	for (int i=0; i<d_neurons->pSynapsesNum[nid]; i++) {
		int loc = d_neurons->pSynapsesLoc[nid];
		int sid = d_neurons->pSynapsesIdx[i+loc];
		int offset = 0;
		int type = get_type(gGpuNet->synapseNums, gGpuNet->sTypeNum, sid, &offset);
		//if (simTime == start_t + get_delay(d_synapses->p_delay[sid]/d_synapses->p__dt[sid]))
		if (simTime == start_t + get_delay[gGpuNet->sTypes[type]](gGpuNet->pSynapses[type], gGpuNet->synapseNums[type+1]-gGpuNet->synapseNums[type], offset))
			gSynapsesFiredTable[d_neurons->pSynapsesIdx[i+loc]] = true;
	}

	return 0;
}

__device__ int update_lif_spike(void *data, int num, int nid, int start_t, int simTime)
{
	if (nid >= num) {
		return -1;
	}

	GLIFNeurons *d_neurons = (GLIFNeurons*)data;
	for (int i=0; i<d_neurons->pSynapsesNum[nid]; i++) {
		int loc = d_neurons->pSynapsesLoc[nid];
		int sid = d_neurons->pSynapsesIdx[i+loc];
		int offset = 0;
		int type = get_type(gGpuNet->synapseNums, gGpuNet->sTypeNum, sid, &offset);
		//if (simTime == start_t + get_delay(d_synapses->p_delay[sid]/d_synapses->p__dt[sid]))
		if (simTime == start_t + get_delay[gGpuNet->sTypes[type]](gGpuNet->pSynapses[type], gGpuNet->synapseNums[type+1]-gGpuNet->synapseNums[type], offset))
			gSynapsesFiredTable[d_neurons->pSynapsesIdx[i+loc]] = true;
	}

	return 0;
}

//UPDATE_NEURON_SPIKE(constant, Constant)
//UPDATE_NEURON_SPIKE(lif, LIF)

__device__ int update_basic_spike(void *data, int num, int sid, int start_t, int simTime)
{
	if (sid >= num) {
		return -1;
	}

	GBasicSynapses *d_synapses = (GBasicSynapses*)data;
	atomicAdd(&(gNeuronInput[d_synapses->pDst[sid]]), d_synapses->p_weight[sid]);

	return 0;
}

__device__ int update_alpha_spike(void *data, int num, int sid, int start_t, int simTime)
{
	if (sid >= num) {
		return -1;
	}

	GAlphaSynapses *d_synapses = (GAlphaSynapses*)data;
	real I_t = d_synapses->p__C2[sid] * d_synapses->p_I_syn[sid] + d_synapses->p__C2[sid] * d_synapses->p_I_tmp[sid];
	d_synapses->p_I_tmp[sid] += d_synapses->p_weight[sid]/d_synapses->p__C1[sid];
	d_synapses->p_I_syn[sid] = (I_t - d_synapses->p__C2[sid] * d_synapses->p_I_tmp[sid])/d_synapses->p__C1[sid];
	//atomicAdd(&(d_neurons->p_i_syn[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
	atomicAdd(&(gNeuronInput[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);

	return 0;
}

__device__ int update_exp_spike(void *data, int num, int sid, int start_t, int simTime)
{
	if (sid >= num) {
		return -1;
	}

	GExpSynapses *d_synapses = (GExpSynapses*)data;
	d_synapses->p_I_syn[sid] += (d_synapses->p_weight[sid]/d_synapses->p__C1[sid]);
	//atomicAdd(&(d_neurons->p_i_syn[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
	atomicAdd(&(gNeuronInput[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);

	return 0;
}


__device__ int (*update_spike[])(void*, int, int, int, int) = { update_constant_spike, update_lif_spike, update_basic_spike, update_alpha_spike, update_exp_spike };

__global__ void update_pre_synapse(GNetwork *d_net, int simTime)
{
	for (int time = 0; time<MAX_DELAY+1; time++) {
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		int start_t = gTimeTable[time];
		for (int idx = tid; idx < gFiredTableSize; idx += blockDim.x*gridDim.x) {
			int nid = idx;
			int offset = 0;
			int type = get_type(d_net->neuronNums, d_net->nTypeNum, nid, &offset);
			int gnid = get_gnid(d_net, type, offset);
			if (gFiredTable[time*gFiredTableSize + gnid] > 0) {
				update_spike[d_net->nTypes[type]](d_net->pNeurons[type], d_net->neuronNums[type+1]-d_net->neuronNums[type], offset, start_t, simTime);
			}
		}
	}
	__syncthreads();
}

__global__ void update_post_synapse(GNetwork *d_net, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int num = d_net->synapseNums[d_net->sTypeNum];
	for (int idx = tid; idx<num; idx += blockDim.x*gridDim.x) {
		int sid = idx;
		int offset = 0;
		int type = get_type(d_net->synapseNums, d_net->sTypeNum, sid, &offset);
		int gsid = get_gsid(d_net, type, offset);
		if (gSynapsesFiredTable[gsid]) {
			update_spike[d_net->sTypes[type]](d_net->pSynapses[type], d_net->synapseNums[type+1]-d_net->neuronNums[type], offset, 0, simTime);
			gSynapsesFiredTable[gsid] = false;
		}
	}
}

__global__ void update_pre_synapse(GLIFNeurons *d_neurons, GExpSynapses* d_synapses, int simTime)
{
	for (int time = 0; time<MAX_DELAY+1; time++) {
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		int start_t = gTimeTable[time];
		for (int idx = tid; idx < gFiredTableSize; idx += blockDim.x*gridDim.x) {
			int nid = idx;
			if (gFiredTable[time*gFiredTableSize + nid]) {
				for (int i=0; i<d_neurons->pSynapsesNum[nid]; i++) {
					int loc = d_neurons->pSynapsesLoc[nid];
					int sid = d_neurons->pSynapsesIdx[i+loc];
					if (simTime == start_t + d_synapses->p_delay_steps[sid])
						gSynapsesFiredTable[d_neurons->pSynapsesIdx[i+loc]] = true;
				}
			}
		}
	}
	__syncthreads();
}

__global__ void update_constant_neuron(GConstantNeurons *d_neurons, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int nid = 0;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		nid = idx;
		if (nid < num) {
			int fired = (simTime * d_neurons->p_fire_rate[nid]) > (d_neurons->p_fire_count[nid]);
			int gnid = type2gnid(gGpuNet, Constant, nid);
			if (fired) {
				d_neurons->p_fire_count[nid]++;
			}
			updateFiredTable(gnid, fired, simTime);
		}
	}
}

__global__ void update_lif_neuron(GLIFNeurons *d_neurons, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int nid = 0;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		nid = idx;
		if (nid < num) {
			int fired = 0;
			//real I = d_neurons->p_i_syn[nid] + d_neurons->p_i_tmp[nid];
			int gnid = type2gnid(gGpuNet, LIF, nid);
			real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
			d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
			//d_neurons->p_i_syn[nid] = 0;
			gNeuronInput[gnid] = 0;

			if (d_neurons->p_refrac_step[nid] > 0) {
				d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
				d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
			} else if (d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid]) {
				fired = 1;
				d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
				d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
			}

			updateFiredTable(gnid, fired, simTime);
		}
	}
	__syncthreads();

	updateTimeTable(simTime);
	__syncthreads();
}

__global__ void update_basic_synapse(GBasicSynapses *d_synapses, int num, int simTime)
{
	__syncthreads();
}

__global__ void update_alpha_synapse(GAlphaSynapses *d_synapses, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
		int sid = idx;
		if (sid < num) {
			d_synapses->p_I_syn[sid] = d_synapses->p_C1[sid] * d_synapses->p_I_syn[sid] + d_synapses->p_C2[sid] * d_synapses->p_I_tmp[sid];
			d_synapses->p_I_tmp[sid] *= d_synapses->p_C1[sid];

			
			//if (gSynapsesFiredTable[sid]) {
			//	real I_t = d_synapses->p__C2[sid] * d_synapses->p_I_syn[sid] + d_synapses->p__C2[sid] * d_synapses->p_I_tmp[sid];
			//	d_synapses->p_I_tmp[sid] += d_synapses->p_weight[sid]/d_synapses->p__C1[sid];
			//	d_synapses->p_I_syn[sid] = (I_t - d_synapses->p__C2[sid] * d_synapses->p_I_tmp[sid])/d_synapses->p__C1[sid];
			//	atomicAdd(&(d_neurons->p_i_syn[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
			//	gSynapsesFiredTable[sid] = false;
			//}
		}
	}
	__syncthreads();
}

__global__ void update_exp_synapse(GExpSynapses *d_synapses, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
		int sid = idx;
		if (sid < num) {
			d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
		}

		//if (gSynapsesFiredTable[sid]) {
		//	d_synapses->p_I_syn[sid] += (d_synapses->p_weight[sid]/d_synapses->p__C1[sid]);
		//	atomicAdd(&(d_neurons->p_i_syn[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
		//	gSynapsesFiredTable[sid] = false;
		//}
	}
	__syncthreads();
}
