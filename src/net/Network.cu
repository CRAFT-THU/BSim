
#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "Network.h"

//Alloc cross device gmem, will set device to 0 before return
CrossNodeDataGPU* Network::arrangeCrossNodeDataGPU(int node_num)
{
	CrossNodeDataGPU * cross_data = static_cast<CrossNodeDataGPU*>(malloc(sizeof(CrossNodeDataGPU)));
	assert(cross_data != NULL);

	cross_data->_maxNum = static_cast<int*>(malloc(sizeof(int)*node_num*node_num));
	assert(cross_data->_maxNum != NULL);
	cross_data->_firedArrays = static_cast<int**>(malloc(sizeof(int*)*node_num*node_num));
	assert(cross_data->_firedArrays != NULL);

	checkCudaErrors(hipHostMalloc((void**)&(cross_data->_firedNum), sizeof(int)*node_num*node_num));
	checkCudaErrors(hipMemset(cross_data->_firedNum, 0, sizeof(int)*node_num*node_num));


	for (int i=0; i<_nodeNum; i++) {
		for (int j=0; j<_nodeNum; j++) {
			// i->j 
			checkCudaErrors(hipSetDevice(j));
			int i2j = i * _nodeNum + j;
			cross_data->_firedNum[i2j] = 0;

			int count = 0;
			for (auto iter = _crossnodeNeuronsSend[i].begin(); iter != _crossnodeNeuronsSend[i].end(); iter++) {
				if (_crossnodeNeuronsRecv[j].find(*iter) != _crossnodeNeuronsRecv[j].end()) {
					count++;
				}
			}
			cross_data->_maxNum[i2j] = count;
			if (count > 0) {
				checkCudaErrors(hipMalloc((void**)&(cross_data->_firedArrays[i2j]), sizeof(int)*count));
			} else {
				cross_data->_firedArrays[i2j] = NULL;
			}
		}
	}

	checkCudaErrors(hipSetDevice(0));

	return cross_data;
}

