#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../neuron/lif/GLIFNeurons.h"
#include "../utils/TypeFunc.h"
#include "../utils/macros.h"
#include "../gpu_utils/mem_op.h"
#include "GNetwork.h"

GNetwork* copyNetworkToGPU(GNetwork *pCpuNet)
{
	if (pCpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		return NULL;
	}

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;

	GNetwork *tmp = allocNetwork(nTypeNum, sTypeNum);

	// tmp->maxDelay = pCpuNet->maxDelay;
	// tmp->minDelay = pCpuNet->minDelay;

	for (int i=0; i<nTypeNum; i++) {
		tmp->pNTypes[i] = pCpuNet->pNTypes[i];
		tmp->pNeuronNums[i] = pCpuNet->pNeuronNums[i];
	}
	tmp->pNeuronNums[nTypeNum] = pCpuNet->pNeuronNums[nTypeNum];

	for (int i=0; i<sTypeNum; i++) {
		tmp->pSTypes[i] = pCpuNet->pSTypes[i];
		tmp->pSynapseNums[i] = pCpuNet->pSynapseNums[i];
	}
	tmp->pSynapseNums[sTypeNum] = pCpuNet->pSynapseNums[sTypeNum];



	for (int i=0; i<nTypeNum; i++) {
		tmp->ppNeurons[i] = cudaAllocType[pCpuNet->pNTypes[i]](pCpuNet->ppNeurons[i], pCpuNet->pNeuronNums[i+1]-pCpuNet->pNeuronNums[i]);
	}

	for (int i=0; i<sTypeNum; i++) {
		tmp->ppSynapses[i] = cudaAllocType[pCpuNet->pSTypes[i]](pCpuNet->ppSynapses[i], pCpuNet->pSynapseNums[i+1]-pCpuNet->pSynapseNums[i]);
	}

	tmp->pConnection = cudaAllocConnection(pCpuNet->pConnection);

	return tmp;
}

int fetchNetworkFromGPU(GNetwork *pCpuNet, GNetwork *pGpuNet)
{
	if (pCpuNet == NULL && pGpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		return -1;
	}

	int nTypeNum = pCpuNet->nTypeNum;

	//TODO support multitype N and S
	for (int i=0; i<nTypeNum; i++) {
		//TODO: cudaFetchType
		//cudaFetchType[pCpuNet->pNTypes[i]](pGpuNet->ppNeurons[i], pCpuNet->ppNeurons[i], pCpuNet->pNeuronNums[i+1]-pCpuNet->pNeuronNums[i]);
	}

	return 0;
}

int freeNetworkGPU(GNetwork *pGpuNet)
{
	GNetwork *pTmp = pGpuNet;

	int nTypeNum = pTmp->nTypeNum;
	int sTypeNum = pTmp->sTypeNum;

	for (int i=0; i<nTypeNum; i++) {
		void *pTmpN = mallocType[pTmp->pNTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpN, pTmp->ppNeurons[i], getTypeSize[pTmp->pNTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmp->pNTypes[i]](pTmpN);
		checkCudaErrors(hipFree(pTmp->ppNeurons[i]));
		free(pTmpN);
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pTmpS = mallocType[pTmp->pSTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpS, pTmp->ppSynapses[i], getTypeSize[pTmp->pSTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmp->pSTypes[i]](pTmpS);
		checkCudaErrors(hipFree(pTmp->ppSynapses[i]));
		free(pTmpS);
	}

	Connection * pConnection = (Connection*)malloc(sizeof(Connection));
	checkCudaErrors(hipMemcpy(pConnection, pTmp->pConnection, sizeof(Connection), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipFree(pConnection->ppSynapsesIdx));
	checkCudaErrors(hipFree(pConnection->pDelayStart));
	checkCudaErrors(hipFree(pConnection->pDelayNum));
	checkCudaErrors(hipFree(pTmp->pConnection));
	free(pConnection);

	free(pTmp->ppNeurons);
	free(pTmp->ppSynapses);
	free(pTmp);

	return 0;
}



int checkGPUNetwork(GNetwork *g, GNetwork *c)
{
	int ret = -1;

	CHECK(g, c, nTypeNum);
	CHECK(g, c, sTypeNum);
	CHECK(g, c, pNeuronNums);
	CHECK(g, c, pSynapseNums);

	CHECK(g, c, pNTypes);
	CHECK(g, c, pSTypes);

	ret = 1;

	//int totalNeuronNum = g->pNeuronNums[g->nTypeNum+1];
	//int totalSynapseNum = g->pSynapseNums[g->sTypeNum+1];
	// Connection p;
	// checkCudaErrors(hipMemcpy(&p, g->pConnection, sizeof(Connection), hipMemcpyDeviceToHost));

	// CHECK_GPU_TO_CPU_ARRAY(p.delayStart, c->pConnection->delayStart, sizeof(int)*(c->pConnection->nNum)*(maxDelay-minDelay+1));
	// CHECK_GPU_TO_CPU_ARRAY(p.delayNum, c->pConnection->delayNum, sizeof(int)*(c->pConnection->nNum)*(maxDelay-minDelay+1));

	ret = 2;

	return ret;
}
