#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../neuron/lif/LIFData.h"
#include "../utils/TypeFunc.h"
#include "../utils/macros.h"
#include "../gpu_utils/mem_op.h"
#include "GNetwork.h"

GNetwork* copyNetworkToGPU(GNetwork *pCpuNet)
{
	if (pCpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		return NULL;
	}

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;

	GNetwork *tmp = allocNetwork(nTypeNum, sTypeNum);

	// tmp->maxDelay = pCpuNet->maxDelay;
	// tmp->minDelay = pCpuNet->minDelay;

	for (int i=0; i<nTypeNum; i++) {
		tmp->pNTypes[i] = pCpuNet->pNTypes[i];
		tmp->pNeuronNums[i] = pCpuNet->pNeuronNums[i];
	}
	tmp->pNeuronNums[nTypeNum] = pCpuNet->pNeuronNums[nTypeNum];

	for (int i=0; i<sTypeNum; i++) {
		tmp->pSTypes[i] = pCpuNet->pSTypes[i];
		tmp->pSynapseNums[i] = pCpuNet->pSynapseNums[i];
	}
	tmp->pSynapseNums[sTypeNum] = pCpuNet->pSynapseNums[sTypeNum];


	for (int i=0; i<nTypeNum; i++) {
		tmp->ppNeurons[i] = cudaAllocType[pCpuNet->pNTypes[i]](pCpuNet->ppNeurons[i], pCpuNet->pNeuronNums[i+1]-pCpuNet->pNeuronNums[i]);
	}

	for (int i=0; i<sTypeNum; i++) {
		tmp->ppSynapses[i] = cudaAllocType[pCpuNet->pSTypes[i]](pCpuNet->ppSynapses[i], pCpuNet->pSynapseNums[i+1]-pCpuNet->pSynapseNums[i]);
	}

	tmp->pConnection = cudaAllocConnection(pCpuNet->pConnection);
	tmp->pLolConnection = cudaAllocLolConnection(pCpuNet->pLolConnection);

	return tmp;
}

int fetchNetworkFromGPU(GNetwork *pCpuNet, GNetwork *pGpuNet)
{
	if (pCpuNet == NULL || pGpuNet == NULL) {
		printf("NULL POINTER: FETCH GNETWORK\n");
		return -1;
	}

	int nTypeNum = pGpuNet->nTypeNum;
	int sTypeNum = pGpuNet->sTypeNum;

	assert(pCpuNet->nTypeNum == nTypeNum);
	assert(pCpuNet->sTypeNum == sTypeNum);

	for (int i=0; i<nTypeNum; i++) {
		pCpuNet->pNTypes[i] = pGpuNet->pNTypes[i];
		pCpuNet->pNeuronNums[i] = pGpuNet->pNeuronNums[i];
	}
	pCpuNet->pNeuronNums[nTypeNum] = pGpuNet->pNeuronNums[nTypeNum];

	for (int i=0; i<sTypeNum; i++) {
		pCpuNet->pSTypes[i] = pGpuNet->pSTypes[i];
		pCpuNet->pSynapseNums[i] = pGpuNet->pSynapseNums[i];
	}
	pCpuNet->pSynapseNums[sTypeNum] = pGpuNet->pSynapseNums[sTypeNum];

	//TODO support multitype N and S
	for (int i=0; i<nTypeNum; i++) {
		//TODO: cudaFetchType
		cudaFetchType[pCpuNet->pNTypes[i]](pCpuNet->ppNeurons[i], pGpuNet->ppNeurons[i], pCpuNet->pNeuronNums[i+1]-pCpuNet->pNeuronNums[i]);
	}
	for (int i=0; i<sTypeNum; i++) {
		//TODO: cudaFetchType
		cudaFetchType[pCpuNet->pSTypes[i]](pCpuNet->ppSynapses[i], pGpuNet->ppSynapses[i], pCpuNet->pSynapseNums[i+1]-pCpuNet->pSynapseNums[i]);
	}

	cudaFetchConnection(pCpuNet->pConnection, pGpuNet->pConnection);
	return 0;
}

int freeNetworkGPU(GNetwork *pGpuNet)
{
	GNetwork *pTmp = pGpuNet;

	int nTypeNum = pTmp->nTypeNum;
	int sTypeNum = pTmp->sTypeNum;

	for (int i=0; i<nTypeNum; i++) {
		cudaFreeType[pTmp->pNTypes[i]](pTmp->ppNeurons[i]);
	}

	for (int i=0; i<sTypeNum; i++) {
		cudaFreeType[pTmp->pSTypes[i]](pTmp->ppSynapses[i]);
	}

	cudaFreeConnection(pTmp->pConnection);
	cudaFreeLolConnection(pTmp->pLolConnection);

	free(pTmp->ppNeurons);
	free(pTmp->ppSynapses);
	free(pTmp);

	return 0;
}



int checkNetworkGPU(GNetwork *g, GNetwork *c)
{
	int ret = -1;

	CHECK(g, c, nTypeNum);
	CHECK(g, c, sTypeNum);
	CHECK(g, c, pNeuronNums);
	CHECK(g, c, pSynapseNums);

	CHECK(g, c, pNTypes);
	CHECK(g, c, pSTypes);

	ret = 1;

	//int totalNeuronNum = g->pNeuronNums[g->nTypeNum+1];
	//int totalSynapseNum = g->pSynapseNums[g->sTypeNum+1];
	// Connection p;
	// checkCudaErrors(hipMemcpy(&p, g->pConnection, sizeof(Connection), hipMemcpyDeviceToHost));

	// CHECK_GPU_TO_CPU_ARRAY(p.delayStart, c->pConnection->delayStart, sizeof(int)*(c->pConnection->nNum)*(maxDelay-minDelay+1));
	// CHECK_GPU_TO_CPU_ARRAY(p.delayNum, c->pConnection->delayNum, sizeof(int)*(c->pConnection->nNum)*(maxDelay-minDelay+1));

	ret = 2;

	return ret;
}
