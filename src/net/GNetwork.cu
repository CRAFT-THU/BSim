#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../neuron/lif/GLIFNeurons.h"
#include "../utils/TypeFunc.h"
#include "../utils/macros.h"
#include "../gpu_utils/mem_op.h"
#include "GNetwork.h"

GNetwork* copyNetworkToGPU(GNetwork *pCpuNet)
{
	if (pCpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		return NULL;
	}

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;

	GNetwork *tmp = allocNetwork(nTypeNum, sTypeNum);

	// tmp->maxDelay = pCpuNet->maxDelay;
	// tmp->minDelay = pCpuNet->minDelay;

	for (int i=0; i<nTypeNum; i++) {
		tmp->pNTypes[i] = pCpuNet->pNTypes[i];
		tmp->pNeuronNums[i] = pCpuNet->pNeuronNums[i];
	}
	tmp->pNeuronNums[nTypeNum] = pCpuNet->pNeuronNums[nTypeNum];

	for (int i=0; i<sTypeNum; i++) {
		tmp->pSTypes[i] = pCpuNet->pSTypes[i];
		tmp->pSynapseNums[i] = pCpuNet->pSynapseNums[i];
	}
	tmp->pSynapseNums[sTypeNum] = pCpuNet->pSynapseNums[sTypeNum];



	for (int i=0; i<nTypeNum; i++) {
		tmp->ppNeurons[i] = cudaAllocType[pCpuNet->pNTypes[i]](pCpuNet->ppNeurons[i], pCpuNet->pNeuronNums[i+1]-pCpuNet->pNeuronNums[i]);
	}

	for (int i=0; i<sTypeNum; i++) {
		tmp->ppSynapses[i] = cudaAllocType[pCpuNet->pSTypes[i]](pCpuNet->ppSynapses[i], pCpuNet->pSynapseNums[i+1]-pCpuNet->pSynapseNums[i]);
	}

	tmp->pConnection = copyConnectionToGPU(pCpuNet->pConnection);

	return tmp;
}

int fetchNetworkFromGPU(GNetwork *pCpuNet, GNetwork *pGpuNet)
{
	if (pCpuNet == NULL && pGpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		return -1;
	}

	int nTypeNum = pCpuNet->nTypeNum;

	//TODO support multitype N and S
	for (int i=0; i<nTypeNum; i++) {
		//TODO: cudaFetchType
		//cudaFetchType[pCpuNet->nTypes[i]](pGpuNet->pNeurons[i], pCpuNet->pNeurons[i], pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i]);
	}

	return 0;
}

int freeNetworkGPU(GNetwork *pGpuNet)
{
	GNetwork *pTmp = pGpuNet;

	int nTypeNum = pTmp->nTypeNum;
	int sTypeNum = pTmp->sTypeNum;

	for (int i=0; i<nTypeNum; i++) {
		void *pTmpN = mallocType[pTmp->pNTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpN, pTmp->ppNeurons[i], getTypeSize[pTmp->pNTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmp->nTypes[i]](pTmpN);
		checkCudaErrors(hipFree(pTmp->pNeurons[i]));
		free(pTmpN);
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pTmpS = mallocType[pTmp->sTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpS, pTmp->pSynapses[i], getTypeSize[pTmp->sTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmp->sTypes[i]](pTmpS);
		checkCudaErrors(hipFree(pTmp->pSynapses[i]));
		free(pTmpS);
	}

	N2SConnection * pConnection = (N2SConnection*)malloc(sizeof(N2SConnection));
	checkCudaErrors(hipMemcpy(pConnection, pTmp->pN2SConnection, sizeof(N2SConnection), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipFree(pConnection->pSynapsesIdx));
	checkCudaErrors(hipFree(pConnection->delayStart));
	checkCudaErrors(hipFree(pConnection->delayNum));
	checkCudaErrors(hipFree(pTmp->pN2SConnection));
	free(pConnection);

	free(pTmp->ppNeurons);
	free(pTmp->ppSynapses);
	free(pTmp);

	return 0;
}



int checkGPUNetwork(GNetwork *g, GNetwork *c)
{
	int ret = -1;

	CHECK(g, c, nTypeNum);
	CHECK(g, c, sTypeNum);
	CHECK(g, c, neuronNums);
	CHECK(g, c, synapseNums);

	CHECK(g, c, nTypes);
	CHECK(g, c, sTypes);

	ret = 1;

	//int totalNeuronNum = g->neuronNums[g->nTypeNum+1];
	//int totalSynapseNum = g->synapseNums[g->sTypeNum+1];
	int maxDelay = c->maxDelay;
	int minDelay = c->minDelay;

	N2SConnection p;
	checkCudaErrors(hipMemcpy(&p, g->pN2SConnection, sizeof(N2SConnection), hipMemcpyDeviceToHost));

	//CHECK_CROSS_ARRAY(p.pSynapsesIdx, c->pN2SConnection->pSynapsesIdx, sizeof(int)*(c->pN2SConnection->s_num));
	CHECK_GPU_TO_CPU_ARRAY(p.delayStart, c->pN2SConnection->delayStart, sizeof(int)*(c->pN2SConnection->nNum)*(maxDelay-minDelay+1);
	CHECK_GPU_TO_CPU_ARRAY(p.delayNum, c->pN2SConnection->delayNum, sizeof(int)*(c->pN2SConnection->nNum)*(maxDelay-minDelay+1));

	ret = 2;

	return ret;
}
