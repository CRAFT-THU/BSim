#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../neuron/lif/GLIFNeurons.h"
#include "../utils/TypeFunc.h"
#include "../utils/macros.h"
#include "../gpu_utils/mem_op.h"
#include "GNetwork.h"

GNetwork* copyNetworkToGPU(GNetwork *pCpuNet)
{
	if (pCpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		return NULL;
	}

	GNetwork *tmpNet = (GNetwork*)malloc(sizeof(GNetwork));
	memcpy(tmpNet, pCpuNet, sizeof(GNetwork));

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;

	//TODO support multitype N and S
	void **pNs = (void**)malloc(sizeof(void*)*nTypeNum);
	void **pSs = (void**)malloc(sizeof(void*)*sTypeNum);

	for (int i=0; i<nTypeNum; i++) {
		pNs[i] = cudaAllocType[pCpuNet->nTypes[i]](pCpuNet->pNeurons[i], pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i]);
	}

	for (int i=0; i<sTypeNum; i++) {
		pSs[i] = cudaAllocType[pCpuNet->sTypes[i]](pCpuNet->pSynapses[i], pCpuNet->synapseNums[i+1]-pCpuNet->synapseNums[i]);
	}

	tmpNet->pNeurons = pNs;
	tmpNet->pSynapses = pSs;
	tmpNet->pN2SConnection = copyConnectionToGPU(pCpuNet->pN2SConnection);
	tmpNet->neuronNums = pCpuNet->neuronNums;
	tmpNet->synapseNums = pCpuNet->synapseNums;

	tmpNet->nTypeNum = pCpuNet->nTypeNum;
	tmpNet->sTypeNum = pCpuNet->sTypeNum;

	tmpNet->nTypes = pCpuNet->nTypes;
	tmpNet->sTypes = pCpuNet->sTypes;

	tmpNet->maxDelay = pCpuNet->maxDelay;
	tmpNet->minDelay = pCpuNet->minDelay;

	return tmpNet;
}

int fetchNetworkFromGPU(GNetwork *pCpuNet, GNetwork *pGpuNet)
{
	if (pCpuNet == NULL && pGpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		return -1;
	}

	int nTypeNum = pCpuNet->nTypeNum;

	//TODO support multitype N and S
	for (int i=0; i<nTypeNum; i++) {
		if (pCpuNet->nTypes[i] == LIF) {
			//cudaFetchLIF(pGpuNet->pNeurons[i], pCpuNet->pNeurons[i], pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i]);
		}
		//TODO: cudaFetchType
		//cudaFetchType[pCpuNet->nTypes[i]](pGpuNet->pNeurons[i], pCpuNet->pNeurons[i], pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i]);
	}

	return 0;
}

int freeNetworkGPU(GNetwork *pGpuNet)
{
	GNetwork *pTmpNet = pGpuNet;

	int nTypeNum = pTmpNet->nTypeNum;
	int sTypeNum = pTmpNet->sTypeNum;

	for (int i=0; i<nTypeNum; i++) {
		void *pTmpN = mallocType[pTmpNet->nTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpN, pTmpNet->pNeurons[i], getTypeSize[pTmpNet->nTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpNet->nTypes[i]](pTmpN);
		checkCudaErrors(hipFree(pTmpNet->pNeurons[i]));
		free(pTmpN);
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pTmpS = mallocType[pTmpNet->sTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpS, pTmpNet->pSynapses[i], getTypeSize[pTmpNet->sTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpNet->sTypes[i]](pTmpS);
		checkCudaErrors(hipFree(pTmpNet->pSynapses[i]));
		free(pTmpS);
	}

	N2SConnection * pConnection = (N2SConnection*)malloc(sizeof(N2SConnection));
	checkCudaErrors(hipMemcpy(pConnection, pTmpNet->pN2SConnection, sizeof(N2SConnection), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipFree(pConnection->pSynapsesIdx));
	checkCudaErrors(hipFree(pConnection->delayStart));
	checkCudaErrors(hipFree(pConnection->delayNum));
	checkCudaErrors(hipFree(pTmpNet->pN2SConnection));
	free(pConnection);

	//checkCudaErrors(hipFree(pTmpNet->nTypes));
	//checkCudaErrors(hipFree(pTmpNet->sTypes));
	//free(pTmpNT);
	//free(pTmpST);

	//checkCudaErrors(hipFree(pTmpNet->gNeuronNums));
	//checkCudaErrors(hipFree(pTmpNet->gSynapseNums));

	//checkCudaErrors(hipFree(pTmpNet->neuronNums));
	//checkCudaErrors(hipFree(pTmpNet->synapseNums));

	//checkCudaErrors(hipFree(pTmpNet->nOffsets));
	//checkCudaErrors(hipFree(pTmpNet->sOffsets));

	free(pTmpNet->pNeurons);
	free(pTmpNet->pSynapses);
	//free(pTmpNs);
	//free(pTmpSs);
	free(pTmpNet);

	return 0;
}



int checkGPUNetwork(GNetwork *g, GNetwork *c)
{
	int ret = -1;

	CHECK(g, c, nTypeNum);
	CHECK(g, c, sTypeNum);
	CHECK(g, c, neuronNums);
	CHECK(g, c, synapseNums);

	CHECK(g, c, nTypes);
	CHECK(g, c, sTypes);

	ret = 1;

	//int totalNeuronNum = g->neuronNums[g->nTypeNum+1];
	//int totalSynapseNum = g->synapseNums[g->sTypeNum+1];
	int maxDelay = c->maxDelay;
	int minDelay = c->minDelay;

	N2SConnection p;
	checkCudaErrors(hipMemcpy(&p, g->pN2SConnection, sizeof(N2SConnection), hipMemcpyDeviceToHost));

	//CHECK_CROSS_ARRAY(p.pSynapsesIdx, c->pN2SConnection->pSynapsesIdx, sizeof(int)*(c->pN2SConnection->s_num));
	CHECK_GPU_TO_CPU_ARRAY(p.delayStart, c->pN2SConnection->delayStart, sizeof(int)*(c->pN2SConnection->nNum)*(maxDelay-minDelay+1);
	CHECK_GPU_TO_CPU_ARRAY(p.delayNum, c->pN2SConnection->delayNum, sizeof(int)*(c->pN2SConnection->nNum)*(maxDelay-minDelay+1));

	ret = 2;

	return ret;
}
