#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../neuron/lif/GLIFNeurons.h"
#include "../utils/TypeFunc.h"
#include "../utils/macros.h"
#include "../gpu_utils/mem_op.h"
#include "GNetwork.h"

GNetwork* copyNetworkToGPU(GNetwork *pCpuNet)
{
	if (pCpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		return NULL;
	}

	GNetwork *tmpNet = (GNetwork*)malloc(sizeof(GNetwork));
	memcpy(tmpNet, pCpuNet, sizeof(GNetwork));

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	//int totalNeuronNum = pCpuNet->neuronNums[pCpuNet->nTypeNum];
	//int totalSynapseNum = pCpuNet->synapseNums[pCpuNet->sTypeNum];
	//int MAX_DELAY = pCpuNet->MAX_DELAY;

	//Type *g_nTypes = NULL, *g_sTypes = NULL;
	//checkCudaErrors(hipMalloc((void**)&(g_nTypes), sizeof(Type)*nTypeNum));
	//checkCudaErrors(hipMemcpy(g_nTypes, pCpuNet->nTypes, sizeof(Type)*nTypeNum, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(sTypes), sizeof(Type)*sTypeNum));
	//checkCudaErrors(hipMemcpy(g_sTypes, pCpuNet->sTypes, sizeof(Type)*sTypeNum, hipMemcpyHostToDevice));

	//int *g_neuronNums = NULL, *g_synapseNums = NULL;
	//checkCudaErrors(hipMalloc((void**)&(g_neuronNums), sizeof(int)*(nTypeNum+1)));
	//checkCudaErrors(hipMemcpy(g_neuronNums, pCpuNet->neuronNums, sizeof(int)*(nTypeNum+1), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(g_synapseNums), sizeof(int)*(sTypeNum+1)));
	//checkCudaErrors(hipMemcpy(g_synapseNums, pCpuNet->synapseNums, sizeof(int)*(sTypeNum+1), hipMemcpyHostToDevice));

	//int *g_gNeuronNums = NULL, *g_gSynapseNums = NULL;
	//checkCudaErrors(hipMalloc((void**)&(g_gNeuronNums), sizeof(int)*(nTypeNum+1)));
	//checkCudaErrors(hipMemcpy(g_gNeuronNums, pCpuNet->gNeuronNums, sizeof(int)*(nTypeNum+1), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(g_gSynapseNums), sizeof(int)*(sTypeNum+1)));
	//checkCudaErrors(hipMemcpy(g_gSynapseNums, pCpuNet->gSynapseNums, sizeof(int)*(sTypeNum+1), hipMemcpyHostToDevice));

	//int *nOffsets = NULL, *sOffsets = NULL;
	//checkCudaErrors(hipMalloc((void**)&(g_nOffsets), sizeof(int)*(nTypeNum)));
	//checkCudaErrors(hipMemcpy(g_nOffsets, pCpuNet->nOffsets, sizeof(int)*(nTypeNum), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(g_sOffsets), sizeof(int)*(sTypeNum)));
	//checkCudaErrors(hipMemcpy(g_sOffsets, pCpuNet->sOffsets, sizeof(int)*(sTypeNum), hipMemcpyHostToDevice));

	//TODO support multitype N and S
	void **pNs = (void**)malloc(sizeof(void*)*nTypeNum);
	void **pSs = (void**)malloc(sizeof(void*)*sTypeNum);

	for (int i=0; i<nTypeNum; i++) {
		void *pNTmp = createType[pCpuNet->nTypes[i]]();
		memcpy(pNTmp, pCpuNet->pNeurons[i], getTypeSize[pCpuNet->nTypes[i]]());
		cudaAllocType[pCpuNet->nTypes[i]](pCpuNet->pNeurons[i], pNTmp, pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i]);
		void *pNGpu;
		checkCudaErrors(hipMalloc((void**)&(pNGpu), getTypeSize[pCpuNet->nTypes[i]]()));
		checkCudaErrors(hipMemcpy(pNGpu, pNTmp, getTypeSize[pCpuNet->nTypes[i]](), hipMemcpyHostToDevice));
		free(pNTmp);
		pNs[i] = pNGpu;
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pSTmp = createType[pCpuNet->sTypes[i]]();
		memcpy(pSTmp, pCpuNet->pSynapses[i], getTypeSize[pCpuNet->sTypes[i]]());
		cudaAllocType[pCpuNet->sTypes[i]](pCpuNet->pSynapses[i], pSTmp, pCpuNet->synapseNums[i+1]-pCpuNet->synapseNums[i]);
		void *pSGpu;
		checkCudaErrors(hipMalloc((void**)&(pSGpu), getTypeSize[pCpuNet->sTypes[i]]()));
		checkCudaErrors(hipMemcpy(pSGpu, pSTmp, getTypeSize[pCpuNet->sTypes[i]](), hipMemcpyHostToDevice));
		free(pSTmp);
		pSs[i] = pSGpu;
	}


	tmpNet->pNeurons = pNs;
	tmpNet->pSynapses = pSs;
	tmpNet->pN2SConnection = copyConnectionToGPU(pCpuNet->pN2SConnection);
	//tmpNet->nOffsets = pCpuNet->nOffsets;
	//tmpNet->sOffsets = pCpuNet->sOffsets;
	tmpNet->neuronNums = pCpuNet->neuronNums;
	tmpNet->synapseNums = pCpuNet->synapseNums;

	tmpNet->nTypeNum = pCpuNet->nTypeNum;
	tmpNet->sTypeNum = pCpuNet->sTypeNum;

	tmpNet->nTypes = pCpuNet->nTypes;
	tmpNet->sTypes = pCpuNet->sTypes;
	//tmpNet->gNeuronNums = pCpuNet->gNeuronNums;
	//tmpNet->gSynapseNums = pCpuNet->gSynapseNums;

	tmpNet->MAX_DELAY = pCpuNet->MAX_DELAY;

	return tmpNet;
}

int fetchNetworkFromGPU(GNetwork *pCpuNet, GNetwork *pGpuNet)
{
	if (pCpuNet == NULL && pGpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		return -1;
	}

	int nTypeNum = pCpuNet->nTypeNum;
	//int sTypeNum = pCpuNet->sTypeNum;
	//int MAX_DELAY = pCpuNet->MAX_DELAY;

	//TODO support multitype N and S
	for (int i=0; i<nTypeNum; i++) {
		if (pCpuNet->nTypes[i] == LIF) {
			cudaFetchLIF(pGpuNet->pNeurons[i], pCpuNet->pNeurons[i], pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i]);
		}
		//TODO: cudaFetchType
		//cudaFetchType[pCpuNet->nTypes[i]](pGpuNet->pNeurons[i], pCpuNet->pNeurons[i], pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i]);
	}

	return 0;
}

int freeGPUNetwork(GNetwork *pGpuNet)
{
	GNetwork *pTmpNet = pGpuNet;

	int nTypeNum = pTmpNet->nTypeNum;
	int sTypeNum = pTmpNet->sTypeNum;

	//Type * pTmpNT = (Type*)malloc(sizeof(Type)*nTypeNum);
	//checkCudaErrors(hipMemcpy(pTmpNT, pTmpNet->nTypes, sizeof(Type)*nTypeNum, hipMemcpyDeviceToHost));
	//Type * pTmpST = (Type*)malloc(sizeof(Type)*sTypeNum);
	//checkCudaErrors(hipMemcpy(pTmpST, pTmpNet->sTypes, sizeof(Type)*sTypeNum, hipMemcpyDeviceToHost));

	for (int i=0; i<nTypeNum; i++) {
		void *pTmpN = createType[pTmpNet->nTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpN, pTmpNet->pNeurons[i], getTypeSize[pTmpNet->nTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpNet->nTypes[i]](pTmpN);
		checkCudaErrors(hipFree(pTmpNet->pNeurons[i]));
		free(pTmpN);
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pTmpS = createType[pTmpNet->sTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpS, pTmpNet->pSynapses[i], getTypeSize[pTmpNet->sTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpNet->sTypes[i]](pTmpS);
		checkCudaErrors(hipFree(pTmpNet->pSynapses[i]));
		free(pTmpS);
	}

	N2SConnection * pConnection = (N2SConnection*)malloc(sizeof(N2SConnection));
	checkCudaErrors(hipMemcpy(pConnection, pTmpNet->pN2SConnection, sizeof(N2SConnection), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipFree(pConnection->pSynapsesIdx));
	checkCudaErrors(hipFree(pConnection->delayStart));
	checkCudaErrors(hipFree(pConnection->delayNum));
	checkCudaErrors(hipFree(pTmpNet->pN2SConnection));
	free(pConnection);

	//checkCudaErrors(hipFree(pTmpNet->nTypes));
	//checkCudaErrors(hipFree(pTmpNet->sTypes));
	//free(pTmpNT);
	//free(pTmpST);

	//checkCudaErrors(hipFree(pTmpNet->gNeuronNums));
	//checkCudaErrors(hipFree(pTmpNet->gSynapseNums));

	//checkCudaErrors(hipFree(pTmpNet->neuronNums));
	//checkCudaErrors(hipFree(pTmpNet->synapseNums));

	//checkCudaErrors(hipFree(pTmpNet->nOffsets));
	//checkCudaErrors(hipFree(pTmpNet->sOffsets));

	free(pTmpNet->pNeurons);
	free(pTmpNet->pSynapses);
	//free(pTmpNs);
	//free(pTmpSs);
	free(pTmpNet);

	return 0;
}



int checkGPUNetwork(GNetwork *g, GNetwork *c)
{
	int ret = -1;

	CHECK(g, c, nTypeNum);
	CHECK(g, c, sTypeNum);
	CHECK(g, c, neuronNums);
	CHECK(g, c, synapseNums);

	CHECK(g, c, nTypes);
	CHECK(g, c, sTypes);

	ret = 1;

	//int totalNeuronNum = g->neuronNums[g->nTypeNum+1];
	//int totalSynapseNum = g->synapseNums[g->sTypeNum+1];
	int MAX_DELAY = c->MAX_DELAY;

	N2SConnection p;
	checkCudaErrors(hipMemcpy(&p, g->pN2SConnection, sizeof(N2SConnection), hipMemcpyDeviceToHost));

	//CHECK_CROSS_ARRAY(p.pSynapsesIdx, c->pN2SConnection->pSynapsesIdx, sizeof(int)*(c->pN2SConnection->s_num));
	CHECK_GPU_TO_CPU_ARRAY(p.delayStart, c->pN2SConnection->delayStart, sizeof(int)*(c->pN2SConnection->n_num)*MAX_DELAY);
	CHECK_GPU_TO_CPU_ARRAY(p.delayNum, c->pN2SConnection->delayNum, sizeof(int)*(c->pN2SConnection->n_num)*MAX_DELAY);

        ret = 2;

	return ret;
}
