
#include <stdlib.h>
#include <assert.h>

#include "../gpu_utils/mem_op.h"
#include "Connection.h"

Connection * cudaAllocConnection(Connection * pCPU)
{
	int nNum = pCPU->nNum;
	int sNum = pCPU->sNum;
	int length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;

	Connection * pGPU = NULL;
	Connection *pTmp = (Connection*)malloc(sizeof(Connection));
	pTmp->nNum = nNum;
	pTmp->sNum = sNum;
	pTmp->maxDelay = pCPU->maxDelay;
	pTmp->minDelay = pCPU->minDelay;

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayStart), sizeof(int)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayStart, pCPU->pDelayStart, sizeof(int)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayNum), sizeof(int)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayNum, pCPU->pDelayNum, sizeof(int)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayStartRev), sizeof(int)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayStartRev, pCPU->pDelayStartRev, sizeof(int)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayNumRev), sizeof(int)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayNumRev, pCPU->pDelayNumRev, sizeof(int)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pSidMapRev), sizeof(int)*sNum));
	checkCudaErrors(hipMemcpy(pTmp->pSidMapRev, pCPU->pSidMapRev, sizeof(int)*sNum, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGPU), sizeof(Connection)));
	checkCudaErrors(hipMemcpy(pGPU, pTmp, sizeof(Connection), hipMemcpyHostToDevice));

	free(pTmp);
	pTmp = NULL;

	return pGPU;
}

int cudaFetchConnection(Connection *pCPU, Connection *pGPU)
{
	int nNum = pCPU->nNum;
	int sNum = pCPU->sNum;
	int length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;

	Connection *pTmp = (Connection*)malloc(sizeof(Connection));
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(Connection), hipMemcpyDeviceToHost));
	assert(nNum == pTmp->nNum);
	assert(sNum == pTmp->sNum);
	assert(pCPU->maxDelay == pTmp->maxDelay);
	assert(pCPU->minDelay == pTmp->minDelay);

	checkCudaErrors(hipMemcpy(pCPU->pDelayStart, pTmp->pDelayStart, sizeof(int)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pDelayNum, pTmp->pDelayNum, sizeof(int)*length, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pCPU->pDelayStartRev, pTmp->pDelayStartRev, sizeof(int)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pDelayNumRev, pTmp->pDelayNumRev, sizeof(int)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pSidMapRev, pTmp->pSidMapRev, sizeof(int)*sNum, hipMemcpyDeviceToHost));

	free(pTmp);
	pTmp = NULL;

	return 0;
}
