
#include <stdlib.h>

#include "../gpu_utils/mem_op.h"
#include "Connection.h"

N2SConnection * copyConnectionToGPU(N2SConnection * pCPU)
{
	int n_num = pCPU->n_num;
	int MAX_DELAY = pCPU->MAX_DELAY;

	N2SConnection * pTmp = (N2SConnection*)malloc(sizeof(N2SConnection));
	N2SConnection * pGPU = NULL;
	int *g_delayStart = NULL, *g_delayNum = NULL;
	//int *g_pSynapsesIdx = NULL;
	//checkCudaErrors(hipMalloc((void**)&(g_pSynapsesIdx), sizeof(int)*conn_s_num));
	//checkCudaErrors(hipMemcpy(g_pSynapsesIdx, pCPU->pSynapsesIdx, sizeof(int)*conn_s_num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(g_delayStart), sizeof(int)*n_num*MAX_DELAY));
	checkCudaErrors(hipMemcpy(g_delayStart, pCPU->delayStart, sizeof(int)*n_num*MAX_DELAY, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(g_delayNum), sizeof(int)*n_num*MAX_DELAY));
	checkCudaErrors(hipMemcpy(g_delayNum, pCPU->delayNum, sizeof(int)*n_num*MAX_DELAY, hipMemcpyHostToDevice));
	//pTmp->pSynapsesIdx = g_pSynapsesIdx;
	pTmp->delayStart = g_delayStart;
	pTmp->delayNum = g_delayNum;
	checkCudaErrors(hipMalloc((void**)&(pGPU), sizeof(N2SConnection)));
	checkCudaErrors(hipMemcpy(pGPU, pTmp, sizeof(N2SConnection), hipMemcpyHostToDevice));
	free(pTmp);

	return pGPU;
}
