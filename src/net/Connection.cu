
#include <stdlib.h>
#include <assert.h>

#include "../gpu_utils/mem_op.h"
#include "Connection.h"

Connection * cudaAllocConnection(Connection * pCPU)
{
	int nNum = pCPU->nNum;
	int sNum = pCPU->sNum;
	int length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;

	Connection * pGPU = NULL;
	Connection *pTmp = (Connection*)malloc(sizeof(Connection));
	pTmp->nNum = nNum;
	pTmp->sNum = sNum;
	pTmp->maxDelay = pCPU->maxDelay;
	pTmp->minDelay = pCPU->minDelay;

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayStart), sizeof(int)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayStart, pCPU->pDelayStart, sizeof(int)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayNum), sizeof(int)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayNum, pCPU->pDelayNum, sizeof(int)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayStartRev), sizeof(int)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayStartRev, pCPU->pDelayStartRev, sizeof(int)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayNumRev), sizeof(int)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayNumRev, pCPU->pDelayNumRev, sizeof(int)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pSidMapRev), sizeof(int)*sNum));
	checkCudaErrors(hipMemcpy(pTmp->pSidMapRev, pCPU->pSidMapRev, sizeof(int)*sNum, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGPU), sizeof(Connection)));
	checkCudaErrors(hipMemcpy(pGPU, pTmp, sizeof(Connection), hipMemcpyHostToDevice));

	free(pTmp);
	pTmp = NULL;

	return pGPU;
}

int cudaFetchConnection(Connection *pCPU, Connection *pGPU)
{
	int nNum = pCPU->nNum;
	int sNum = pCPU->sNum;
	int length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;

	Connection *pTmp = (Connection*)malloc(sizeof(Connection));
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(Connection), hipMemcpyDeviceToHost));
	assert(nNum == pTmp->nNum);
	assert(sNum == pTmp->sNum);
	assert(pCPU->maxDelay == pTmp->maxDelay);
	assert(pCPU->minDelay == pTmp->minDelay);

	checkCudaErrors(hipMemcpy(pCPU->pDelayStart, pTmp->pDelayStart, sizeof(int)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pDelayNum, pTmp->pDelayNum, sizeof(int)*length, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pCPU->pDelayStartRev, pTmp->pDelayStartRev, sizeof(int)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pDelayNumRev, pTmp->pDelayNumRev, sizeof(int)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pSidMapRev, pTmp->pSidMapRev, sizeof(int)*sNum, hipMemcpyDeviceToHost));

	free(pTmp);
	pTmp = NULL;

	return 0;
}

int cudaFreeConnection(Connection *pGPU)
{
	Connection * pTmp = (Connection*)malloc(sizeof(Connection));
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(Connection), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(pTmp->pDelayStart));
	checkCudaErrors(hipFree(pTmp->pDelayNum));
	checkCudaErrors(hipFree(pTmp->pDelayStartRev));
	checkCudaErrors(hipFree(pTmp->pDelayNumRev));
	checkCudaErrors(hipFree(pTmp->pSidMapRev));
	free(pTmp);
	pTmp = NULL;
	return 0;
}

LolConnection * cudaAllocLolConnection(LolConnection * pCPU)
{
	int nNum = pCPU->nNum;
	int sNum = pCPU->sNum;
	int length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;

	LolConnection * pGPU = NULL;
	LolConnection *pTmp = (LolConnection*)malloc(sizeof(LolConnection));
	pTmp->nNum = nNum;
	pTmp->sNum = sNum;
	pTmp->maxDelay = pCPU->maxDelay;
	pTmp->minDelay = pCPU->minDelay;

	int **pIdx = (int**)malloc(sizeof(int*)*length); 
	for (int i=0; i<length; i++) {
		if (pCPU->pNum[i] > 0) {
			checkCudaErrors(hipMalloc((void**)&(pIdx[i]), sizeof(int)*(pCPU->pNum[i])));
			checkCudaErrors(hipMemcpy(pIdx[i], pCPU->pIndex[i], sizeof(int)*(pCPU->pNum[i]), hipMemcpyHostToDevice));
		} else {
			pIdx[i] = NULL;
		}
	}

	checkCudaErrors(hipMalloc((void**)&(pTmp->pIndex), sizeof(int*)*length));
	checkCudaErrors(hipMemcpy(pTmp->pIndex, pIdx, sizeof(int*)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pNum), sizeof(int)*length));
	checkCudaErrors(hipMemcpy(pTmp->pNum, pCPU->pNum, sizeof(int)*length, hipMemcpyHostToDevice));


	checkCudaErrors(hipMalloc((void**)&(pGPU), sizeof(LolConnection)));
	checkCudaErrors(hipMemcpy(pGPU, pTmp, sizeof(LolConnection), hipMemcpyHostToDevice));

	free(pTmp);
	pTmp = NULL;
	free(pIdx);
	pIdx = NULL;

	return pGPU;
}

// int cudaFetchLolConnection(LolConnection *pCPU, LolConnection *pGPU)
// {
// 	int nNum = pCPU->nNum;
// 	int sNum = pCPU->sNum;
// 	int length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;
// 
// 	Connection *pTmp = (Connection*)malloc(sizeof(Connection));
// 	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(Connection), hipMemcpyDeviceToHost));
// 	assert(nNum == pTmp->nNum);
// 	assert(sNum == pTmp->sNum);
// 	assert(pCPU->maxDelay == pTmp->maxDelay);
// 	assert(pCPU->minDelay == pTmp->minDelay);
// 
// 	checkCudaErrors(hipMemcpy(pCPU->pDelayStart, pTmp->pDelayStart, sizeof(int)*length, hipMemcpyDeviceToHost));
// 	checkCudaErrors(hipMemcpy(pCPU->pDelayNum, pTmp->pDelayNum, sizeof(int)*length, hipMemcpyDeviceToHost));
// 
// 	free(pTmp);
// 	pTmp = NULL;
// 
// 	return 0;
// }

int cudaFreeLolConnection(LolConnection *pGPU)
{
	LolConnection * pTmp = (LolConnection*)malloc(sizeof(LolConnection));
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(LolConnection), hipMemcpyDeviceToHost));
	int length = (pTmp->maxDelay - pTmp->minDelay + 1) * pTmp->nNum;

	int *num = (int*)malloc(sizeof(int)*length);
	int **index = (int**)malloc(sizeof(int*)*length);
	checkCudaErrors(hipMemcpy(num, pTmp->pNum, sizeof(int)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(index, pTmp->pIndex, sizeof(int*)*length, hipMemcpyDeviceToHost));
	for (int i=0; i<length; i++) {
		if (num[i] > 0) {
			checkCudaErrors(hipFree(index[i]));
		}
	}
	checkCudaErrors(hipFree(pTmp->pIndex));
	checkCudaErrors(hipFree(pTmp->pNum));
	free(pTmp);
	pTmp = NULL;
	free(index);
	index = NULL;
	free(num);
	num = NULL;
	// checkCudaErrors(hipFree(pGPU));
	return 0;
}
