
#include <stdlib.h>

#include "../gpu_utils/mem_op.h"
#include "Connection.h"

N2SConnection * copyConnectionToGPU(N2SConnection * pCPU)
{
	int n_num = pCPU->nNum;
	int maxDelay = pCPU->maxDelay;

	N2SConnection * pGPU = NULL;
	N2SConnection * pTmp = (N2SConnection*)malloc(sizeof(N2SConnection));

	checkCudaErrors(hipMalloc((void**)&(pTmp->delayStart), sizeof(int)*n_num*maxDelay));
	checkCudaErrors(hipMemcpy(pTmp->delayStart, pCPU->delayStart, sizeof(int)*n_num*maxDelay, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->delayNum), sizeof(int)*n_num*maxDelay));
	checkCudaErrors(hipMemcpy(pTmp->delayNum, pCPU->delayNum, sizeof(int)*n_num*maxDelay, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGPU), sizeof(N2SConnection)));
	checkCudaErrors(hipMemcpy(pGPU, pTmp, sizeof(N2SConnection), hipMemcpyHostToDevice));
	free(pTmp);
	pTmp = NULL;

	return pGPU;
}
