
#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "MultiNetwork.h"

//Alloc cross device gmem, will set device to 0 before return
CrossNodeDataGPU* MultiNetwork::arrangeCrossNodeDataGPU(int node_num)
{
	CrossNodeDataGPU * cross_data = static_cast<CrossNodeDataGPU*>(malloc(sizeof(CrossNodeDataGPU)));
	assert(cross_data != NULL);

	cross_data->_max_num = static_cast<int*>(malloc(sizeof(int)*node_num*node_num));
	assert(cross_data->_max_num != NULL);
	cross_data->_fired_arrays = static_cast<int**>(malloc(sizeof(int*)*node_num*node_num));
	assert(cross_data->_fired_arrays != NULL);

	checkCudaErrors(hipHostMalloc((void**)&(cross_data->_fired_num), sizeof(int)*node_num*node_num));
	checkCudaErrors(hipMemset(cross_data->_fired_num, 0, sizeof(int)*node_num*node_num));


	for (int i=0; i<_node_num; i++) {
		for (int j=0; j<_node_num; j++) {
			// i->j 
			int i2j = i * _node_num + j;
			cross_data._fired_num = 0;

			int count = 0;
			for (auto iter = _crossnode_neurons_send[i].begin(); iter != _crossnode_neurons_send[i].end(); iter++) {
				if (_crossnode_neurons_recv[j].find(*iter) != _crossnode_neurons_recv[j].end()) {
					count++;
				}
			}
			cross_data->_max_num[i2j] = count;
			if (count > 0) {
				checkCudaErrors(hipMalloc((void**)&(cross_data->_fired_arrays[i2j]), sizeof(int)*count));
			} else {
				cross_data->_fired_arrays[i2j] = NULL;
			}
		}
	}

	return cross_data;
}

