#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat March 12 2016
 */

#include "../utils/utils.h"
#include "../neuron/lif/lif.h"
#include "../neuron/array/array.h"
#include "../neuron/constant/constants.h"
#include "../neuron/decide/decide.h"
#include "../neuron/fft/fft.h"
#include "../neuron/max/max.h"
#include "../neuron/mem/mem.h"
#include "../neuron/poisson/poisson.h"
#include "../neuron/tj/tj.h"
#include "../synapse/static/static.h"
#include "runtime.h"
#include "gpu_func.h"


int addCrossNeurons(int *ids, int num, int time)
{
	add_cross_neuron<<<(num+MAXBLOCKSIZE-1)/MAXBLOCKSIZE, MAXBLOCKSIZE>>>(ids, num, time);
	return 0;
}


int cudaDeliverNeurons(int *idx2index, int *crossnode_index2idx, int *global_cross_data, int *fired_n_num, int node_num, int neuron_num, int time)
{
	deliver_neurons<<<(neuron_num + MAXBLOCKSIZE-1)/MAXBLOCKSIZE, MAXBLOCKSIZE>>>(idx2index, crossnode_index2idx, global_cross_data, fired_n_num, node_num, time);

	return 0;
}

BlockSize * getBlockSize(int nSize, int sSize)
{
	BlockSize *ret = (BlockSize*)malloc(sizeof(BlockSize)*TYPESIZE);
	memset(ret, 0, sizeof(BlockSize)*TYPESIZE);

	hipOccupancyMaxPotentialBlockSize(&(ret[Constant].minGridSize), &(ret[Constant].blockSize), update_constant_neuron, 0, nSize); 
	ret[Constant].gridSize = (upzero_else_set_one(nSize) + (ret[Constant].blockSize) - 1) / (ret[Constant].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Poisson].minGridSize), &(ret[Poisson].blockSize), update_poisson_neuron, 0, nSize); 
	ret[Poisson].gridSize = (upzero_else_set_one(nSize) + (ret[Poisson].blockSize) - 1) / (ret[Poisson].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Array].minGridSize), &(ret[Array].blockSize), update_array_neuron, 0, nSize); 
	ret[Array].gridSize = (upzero_else_set_one(nSize) + (ret[Array].blockSize) - 1) / (ret[Array].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Decide].minGridSize), &(ret[Decide].blockSize), update_max_neuron, 0, nSize); 
	ret[Decide].gridSize = (upzero_else_set_one(nSize) + (ret[Decide].blockSize) - 1) / (ret[Decide].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[FFT].minGridSize), &(ret[FFT].blockSize), update_fft_neuron, 0, nSize); 
	ret[FFT].gridSize = (upzero_else_set_one(nSize) + (ret[FFT].blockSize) - 1) / (ret[FFT].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Mem].minGridSize), &(ret[Mem].blockSize), update_mem_neuron, 0, nSize); 
	ret[Mem].gridSize = (upzero_else_set_one(nSize) + (ret[Mem].blockSize) - 1) / (ret[Mem].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Max].minGridSize), &(ret[Max].blockSize), update_max_neuron, 0, nSize); 
	ret[Max].gridSize = (upzero_else_set_one(nSize) + (ret[Max].blockSize) - 1) / (ret[Max].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[LIFE].minGridSize), &(ret[LIFE].blockSize), update_life_neuron, 0, nSize); 
	ret[LIFE].gridSize = (upzero_else_set_one(nSize) + (ret[LIFE].blockSize) - 1) / (ret[LIFE].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[TJ].minGridSize), &(ret[TJ].blockSize), update_tj_neuron, 0, nSize); 
	ret[TJ].gridSize = (upzero_else_set_one(nSize) + (ret[TJ].blockSize) - 1) / (ret[TJ].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Static].minGridSize), &(ret[Static].blockSize), update_static_hit, 0, sSize); 
	ret[Static].blockSize = 128;
	ret[Static].gridSize = (upzero_else_set_one(nSize) + (ret[Static].blockSize) - 1) / (ret[Static].blockSize);

	return ret;
}
