#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat March 12 2016
 */

#include "../neuron/lif/GLIF.h"
#include "../neuron/tj/GTJ.h"
#include "../neuron/max/GMax.h"
#include "../neuron/poisson/GPoisson.h"
#include "../neuron/array/GArray.h"
#include "../neuron/constant/GConstant.h"

#include "../synapse/static/GStatic.h"

//#include "../../include/GNeuron.h"
//#include "../../include/GSynapse.h"

#include "../utils/utils.h"
#include "runtime.h"
#include "gpu_func.h"


int cudaUpdateConstant(void *data, int num, int start_id, int t, BlockSize *pSize)
{
	update_constant_neuron<<<pSize->gridSize, pSize->blockSize>>>((GConstantNeurons*)data, num, start_id, t);

	return 0;
}

int cudaUpdatePoisson(void *data, int num, int start_id, int t, BlockSize *pSize)
{
	update_poisson_neuron<<<pSize->gridSize, pSize->blockSize>>>((GPoissonNeurons*)data, num, start_id, t);

	return 0;
}

int cudaUpdateArray(void *data, int num, int start_id, int t, BlockSize *pSize)
{
	update_array_neuron<<<pSize->gridSize, pSize->blockSize>>>((GArrayNeurons*)data, num, start_id, t);

	return 0;
}

int cudaUpdateLIFE(void *data, int num, int start_id, int t, BlockSize *pSize)
{
	find_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFENeurons*)data, num, start_id);
	update_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFENeurons*)data, num, start_id, t);
	//update_dense_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFENeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateTJ(void *data, int num, int start_id, int t, BlockSize *pSize)
{
	update_tj_neuron<<<pSize->gridSize, pSize->blockSize>>>((GTJNeurons*)data, num, start_id, t);

	return 0;
}

int cudaUpdateMax(void *data, int num, int start_id, int t, BlockSize *pSize)
{
	update_max_neuron<<<pSize->gridSize, pSize->blockSize>>>((GMaxNeurons*)data, num, start_id, t);

	return 0;
}

int cudaUpdateStatic(void *data, int num, int start_id, int t, BlockSize *pSize)
{
	//update_static_hit<<<pSize->gridSize, pSize->blockSize>>>((GStaticSynapses*)data, num, start_id);
	//reset_active_synapse<<<1, 1>>>();
	update_dense_static_hit<<<pSize->gridSize, pSize->blockSize>>>((GStaticSynapses*)data, num, start_id, t);

	return 0;
}

int addCrossNeurons(int *ids, int num)
{
	add_cross_neuron<<<(num+MAXBLOCKSIZE-1)/MAXBLOCKSIZE, MAXBLOCKSIZE>>>(ids, num);
	return 0;
}


int cudaDeliverNeurons(int *idx2index, int *crossnode_index2idx, int *global_cross_data, int *fired_n_num, int node_num, int neuron_num)
{
	deliver_neurons<<<(neuron_num + MAXBLOCKSIZE-1)/MAXBLOCKSIZE, MAXBLOCKSIZE>>>(idx2index, crossnode_index2idx, global_cross_data, fired_n_num, node_num);

	return 0;
}

BlockSize * getBlockSize(int nSize, int sSize)
{
	BlockSize *ret = (BlockSize*)malloc(sizeof(BlockSize)*TYPESIZE);
	memset(ret, 0, sizeof(BlockSize)*TYPESIZE);

	hipOccupancyMaxPotentialBlockSize(&(ret[Constant].minGridSize), &(ret[Constant].blockSize), update_constant_neuron, 0, nSize); 
	ret[Constant].gridSize = (upzero_else_set_one(nSize) + (ret[Constant].blockSize) - 1) / (ret[Constant].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Poisson].minGridSize), &(ret[Poisson].blockSize), update_poisson_neuron, 0, nSize); 
	ret[Poisson].gridSize = (upzero_else_set_one(nSize) + (ret[Poisson].blockSize) - 1) / (ret[Poisson].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Array].minGridSize), &(ret[Array].blockSize), update_poisson_neuron, 0, nSize); 
	ret[Array].gridSize = (upzero_else_set_one(nSize) + (ret[Array].blockSize) - 1) / (ret[Array].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Decide].minGridSize), &(ret[Decide].blockSize), update_poisson_neuron, 0, nSize); 
	ret[Decide].gridSize = (upzero_else_set_one(nSize) + (ret[Decide].blockSize) - 1) / (ret[Decide].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[FFT].minGridSize), &(ret[FFT].blockSize), update_poisson_neuron, 0, nSize); 
	ret[FFT].gridSize = (upzero_else_set_one(nSize) + (ret[FFT].blockSize) - 1) / (ret[FFT].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Mem].minGridSize), &(ret[Mem].blockSize), update_poisson_neuron, 0, nSize); 
	ret[Mem].gridSize = (upzero_else_set_one(nSize) + (ret[Mem].blockSize) - 1) / (ret[Mem].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Max].minGridSize), &(ret[Max].blockSize), update_tj_neuron, 0, nSize); 
	ret[Max].gridSize = (upzero_else_set_one(nSize) + (ret[Max].blockSize) - 1) / (ret[Max].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[LIFE].minGridSize), &(ret[LIFE].blockSize), update_life_neuron, 0, nSize); 
	ret[LIFE].gridSize = (upzero_else_set_one(nSize) + (ret[LIFE].blockSize) - 1) / (ret[LIFE].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[TJ].minGridSize), &(ret[TJ].blockSize), update_life_neuron, 0, nSize); 
	ret[TJ].gridSize = (upzero_else_set_one(nSize) + (ret[TJ].blockSize) - 1) / (ret[TJ].blockSize);

	//hipOccupancyMaxPotentialBlockSize(&(ret[Static].minGridSize), &(ret[Static].blockSize), update_static_hit, 0, sSize); 
	ret[Static].blockSize = 128;
	ret[Static].gridSize = (upzero_else_set_one(nSize) + (ret[Static].blockSize) - 1) / (ret[Static].blockSize);

	return ret;
}
