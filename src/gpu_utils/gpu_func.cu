/* This program is writen by qp09.
 * usually just for fun.
 * Sat March 12 2016
 */

#include "../neuron/GNeuron.h"
#include "gpu_kernel.h"
#include "gpu_func.h"

int cudaUpdatePoisson(void *data, int num, int start_id, BlockSize *pSize)
{
	update_poisson_neuron<<<pSize->gridSize, pSize->blockSize>>>((GPoissonNeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateConstant(void *data, int num, int start_id, BlockSize *pSize)
{
	update_constant_neuron<<<pSize->gridSize, pSize->blockSize>>>((GConstantNeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateLIF(void *data, int num, int start_id, BlockSize *pSize)
{
	find_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, num, start_id);
	update_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateExp(void *data, int num, int start_id, BlockSize *pSize)
{
	update_exp_hit<<<pSize->gridSize, pSize->blockSize>>>((GExpSynapses*)data, num, start_id);
	reset_active_synapse<<<1, 1>>>();
	find_exp_synapse<<<pSize->gridSize, pSize->blockSize>>>((GExpSynapses*)data, num, start_id);
	update_exp_synapse<<<pSize->gridSize, pSize->blockSize>>>((GExpSynapses*)data, num, start_id);

	return 0;
}

int addCrossNeurons(int *ids, int num)
{
	add_cross_neuron<<<(num+MAXBLOCKSIZE-1)/MAXBLOCKSIZE, MAXBLOCKSIZE>>>(ids, num);
	return 0;
}

//int cudaUpdateAlpha(void *data, int num, int start_id, BlockSize *pSize)
//{
//	update_alpha_synapse<<<pSize->gridSize, pSize->blockSize>>>((GAlphaSynapses*)data, num, start_id);
//
//	return 0;
//}
//
//int cudaUpdateBasic(void *data, int num, int start_id, BlockSize *pSize)
//{
//	update_basic_synapse<<<pSize->gridSize, pSize->blockSize>>>((GBasicSynapses*)data, num, start_id);
//
//	return 0;
//}

BlockSize * getBlockSize(int nSize, int sSize)
{
	BlockSize *ret = (BlockSize*)malloc(sizeof(BlockSize)*TYPESIZE);
	hipOccupancyMaxPotentialBlockSize(&(ret[Constant].minGridSize), &(ret[Constant].blockSize), update_constant_neuron, 0, nSize); 
	ret[Constant].gridSize = (nSize + (ret[Constant].blockSize) - 1) / (ret[Constant].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Poisson].minGridSize), &(ret[Poisson].blockSize), update_poisson_neuron, 0, nSize); 
	ret[Poisson].gridSize = (nSize + (ret[Poisson].blockSize) - 1) / (ret[Poisson].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[LIF].minGridSize), &(ret[LIF].blockSize), update_lif_neuron, 0, nSize); 
	ret[LIF].gridSize = (nSize + (ret[LIF].blockSize) - 1) / (ret[LIF].blockSize);

	//hipOccupancyMaxPotentialBlockSize(&(ret[Basic].minGridSize), &(ret[Basic].blockSize), update_basic_synapse, 0, sSize); 
	//ret[Basic].gridSize = (sSize + (ret[Basic].blockSize) - 1) / (ret[Basic].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Exp].minGridSize), &(ret[Exp].blockSize), update_exp_synapse, 0, sSize); 
	ret[Exp].gridSize = (sSize + (ret[Exp].blockSize) - 1) / (ret[Exp].blockSize);

	//hipOccupancyMaxPotentialBlockSize(&(ret[Alpha].minGridSize), &(ret[Alpha].blockSize), update_alpha_synapse, 0, sSize); 
	//ret[Alpha].gridSize = (sSize + (ret[Alpha].blockSize) - 1) / (ret[Alpha].blockSize);

	return ret;
}
