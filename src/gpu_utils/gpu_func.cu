/* This program is writen by qp09.
 * usually just for fun.
 * Sat March 12 2016
 */

#include "../../include/GNeuron.h"
#include "../utils/utils.h"
#include "gpu_kernel.h"
#include "gpu_func.h"


int cudaUpdateConstant(void *data, int num, int start_id, BlockSize *pSize)
{
	update_constant_neuron<<<pSize->gridSize, pSize->blockSize>>>((GConstantNeurons*)data, num, start_id);

	return 0;
}

int cudaUpdatePoisson(void *data, int num, int start_id, BlockSize *pSize)
{
	update_poisson_neuron<<<pSize->gridSize, pSize->blockSize>>>((GPoissonNeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateArray(void *data, int num, int start_id, BlockSize *pSize)
{
	update_array_neuron<<<pSize->gridSize, pSize->blockSize>>>((GArrayNeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateLIF(void *data, int num, int start_id, BlockSize *pSize)
{
	find_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, num, start_id);
	update_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateAllLIF(void *data, int num, int start_id, BlockSize *pSize)
{
	update_all_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateLIFE(void *data, int num, int start_id, BlockSize *pSize)
{
	find_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFENeurons*)data, num, start_id);
	update_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFENeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateAllLIFE(void *data, int num, int start_id, BlockSize *pSize)
{
	//update_all_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFENeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateMax(void *data, int num, int start_id, BlockSize *pSize)
{
	update_max_neuron<<<pSize->gridSize, pSize->blockSize>>>((GMaxNeurons*)data, num, start_id);

	return 0;
}

int cudaUpdateExp(void *data, int num, int start_id, BlockSize *pSize)
{
	update_exp_hit<<<pSize->gridSize, pSize->blockSize>>>((GExpSynapses*)data, num, start_id);
	reset_active_synapse<<<1, 1>>>();
	find_exp_synapse<<<pSize->gridSize, pSize->blockSize>>>((GExpSynapses*)data, num, start_id);
	update_exp_synapse<<<pSize->gridSize, pSize->blockSize>>>((GExpSynapses*)data, num, start_id);

	return 0;
}

int cudaUpdateAllExp(void *data, int num, int start_id, BlockSize *pSize)
{
	update_exp_hit<<<pSize->gridSize, pSize->blockSize>>>((GExpSynapses*)data, num, start_id);
	reset_active_synapse<<<1, 1>>>();
	update_all_exp_synapse<<<pSize->gridSize, pSize->blockSize>>>((GExpSynapses*)data, num, start_id);

	return 0;
}

int cudaUpdateStatic(void *data, int num, int start_id, BlockSize *pSize)
{
	update_static_hit<<<pSize->gridSize, pSize->blockSize>>>((GStaticSynapses*)data, num, start_id);
	reset_active_synapse<<<1, 1>>>();

	return 0;
}

int cudaUpdateAllStatic(void *data, int num, int start_id, BlockSize *pSize)
{
	update_static_hit<<<pSize->gridSize, pSize->blockSize>>>((GStaticSynapses*)data, num, start_id);
	reset_active_synapse<<<1, 1>>>();

	return 0;
}

int addCrossNeurons(int *ids, int num)
{
	add_cross_neuron<<<(num+MAXBLOCKSIZE-1)/MAXBLOCKSIZE, MAXBLOCKSIZE>>>(ids, num);
	return 0;
}

//int cudaUpdateAlpha(void *data, int num, int start_id, BlockSize *pSize)
//{
//	update_alpha_synapse<<<pSize->gridSize, pSize->blockSize>>>((GAlphaSynapses*)data, num, start_id);
//
//	return 0;
//}
//
//int cudaUpdateBasic(void *data, int num, int start_id, BlockSize *pSize)
//{
//	update_basic_synapse<<<pSize->gridSize, pSize->blockSize>>>((GBasicSynapses*)data, num, start_id);
//
//	return 0;
//}

BlockSize * getBlockSize(int nSize, int sSize)
{
	BlockSize *ret = (BlockSize*)malloc(sizeof(BlockSize)*TYPESIZE);
	hipOccupancyMaxPotentialBlockSize(&(ret[Constant].minGridSize), &(ret[Constant].blockSize), update_constant_neuron, 0, nSize); 
	ret[Constant].gridSize = (upzero_else_set_one(nSize) + (ret[Constant].blockSize) - 1) / (ret[Constant].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Poisson].minGridSize), &(ret[Poisson].blockSize), update_poisson_neuron, 0, nSize); 
	ret[Poisson].gridSize = (upzero_else_set_one(nSize) + (ret[Poisson].blockSize) - 1) / (ret[Poisson].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Array].minGridSize), &(ret[Array].blockSize), update_poisson_neuron, 0, nSize); 
	ret[Array].gridSize = (upzero_else_set_one(nSize) + (ret[Array].blockSize) - 1) / (ret[Array].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[LIF].minGridSize), &(ret[LIF].blockSize), update_lif_neuron, 0, nSize); 
	ret[LIF].gridSize = (upzero_else_set_one(nSize) + (ret[LIF].blockSize) - 1) / (ret[LIF].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[LIFE].minGridSize), &(ret[LIFE].blockSize), update_life_neuron, 0, nSize); 
	ret[LIFE].gridSize = (upzero_else_set_one(nSize) + (ret[LIFE].blockSize) - 1) / (ret[LIFE].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Max].minGridSize), &(ret[Max].blockSize), update_max_neuron, 0, nSize); 
	ret[Max].gridSize = (upzero_else_set_one(nSize) + (ret[Max].blockSize) - 1) / (ret[Max].blockSize);

	//hipOccupancyMaxPotentialBlockSize(&(ret[Basic].minGridSize), &(ret[Basic].blockSize), update_basic_synapse, 0, sSize); 
	//ret[Basic].gridSize = (sSize + (ret[Basic].blockSize) - 1) / (ret[Basic].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Exp].minGridSize), &(ret[Exp].blockSize), update_exp_synapse, 0, sSize); 
	ret[Exp].gridSize = (upzero_else_set_one(sSize) + (ret[Exp].blockSize) - 1) / (ret[Exp].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[Static].minGridSize), &(ret[Static].blockSize), update_static_hit, 0, sSize); 
	ret[Static].gridSize = (upzero_else_set_one(sSize) + (ret[Static].blockSize) - 1) / (ret[Static].blockSize);

	//hipOccupancyMaxPotentialBlockSize(&(ret[Alpha].minGridSize), &(ret[Alpha].blockSize), update_alpha_synapse, 0, sSize); 
	//ret[Alpha].gridSize = (sSize + (ret[Alpha].blockSize) - 1) / (ret[Alpha].blockSize);

	return ret;
}
