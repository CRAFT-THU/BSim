#include "hip/hip_runtime.h"

#include "array.h"


__global__ void update_array_neuron(GArrayNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		fired = (d_neurons->p_start[idx] < d_neurons->p_end[idx]) &&  (gCurrentCycle >= d_neurons->p_fire_time[d_neurons->p_start[idx]]);
		gFireCount[start_id + idx] += fired;

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = start_id + idx;
					d_neurons->p_start[idx] = d_neurons->p_start[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
				//advance_array_neuron(d_neurons, fire_table_t, MAXBLOCKSIZE, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		//advance_array_neuron(d_neurons, fire_table_t, fire_cnt, start_id);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

