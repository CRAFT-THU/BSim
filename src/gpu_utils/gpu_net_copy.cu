
#include "../utils/TypeFunc.h"
#include "mem_op.h"
#include "gpu_func.h"
#include "gpu_utils.h"

GNetwork* copyNetworkToGPU(GNetwork *pCpuNet)
{
	if (pCpuNet == NULL) {
		printf("NULL POINTER: GNETWORK\n");
		exit(-1);
	}

	GNetwork *tmpNet = (GNetwork*)malloc(sizeof(GNetwork));
	memcpy(tmpNet, pCpuNet, sizeof(GNetwork));

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[pCpuNet->nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[pCpuNet->sTypeNum];
	int MAX_DELAY = pCpuNet->MAX_DELAY;

	//Type *g_nTypes = NULL, *g_sTypes = NULL;
	//checkCudaErrors(hipMalloc((void**)&(g_nTypes), sizeof(Type)*nTypeNum));
	//checkCudaErrors(hipMemcpy(g_nTypes, pCpuNet->nTypes, sizeof(Type)*nTypeNum, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(sTypes), sizeof(Type)*sTypeNum));
	//checkCudaErrors(hipMemcpy(g_sTypes, pCpuNet->sTypes, sizeof(Type)*sTypeNum, hipMemcpyHostToDevice));

	//int *g_neuronNums = NULL, *g_synapseNums = NULL;
	//checkCudaErrors(hipMalloc((void**)&(g_neuronNums), sizeof(int)*(nTypeNum+1)));
	//checkCudaErrors(hipMemcpy(g_neuronNums, pCpuNet->neuronNums, sizeof(int)*(nTypeNum+1), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(g_synapseNums), sizeof(int)*(sTypeNum+1)));
	//checkCudaErrors(hipMemcpy(g_synapseNums, pCpuNet->synapseNums, sizeof(int)*(sTypeNum+1), hipMemcpyHostToDevice));

	//int *g_gNeuronNums = NULL, *g_gSynapseNums = NULL;
	//checkCudaErrors(hipMalloc((void**)&(g_gNeuronNums), sizeof(int)*(nTypeNum+1)));
	//checkCudaErrors(hipMemcpy(g_gNeuronNums, pCpuNet->gNeuronNums, sizeof(int)*(nTypeNum+1), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(g_gSynapseNums), sizeof(int)*(sTypeNum+1)));
	//checkCudaErrors(hipMemcpy(g_gSynapseNums, pCpuNet->gSynapseNums, sizeof(int)*(sTypeNum+1), hipMemcpyHostToDevice));

	//int *nOffsets = NULL, *sOffsets = NULL;
	//checkCudaErrors(hipMalloc((void**)&(g_nOffsets), sizeof(int)*(nTypeNum)));
	//checkCudaErrors(hipMemcpy(g_nOffsets, pCpuNet->nOffsets, sizeof(int)*(nTypeNum), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(g_sOffsets), sizeof(int)*(sTypeNum)));
	//checkCudaErrors(hipMemcpy(g_sOffsets, pCpuNet->sOffsets, sizeof(int)*(sTypeNum), hipMemcpyHostToDevice));

	//TODO support multitype N and S
	void **pNs = (void**)malloc(sizeof(void*)*nTypeNum);
	void **pSs = (void**)malloc(sizeof(void*)*sTypeNum);

	for (int i=0; i<nTypeNum; i++) {
		void *pNTmp = createType[pCpuNet->nTypes[i]]();
		memcpy(pNTmp, pCpuNet->pNeurons[i], getTypeSize[pCpuNet->nTypes[i]]());
		cudaAllocType[pCpuNet->nTypes[i]](pCpuNet->pNeurons[i], pNTmp, pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i]);
		void *pNGpu;
		checkCudaErrors(hipMalloc((void**)&(pNGpu), getTypeSize[pCpuNet->nTypes[i]]()));
		checkCudaErrors(hipMemcpy(pNGpu, pNTmp, getTypeSize[pCpuNet->nTypes[i]](), hipMemcpyHostToDevice));
		free(pNTmp);
		pNs[i] = pNGpu;
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pSTmp = createType[pCpuNet->sTypes[i]]();
		memcpy(pSTmp, pCpuNet->pSynapses[i], getTypeSize[pCpuNet->sTypes[i]]());
		cudaAllocType[pCpuNet->sTypes[i]](pCpuNet->pSynapses[i], pSTmp, pCpuNet->synapseNums[i+1]-pCpuNet->synapseNums[i]);
		void *pSGpu;
		checkCudaErrors(hipMalloc((void**)&(pSGpu), getTypeSize[pCpuNet->sTypes[i]]()));
		checkCudaErrors(hipMemcpy(pSGpu, pSTmp, getTypeSize[pCpuNet->sTypes[i]](), hipMemcpyHostToDevice));
		free(pSTmp);
		pSs[i] = pSGpu;

	}

	N2SConnection * pConnection = (N2SConnection*)malloc(sizeof(N2SConnection));
	N2SConnection * g_pConnection = NULL;
	int *g_pSynapsesIdx = NULL, *g_delayStart = NULL, *g_delayNum = NULL;
	checkCudaErrors(hipMalloc((void**)&(g_pSynapsesIdx), sizeof(int)*totalSynapseNum));
	checkCudaErrors(hipMemcpy(g_pSynapsesIdx, pCpuNet->pN2SConnection->pSynapsesIdx, sizeof(int)*totalSynapseNum, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(g_delayStart), sizeof(int)*totalNeuronNum*MAX_DELAY));
	checkCudaErrors(hipMemcpy(g_delayStart, pCpuNet->pN2SConnection->delayStart, sizeof(int)*totalNeuronNum*MAX_DELAY, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(g_delayNum), sizeof(int)*totalNeuronNum*MAX_DELAY));
	checkCudaErrors(hipMemcpy(g_delayNum, pCpuNet->pN2SConnection->delayNum, sizeof(int)*totalNeuronNum*MAX_DELAY, hipMemcpyHostToDevice));
	pConnection->pSynapsesIdx = g_pSynapsesIdx;
	pConnection->delayStart = g_delayStart;
	pConnection->delayNum = g_delayNum;
	checkCudaErrors(hipMalloc((void**)&(g_pConnection), sizeof(N2SConnection)));
	checkCudaErrors(hipMemcpy(g_pConnection, pConnection, sizeof(N2SConnection), hipMemcpyHostToDevice));
	free(pConnection);

	tmpNet->pNeurons = pNs;
	tmpNet->pSynapses = pSs;
	tmpNet->pN2SConnection = g_pConnection;
	//tmpNet->nOffsets = pCpuNet->nOffsets;
	//tmpNet->sOffsets = pCpuNet->sOffsets;
	tmpNet->neuronNums = pCpuNet->neuronNums;
	tmpNet->synapseNums = pCpuNet->synapseNums;

	tmpNet->nTypes = pCpuNet->nTypes;
	tmpNet->sTypes = pCpuNet->sTypes;
	//tmpNet->gNeuronNums = pCpuNet->gNeuronNums;
	//tmpNet->gSynapseNums = pCpuNet->gSynapseNums;

	return tmpNet;
}

int freeGPUNetwork(GNetwork *pGpuNet)
{
	GNetwork *pTmpNet = pGpuNet;

	int nTypeNum = pTmpNet->nTypeNum;
	int sTypeNum = pTmpNet->sTypeNum;

	//Type * pTmpNT = (Type*)malloc(sizeof(Type)*nTypeNum);
	//checkCudaErrors(hipMemcpy(pTmpNT, pTmpNet->nTypes, sizeof(Type)*nTypeNum, hipMemcpyDeviceToHost));
	//Type * pTmpST = (Type*)malloc(sizeof(Type)*sTypeNum);
	//checkCudaErrors(hipMemcpy(pTmpST, pTmpNet->sTypes, sizeof(Type)*sTypeNum, hipMemcpyDeviceToHost));

	for (int i=0; i<nTypeNum; i++) {
		void *pTmpN = createType[pTmpNet->nTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpN, pTmpNet->pNeurons[i], getTypeSize[pTmpNet->nTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpNet->nTypes[i]](pTmpN);
		checkCudaErrors(hipFree(pTmpNet->pNeurons[i]));
		free(pTmpN);
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pTmpS = createType[pTmpNet->sTypes[i]]();
		checkCudaErrors(hipMemcpy(pTmpS, pTmpNet->pSynapses[i], getTypeSize[pTmpNet->sTypes[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpNet->sTypes[i]](pTmpS);
		checkCudaErrors(hipFree(pTmpNet->pSynapses[i]));
		free(pTmpS);
	}

	N2SConnection * pConnection = (N2SConnection*)malloc(sizeof(N2SConnection));
	checkCudaErrors(hipMemcpy(pConnection, pTmpNet->pN2SConnection, sizeof(N2SConnection), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(pConnection->pSynapsesIdx));
	checkCudaErrors(hipFree(pConnection->delayStart));
	checkCudaErrors(hipFree(pConnection->delayNum));
	checkCudaErrors(hipFree(pTmpNet->pN2SConnection));
	free(pConnection);

	//checkCudaErrors(hipFree(pTmpNet->nTypes));
	//checkCudaErrors(hipFree(pTmpNet->sTypes));
	//free(pTmpNT);
	//free(pTmpST);

	//checkCudaErrors(hipFree(pTmpNet->gNeuronNums));
	//checkCudaErrors(hipFree(pTmpNet->gSynapseNums));

	//checkCudaErrors(hipFree(pTmpNet->neuronNums));
	//checkCudaErrors(hipFree(pTmpNet->synapseNums));

	//checkCudaErrors(hipFree(pTmpNet->nOffsets));
	//checkCudaErrors(hipFree(pTmpNet->sOffsets));

	free(pTmpNet->pNeurons);
	free(pTmpNet->pSynapses);
	//free(pTmpNs);
	//free(pTmpSs);
	free(pTmpNet);

	return 0;
}
