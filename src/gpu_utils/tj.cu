#include "hip/hip_runtime.h"

#include "tj.h"

__device__ void reset_tj_neuron(GTJNeurons *d_neurons, int *shared_buf, volatile int size, int start_id) 
{
	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		int nid = shared_buf[idx] - start_id;
		d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
		d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
	}
}

__global__ void update_tj_neuron(GTJNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int gnid = start_id + idx; 
		bool actived = d_neurons->p_refrac_step[idx] <= 0;
		//real DT = 0.001;
		if (actived) {
			real I = gNeuronInput[gnid] + gNeuronInput_I[gnid] + d_neurons->p_i_tmp[idx];
			d_neurons->p_vm[idx] = d_neurons->p_vm[idx] + DT * I/d_neurons->p_cm[idx];
			gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];

			fired = d_neurons->p_vm[idx] >= d_neurons->p_v_thresh[idx];

			gFireCount[gnid] += fired;

			for (int i=0; i<2; i++) {
				if (fired) {
					test_loc = atomicAdd((int*)&fire_cnt, 1);
					if (test_loc < MAXBLOCKSIZE) {
						fire_table_t[test_loc] = gnid;
						fired = false;
					}
				}
				__syncthreads();
				if (fire_cnt >= MAXBLOCKSIZE) {
					commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
					reset_tj_neuron(d_neurons, fire_table_t, MAXBLOCKSIZE, start_id);
					if (threadIdx.x == 0) {
						fire_cnt = 0;
					}
				}
				__syncthreads();
			}
			__syncthreads();

			if (fire_cnt > 0) {
				commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
				reset_tj_neuron(d_neurons, fire_table_t, fire_cnt, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
		} else {
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		gNeuronInput[start_id + idx] = 0;
		gNeuronInput_I[start_id + idx] = 0;
	}
	__syncthreads();
}

