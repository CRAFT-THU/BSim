#include "hip/hip_runtime.h"

#include "array.h"

__global__ void update_max_neuron(GMaxNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int input = (int)gNeuronInput[start_id + idx];
		gNeuronInput[start_id + idx] = 0;
		int test = 1;
		int record_offset = idx*d_neurons->max_N; 
		for (int i=0; i<d_neurons->p_N[idx]; i++) {
			if (input & test) {
				d_neurons->p_record[record_offset + i]++;
				if (d_neurons->p_record[record_offset + i] > d_neurons->p_count[idx]) {

					fired = true;
				}
			}
			test = test << 1;
		}

		gFireCount[start_id + idx] += fired;

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = start_id + idx;
					d_neurons->p_count[idx] = d_neurons->p_count[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
				//advance_array_neuron(d_neurons, fire_table_t, MAXBLOCKSIZE, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		//advance_array_neuron(d_neurons, fire_table_t, fire_cnt, start_id);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}

}
