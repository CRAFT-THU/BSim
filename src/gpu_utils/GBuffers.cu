
#include "GBuffers.h"

GBuffers* alloc_buffers(int neuron_num, int synapse_num, int max_delay, real dt) 
{
	GBuffers *ret = (GBuffers*)malloc(sizeof(GBuffers));
	memset(ret, 0, sizeof(GBuffers));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gNeuronInput), sizeof(real)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gNeuronInput, 0, sizeof(real)*(neuron_num)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gNeuronInput_I), sizeof(real)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gNeuronInput_I, 0, sizeof(real)*(neuron_num)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gFiredTable), sizeof(int)*((neuron_num)*(max_delay+1))));
	checkCudaErrors(hipMemset(ret->c_gFiredTable, 0, sizeof(int)*((neuron_num)*(max_delay+1))));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gFiredTableSizes), sizeof(int)*(max_delay+1)));
	checkCudaErrors(hipMemset(ret->c_gFiredTableSizes, 0, sizeof(int)*(max_delay+1)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gActiveTable), sizeof(int)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gActiveTable, 0, sizeof(int)*(neuron_num)));

	//checkCudaErrors(hipMalloc((void**)&(ret->c_gSynapsesActiveTable), sizeof(int)*(synapse_num)));
	//checkCudaErrors(hipMemset(ret->c_gSynapsesActiveTable, 0, sizeof(int)*(synapse_num)));

	//checkCudaErrors(hipMalloc((void**)&(ret->c_gSynapsesLogTable), sizeof(int)*(synapse_num)));
	//checkCudaErrors(hipMemset(ret->c_gSynapsesLogTable, 0, sizeof(int)*(synapse_num)));

	ret->c_gLayerInput = gpuMalloc<int>(neuron_num);
	ret->c_gXInput = gpuMalloc<real>(neuron_num);
	ret->c_gFireCount = gpuMalloc<int>(neuron_num);

	int timeTableCap = max_delay+1;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(MAX_DELAY), &max_delay, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gTimeTableCap), &timeTableCap, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gFiredTableCap), &neuron_num, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gSynapsesTableCap), &synapse_num, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DT), &dt, sizeof(real)));
	//checkCudaErrors(hipMalloc((void**)&ret->c_gTimeTable, sizeof(int)*(max_delay+1)));
	//checkCudaErrors(hipMemset(ret->c_gTimeTable, 0, sizeof(int)*(max_delay+1)));

	checkCudaErrors(hipHostMalloc((void**)(&ret->c_neuronsFired), sizeof(int)*(neuron_num)));
	checkCudaErrors(hipHostMalloc((void**)(&ret->c_synapsesFired), sizeof(int)*(synapse_num)));

	init_buffers<<<1, 1, 0>>>(/*ret->c_gTimeTable,*/ ret->c_gNeuronInput, ret->c_gNeuronInput_I, ret->c_gFiredTable, ret->c_gFiredTableSizes, ret->c_gActiveTable, ret->c_gSynapsesActiveTable, ret->c_gSynapsesLogTable);

	init_log_buffers<<<1, 1, 0>>>(ret->c_gLayerInput, ret->c_gXInput, ret->c_gFireCount);

	return ret;
}

void init_buffers(GBuffers * buf) {
	init_buffers<<<1, 1, 0>>>(/*buf->c_gTimeTable,*/ buf->c_gNeuronInput, buf->c_gNeuronInput_I, buf->c_gFiredTable, buf->c_gFiredTableSizes, buf->c_gActiveTable, buf->c_gSynapsesActiveTable, buf->c_gSynapsesLogTable);

	init_log_buffers<<<1, 1, 0>>>(buf->c_gLayerInput, buf->c_gXInput, buf->c_gFireCount);
}

int free_buffers(GBuffers *buf) 
{
	checkCudaErrors(hipFree(buf->c_gNeuronInput));
	checkCudaErrors(hipFree(buf->c_gNeuronInput_I));
	checkCudaErrors(hipFree(buf->c_gFiredTable));
	checkCudaErrors(hipFree(buf->c_gFiredTableSizes));
	checkCudaErrors(hipFree(buf->c_gActiveTable));
	checkCudaErrors(hipFree(buf->c_gSynapsesActiveTable));
	checkCudaErrors(hipFree(buf->c_gSynapsesLogTable));

	checkCudaErrors(hipHostFree(buf->c_neuronsFired));
	checkCudaErrors(hipHostFree(buf->c_synapsesFired));

	return 0;
}
