#include "hip/hip_runtime.h"

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "mem_op.h"
// #include "gpu_macros.h"
#include "runtime.h"

#include "GBuffers.h"

__global__ void init_buffers(/*int *c_gTimeTable, real *c_gNeuronInput, real *c_gNeuronInput_I, int *c_gFiredTable, int *c_gFiredTableSizes, */int *c_gActiveTable/*, int *c_gSynapsesActiveTable, int *c_gSynapsesLogTable*/) 
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		// gCurrentIdx = 0;
		// gCurrentCycle = 0;

		// gTimeTable = c_gTimeTable;
		// gNeuronInput = c_gNeuronInput;
		// gNeuronInput_I = c_gNeuronInput_I;
		// gFiredTable = c_gFiredTable;
		// gFiredTableSize = 0;
		// gFiredTableSizes = c_gFiredTableSizes;
		gActiveTable = c_gActiveTable;
		gActiveTableSize = 0;
		// gSynapsesActiveTable = c_gSynapsesActiveTable;
		// gSynapsesActiveTableSize = 0;
		// gSynapsesLogTable = c_gSynapsesLogTable;
	}
}

__global__ void init_log_buffers(int *layer_input, real *x_input, int *fire_count)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gLayerInput = layer_input;
		gXInput = x_input;
		gFireCount = fire_count;
	}
}


GBuffers* alloc_buffers(int neuron_num, int synapse_num, int max_delay, real dt) 
{
	GBuffers *ret = (GBuffers*)malloc(sizeof(GBuffers));
	memset(ret, 0, sizeof(GBuffers));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gNeuronInput), sizeof(real)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gNeuronInput, 0, sizeof(real)*(neuron_num)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gNeuronInput_I), sizeof(real)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gNeuronInput_I, 0, sizeof(real)*(neuron_num)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gFiredTable), sizeof(int)*((neuron_num)*(max_delay+1))));
	checkCudaErrors(hipMemset(ret->c_gFiredTable, 0, sizeof(int)*((neuron_num)*(max_delay+1))));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gFiredTableSizes), sizeof(int)*(max_delay+1)));
	checkCudaErrors(hipMemset(ret->c_gFiredTableSizes, 0, sizeof(int)*(max_delay+1)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gActiveTable), sizeof(int)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gActiveTable, 0, sizeof(int)*(neuron_num)));

	// checkCudaErrors(hipMalloc((void**)&(ret->c_gSynapsesActiveTable), sizeof(int)*(synapse_num)));
	// checkCudaErrors(hipMemset(ret->c_gSynapsesActiveTable, 0, sizeof(int)*(synapse_num)));

	// checkCudaErrors(hipMalloc((void**)&(ret->c_gSynapsesLogTable), sizeof(int)*(synapse_num)));
	// checkCudaErrors(hipMemset(ret->c_gSynapsesLogTable, 0, sizeof(int)*(synapse_num)));

	ret->c_gLayerInput = gpuMalloc<int>(neuron_num);
	ret->c_gXInput = gpuMalloc<real>(neuron_num);
	ret->c_gFireCount = gpuMalloc<int>(neuron_num);

	// int timeTableCap = max_delay+1;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(MAX_DELAY), &max_delay, sizeof(int)));
	// checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gTimeTableCap), &timeTableCap, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gFiredTableCap), &neuron_num, sizeof(int)));
	// checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gSynapsesTableCap), &synapse_num, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DT), &dt, sizeof(real)));
	//checkCudaErrors(hipMalloc((void**)&ret->c_gTimeTable, sizeof(int)*(max_delay+1)));
	//checkCudaErrors(hipMemset(ret->c_gTimeTable, 0, sizeof(int)*(max_delay+1)));

	checkCudaErrors(hipHostMalloc((void**)(&ret->c_neuronsFired), sizeof(int)*(neuron_num)));
	checkCudaErrors(hipHostMalloc((void**)(&ret->c_synapsesFired), sizeof(int)*(synapse_num)));

	// init_buffers<<<1, 1, 0>>>(/*ret->c_gTimeTable,*/ ret->c_gNeuronInput, ret->c_gNeuronInput_I, ret->c_gFiredTable, ret->c_gFiredTableSizes, ret->c_gActiveTable/*, ret->c_gSynapsesActiveTable, ret->c_gSynapsesLogTable*/);
	init_buffers<<<1, 1, 0>>>(ret->c_gActiveTable);

	init_log_buffers<<<1, 1, 0>>>(ret->c_gLayerInput, ret->c_gXInput, ret->c_gFireCount);

	return ret;
}

// void init_buffers(GBuffers * buf) {
// 	init_buffers<<<1, 1, 0>>>(/*buf->c_gTimeTable,*/ buf->c_gNeuronInput, buf->c_gNeuronInput_I, buf->c_gFiredTable, buf->c_gFiredTableSizes, buf->c_gActiveTable/*, buf->c_gSynapsesActiveTable, buf->c_gSynapsesLogTable*/);
// 
// 	init_log_buffers<<<1, 1, 0>>>(buf->c_gLayerInput, buf->c_gXInput, buf->c_gFireCount);
// }

int free_buffers(GBuffers *buf) 
{
	checkCudaErrors(hipFree(buf->c_gNeuronInput));
	checkCudaErrors(hipFree(buf->c_gNeuronInput_I));
	checkCudaErrors(hipFree(buf->c_gFiredTable));
	checkCudaErrors(hipFree(buf->c_gFiredTableSizes));
	checkCudaErrors(hipFree(buf->c_gActiveTable));
	// checkCudaErrors(hipFree(buf->c_gSynapsesActiveTable));
	// checkCudaErrors(hipFree(buf->c_gSynapsesLogTable));

	checkCudaErrors(hipHostFree(buf->c_neuronsFired));
	checkCudaErrors(hipHostFree(buf->c_synapsesFired));

	return 0;
}
