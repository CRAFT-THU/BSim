#include "hip/hip_runtime.h"

#include <assert.h>

#include "../utils/utils.h"
#include "../neuron/lif/lif.h"
// #include "../neuron/array/array.h"
// #include "../synapse/static/static.h"
// #include "../neuron/constant/constants.h"
// #include "../neuron/decide/decide.h"
// #include "../neuron/fft/fft.h"
// #include "../neuron/max/max.h"
// #include "../neuron/mem/mem.h"
// #include "../neuron/poisson/poisson.h"
// #include "../neuron/tj/tj.h"

#include "mem_op.h"
#include "runtime.h"
// #include "gpu_func.h"

#include "../third_party/cuda/hip/hip_runtime_api.h"

#define MAXBLOCKSIZE 1024


//#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
//#else
//__device__ double atomicAdd(double* address, double val)
//{
//	unsigned long long int* address_as_ull = (unsigned long long int*)address;
//	unsigned long long int old = *address_as_ull, assumed;
//	do {
//		assumed = old;
//		old = atomicCAS(address_as_ull, assumed,
//				__double_as_longlong(val + __longlong_as_double(assumed)));
//	} while (assumed != old);
//	return __longlong_as_double(old);
//}
//#endif


// __constant__ int MAX_DELAY;
__constant__ int gTimeTableCap;
__constant__ int gFiredTableCap;
// __constant__ int gSynapsesTableCap;
__constant__ real DT;

// Variable
// __device__ int gCurrentIdx;
// __device__ int gCurrentCycle;
// __device__ int gFiredTableSize;
// __device__ int gSynapsesActiveTableSize;

// Arrays
//__device__ int *gTimeTable;

// Neuron Arrays
// __device__ real *gNeuronInput;
// __device__ real *gNeuronInput_I;

// Neuron Tables
// __device__ int *gFiredTable;
// __device__ int *gFiredTableSizes;
__device__ int *gActiveTable;
__device__ int gActiveTableSize;

// Synapse Tables
//__device__ int *gSynapsesActiveTable;
//__device__ int *gSynapsesLogTable;

// Log Arrays
__device__ int *gLayerInput;
__device__ real *gXInput;
__device__ int *gFireCount;

// Connection
// __device__ N2SConnection *gConnection;


__device__ int commit2globalTable(int *shared_buf, volatile unsigned int size, int *global_buf, int * global_size, int offset) 
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicAdd(global_size, (int)size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx] = shared_buf[idx];
	}

	return 0;
}

__global__ void update_time(Connection *connection, int time, int *firedTableSizes)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		// gCurrentCycle = gCurrentCycle + 1;
		// gCurrentIdx = (gCurrentIdx +1)%(MAX_DELAY + 1);
		int currentIdx = time % (connection->maxDelay-connection->minDelay + 1);
		gActiveTableSize = 0;
		firedTableSizes[currentIdx] = 0;
		// gSynapsesActiveTableSize = 0;
	}
	__syncthreads();
}
// 
// __global__ void init_time(int gCurrentCycle)
// {
// 	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
// 		//gTimeTable[gCurrentIdx] = simTime;
// 		gCurrentCycle = gCurrentCycle;
// 		gCurrentIdx = (gCurrentCycle)%(MAX_DELAY + 1);
// 		gActiveTableSize = 0;
// 		gFiredTableSizes[gCurrentIdx] = 0;
// 		gSynapsesActiveTableSize = 0;
// 	}
// 	__syncthreads();
// }

// __global__ void reset_active_synapse()
// {
// 	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
// 		gSynapsesActiveTableSize = 0;
// 	}
// 	__syncthreads();
// 
// }

__global__ void curand_setup_kernel(hiprandState *state, int num)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < num) {
		hiprand_init(1234, id, 0, &state[id]); 
	}
}

__global__ void cudaAddCrossNeurons(Connection *connection, int *firedTable, int *firedTableSizes, int *ids, int num, int time)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int delayIdx = time % (connection->maxDelay-connection->minDelay+1);
	if (tid < num) {
		firedTable[gFiredTableCap*delayIdx + firedTableSizes[delayIdx] + tid] = ids[tid];
	}
	__syncthreads();

	if (tid == 0) {
		firedTableSizes[delayIdx] += num;
	}
}

__global__ void cudaDeliverNeurons(Connection *conn, int *firedTable, int *firedTableSizes, int *idx2index, int *crossnode_index2idx, int *global_cross_data, int *fired_n_num, int node_num, int time)
{
	__shared__ int cross_neuron_id[MAXBLOCKSIZE];
	__shared__ volatile int cross_cnt;

	if (threadIdx.x == 0) {
		cross_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int delayIdx = time % (conn->maxDelay-conn->minDelay+1);
	int fired_size = firedTableSizes[delayIdx];
	for (int node = 0; node < node_num; node++) {
		for (int idx = tid; idx < fired_size; idx += blockDim.x * gridDim.x) {
			int nid = firedTable[gFiredTableCap*delayIdx + idx];
			int tmp = idx2index[nid];
			if (tmp >= 0) {
				int map_nid = crossnode_index2idx[tmp*node_num + node];
				if (map_nid >= 0) {
					int test_loc = atomicAdd((int*)&cross_cnt, 1);
					if (test_loc < MAXBLOCKSIZE) {
						cross_neuron_id[test_loc] = map_nid;
					}
				}
			}
			__syncthreads();

			if (cross_cnt > 0) {
				commit2globalTable(cross_neuron_id, cross_cnt, global_cross_data, &fired_n_num[node], gFiredTableCap*node);
				if (threadIdx.x == 0) {
					cross_cnt = 0;
				}
			}
			__syncthreads();
		}
		__syncthreads();
	}
}

// __global__ void init_connection(N2SConnection *pConnection)
// {
// 	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
// 		gConnection = pConnection;
// 	}
// }

BlockSize * getBlockSize(int nSize, int sSize)
{
	BlockSize *ret = (BlockSize*)malloc(sizeof(BlockSize)*TYPESIZE);
	memset(ret, 0, sizeof(BlockSize)*TYPESIZE);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Array].minGridSize), &(ret[Array].blockSize), update_array_neuron, 0, nSize); 
	// ret[Array].gridSize = (upzero_else_set_one(nSize) + (ret[Array].blockSize) - 1) / (ret[Array].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[LIF].minGridSize), &(ret[LIF].blockSize), update_lif_neuron, 0, nSize); 
	ret[LIF].gridSize = (upzero_else_set_one(nSize) + (ret[LIF].blockSize) - 1) / (ret[LIF].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Constant].minGridSize), &(ret[Constant].blockSize), update_constant_neuron, 0, nSize); 
	// ret[Constant].gridSize = (upzero_else_set_one(nSize) + (ret[Constant].blockSize) - 1) / (ret[Constant].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Poisson].minGridSize), &(ret[Poisson].blockSize), update_poisson_neuron, 0, nSize); 
	// ret[Poisson].gridSize = (upzero_else_set_one(nSize) + (ret[Poisson].blockSize) - 1) / (ret[Poisson].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Decide].minGridSize), &(ret[Decide].blockSize), update_max_neuron, 0, nSize); 
	// ret[Decide].gridSize = (upzero_else_set_one(nSize) + (ret[Decide].blockSize) - 1) / (ret[Decide].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[FFT].minGridSize), &(ret[FFT].blockSize), update_fft_neuron, 0, nSize); 
	// ret[FFT].gridSize = (upzero_else_set_one(nSize) + (ret[FFT].blockSize) - 1) / (ret[FFT].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Mem].minGridSize), &(ret[Mem].blockSize), update_mem_neuron, 0, nSize); 
	// ret[Mem].gridSize = (upzero_else_set_one(nSize) + (ret[Mem].blockSize) - 1) / (ret[Mem].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Max].minGridSize), &(ret[Max].blockSize), update_max_neuron, 0, nSize); 
	// ret[Max].gridSize = (upzero_else_set_one(nSize) + (ret[Max].blockSize) - 1) / (ret[Max].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[TJ].minGridSize), &(ret[TJ].blockSize), update_tj_neuron, 0, nSize); 
	// ret[TJ].gridSize = (upzero_else_set_one(nSize) + (ret[TJ].blockSize) - 1) / (ret[TJ].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Static].minGridSize), &(ret[Static].blockSize), update_static_hit, 0, sSize); 
	ret[Static].blockSize = 128;
	ret[Static].gridSize = (upzero_else_set_one(nSize) + (ret[Static].blockSize) - 1) / (ret[Static].blockSize);

	return ret;
}
