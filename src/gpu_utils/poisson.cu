#include "hip/hip_runtime.h"

#include "poisson.h"

__device__ void reset_poisson_neuron(GPoissonNeurons *d_neurons, int *shared_buf, volatile int size, int start_id) 
{
	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		int nid = shared_buf[idx] - start_id;

		hiprandState localState = d_neurons->p_state[nid];
		int tmp = hiprand_poisson(&localState, d_neurons->p_rate[nid]);
		d_neurons->p_fire_cycle[nid] = d_neurons->p_fire_cycle[nid] + 1 + tmp + d_neurons->p_refrac_step[nid];
		d_neurons->p_state[nid] = localState;
	}
}

__global__ void update_poisson_neuron(GPoissonNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		fired = (gCurrentCycle == d_neurons->p_fire_cycle[idx]) && (gCurrentCycle <= d_neurons->p_end_cycle[idx]);
		gFireCount[start_id + idx] += fired;

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = start_id + idx;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
				reset_poisson_neuron(d_neurons, fire_table_t, MAXBLOCKSIZE, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		reset_poisson_neuron(d_neurons, fire_table_t, fire_cnt, start_id);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}
