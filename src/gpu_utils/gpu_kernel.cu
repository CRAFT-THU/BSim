#include "hip/hip_runtime.h"

#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "gpu_macros.h"
#include "gpu_kernel.h"

#define SHARED_SIZE 512

// Constant
__constant__ int MAX_DELAY;
__constant__ int gTimeTableCap;
__constant__ int gFiredTableCap;
__constant__ int gSynapsesTableCap;

// Variable
__device__ int gCurrentIdx;
__device__ int gCurrentCycle;
__device__ int gFiredTableSize;
__device__ int gActiveTableSize;
__device__ int gSynapsesActiveTableSize;

// Arrays
//__device__ int *gTimeTable;

// Neuron Arrays
__device__ real *gNeuronInput;

// Neuron Tables
__device__ int *gFiredTable;
__device__ int *gFiredTableSizes;

__device__ int *gActiveTable;

// Synapse Tables
__device__ int *gSynapsesActiveTable;

__device__ int *gSynapsesLogTable;



__device__ int commit2globalTable(int *shared_buf, volatile unsigned int size, int *global_buf, int * global_size, int offset) 
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicAdd(global_size, (int)size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx] = shared_buf[idx];
	}

	return 0;
}

__global__ void update_time()
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		//gTimeTable[gCurrentIdx] = simTime;
		gCurrentCycle = gCurrentCycle + 1;
		gCurrentIdx = (gCurrentIdx +1)%(MAX_DELAY + 1);
		gActiveTableSize = 0;
		gFiredTableSizes[gCurrentIdx] = 0;
		gSynapsesActiveTableSize = 0;
	}
	__syncthreads();
}

__global__ void update_constant_neuron(GConstantNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = (gCurrentCycle * d_neurons->p_fire_rate[idx]) > (d_neurons->p_fire_count[idx]);
		}

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < SHARED_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= SHARED_SIZE) {
				commit2globalTable(fire_table_t, SHARED_SIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

__global__ void find_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = d_neurons->p_refrac_step[idx] <= 0;
		}

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < SHARED_SIZE) {
				fire_table_t[test_loc] = start_id + idx;
				fired = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		__syncthreads();

		if (fire_cnt >= SHARED_SIZE) {
			commit2globalTable(fire_table_t, SHARED_SIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < SHARED_SIZE) {
				fire_table_t[test_loc] = start_id + idx;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= SHARED_SIZE) {
			commit2globalTable(fire_table_t, SHARED_SIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
	}
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gActiveTable, &gActiveTableSize, 0);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();

}

__device__ void reset_lif_neuron(GLIFNeurons *d_neurons, int *shared_buf, volatile int size) 
{
	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		int nid = shared_buf[idx];
		d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
		d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
	}
}

__global__ void update_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		if (idx < gActiveTableSize) {
			int nid = gActiveTable[idx];
			int gnid = start_id + gActiveTable[idx];
			real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
			d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
			//d_neurons->p_i_syn[nid] = 0;
			gNeuronInput[gnid] = 0;

			fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

			for (int i=0; i<2; i++) {
				if (fired) {
					test_loc = atomicAdd((int*)&fire_cnt, 1);
					if (test_loc < SHARED_SIZE) {
						fire_table_t[test_loc] = nid;
						fired = false;
					}
				}
				__syncthreads();
				if (fire_cnt >= SHARED_SIZE) {
					commit2globalTable(fire_table_t, SHARED_SIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
					reset_lif_neuron(d_neurons, fire_table_t, SHARED_SIZE);
					if (threadIdx.x == 0) {
						fire_cnt = 0;
					}
				}
				__syncthreads();
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			reset_lif_neuron(d_neurons, fire_table_t, SHARED_SIZE);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
	}
	__syncthreads();
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		gActiveTableSize = 0;
	}
}

__global__ void update_pre_synapse(N2SConnection *pConnection)
{
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
		int time_idx = (gCurrentIdx+MAX_DELAY-delta_t)%(MAX_DELAY+1);
		int firedSize = gFiredTableSizes[time_idx];
		for (int idx = tid; idx < firedSize; idx += blockDim.x*gridDim.x) {
			int nid = gFiredTable[time_idx*gFiredTableCap + idx];
			int start_loc = pConnection->delayStart[delta_t + nid * MAX_DELAY];
			int synapseNum = pConnection->delayNum[delta_t + nid * MAX_DELAY];
			int offset = atomicAdd(&gSynapsesActiveTableSize, synapseNum);
			for (int i=0; i<synapseNum; i++) {
				gSynapsesActiveTable[offset+i] = pConnection->pSynapsesIdx[i+start_loc];
			}
		}
	}
	__syncthreads();
}

__global__ void update_exp_hit(GExpSynapses *d_synapses, int num, int start_id)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gSynapsesActiveTableSize; idx += blockDim.x*gridDim.x) {
		int sid = gSynapsesActiveTable[idx];
		d_synapses->p_I_syn[sid] += d_synapses->p_weight[sid];
		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
		gSynapsesLogTable[sid] = gCurrentCycle;
		atomicAdd(&(gNeuronInput[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);
	}
	__syncthreads();

}

__global__ void find_exp_synapse(GLIFNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = gSynapsesLogTable[start_id + idx] <= gCurrentCycle;
		}

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < SHARED_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					fired = false;
				}
			}
			__syncthreads();

			if (fire_cnt >= SHARED_SIZE) {
				commit2globalTable(fire_table_t, SHARED_SIZE, gSynapsesActiveTable, &gSynapsesActiveTableSize, 0);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gSynapsesActiveTable, &gSynapsesActiveTableSize, 0);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();

}

__global__ void update_exp_synapse(GExpSynapses *d_synapses, int num, int start_id)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gSynapsesActiveTableSize; idx += blockDim.x*gridDim.x) {
		int sid = gSynapsesActiveTable[idx];
		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
		atomicAdd(&(gNeuronInput[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid]);

	}
	__syncthreads();
}

//__global__ void update_basic_synapse(GBasicSynapses *d_synapses, int num, int start_id)
//{
//	__syncthreads();
//}
//
//__global__ void update_alpha_synapse(GAlphaSynapses *d_synapses, int num, int start_id)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
//		int sid = idx;
//		if (sid < num) {
//			d_synapses->p_I_syn[sid] = d_synapses->p_C1[sid] * d_synapses->p_I_syn[sid] + d_synapses->p_C2[sid] * d_synapses->p_I_tmp[sid];
//			d_synapses->p_I_tmp[sid] *= d_synapses->p_C1[sid];
//		}
//	}
//	__syncthreads();
//}


__global__ void init_buffers(/*int *c_gTimeTable,*/ real *c_gNeuronInput, int *c_gFiredTable, int *c_gFiredTableSizes, int *c_gActiveTable, int *c_gSynapsesActiveTable, int *c_gSynapsesLogTable) 
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gCurrentIdx = 0;
		gCurrentCycle = 0;
		gFiredTableSize = 0;
		gActiveTableSize = 0;
		gSynapsesActiveTableSize = 0;

		//gTimeTable = c_gTimeTable;
		gNeuronInput = c_gNeuronInput;
		gFiredTable = c_gFiredTable;
		gFiredTableSizes = c_gFiredTableSizes;
		gActiveTable = c_gActiveTable;
		gSynapsesActiveTable = c_gSynapsesActiveTable;
		gSynapsesLogTable = c_gSynapsesLogTable;
	}
}

GBuffers* alloc_buffers(int neuron_num, int synapse_num, int max_delay) 
{
	GBuffers *ret = (GBuffers*)malloc(sizeof(GBuffers));
	
	checkCudaErrors(hipMalloc((void**)&(ret->c_gNeuronInput), sizeof(real)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gNeuronInput, 0, sizeof(real)*(neuron_num)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gFiredTable), sizeof(int)*((neuron_num)*(max_delay+1))));
	checkCudaErrors(hipMemset(ret->c_gFiredTable, 0, sizeof(int)*((neuron_num)*(max_delay+1))));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gFiredTableSizes), sizeof(int)*(max_delay+1)));
	checkCudaErrors(hipMemset(ret->c_gFiredTableSizes, 0, sizeof(int)*(max_delay+1)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gActiveTable), sizeof(int)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gActiveTable, 0, sizeof(int)*(neuron_num)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gSynapsesActiveTable), sizeof(int)*(synapse_num)));
	checkCudaErrors(hipMemset(ret->c_gSynapsesActiveTable, 0, sizeof(int)*(synapse_num)));
	
	checkCudaErrors(hipMalloc((void**)&(ret->c_gSynapsesLogTable), sizeof(int)*(synapse_num)));
	checkCudaErrors(hipMemset(ret->c_gSynapsesLogTable, 0, sizeof(int)*(synapse_num)));

	int timeTableCap = max_delay+1;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(MAX_DELAY), &max_delay, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gTimeTableCap), &timeTableCap, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gFiredTableCap), &neuron_num, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gSynapsesTableCap), &synapse_num, sizeof(int)));
	//checkCudaErrors(hipMalloc((void**)&ret->c_gTimeTable, sizeof(int)*(max_delay+1)));
	//checkCudaErrors(hipMemset(ret->c_gTimeTable, 0, sizeof(int)*(max_delay+1)));

	checkCudaErrors(hipHostMalloc((void**)(&ret->c_neuronsFired), sizeof(int)*(neuron_num)));
	checkCudaErrors(hipHostMalloc((void**)(&ret->c_synapsesFired), sizeof(int)*(synapse_num)));

	init_buffers<<<1, 1, 0>>>(/*ret->c_gTimeTable,*/ ret->c_gNeuronInput, ret->c_gFiredTable, ret->c_gFiredTableSizes, ret->c_gActiveTable, ret->c_gSynapsesActiveTable, ret->c_gSynapsesLogTable);

	return ret;
}

int free_buffers(GBuffers *buf) 
{
	checkCudaErrors(hipFree(buf->c_gNeuronInput));
	checkCudaErrors(hipFree(buf->c_gFiredTable));
	checkCudaErrors(hipFree(buf->c_gFiredTableSizes));
	checkCudaErrors(hipFree(buf->c_gActiveTable));
	checkCudaErrors(hipFree(buf->c_gSynapsesActiveTable));
	checkCudaErrors(hipFree(buf->c_gSynapsesLogTable));

	checkCudaErrors(hipHostFree(buf->c_neuronsFired));
	checkCudaErrors(hipHostFree(buf->c_synapsesFired));

	return 0;
}
