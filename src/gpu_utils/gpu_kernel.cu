#include "hip/hip_runtime.h"

#include "gpu_macros.h"
#include "gpu_kernel.h"

#define SHARED_SIZE 512

__device__ int MAX_DELAY;
__device__ int gCurrentTime;
__device__ int *gTimeTable;
__device__ int gTimeTableSize;
__device__ real *gNeuronInput;
__device__ int *gFiredTable;
__device__ int gFiredTableLoc;
__device__ int gFiredTableSize;
__device__ int *gActiveFiredTableSize;

__device__ int *gActiveTable;
__device__ int gActiveTableLoc;
__device__ int *gSynapsesFiredTable;
__device__ int gSynapsesFiredTableSize;
__device__ int *gSynapsesLogTable;
__device__ int gSynapsesLogTableSize;

__device__ real *gNeuronInput;


__device__ int commit2globalTable(int *shared_buf, volatile unsigned int size, int *global_buf, int * global_size, int offset) 
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicadd(global_size, size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx] = shared_buf[idx];
	}
}

__device__ int updateTimeTable(int simTime)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gTimeTable[gCurrentTime] = simTime;
		gCurrentTime = (gCurrentTime +1)%(MAX_DELAY + 1);
	}
	__syncthreads();
	return 0;
}

__global__ void update_constant_neuron(GConstantNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = (simTime * d_neurons->p_fire_rate[nid]) > (d_neurons->p_fire_count[nid]);
		}

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicadd(&fire_cnt, 1);
				if (test_loc < SHARED_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= SHARED_SIZE) {
				commit2globalTable(fire_table_t, SHARED_SIZE, gFiredTable,&gFiredTableLoc, gSynapsesFiredTableSize*gCurrentTime);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable,&gFiredTableLoc, gSynapsesFiredTableSize*gCurrentTime);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

__global__ void find_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = d_neurons->p_refrac_step[nid] <= 0;
		}

		if (fired) {
			test_loc = atomicadd(fire_cnt, 1);
			if (test_loc < SHARED_SIZE) {
				fire_table_t[test_loc] = start_id + idx;
				fired = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_step[nid] - 1;
		}
		__syncthreads();

		if (fire_cnt >= SHARED_SIZE) {
			commit2globalTable(fire_table_t, SHARED_SIZE, gActiveTable, &gActiveTableLoc, 0);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			test_loc = atomicadd(fire_cnt, 1);
			if (test_loc < SHARED_SIZE) {
				fire_table_t[test_loc] = start_id + idx;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= SHARED_SIZE) {
			commit2globalTable(fire_table_t, SHARED_SIZE, gActiveTable, &gActiveTableLoc, 0);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
	}
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gActiveTable, &gActiveTableLoc, 0);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();

}

__device__ void reset_lif_neuron(GLIFNeurons *d_neurons, int *shared_buf, volatile unsigned int size) 
{
	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		int nid = shared_buf[idx];
		d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
		d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
	}
}

__global__ void update_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		if (idx < gActiveTableLoc) {
			int nid = gActiveTable[idx];
			int gnid = start_id + gActiveTable[idx];
			real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
			d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
			//d_neurons->p_i_syn[nid] = 0;
			gNeuronInput[gnid] = 0;

			fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

			for (int i=0; i<2; i++) {
				if (fired) {
					test_loc = atomicadd(&fire_cnt, 1);
					if (test_loc < SHARED_SIZE) {
						fire_table_t[test_loc] = nid;
						fired = false;
					}
				}
				__syncthreads();
				if (fire_cnt >= SHARED_SIZE) {
					commit2globalTable(fire_table_t, SHARED_SIZE, gFiredTable,&gFiredTableLoc, gSynapsesFiredTableSize*gCurrentTime);
					reset_lif_neuron(d_neurons, fire_table_t, SHARED_SIZE);
					if (threadIdx.x == 0) {
						fire_cnt = 0;
					}
				}
				__syncthreads();
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable,&gFiredTableLoc, gSynapsesFiredTableSize*gCurrentTime);
			reset_lif_neuron(d_neurons, fire_cnt, SHARED_SIZE);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
	}
	__syncthreads();
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		gActiveTableLoc = 0;
	}
}

__global__ void update_pre_synapse(GNetwork *d_net, int simTime)
{
	updateTimeTable(simTime);
	__syncthreads();
	for (int time = 0; time<MAX_DELAY+1; time++) {
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		int firedSize = gActiveFiredTableSize[time];
		int delta_t = simTime - gTimeTable[time]-1;
		if (delta_t < 0) {
			continue;
		}
		for (int idx = tid; idx < firedSize; idx += blockDim.x*gridDim.x) {
			int nid = gFiredTable[time*gFiredTableSize + idx];
			int start_loc = d_net->pN2SConnections[nid].delayStart[delta_t];
			int synapseNum = d_net->pN2SConnections[nid].delayNum[delta_t];
			int offset = atomicadd(&gActiveFiredTableLoc, synapseNum);
			for (int i=0; i<synapseNum; i++) {
				gSynapsesFiredTable[offset+i] = d_net->pN2SConnections[nid].pSynapsesIdx[i+start_loc];
			}
		}
	}
	__syncthreads();
}

__global__ void update_exp_hit(GExpSynapses *d_synapses, int num, int start_id, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gSynapsesFiredTableLoc; idx += blockDim.x*gridDim.x) {
		int sid = gSynapsesFiredTable[idx];
		d_synapses->p_I_syn[sid] += d_synapses->p_weight[sid];
		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
		gSynapsesLogTable[sid] = simTime;
		atomicAdd(&(gNeuronInput[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid])
	}
	__syncthreads();

}

__global__ void find_exp_synapse(GLIFNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = gSynapsesLogTable[start_id + idx] <= simTime;
		}

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicadd(fire_cnt, 1);
				if (test_loc < SHARED_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					fired = false;
				}
			}
			__syncthreads();

			if (fire_cnt >= SHARED_SIZE) {
				commit2globalTable(fire_table_t, SHARED_SIZE, gSynapsesFiredTable, &gSynapsesFiredTableLoc, 0);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gSynapsesFiredTable, &gSynapsesFiredTableLoc, 0);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();

}

__global__ void update_exp_synapse(GExpSynapses *d_synapses, int num, int start_id, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gSynapsesFiredTableLoc; idx += blockDim.x*gridDim.x) {
		int sid = gSynapsesFiredTable[idx];
		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
		atomicAdd(&(gNeuronInput[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid])

	}
	__syncthreads();
}

__global__ void update_basic_synapse(GBasicSynapses *d_synapses, int num, int simTime)
{
	__syncthreads();
}

__global__ void update_alpha_synapse(GAlphaSynapses *d_synapses, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
		int sid = idx;
		if (sid < num) {
			d_synapses->p_I_syn[sid] = d_synapses->p_C1[sid] * d_synapses->p_I_syn[sid] + d_synapses->p_C2[sid] * d_synapses->p_I_tmp[sid];
			d_synapses->p_I_tmp[sid] *= d_synapses->p_C1[sid];
		}
	}
	__syncthreads();
}


__global__ void init_global(int max_delay, int *c_gTimeTable, real *c_gNeuronInput, int *c_gFiredTable, int c_gFiredTableSize, bool *c_gSynapsesFiredTable, int c_gSynapsesFiredTableSize, GNetwork* network) 
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		MAX_DELAY = max_delay;
		gCurrentTime = 0;
		gTimeTable = c_gTimeTable;
		gTimeTableSize = MAX_DELAY + 1;
		gNeuronInput = c_gNeuronInput;
		gNeuronNum = c_gFiredTableSize;
		gFiredTable = c_gFiredTable;
		gFiredTableLoc = 0;
		gFiredTableSize = c_gFiredTableSize;
		gSynapsesFiredTable = c_gSynapsesFiredTable;
		gSynapsesFiredTableSize = c_gSynapsesFiredTableSize;
		gFiredCnt = 0;
		gFiredCntTest = 0;
		//gGpuNet = network;
	}
}
