#include "hip/hip_runtime.h"

#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "mem_op.h"
#include "gpu_macros.h"
#include "gpu_kernel.h"

#define MAXBLOCKSIZE 1024

// Constant
__constant__ int MAX_DELAY;
__constant__ int gTimeTableCap;
__constant__ int gFiredTableCap;
__constant__ int gSynapsesTableCap;

// Variable
__device__ int gCurrentIdx;
__device__ int gCurrentCycle;
__device__ int gFiredTableSize;
__device__ int gActiveTableSize;
__device__ int gSynapsesActiveTableSize;

// Arrays
//__device__ int *gTimeTable;

// Neuron Arrays
__device__ real *gNeuronInput;
__device__ real *gNeuronInput_I;

// Neuron Tables
__device__ int *gFiredTable;
__device__ int *gFiredTableSizes;
__device__ int *gActiveTable;

// Synapse Tables
//__device__ int *gSynapsesActiveTable;
//__device__ int *gSynapsesLogTable;

// Log Arrays
__device__ int *gLayerInput;
__device__ real *gXInput;
__device__ int *gFireCount;

// Connection
__device__ N2SConnection *gConnection;

//#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
//#else
//__device__ double atomicAdd(double* address, double val)
//{
//	unsigned long long int* address_as_ull = (unsigned long long int*)address;
//	unsigned long long int old = *address_as_ull, assumed;
//	do {
//		assumed = old;
//		old = atomicCAS(address_as_ull, assumed,
//				__double_as_longlong(val + __longlong_as_double(assumed)));
//	} while (assumed != old);
//	return __longlong_as_double(old);
//}
//#endif


__device__ int commit2globalTable(int *shared_buf, volatile unsigned int size, int *global_buf, int * global_size, int offset) 
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicAdd(global_size, (int)size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx] = shared_buf[idx];
	}

	return 0;
}

__global__ void update_time()
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		//gTimeTable[gCurrentIdx] = simTime;
		gCurrentCycle = gCurrentCycle + 1;
		gCurrentIdx = (gCurrentIdx +1)%(MAX_DELAY + 1);
		gActiveTableSize = 0;
		gFiredTableSizes[gCurrentIdx] = 0;
		gSynapsesActiveTableSize = 0;
	}
	__syncthreads();
}

__global__ void reset_active_synapse()
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gSynapsesActiveTableSize = 0;
	}
	__syncthreads();

}

__global__ void update_constant_neuron(GConstantNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		fired = (gCurrentCycle * d_neurons->p_fire_rate[idx]) > (d_neurons->p_fire_count[idx]);

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = start_id + idx;
					d_neurons->p_fire_count[idx] = d_neurons->p_fire_count[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

__global__ void curand_setup_kernel(hiprandState *state, int num)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < num) {
		hiprand_init(1234, id, 0, &state[id]); 
	}
}

__device__ void reset_poisson_neuron(GPoissonNeurons *d_neurons, int *shared_buf, volatile int size, int start_id) 
{
	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		int nid = shared_buf[idx] - start_id;

		hiprandState localState = d_neurons->p_state[nid];
		int tmp = hiprand_poisson(&localState, d_neurons->p_rate[nid]);
		d_neurons->p_fire_cycle[nid] = d_neurons->p_fire_cycle[nid] + 1 + tmp + d_neurons->p_refrac_step[nid];
		d_neurons->p_state[nid] = localState;
	}
}

__global__ void update_poisson_neuron(GPoissonNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		fired = (gCurrentCycle == d_neurons->p_fire_cycle[idx]) && (gCurrentCycle <= d_neurons->p_end_cycle[idx]);

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = start_id + idx;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
				reset_poisson_neuron(d_neurons, fire_table_t, MAXBLOCKSIZE, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		reset_poisson_neuron(d_neurons, fire_table_t, fire_cnt, start_id);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

//__device__ void advance_array_neuron(GArrayNeurons *d_neurons, int *shared_buf, volatile int size, int start_id) 
//{
//	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
//		int offset = shared_buf[idx] - start_id;
//		d_neurons->p_start[offset] = d_neurons->p_start[offset] + 1;
//	}
//}

__global__ void update_array_neuron(GArrayNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		fired = (d_neurons->p_start[idx] < d_neurons->p_end[idx]) &&  (gCurrentCycle >= d_neurons->p_fire_time[d_neurons->p_start[idx]]);

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = start_id + idx;
					d_neurons->p_start[idx] = d_neurons->p_start[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
				//advance_array_neuron(d_neurons, fire_table_t, MAXBLOCKSIZE, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		//advance_array_neuron(d_neurons, fire_table_t, fire_cnt, start_id);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

__global__ void update_max_neuron(GMaxNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int input = (int)gNeuronInput[start_id + idx];
		gNeuronInput[start_id + idx] = 0;
		int test = 1;
		int record_offset = idx*d_neurons->max_N; 
		for (int i=0; i<d_neurons->p_N[idx]; i++) {
			if (input & test) {
				d_neurons->p_record[record_offset + i]++;
				if (d_neurons->p_record[record_offset + i] > d_neurons->p_count[idx]) {

					fired = true;
				}
			}
			test = test << 1;
		}

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = start_id + idx;
					d_neurons->p_count[idx] = d_neurons->p_count[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
				//advance_array_neuron(d_neurons, fire_table_t, MAXBLOCKSIZE, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		//advance_array_neuron(d_neurons, fire_table_t, fire_cnt, start_id);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}

}

__global__ void find_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id)
{
	__shared__ int active_table_t[MAXBLOCKSIZE];
	__shared__ volatile int active_cnt;

	if (threadIdx.x == 0) {
		active_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool actived = false;
		int test_loc = 0;

		actived = d_neurons->p_refrac_step[idx] <= 0;

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		__syncthreads();

		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		}
		__syncthreads();
		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (active_cnt > 0) {
			commit2globalTable(active_table_t, active_cnt, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
	}
}

__device__ void reset_lif_neuron(GLIFNeurons *d_neurons, int *shared_buf, volatile int size, int start_id) 
{
	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		int nid = shared_buf[idx] - start_id;
		d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
		d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
	}
}

__global__ void update_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int nid = gActiveTable[idx];
		int gnid = start_id + nid; 
		real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
		d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
		//d_neurons->p_i_syn[nid] = 0;
		gXInput[gnid] += gNeuronInput[gnid];
		gNeuronInput[gnid] = 0;

		fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

		gFireCount[gnid] += fired;

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = gnid;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
				reset_lif_neuron(d_neurons, fire_table_t, MAXBLOCKSIZE, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			reset_lif_neuron(d_neurons, fire_table_t, fire_cnt, start_id);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
	}
	//__syncthreads();
	//if (threadIdx.x == 0 && blockIdx.x == 0) {
	//	gActiveTableSize = 0;
	//}
}

__global__ void update_all_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int gnid = start_id + idx; 
		bool actived = d_neurons->p_refrac_step[idx] <= 0;
		if (actived) {
			real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[idx];
			d_neurons->p_vm[idx] = d_neurons->p_vm[idx] * d_neurons->p_C1[idx] + d_neurons->p_C2[idx] * I;
			//d_neurons->p_i_syn[idx] = 0;
			gXInput[gnid] += gNeuronInput[gnid];

			fired = d_neurons->p_vm[idx] >= d_neurons->p_v_thresh[idx];

			gFireCount[gnid] += fired;

			for (int i=0; i<2; i++) {
				if (fired) {
					test_loc = atomicAdd((int*)&fire_cnt, 1);
					if (test_loc < MAXBLOCKSIZE) {
						fire_table_t[test_loc] = gnid;
						fired = false;
					}
				}
				__syncthreads();
				if (fire_cnt >= MAXBLOCKSIZE) {
					commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
					reset_lif_neuron(d_neurons, fire_table_t, MAXBLOCKSIZE, start_id);
					if (threadIdx.x == 0) {
						fire_cnt = 0;
					}
				}
				__syncthreads();
			}
			__syncthreads();

			if (fire_cnt > 0) {
				commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
				reset_lif_neuron(d_neurons, fire_table_t, fire_cnt, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
		} else {
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		gNeuronInput[start_id + idx] = 0;
	}
	__syncthreads();
}

__global__ void find_life_neuron(GLIFENeurons *d_neurons, int num, int start_id)
{
	__shared__ int active_table_t[MAXBLOCKSIZE];
	__shared__ volatile int active_cnt;

	if (threadIdx.x == 0) {
		active_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		//bool actived = false;
		int test_loc = 0;

		bool actived = d_neurons->p_refrac_step[idx] <= 0;

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			gNeuronInput_I[start_id + idx] = 0;
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		__syncthreads();

		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		}
		__syncthreads();
		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
		if (active_cnt > 0) {
			commit2globalTable(active_table_t, active_cnt, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
	}
}

__global__ void update_life_neuron(GLIFENeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int nid = gActiveTable[idx];
		int gnid = start_id + nid; 

		//real I = sqrtf(d_neurons->p_CE[nid]) * d_neurons->p_i_E[nid] + sqrtf(d_neurons->p_CI[nid]) * d_neurons->p_i_I[nid] + d_neurons->p_i_tmp[nid];

		//real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
		//d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;

		d_neurons->p_vm[nid] = d_neurons->p_Cm[nid] * d_neurons->p_vm[nid] + d_neurons->p_v_tmp[nid] + d_neurons->p_i_E[nid] * d_neurons->p_C_E[nid] + d_neurons->p_i_I[nid] * d_neurons->p_C_I[nid];

		//d_neurons->p_i_syn[nid] = 0;

		d_neurons->p_i_E[nid] *= d_neurons->p_CE[nid];
		d_neurons->p_i_I[nid] *= d_neurons->p_CI[nid];

		fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

		gFireCount[gnid] += fired;

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}

			d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
			d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
		} else {
			gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];
			d_neurons->p_i_E[nid] += gNeuronInput[gnid];
			d_neurons->p_i_I[nid] += gNeuronInput_I[gnid];
		}

		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;

		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		__syncthreads();

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

	}
	//__syncthreads();
	//if (threadIdx.x == 0 && blockIdx.x == 0) {
	//	gActiveTableSize = 0;
	//}
}

__global__ void update_all_life_neuron(GLIFENeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int nid = idx;
		int gnid = start_id + idx; 
		bool actived = d_neurons->p_refrac_step[idx] <= 0;

		if (actived) {
			//real I = sqrtf(d_neurons->p_CE[nid]) * d_neurons->p_i_E[nid] + sqrtf(d_neurons->p_CI[nid]) * d_neurons->p_i_I[nid] + d_neurons->p_i_tmp[nid];

			//real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
			//d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
			//d_neurons->p_i_syn[nid] = 0;

			d_neurons->p_vm[nid] = d_neurons->p_Cm[nid] * d_neurons->p_vm[nid] + d_neurons->p_v_tmp[nid] + d_neurons->p_i_E[nid] * d_neurons->p_C_E[nid] + d_neurons->p_i_I[nid] * d_neurons->p_C_I[nid];

			gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];

			d_neurons->p_i_E[nid] *= d_neurons->p_CE[nid];
			d_neurons->p_i_I[nid] *= d_neurons->p_CI[nid];

			fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

			gFireCount[gnid] += fired;

			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = gnid;
					fired = false;
				}

				d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
				d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
			} else {
				d_neurons->p_i_E[nid] += gNeuronInput[gnid];
				d_neurons->p_i_I[nid] += gNeuronInput_I[gnid];
			}

			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}

			__syncthreads();

			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = gnid;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();

			if (fire_cnt > 0) {
				commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
		} else {
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;
	}
	__syncthreads();
}

__global__ void update_dense_life_neuron(GLIFENeurons *d_neurons, int num, int start_id)
{
	//__shared__ int fire_table_t[MAXBLOCKSIZE];
	//__shared__ volatile int fire_cnt;

	//if (threadIdx.x == 0) {
	//	fire_cnt = 0;
	//}
	//__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		//bool fired = false;
		//int test_loc = 0;

		int nid = idx;
		int gnid = start_id + idx; 
		bool actived = d_neurons->p_refrac_step[idx] <= 0;

		if (actived) {
			d_neurons->p_vm[nid] = d_neurons->p_Cm[nid] * d_neurons->p_vm[nid] + d_neurons->p_v_tmp[nid] + d_neurons->p_i_E[nid] * d_neurons->p_C_E[nid] + d_neurons->p_i_I[nid] * d_neurons->p_C_I[nid];

			d_neurons->p_i_E[nid] *= d_neurons->p_CE[nid];
			d_neurons->p_i_I[nid] *= d_neurons->p_CI[nid];

			bool fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

			gFiredTable[gFiredTableCap*gCurrentIdx + gnid] = fired;

			gFireCount[gnid] += fired;

			if (fired) {
				d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
				d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];

			} else {
				gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];
				d_neurons->p_i_E[nid] += gNeuronInput[gnid];
				d_neurons->p_i_I[nid] += gNeuronInput_I[gnid];
				//real input = 0, input_I = 0;
				//for (int i=d_neurons->p_start_E[nid]; i<d_neurons->p_start_I[nid]; i++) {
				//	input += gNeuronInput[i];
				//}
				//for (int i=d_neurons->p_start_I[nid]; i<d_neurons->p_end[nid]; i++) {
				//	input_I += gNeuronInput[i];
				//}
				//d_neurons->p_i_E[nid] += input;
				//d_neurons->p_i_I[nid] += input_I;
				//gXInput[gnid] += input + input_I;
			}

		} else {
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
			gFiredTable[gFiredTableCap*gCurrentIdx + gnid] = 0;
		}
		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;
	}
	__syncthreads();
}

__global__ void update_pre_synapse(N2SConnection *pConnection)
{
	__shared__ int fire_neuron_id[MAXBLOCKSIZE];
	__shared__ int fire_synapse_id[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;
	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int block_idx = blockIdx.x;
	for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
		int time_idx = (gCurrentIdx+MAX_DELAY-delta_t)%(MAX_DELAY+1);
		int firedSize = gFiredTableSizes[time_idx];
		int block_nums = (firedSize + blockDim.x - 1) / blockDim.x;
		for (int idx = tid; idx < firedSize; idx += blockDim.x*gridDim.x) {
			block_idx += gridDim.x;
			fire_neuron_id[threadIdx.x] = gFiredTable[time_idx*gFiredTableCap + idx];
			__syncthreads();

			int size = 0;
			if (block_idx == block_nums - 1) {
				size = firedSize - block_idx * blockDim.x;
			} else if (block_idx < block_nums - 1) {
				size = blockDim.x;
			} else {
				size = 0;
			}

			for (int i=0; i<size; i++) {
				int nid =  fire_neuron_id[i];
				int start_loc = pConnection->delayStart[delta_t + nid * MAX_DELAY];
				int synapseNum = pConnection->delayNum[delta_t + nid * MAX_DELAY];
				for (int j=tid; j<synapseNum; j += blockDim.x) {
					int test_loc = atomicAdd((int*)&fire_cnt, 1);
					//fire_synapse_id[threadIdx.x] = pConnection->pSynapsesIdx[j+start_loc];
					fire_synapse_id[threadIdx.x] = j + start_loc;
					__syncthreads();
					//TODO
				}
			}

			//int offset = atomicAdd(&gSynapsesActiveTableSize, synapseNum);
			//gLayerInput[nid]++;
			//for (int i=0; i<synapseNum; i++) {
			//	gSynapsesActiveTable[offset+i] = pConnection->pSynapsesIdx[i+start_loc];
			//}
		}
	}

	//for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
	//	int time_idx = (gCurrentIdx+MAX_DELAY-delta_t)%(MAX_DELAY+1);
	//	int firedSize = gFiredTableSizes[time_idx];
	//	for (int idx = tid; idx < firedSize; idx += blockDim.x*gridDim.x) {
	//		int nid = gFiredTable[time_idx*gFiredTableCap + idx];
	//		int start_loc = pConnection->delayStart[delta_t + nid * MAX_DELAY];
	//		int synapseNum = pConnection->delayNum[delta_t + nid * MAX_DELAY];
	//		int offset = atomicAdd(&gSynapsesActiveTableSize, synapseNum);
	//		gLayerInput[nid]++;
	//		for (int i=0; i<synapseNum; i++) {
	//			gSynapsesActiveTable[offset+i] = pConnection->pSynapsesIdx[i+start_loc];
	//		}
	//	}
	//}
	__syncthreads();
}

//__global__ void update_exp_hit(GExpSynapses *d_synapses, int num, int start_id)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int idx = tid; idx < gSynapsesActiveTableSize; idx += blockDim.x*gridDim.x) {
//		int sid = gSynapsesActiveTable[idx];
//		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
//		d_synapses->p_I_syn[sid] += d_synapses->p_weight[sid];
//		gSynapsesLogTable[start_id + sid] = gCurrentCycle;
//		atomicAdd(&(gNeuronInput[d_synapses->p_dst[sid]]), d_synapses->p_I_syn[sid]);
//	}
//	__syncthreads();
//
//}
//
//__global__ void find_exp_synapse(GExpSynapses *d_synapses, int num, int start_id)
//{
//	__shared__ int active_table_t[MAXBLOCKSIZE];
//	__shared__ volatile unsigned int active_cnt;
//
//	if (threadIdx.x == 0) {
//		active_cnt = 0;
//	}
//	__syncthreads();
//
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
//		bool actived = false;
//		int test_loc = 0;
//
//		actived = (gSynapsesLogTable[start_id + idx] < gCurrentCycle);
//
//		for (int i=0; i<2; i++) {
//			if (actived) {
//				test_loc = atomicAdd((int*)&active_cnt, 1);
//				if (test_loc < MAXBLOCKSIZE) {
//					active_table_t[test_loc] = idx;
//					actived = false;
//				}
//			}
//			__syncthreads();
//
//			if (active_cnt >= MAXBLOCKSIZE) {
//				commit2globalTable(active_table_t, MAXBLOCKSIZE, gSynapsesActiveTable, &gSynapsesActiveTableSize, 0);
//				if (threadIdx.x == 0) {
//					active_cnt = 0;
//				}
//			}
//			__syncthreads();
//		}
//	}
//	
//	if (active_cnt > 0) {
//		commit2globalTable(active_table_t, active_cnt, gSynapsesActiveTable, &gSynapsesActiveTableSize, 0);
//		if (threadIdx.x == 0) {
//			active_cnt = 0;
//		}
//	}
//	__syncthreads();
//
//}
//
//__global__ void update_exp_synapse(GExpSynapses *d_synapses, int num, int start_id)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int idx = tid; idx < gSynapsesActiveTableSize; idx += blockDim.x*gridDim.x) {
//		int sid = gSynapsesActiveTable[idx];
//		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
//
//		if (gSynapsesLogTable[sid] + d_synapses->p_active_steps[sid] <= gCurrentCycle) {
//			atomicAdd(&(gNeuronInput[d_synapses->p_dst[sid]]), d_synapses->p_I_syn[sid]);
//		} else {
//			d_synapses->p_I_syn[sid] = 0; 
//		}
//
//	}
//	__syncthreads();
//}
//
//__global__ void update_all_exp_synapse(GExpSynapses *d_synapses, int num, int start_id)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
//		if (gSynapsesLogTable[idx] < gCurrentCycle) {
//			d_synapses->p_I_syn[idx] *= d_synapses->p_C1[idx];
//
//			if (gSynapsesLogTable[idx] + d_synapses->p_active_steps[idx] <= gCurrentCycle) {
//				atomicAdd(&(gNeuronInput[d_synapses->p_dst[idx]]), d_synapses->p_I_syn[idx]);
//			} else {
//				d_synapses->p_I_syn[idx] = 0; 
//			}
//		}
//
//	}
//	__syncthreads();
//}

__global__ void update_dense_static_hit(GStaticSynapses *d_synapses, int num, int start_id)
{
#ifndef FAST_TEST
	__shared__ int fire_neuron_id[MAXBLOCKSIZE];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
		int block_idx = blockIdx.x;
		int time_idx = (gCurrentIdx+MAX_DELAY-delta_t)%(MAX_DELAY+1);
		int firedSize = gFiredTableSizes[time_idx];
		int block_nums_minus_1 = (firedSize - 1 + blockDim.x) / blockDim.x - 1;
		int grid_nums = (firedSize - 1 + blockDim.x*gridDim.x)/(blockDim.x * gridDim.x);
		int oid = tid;
		for (int idx = 0; idx < grid_nums; idx++) {
			if (oid < firedSize) {
				fire_neuron_id[threadIdx.x] = gFiredTable[time_idx*gFiredTableCap + oid];
			} else {
				fire_neuron_id[threadIdx.x] = -1;
			}
			oid += blockDim.x * gridDim.x;
			__syncthreads();

			int size = 0;
			if (block_idx == block_nums_minus_1) {
				size = firedSize - block_idx * blockDim.x;
			} else if (block_idx < block_nums_minus_1) {
				size = blockDim.x;
			} else {
				size = 0;
			}

			for (int i=0; i<size; i++) {
				int nid = fire_neuron_id[i];
				int start_loc = gConnection->delayStart[delta_t + nid * MAX_DELAY];
				int synapseNum = gConnection->delayNum[delta_t + nid * MAX_DELAY];
				for (int j=threadIdx.x; j<synapseNum; j += blockDim.x) {
					//int sid = gConnection->pSynapsesIdx[j+start_loc];
					int sid = j+start_loc;
					real weight = d_synapses->p_weight[sid];
					if (weight >= 0) {
						atomicAdd(&(gNeuronInput[d_synapses->p_dst[sid]]), weight);
					} else {
						atomicAdd(&(gNeuronInput_I[d_synapses->p_dst[sid]]), weight);
					}
				}
			}
			block_idx += gridDim.x;
			__syncthreads();
		}
		__syncthreads();
	}
#else
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
		int time_idx = (gCurrentIdx+MAX_DELAY-delta_t)%(MAX_DELAY+1);
		int firedSize = gFiredTableSizes[time_idx];
		for (int idx = tid; idx < firedSize; idx += blockDim.x*gridDim.x) {
			int nid = gFiredTable[time_idx*gFiredTableCap + idx];
			int start_loc = gConnection->delayStart[delta_t + nid * MAX_DELAY];
			int synapseNum = gConnection->delayNum[delta_t + nid * MAX_DELAY];
			gLayerInput[nid]++;
			for (int i=0; i<synapseNum; i++) {
				//int sid = gConnection->pSynapsesIdx[i+start_loc];
				int sid = i+start_loc;
				real weight = d_synapses->p_weight[sid];
				if (weight >= 0) {
					atomicAdd(&(gNeuronInput[d_synapses->p_dst[sid]]), weight);
				} else {
					atomicAdd(&(gNeuronInput_I[d_synapses->p_dst[sid]]), weight);
				}
			}
		}
	}
#endif
}

//__global__ void update_static_hit(GStaticSynapses *d_synapses, int num, int start_id)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int idx = tid; idx < gSynapsesActiveTableSize; idx += blockDim.x*gridDim.x) {
//		int sid = gSynapsesActiveTable[idx];
//		real weight = d_synapses->p_weight[sid];
//		gSynapsesLogTable[start_id + sid] = gCurrentCycle;
//		if (weight >= 0) {
//			atomicAdd(&(gNeuronInput[d_synapses->p_dst[sid]]), weight);
//		} else {
//			atomicAdd(&(gNeuronInput_I[d_synapses->p_dst[sid]]), weight);
//		}
//	}
//	__syncthreads();
//
//}

__global__ void add_cross_neuron(int *ids, int num)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num) {
		gFiredTable[gFiredTableCap*gCurrentIdx + gFiredTableSizes[gCurrentIdx] + tid] = ids[tid];
	}
	__syncthreads();

	if (tid == 0) {
		gFiredTableSizes[gCurrentIdx] += num;
	}
}

//__global__ void update_basic_synapse(GBasicSynapses *d_synapses, int num, int start_id)
//{
//	__syncthreads();
//}
//
//__global__ void update_alpha_synapse(GAlphaSynapses *d_synapses, int num, int start_id)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
//		int sid = idx;
//		if (sid < num) {
//			d_synapses->p_I_syn[sid] = d_synapses->p_C1[sid] * d_synapses->p_I_syn[sid] + d_synapses->p_C2[sid] * d_synapses->p_I_tmp[sid];
//			d_synapses->p_I_tmp[sid] *= d_synapses->p_C1[sid];
//		}
//	}
//	__syncthreads();
//}

__global__ void init_connection(N2SConnection *pConnection)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gConnection = pConnection;
	}
}

__global__ void init_buffers(/*int *c_gTimeTable,*/ real *c_gNeuronInput, real *c_gNeuronInput_I, int *c_gFiredTable, int *c_gFiredTableSizes, int *c_gActiveTable, int *c_gSynapsesActiveTable, int *c_gSynapsesLogTable) 
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gCurrentIdx = 0;
		gCurrentCycle = 0;
		gFiredTableSize = 0;
		gActiveTableSize = 0;
		gSynapsesActiveTableSize = 0;

		//gTimeTable = c_gTimeTable;
		gNeuronInput = c_gNeuronInput;
		gNeuronInput_I = c_gNeuronInput_I;
		gFiredTable = c_gFiredTable;
		gFiredTableSizes = c_gFiredTableSizes;
		gActiveTable = c_gActiveTable;
		//gSynapsesActiveTable = c_gSynapsesActiveTable;
		//gSynapsesLogTable = c_gSynapsesLogTable;
	}
}

__global__ void init_log_buffers(int * layer_input, real * x_input, int * fire_count)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gLayerInput = layer_input;
		gXInput = x_input;
		gFireCount = fire_count;
	}
}

GBuffers* alloc_buffers(int neuron_num, int synapse_num, int max_delay) 
{
	GBuffers *ret = (GBuffers*)malloc(sizeof(GBuffers));
	memset(ret, 0, sizeof(GBuffers));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gNeuronInput), sizeof(real)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gNeuronInput, 0, sizeof(real)*(neuron_num)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gNeuronInput_I), sizeof(real)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gNeuronInput_I, 0, sizeof(real)*(neuron_num)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gFiredTable), sizeof(int)*((neuron_num)*(max_delay+1))));
	checkCudaErrors(hipMemset(ret->c_gFiredTable, 0, sizeof(int)*((neuron_num)*(max_delay+1))));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gFiredTableSizes), sizeof(int)*(max_delay+1)));
	checkCudaErrors(hipMemset(ret->c_gFiredTableSizes, 0, sizeof(int)*(max_delay+1)));

	checkCudaErrors(hipMalloc((void**)&(ret->c_gActiveTable), sizeof(int)*(neuron_num)));
	checkCudaErrors(hipMemset(ret->c_gActiveTable, 0, sizeof(int)*(neuron_num)));

	//checkCudaErrors(hipMalloc((void**)&(ret->c_gSynapsesActiveTable), sizeof(int)*(synapse_num)));
	//checkCudaErrors(hipMemset(ret->c_gSynapsesActiveTable, 0, sizeof(int)*(synapse_num)));

	//checkCudaErrors(hipMalloc((void**)&(ret->c_gSynapsesLogTable), sizeof(int)*(synapse_num)));
	//checkCudaErrors(hipMemset(ret->c_gSynapsesLogTable, 0, sizeof(int)*(synapse_num)));

	ret->c_gLayerInput = gpuMalloc<int>(neuron_num);
	ret->c_gXInput = gpuMalloc<real>(neuron_num);
	ret->c_gFireCount = gpuMalloc<int>(neuron_num);

	int timeTableCap = max_delay+1;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(MAX_DELAY), &max_delay, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gTimeTableCap), &timeTableCap, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gFiredTableCap), &neuron_num, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gSynapsesTableCap), &synapse_num, sizeof(int)));
	//checkCudaErrors(hipMalloc((void**)&ret->c_gTimeTable, sizeof(int)*(max_delay+1)));
	//checkCudaErrors(hipMemset(ret->c_gTimeTable, 0, sizeof(int)*(max_delay+1)));

	checkCudaErrors(hipHostMalloc((void**)(&ret->c_neuronsFired), sizeof(int)*(neuron_num)));
	checkCudaErrors(hipHostMalloc((void**)(&ret->c_synapsesFired), sizeof(int)*(synapse_num)));

	init_buffers<<<1, 1, 0>>>(/*ret->c_gTimeTable,*/ ret->c_gNeuronInput, ret->c_gNeuronInput_I, ret->c_gFiredTable, ret->c_gFiredTableSizes, ret->c_gActiveTable, ret->c_gSynapsesActiveTable, ret->c_gSynapsesLogTable);

	init_log_buffers<<<1, 1, 0>>>(ret->c_gLayerInput, ret->c_gXInput, ret->c_gFireCount);

	return ret;
}

void init_buffers(GBuffers * buf) {
	init_buffers<<<1, 1, 0>>>(/*buf->c_gTimeTable,*/ buf->c_gNeuronInput, buf->c_gNeuronInput_I, buf->c_gFiredTable, buf->c_gFiredTableSizes, buf->c_gActiveTable, buf->c_gSynapsesActiveTable, buf->c_gSynapsesLogTable);

	init_log_buffers<<<1, 1, 0>>>(buf->c_gLayerInput, buf->c_gXInput, buf->c_gFireCount);
}

int free_buffers(GBuffers *buf) 
{
	checkCudaErrors(hipFree(buf->c_gNeuronInput));
	checkCudaErrors(hipFree(buf->c_gNeuronInput_I));
	checkCudaErrors(hipFree(buf->c_gFiredTable));
	checkCudaErrors(hipFree(buf->c_gFiredTableSizes));
	checkCudaErrors(hipFree(buf->c_gActiveTable));
	checkCudaErrors(hipFree(buf->c_gSynapsesActiveTable));
	checkCudaErrors(hipFree(buf->c_gSynapsesLogTable));

	checkCudaErrors(hipHostFree(buf->c_neuronsFired));
	checkCudaErrors(hipHostFree(buf->c_synapsesFired));

	return 0;
}
