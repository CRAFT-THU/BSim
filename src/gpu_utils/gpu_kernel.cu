#include "hip/hip_runtime.h"

#include "gpu_macros.h"
#include "gpu_kernel.h"

#define SHARED_SIZE 512

__device__ int MAX_DELAY;
__device__ int gCurrentTime;
__device__ int *gTimeTable;
__device__ int gTimeTableSize;
__device__ real *gNeuronInput;
__device__ int gNeuronNum;
__device__ int *gFiredTable;
__device__ int gFiredTableLoc;
__device__ int gFiredTableSize;
__device__ int gFiredCnt;
__device__ int gFiredCntTest;
//__device__ bool *gSynapsesFiredTable;
//__device__ int gSynapsesFiredTableSize;
__device__ GNetwork *gGpuNet;

__device__ int *gActiveTable;
__device__ int gActiveTableLoc;
__device__ int *gSynapsesFiredTable;
__device__ int *gSynapsesLogTable;
__device__ int gSynapsesLogTableSize;


__device__ int commit2globalTable(int *shared_buf, volatile unsigned int size, int *global_buf, int * global_size, int offset) 
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicadd(global_size, size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx] = shared_buf[idx];
	}

}

__device__ int updateTimeTable(int simTime)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gTimeTable[gCurrentTime] = simTime;
		gCurrentTime = (gCurrentTime +1)%(MAX_DELAY + 1);
	}
	__syncthreads();
	return 0;
}

__device__ int updateFiredTable(int firedID, int fired, int simTime)
{
	gFiredTable[gCurrentTime*gFiredTableSize + firedID] = fired;

	return 0;
}

__global__ void update_constant_neuron(GConstantNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = (simTime * d_neurons->p_fire_rate[nid]) > (d_neurons->p_fire_count[nid]);
		}

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicadd(&fire_cnt, 1);
				if (test_loc < SHARED_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= SHARED_SIZE) {
				commit2globalTable(fire_table_t, SHARED_SIZE, gFiredTable,&gFiredTableLoc, gSynapsesFiredTableSize*gCurrentTime);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable,&gFiredTableLoc, gSynapsesFiredTableSize*gCurrentTime);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

__global__ void find_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = d_neurons->p_refrac_step[nid] <= 0;
		}

		if (fired) {
			test_loc = atomicadd(fire_cnt, 1);
			if (test_loc < SHARED_SIZE) {
				fire_table_t[test_loc] = start_id + idx;
				fired = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_step[nid] - 1;
		}
		__syncthreads();

		if (fire_cnt >= SHARED_SIZE) {
			commit2globalTable(fire_table_t, SHARED_SIZE, gActiveTable, &gActiveTableLoc, 0);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			test_loc = atomicadd(fire_cnt, 1);
			if (test_loc < SHARED_SIZE) {
				fire_table_t[test_loc] = start_id + idx;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= SHARED_SIZE) {
			commit2globalTable(fire_table_t, SHARED_SIZE, gActiveTable, &gActiveTableLoc, 0);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
	}
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gActiveTable, &gActiveTableLoc, 0);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();

}

__device__ void reset_lif_neuron(GLIFNeurons *d_neurons, int *shared_buf, volatile unsigned int size) 
{
	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		int nid = shared_buf[idx];
		d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
		d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
	}
}

__global__ void update_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		if (idx < gActiveTableLoc) {
			int nid = gActiveTable[idx];
			int gnid = start_id + gActiveTable[idx];
			real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
			d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
			//d_neurons->p_i_syn[nid] = 0;
			gNeuronInput[gnid] = 0;

			fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

			for (int i=0; i<2; i++) {
				if (fired) {
					test_loc = atomicadd(&fire_cnt, 1);
					if (test_loc < SHARED_SIZE) {
						fire_table_t[test_loc] = nid;
						fired = false;
					}
				}
				__syncthreads();
				if (fire_cnt >= SHARED_SIZE) {
					commit2globalTable(fire_table_t, SHARED_SIZE, gFiredTable,&gFiredTableLoc, gSynapsesFiredTableSize*gCurrentTime);
					reset_lif_neuron(d_neurons, fire_table_t, SHARED_SIZE);
					if (threadIdx.x == 0) {
						fire_cnt = 0;
					}
				}
				__syncthreads();
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable,&gFiredTableLoc, gSynapsesFiredTableSize*gCurrentTime);
			reset_lif_neuron(d_neurons, fire_cnt, SHARED_SIZE);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
	}
}
__global__ void init_global(int max_delay, int *c_gTimeTable, real *c_gNeuronInput, int *c_gFiredTable, int c_gFiredTableSize, bool *c_gSynapsesFiredTable, int c_gSynapsesFiredTableSize, GNetwork* network) 
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		MAX_DELAY = max_delay;
		gCurrentTime = 0;
		gTimeTable = c_gTimeTable;
		gTimeTableSize = MAX_DELAY + 1;
		gNeuronInput = c_gNeuronInput;
		gNeuronNum = c_gFiredTableSize;
		gFiredTable = c_gFiredTable;
		gFiredTableLoc = 0;
		gFiredTableSize = c_gFiredTableSize;
		gSynapsesFiredTable = c_gSynapsesFiredTable;
		gSynapsesFiredTableSize = c_gSynapsesFiredTableSize;
		gFiredCnt = 0;
		gFiredCntTest = 0;
		gGpuNet = network;
	}
}

__global__ void update_pre_synapse(GNetwork *d_net, int simTime)
{
	for (int time = 0; time<MAX_DELAY+1; time++) {
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		int start_t = gTimeTable[time];
		for (int idx = tid; idx < gFiredTableSize; idx += blockDim.x*gridDim.x) {
			int nid = idx;
			int offset = 0;
			int type = get_type(d_net->neuronNums, d_net->nTypeNum, nid, &offset);
			int gnid = get_gnid(d_net, type, offset);
			if (gFiredTable[time*gFiredTableSize + gnid] > 0) {
				update_spike[d_net->nTypes[type]](d_net->pNeurons[type], d_net->neuronNums[type+1]-d_net->neuronNums[type], offset, start_t, simTime);
			}
		}
	}
	__syncthreads();
}

__global__ void update_post_synapse(GNetwork *d_net, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int num = d_net->synapseNums[d_net->sTypeNum];
	for (int idx = tid; idx<num; idx += blockDim.x*gridDim.x) {
		int sid = idx;
		int offset = 0;
		int type = get_type(d_net->synapseNums, d_net->sTypeNum, sid, &offset);
		int gsid = get_gsid(d_net, type, offset);
		if (gSynapsesFiredTable[gsid]) {
			update_spike[d_net->sTypes[type]](d_net->pSynapses[type], d_net->synapseNums[type+1]-d_net->neuronNums[type], offset, 0, simTime);
			gSynapsesFiredTable[gsid] = false;
		}
	}
}

__global__ void update_pre_synapse(GLIFNeurons *d_neurons, GExpSynapses* d_synapses, int simTime)
{
	for (int time = 0; time<MAX_DELAY+1; time++) {
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		int start_t = gTimeTable[time];
		for (int idx = tid; idx < gFiredTableSize; idx += blockDim.x*gridDim.x) {
			int nid = idx;
			if (gFiredTable[time*gFiredTableSize + nid]) {
				for (int i=0; i<d_neurons->pSynapsesNum[nid]; i++) {
					int loc = d_neurons->pSynapsesLoc[nid];
					int sid = d_neurons->pSynapsesIdx[i+loc];
					if (simTime == start_t + d_synapses->p_delay_steps[sid])
						gSynapsesFiredTable[d_neurons->pSynapsesIdx[i+loc]] = true;
				}
			}
		}
	}
	__syncthreads();
}


__global__ void update_basic_synapse(GBasicSynapses *d_synapses, int num, int simTime)
{
	__syncthreads();
}

__global__ void update_alpha_synapse(GAlphaSynapses *d_synapses, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
		int sid = idx;
		if (sid < num) {
			d_synapses->p_I_syn[sid] = d_synapses->p_C1[sid] * d_synapses->p_I_syn[sid] + d_synapses->p_C2[sid] * d_synapses->p_I_tmp[sid];
			d_synapses->p_I_tmp[sid] *= d_synapses->p_C1[sid];
		}
	}
	__syncthreads();
}

__global__ void update_exp_synapse(GExpSynapses *d_synapses, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
		int sid = idx;
		if (sid < num) {
			d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
		}
	}
	__syncthreads();
}
