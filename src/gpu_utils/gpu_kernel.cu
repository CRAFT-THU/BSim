#include "hip/hip_runtime.h"

#include "gpu_macros.h"
#include "gpu_kernel.h"

#define SHARED_SIZE 512

// Constant
__constant__ int MAX_DELAY;
__constant__ int gTimeTableCap;
__constant__ int gFiredTableCap;
__constant__ int gSynapsesActiveTableCap;

// Variable
__device__ int gCurrentTime;
__device__ int gFiredTableSize;

// Arrays
__device__ int *gTimeTable;

// Neuron Arrays
__device__ real *gNeuronInput;

// Neuron Tables
__device__ int *gFiredTable;
__device__ int *gFiredTableSizes;

__device__ int *gActiveTable;
__device__ int gActiveTableSize;

// Synapse Tables
__device__ int *gSynapsesActiveTable;
__device__ int *gSynapsesActiveTableSize;

__device__ int *gSynapsesLogTable;



__device__ int commit2globalTable(int *shared_buf, volatile unsigned int size, int *global_buf, int * global_size, int offset) 
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicadd(global_size, size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx] = shared_buf[idx];
	}
}

__device__ int updateTimeTable(int simTime)
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		gTimeTable[gCurrentTime] = simTime;
		gCurrentTime = (gCurrentTime +1)%(MAX_DELAY + 1);
	}
	__syncthreads();
	return 0;
}

__global__ void update_constant_neuron(GConstantNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = (simTime * d_neurons->p_fire_rate[nid]) > (d_neurons->p_fire_count[nid]);
		}

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicadd(&fire_cnt, 1);
				if (test_loc < SHARED_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= SHARED_SIZE) {
				commit2globalTable(fire_table_t, SHARED_SIZE, gFiredTable,&gFiredTableSize, gFiredTableCap*gCurrentTime);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable,&gFiredTableSize, gFiredTableCap*gCurrentTime);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

__global__ void find_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = d_neurons->p_refrac_step[nid] <= 0;
		}

		if (fired) {
			test_loc = atomicadd(fire_cnt, 1);
			if (test_loc < SHARED_SIZE) {
				fire_table_t[test_loc] = start_id + idx;
				fired = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_step[nid] - 1;
		}
		__syncthreads();

		if (fire_cnt >= SHARED_SIZE) {
			commit2globalTable(fire_table_t, SHARED_SIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			test_loc = atomicadd(fire_cnt, 1);
			if (test_loc < SHARED_SIZE) {
				fire_table_t[test_loc] = start_id + idx;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= SHARED_SIZE) {
			commit2globalTable(fire_table_t, SHARED_SIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
	}
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gActiveTable, &gActiveTableSize, 0);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();

}

__device__ void reset_lif_neuron(GLIFNeurons *d_neurons, int *shared_buf, volatile unsigned int size) 
{
	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		int nid = shared_buf[idx];
		d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
		d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
	}
}

__global__ void update_lif_neuron(GLIFNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		if (idx < gActiveTableSize) {
			int nid = gActiveTable[idx];
			int gnid = start_id + gActiveTable[idx];
			real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
			d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
			//d_neurons->p_i_syn[nid] = 0;
			gNeuronInput[gnid] = 0;

			fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

			for (int i=0; i<2; i++) {
				if (fired) {
					test_loc = atomicadd(&fire_cnt, 1);
					if (test_loc < SHARED_SIZE) {
						fire_table_t[test_loc] = nid;
						fired = false;
					}
				}
				__syncthreads();
				if (fire_cnt >= SHARED_SIZE) {
					commit2globalTable(fire_table_t, SHARED_SIZE, gFiredTable, &gFiredTableSize, gFiredTableCap*gCurrentTime);
					reset_lif_neuron(d_neurons, fire_table_t, SHARED_SIZE);
					if (threadIdx.x == 0) {
						fire_cnt = 0;
					}
				}
				__syncthreads();
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable,&gFiredTableSize, gFiredTableCap*gCurrentTime);
			reset_lif_neuron(d_neurons, fire_cnt, SHARED_SIZE);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
	}
	__syncthreads();
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		gActiveTableSize = 0;
	}
}

__global__ void update_pre_synapse(GNetwork *d_net, int simTime)
{
	updateTimeTable(simTime);
	__syncthreads();
	for (int time = 0; time<MAX_DELAY+1; time++) {
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		int firedSize = gFiredTableSizes[time];
		int delta_t = simTime - gTimeTable[time]-1;
		if (delta_t < 0) {
			continue;
		}
		for (int idx = tid; idx < firedSize; idx += blockDim.x*gridDim.x) {
			int nid = gFiredTable[time*gFiredTableCap + idx];
			int start_loc = d_net->pN2SConnections[nid].delayStart[delta_t];
			int synapseNum = d_net->pN2SConnections[nid].delayNum[delta_t];
			int offset = atomicadd(&gActiveFiredTableLoc, synapseNum);
			for (int i=0; i<synapseNum; i++) {
				gSynapsesActiveTable[offset+i] = d_net->pN2SConnections[nid].pSynapsesIdx[i+start_loc];
			}
		}
	}
	__syncthreads();
}

__global__ void update_exp_hit(GExpSynapses *d_synapses, int num, int start_id, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gSynapsesActiveTableSize; idx += blockDim.x*gridDim.x) {
		int sid = gSynapsesActiveTable[idx];
		d_synapses->p_I_syn[sid] += d_synapses->p_weight[sid];
		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
		gSynapsesLogTable[sid] = simTime;
		atomicAdd(&(gNeuronInput[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid])
	}
	__syncthreads();

}

__global__ void find_exp_synapse(GLIFNeurons *d_neurons, int num, int start_id, int simTime)
{
	__shared__ int fire_table_t[SHARED_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		if (idx < num) {
			fired = gSynapsesLogTable[start_id + idx] <= simTime;
		}

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicadd(fire_cnt, 1);
				if (test_loc < SHARED_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					fired = false;
				}
			}
			__syncthreads();

			if (fire_cnt >= SHARED_SIZE) {
				commit2globalTable(fire_table_t, SHARED_SIZE, gSynapsesActiveTable, &gSynapsesActiveTableSize, 0);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gSynapsesActiveTable, &gSynapsesActiveTableSize, 0);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();

}

__global__ void update_exp_synapse(GExpSynapses *d_synapses, int num, int start_id, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gSynapsesActiveTableSize; idx += blockDim.x*gridDim.x) {
		int sid = gSynapsesActiveTable[idx];
		d_synapses->p_I_syn[sid] *= d_synapses->p_C1[sid];
		atomicAdd(&(gNeuronInput[d_synapses->pDst[sid]]), d_synapses->p_I_syn[sid])

	}
	__syncthreads();
}

__global__ void update_basic_synapse(GBasicSynapses *d_synapses, int num, int simTime)
{
	__syncthreads();
}

__global__ void update_alpha_synapse(GAlphaSynapses *d_synapses, int num, int simTime)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
		int sid = idx;
		if (sid < num) {
			d_synapses->p_I_syn[sid] = d_synapses->p_C1[sid] * d_synapses->p_I_syn[sid] + d_synapses->p_C2[sid] * d_synapses->p_I_tmp[sid];
			d_synapses->p_I_tmp[sid] *= d_synapses->p_C1[sid];
		}
	}
	__syncthreads();
}


__global__ void init_global(int max_delay, int *c_gTimeTable, real *c_gNeuronInput, int *c_gFiredTable, int c_gFiredTableCap, bool *c_gSynapsesActiveTable, int c_gSynapsesActiveTableCap, GNetwork* network) 
{
	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		MAX_DELAY = max_delay;
		gCurrentTime = 0;
		gTimeTable = c_gTimeTable;
		gTimeTableCap = MAX_DELAY + 1;
		gNeuronInput = c_gNeuronInput;
		gNeuronNum = c_gFiredTableCap;
		gFiredTable = c_gFiredTable;
		gFiredTableSize = 0;
		gFiredTableCap = c_gFiredTableCap;
		gSynapsesActiveTable = c_gSynapsesActiveTable;
		gSynapsesActiveTableCap = c_gSynapsesActiveTableCap;
		gFiredCnt = 0;
		gFiredCntTest = 0;
		//gGpuNet = network;
	}
}
