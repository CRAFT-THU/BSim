
#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "gpu_utils.h"

#define CHECK(p1, p2, para) \
	do { \
		assert((p1->para) == (p2->para)); \
	} while(0)

#define CHECK_ARRAY(p1, p2, n) \
	do { \
		for (int i_t=0; i_t<(n); i++) {\
			assert(((p1)[i_t]) == ((p2)[i_t])); \
		}\
	} while(0)

#define CHECK_CROSS_ARRAY(g, c, size) \
	do { \
		unsigned char * mem_t = (unsigned char*)malloc(size); \
		checkCudaErrors(hipMemcpy(mem_t, (g), size, hipMemcpyDeviceToHost)); \
		assert(0==memcmp(mem_t, (c), (size))); \
		free(mem_t); \
	} while(0)


int checkGPUNetwork(GNetwork *g, GNetwork *c)
{
	CHECK(g, c, nTypeNum);
	CHECK(g, c, sTypeNum);
	CHECK(g, c, noffsets);
	CHECK(g, c, soffsets);
	CHECK(g, c, neuronNums);
	CHECK(g, c, synapseNums);

	CHECK(g, c, nTypes);
	CHECK(g, c, sTypes);
	CHECK(g, c, gNeuronNums);
	CHECK(g, c, gSynapseNums);

	return 1;

	int totalNeuronNum = g->neuronNums[g->nTypeNum+1];
	int totalSynapseNum = g->synapseNums[g->sTypeNum+1];
	int MAX_DELAY = pCpuNet->MAX_DELAY;

	N2SConnection p;
	checkCudaErrors(hipMemcpy(&p, g->pN2SConnection, sizeof(N2SConnection), hipMemcpyDeviceToHost));

	CHECK_CROSS_ARRAY(p.pSynapsesIdx, c->pN2SConnection->pSynapsesIdx, sizeof(int)*totalSynapseNum);
	CHECK_CROSS_ARRAY(p.delayStart, c->pN2SConnection->delayStart, sizeof(int)*totalNeuronNum*MAX_DELAY);
	CHECK_CROSS_ARRAY(p.delayNum, c->pN2SConnection->delayNum, sizeof(int)*totalNeuronNum*MAX_DELAY);

	return 2;

	return 0;
}
