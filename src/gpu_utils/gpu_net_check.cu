
#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "gpu_utils.h"

#define CHECK(p1, p2, para) \
	do { \
		assert((p1->para) == (p2->para)); \
	} while(0)

#define CHECK_ARRAY(p1, p2, n) \
	do { \
		for (int i_t=0; i_t<(n); i++) {\
			assert(((p1)[i_t]) == ((p2)[i_t])); \
		}\
	} while(0)

#define CHECK_CROSS_ARRAY(g, c, size) \
	do { \
		unsigned char * mem_t = (unsigned char*)malloc(size); \
		checkCudaErrors(hipMemcpy(mem_t, (g), size, hipMemcpyDeviceToHost)); \
		assert(0==memcmp(mem_t, (c), (size))); \
		free(mem_t); \
	} while(0)


int checkGPUNetwork(GNetwork *g, GNetwork *c)
{
	int ret = -1;

	CHECK(g, c, nTypeNum);
	CHECK(g, c, sTypeNum);
	CHECK(g, c, neuronNums);
	CHECK(g, c, synapseNums);

	CHECK(g, c, nTypes);
	CHECK(g, c, sTypes);

	ret = 1;

	//int totalNeuronNum = g->neuronNums[g->nTypeNum+1];
	//int totalSynapseNum = g->synapseNums[g->sTypeNum+1];
	int MAX_DELAY = c->MAX_DELAY;

	N2SConnection p;
	checkCudaErrors(hipMemcpy(&p, g->pN2SConnection, sizeof(N2SConnection), hipMemcpyDeviceToHost));

	CHECK_CROSS_ARRAY(p.pSynapsesIdx, c->pN2SConnection->pSynapsesIdx, sizeof(int)*(c->pN2SConnection->s_num));
	CHECK_CROSS_ARRAY(p.delayStart, c->pN2SConnection->delayStart, sizeof(int)*(c->pN2SConnection->n_num)*MAX_DELAY);
	CHECK_CROSS_ARRAY(p.delayNum, c->pN2SConnection->delayNum, sizeof(int)*(c->pN2SConnection->n_num)*MAX_DELAY);

        ret = 2;

	return ret;
}
