/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <stdio.h>

#include "utils/cuda/hip/hip_runtime_api.h"
#include "gpu_kernel.h"
#include "SingleGPUSimulator.h"

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt)
	: SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

int SingleGPUSimulator::run(real time)
{
	int sim_cycle = (int) (time/dt);

	reset();

	PlainNetwork *pCpuNet = network->buildNetwrok();
	PlainNetwork *c_pGpuNet = copyDataToGPU(pCpuNet);

	unsigned int *c_gTimeTable = NULL;
	unsigned int *c_gFiredTable = NULL;
	bool *c_gSynapsesFiredTable = NULL;

	GLIFNeurons *pN = (GLIFNeurons*)pCpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pCpuNet->pSynapses;
	checkCudaErrors(hipMalloc((void**)c_gTimeTable, sizeof(unsigned int)*(1000+pCpuNet->MAX_DELAY)));
	//TODO: need to adapt the length
	checkCudaErrors(hipMalloc((void**)c_gFiredTable, sizeof(unsigned int)*((pN->num)*(pCpuNet->MAX_DELAY)*1000)));
	checkCudaErrors(hipMalloc((void**)c_gSynapsesFiredTable, sizeof(bool)*(pS->num)));

	init_global<<<1, 1>>>(c_pGpuNet, c_gTimeTable, c_gFiredTable, c_gSynapsesFiredTable);

	printf("Start runing for %d cycles\n", sim_cycle);
	for (int i=0; i<sim_cycle; i++) {
		printf("\rCycle: %d", i);
		fflush(stdout);

	}
	printf("\nFinish runing\n");

	return 0;
}

PlainNetwork* SingleGPUSimulator::copyDataToGPU(PlainNetwork *pCpuNet)
{
	PlainNetwork * ret = NULL;

	GLIFNeurons *pN = (GLIFNeurons*)pCpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pCpuNet->pSynapses;

	checkCudaErrors(hipMalloc((void**)ret, sizeof(PlainNetwork)));
	checkCudaErrors(hipMemcpy(ret, pCpuNet, sizeof(PlainNetwork), hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMalloc((void**)ret->pNeurons, sizeof(GLIFNeurons)));
	checkCudaErrors(hipMemcpy(ret->pNeurons, pCpuNet->pNeurons, sizeof(GLIFNeurons), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)ret->pSynapses, sizeof(GExpSynapses)));
	checkCudaErrors(hipMemcpy(ret->pSynapses, pCpuNet->pSynapses, sizeof(GExpSynapses), hipMemcpyHostToDevice));

	pN->allocGNeurons((GLIFNeurons*)ret->pNeurons);
	pS->allocGSynapses((GExpSynapses*)ret->pSynapses);

	return ret;
}
