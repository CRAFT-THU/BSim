#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <stdio.h>

#include "utils/cuda/hip/hip_runtime_api.h"
#include "gpu_kernel.h"
#include "SingleGPUSimulator.h"

#define TEST

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt)
	: SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

GNetwork* copyDataToGPU(GNetwork *);
int freeGPUData(GNetwork *);

int SingleGPUSimulator::run(real time)
{
	findCudaDevice(0, NULL);

	int sim_cycle = (int) (time/dt);

	reset();

	GNetwork *pCpuNet = network->buildNetwork();
	bool * c_n_fired = (bool*)malloc(sizeof(bool)*pCpuNet->neuronNum);

#ifdef TEST
	real * c_n_vm = (real*)malloc(sizeof(real)*pCpuNet->neuronNum);
	real * c_s_vm = (real*)malloc(sizeof(real)*pCpuNet->synapseNum);
#endif 

	FILE *logFile = fopen("SimGPU.log", "w+");
	if (logFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}

#ifdef TEST
	FILE **fileN = (FILE**)malloc(sizeof(FILE*)*pCpuNet->neuronNum);
	char filename[128];
	for (int i=0; i<pCpuNet->neuronNum; i++) {
		sprintf(filename, "GNeuron_%d.log", i);
		fileN[i] = fopen(filename, "w+");
	}
	FILE **fileS = (FILE**)malloc(sizeof(FILE*)*pCpuNet->synapseNum);
	for (int i=0; i<pCpuNet->synapseNum; i++) {
		sprintf(filename, "GSynapse_%d.log", i);
		fileS[i] = fopen(filename, "w+");
	}
#endif

	GNetwork *c_pGpuNet = copyDataToGPU(pCpuNet);
	int *c_gTimeTable = NULL;
	bool *c_gFiredTable = NULL;
	bool *c_gSynapsesFiredTable = NULL;


	int MAX_DELAY = (int)(pCpuNet->MAX_DELAY/dt);
	printf("MAX_DELAY: %lf %lf %lf\n", network->maxDelay, pCpuNet->MAX_DELAY, dt);
	printf("MAX_DELAY: %u\n", MAX_DELAY);

	GLIFNeurons *pN = (GLIFNeurons*)pCpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pCpuNet->pSynapses;
	checkCudaErrors(hipMalloc((void**)&c_gTimeTable, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMemset(c_gTimeTable, 0, sizeof(int)*(MAX_DELAY+1)));
	//TODO: need to adapt the length
	checkCudaErrors(hipMalloc((void**)&c_gFiredTable, sizeof(bool)*((pN->num)*(MAX_DELAY+1))));
	checkCudaErrors(hipMemset(c_gFiredTable, 0, sizeof(bool)*((pN->num)*(MAX_DELAY+1))));
	checkCudaErrors(hipMalloc((void**)&c_gSynapsesFiredTable, sizeof(bool)*(pS->num)));
	checkCudaErrors(hipMemset(c_gSynapsesFiredTable, 0, sizeof(bool)*(pS->num)));

#ifdef TEST
	GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	checkCudaErrors(hipMemcpy(pNTmp, ((GLIFNeurons*)(c_pGpuNet->pNeurons)), sizeof(GLIFNeurons), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pSTmp, ((GExpSynapses*)(c_pGpuNet->pSynapses)), sizeof(GExpSynapses), hipMemcpyDeviceToHost));
	real *g_n_vm = pNTmp->p_vm;
	real *g_s_vm = pSTmp->p_I_syn;
	free(pNTmp);
	free(pSTmp);
#endif

	init_global<<<1, 1, 0>>>(MAX_DELAY, c_gTimeTable, MAX_DELAY+1, c_gFiredTable, pN->num, c_gSynapsesFiredTable, pS->num);

	printf("Start runing for %d cycles\n", sim_cycle);
	for (int time=0; time<sim_cycle; time++) {
		printf("\rCycle: %d", time);
		update_lif_neuron<<<3, 2, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, c_pGpuNet->neuronNum, time);
		update_pre_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, time);
		update_exp_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, c_pGpuNet->synapseNum, time);

		int currentIdx = time%(MAX_DELAY+1);
		checkCudaErrors(hipMemcpy(c_n_fired, c_gFiredTable + (currentIdx*pN->num), sizeof(bool)*pN->num, hipMemcpyDeviceToHost));

#ifdef TEST
		checkCudaErrors(hipMemcpy(c_n_vm, g_n_vm, sizeof(real)*pCpuNet->neuronNum,hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(c_s_vm, g_s_vm, sizeof(real)*pCpuNet->synapseNum,hipMemcpyDeviceToHost));
		for (int i=0; i<pCpuNet->neuronNum; i++) {
			fprintf(fileN[i], "%lf\n", c_n_vm[i]); 
		}
		for (int i=0; i<pCpuNet->synapseNum; i++) {
			fprintf(fileS[i], "%lf\n", c_s_vm[i]); 
		}
#endif

		int count = 0;
		for (int i=0; i<pCpuNet->neuronNum; i++) {
			if (c_n_fired[i]) {
				count++;
			}
		}
		if (count > 0) {
			fprintf(logFile, "Cycle %d: ", time);
			for (int i=0; i<pCpuNet->neuronNum; i++) {
				if (c_n_fired[i]) {
					fprintf(logFile, " %d_%d", pN->pID[i].groupId, pN->pID[i].id);
				}
			}
			fprintf(logFile, "\n");
			fflush(stdout);
		}
	}
	printf("\nFinish runing\n");

	checkCudaErrors(hipFree(c_gTimeTable));
	checkCudaErrors(hipFree(c_gFiredTable));
	checkCudaErrors(hipFree(c_gSynapsesFiredTable));
	freeGPUData(c_pGpuNet);

#ifdef TEST
	for (int i=0; i<pCpuNet->neuronNum; i++) {
		fflush(fileN[i]);
		fclose(fileN[i]);
	}
	for (int i=0; i<pCpuNet->synapseNum; i++) {
		fflush(fileS[i]);
		fclose(fileS[i]);
	}

	free(fileN);
	free(fileS);
#endif

	return 0;
}

GNetwork* copyDataToGPU(GNetwork *pCpuNet)
{
	GNetwork *tmpNet = (GNetwork*)malloc(sizeof(GNetwork));
	memcpy(tmpNet, pCpuNet, sizeof(GNetwork));

	GLIFNeurons *pN = (GLIFNeurons*)pCpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pCpuNet->pSynapses;

	GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	memcpy(pNTmp, pN, sizeof(GLIFNeurons));
	GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	memcpy(pSTmp, pS, sizeof(GExpSynapses));

	GLIFNeurons *pNGpu;
	GExpSynapses *pSGpu;

	pN->allocGNeurons((GLIFNeurons*)pNTmp);
	pS->allocGSynapses((GExpSynapses*)pSTmp);
	
	checkCudaErrors(hipMalloc((void**)&(pNGpu), sizeof(GLIFNeurons)));
	checkCudaErrors(hipMemcpy(pNGpu, pNTmp, sizeof(GLIFNeurons), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pSGpu), sizeof(GExpSynapses)));
	checkCudaErrors(hipMemcpy(pSGpu, pSTmp, sizeof(GExpSynapses), hipMemcpyHostToDevice));
	free(pNTmp);
	free(pSTmp);

	tmpNet->pNeurons = pNGpu;
	tmpNet->pSynapses = pSGpu;

	return tmpNet;
}


int freeGPUData(GNetwork *pGpuNet)
{
	GLIFNeurons *pN = (GLIFNeurons*)pGpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pGpuNet->pSynapses;

	GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	checkCudaErrors(hipMemcpy(pNTmp, pN, sizeof(GLIFNeurons), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pSTmp, pS, sizeof(GExpSynapses), hipMemcpyDeviceToHost));

	freeGNeurons(pNTmp);
	freeGSynapses(pSTmp);

	free(pNTmp);
	free(pSTmp);

	checkCudaErrors(hipFree(pN));
	checkCudaErrors(hipFree(pS));
	free(pGpuNet);

	return 0;
}
