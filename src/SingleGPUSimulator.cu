/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <stdio.h>

#include "utils/cuda/hip/hip_runtime_api.h"
#include "gpu_kernel.h"
#include "SingleGPUSimulator.h"

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt)
	: SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

PlainNetwork* copyDataToGPU(PlainNetwork *);
int freeGPUData(PlainNetwork *);

int SingleGPUSimulator::run(real time)
{
	findCudaDevice(0, NULL);
	FILE *fileN = fopen("GNeurons.txt", "w+");
	FILE *fileS = fopen("GSynapses.txt", "w+");

	int sim_cycle = (int) (time/dt);

	reset();

	PlainNetwork *pCpuNet = network->buildNetwrok();
	real * c_n_vm = (real*)malloc(sizeof(real)*pCpuNet->neuronNum);
	real * c_s_vm = (real*)malloc(sizeof(real)*pCpuNet->synapseNum);

	PlainNetwork *c_pGpuNet = copyDataToGPU(pCpuNet);
	unsigned int *c_gTimeTable = NULL;
	unsigned int *c_gFiredTable = NULL;
	bool *c_gSynapsesFiredTable = NULL;

	GLIFNeurons *pN = (GLIFNeurons*)pCpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pCpuNet->pSynapses;
	checkCudaErrors(hipMalloc((void**)c_gTimeTable, sizeof(unsigned int)*(1000+pCpuNet->MAX_DELAY)));
	//TODO: need to adapt the length
	checkCudaErrors(hipMalloc((void**)c_gFiredTable, sizeof(unsigned int)*((pN->num)*(pCpuNet->MAX_DELAY)*1000)));
	checkCudaErrors(hipMalloc((void**)c_gSynapsesFiredTable, sizeof(bool)*(pS->num)));

	init_global<<<1, 1>>>(c_pGpuNet, c_gTimeTable, c_gFiredTable, c_gSynapsesFiredTable);

	printf("Start runing for %d cycles\n", sim_cycle);
	for (unsigned int time=0; time<sim_cycle; time++) {
		printf("\rCycle: %d", time);
		update_lif_neuron<<<1, 1>>>((GLIFNeurons*)c_pGpuNet->pNeurons, c_pGpuNet->neuronNum, time);
		update_postSynapse<<<1,1>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, time);
		update_exp_synapse<<<1,1>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, c_pGpuNet->synapseNum, time);

		checkCudaErrors(hipMemcpy(c_n_vm, ((GLIFNeurons*)(c_pGpuNet->pNeurons))->p_vm, sizeof(real)*pCpuNet->neuronNum,hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(c_s_vm, ((GExpSynapses*)(c_pGpuNet->pSynapses))->p_I_syn, sizeof(real)*pCpuNet->synapseNum,hipMemcpyDeviceToHost));
		for (int i=0; i<pCpuNet->neuronNum; i++) {
			fprintf(fileN, "%lf ", c_n_vm[i]); 
		}
		fprintf(fileN, "\n");
		for (int i=0; i<pCpuNet->synapseNum; i++) {
			fprintf(fileS, "%lf ", c_s_vm[i]); 
		}
		fprintf(fileS, "\n");

		fflush(stdout);

	}
	printf("\nFinish runing\n");

	checkCudaErrors(hipFree(c_gTimeTable));
	checkCudaErrors(hipFree(c_gFiredTable));
	checkCudaErrors(hipFree(c_gSynapsesFiredTable));
	checkCudaErrors(hipFree(c_gSynapsesFiredTable));
	freeGPUData(c_pGpuNet);

	fflush(fileN);
	fflush(fileS);
	fclose(fileN);
	fclose(fileS);

	return 0;
}

PlainNetwork* copyDataToGPU(PlainNetwork *pCpuNet)
{
	PlainNetwork * ret = NULL;

	GLIFNeurons *pN = (GLIFNeurons*)pCpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pCpuNet->pSynapses;

	checkCudaErrors(hipMalloc((void**)ret, sizeof(PlainNetwork)));
	checkCudaErrors(hipMemcpy(ret, pCpuNet, sizeof(PlainNetwork), hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMalloc((void**)ret->pNeurons, sizeof(GLIFNeurons)));
	checkCudaErrors(hipMemcpy(ret->pNeurons, pCpuNet->pNeurons, sizeof(GLIFNeurons), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)ret->pSynapses, sizeof(GExpSynapses)));
	checkCudaErrors(hipMemcpy(ret->pSynapses, pCpuNet->pSynapses, sizeof(GExpSynapses), hipMemcpyHostToDevice));

	pN->allocGNeurons((GLIFNeurons*)ret->pNeurons);
	pS->allocGSynapses((GExpSynapses*)ret->pSynapses);

	return ret;
}

int freeGPUData(PlainNetwork *pGpuNet)
{
	checkCudaErrors(hipFree(pGpuNet->pSynapses));
	checkCudaErrors(hipFree(pGpuNet->pNeurons));
	checkCudaErrors(hipFree(pGpuNet));

	return 0;
}
