#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <stdio.h>

#include "utils/cuda/hip/hip_runtime_api.h"
#include "GFunc.h"
#include "gpu_kernel.h"
#include "SingleGPUSimulator.h"

#define TEST

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

GNetwork* copyDataToGPU(GNetwork *);
int freeGPUData(GNetwork *);

int SingleGPUSimulator::run(real time)
{
	findCudaDevice(0, NULL);

	int sim_cycle = (int) (time/dt);

	reset();

	GNetwork *pCpuNet = network->buildNetwork();

	FILE *logFile = fopen("SimGPU.log", "w+");
	if (logFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}

	GNetwork *c_pGpuNet = copyDataToGPU(pCpuNet);
	GNetwork *pGpuNet;
	void **c_pNeurons;
	void **c_pSynapses;
	int *c_gTimeTable = NULL;
	int *c_gFiredTable = NULL;
	bool *c_gSynapsesFiredTable = NULL;
	real *c_gNeuronInput = NULL;

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = (int)(pCpuNet->MAX_DELAY/dt);
	printf("MAX_DELAY: %lf %lf %lf\n", network->maxDelay, pCpuNet->MAX_DELAY, dt);
	printf("MAX_DELAY: %u\n", MAX_DELAY);

	int * c_n_fired = (int*)malloc(sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1)));
	checkCudaErrors(hipMalloc((void**)&(pGpuNet), sizeof(GNetwork)));
	checkCudaErrors(hipMemcpy(pGpuNet, c_pGpuNet, sizeof(GNetwork), hipMemcpyHostToDevice));

	c_pNeurons = (void**)malloc(sizeof(void*)*pCpuNet->nTypeNum);
	checkCudaErrors(hipMemcpy(c_pNeurons, c_pGpuNet->pNeurons, sizeof(void*)*(pCpuNet->nTypeNum), hipMemcpyDeviceToHost));
	c_pSynapses = (void**)malloc(sizeof(void*)*pCpuNet->sTypeNum);
	checkCudaErrors(hipMemcpy(c_pSynapses, c_pGpuNet->pSynapses, sizeof(void*)*(pCpuNet->sTypeNum), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMalloc((void**)&c_gTimeTable, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMemset(c_gTimeTable, 0, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMalloc((void**)&c_gFiredTable, sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1))));
	checkCudaErrors(hipMemset(c_gFiredTable, 0, sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1))));
	checkCudaErrors(hipMalloc((void**)&c_gSynapsesFiredTable, sizeof(bool)*(totalSynapseNum)));
	checkCudaErrors(hipMemset(c_gSynapsesFiredTable, 0, sizeof(bool)*(totalSynapseNum)));
	checkCudaErrors(hipMalloc((void**)&c_gNeuronInput, sizeof(real)*(totalNeuronNum)));
	checkCudaErrors(hipMemset(c_gNeuronInput, 0, sizeof(real)*(totalNeuronNum)));

	init_global<<<1, 1, 0>>>(MAX_DELAY, c_gTimeTable, c_gNeuronInput, c_gFiredTable, totalNeuronNum, c_gSynapsesFiredTable, totalSynapseNum, pGpuNet);

	printf("Start runing for %d cycles\n", sim_cycle);
	for (int time=0; time<sim_cycle; time++) {
		printf("\rCycle: %d", time);
		//update_pre_neuron<<<1, 1, 0>>>(pGpuNet, simTime);
		//update_lif_neuron<<<3, 2, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, c_pGpuNet->neuronNum, time);
		for (int i=0; i<nTypeNum; i++) {
			updateType[pCpuNet->nTypes[i]](c_pNeurons[i], pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i], time);
		}

		//update_pre_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, time);
		update_pre_synapse<<<1, 1, 0>>>(pGpuNet, time);
		//update_exp_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, c_pGpuNet->synapseNum, time);
		for (int i=0; i<sTypeNum; i++) {
			updateType[pCpuNet->sTypes[i]](c_pSynapses[i], pCpuNet->synapseNums[i+1]-pCpuNet->synapseNums[i], time);
		}

		update_post_synapse<<<1, 1, 0>>>(pGpuNet, time);

		int currentIdx = time%(MAX_DELAY+1);
		checkCudaErrors(hipMemcpy(c_n_fired, c_gFiredTable, sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1)), hipMemcpyDeviceToHost));

		int count = 0;
		for (int i=0; i<pCpuNet->neuronNums[nTypeNum]; i++) {
			if (c_n_fired[currentIdx*totalNeuronNum+i] > 0) {
				count++;
			}
		}
		if (count > 0) {
			fprintf(logFile, "Cycle %d:", time);
			for (int i=0; i<totalNeuronNum; i++) {
				if (c_n_fired[currentIdx*totalNeuronNum+i] > 0) {
					fprintf(logFile, " %d_%d", network->idx2id[i].groupId, network->idx2id[i].id);
				}
			}
			fprintf(logFile, "\n");
			fflush(stdout);
		}
	}
	printf("\nFinish runing\n");

	checkCudaErrors(hipFree(c_gTimeTable));
	checkCudaErrors(hipFree(c_gFiredTable));
	checkCudaErrors(hipFree(c_gSynapsesFiredTable));

	freeGPUData(c_pGpuNet);
	checkCudaErrors(hipFree(pGpuNet));

	return 0;
}

GNetwork* copyDataToGPU(GNetwork *pCpuNet)
{
	GNetwork *tmpNet = (GNetwork*)malloc(sizeof(GNetwork));
	memcpy(tmpNet, pCpuNet, sizeof(GNetwork));

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;

	Type *nTypes, *sTypes;
	checkCudaErrors(hipMalloc((void**)&(nTypes), sizeof(Type)*nTypeNum));
	checkCudaErrors(hipMemcpy(nTypes, pCpuNet->nTypes, sizeof(Type)*nTypeNum, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(sTypes), sizeof(Type)*sTypeNum));
	checkCudaErrors(hipMemcpy(sTypes, pCpuNet->sTypes, sizeof(Type)*sTypeNum, hipMemcpyHostToDevice));

	int *neuronNums, *synapseNums;
	checkCudaErrors(hipMalloc((void**)&(neuronNums), sizeof(int)*(nTypeNum+1)));
	checkCudaErrors(hipMemcpy(neuronNums, pCpuNet->neuronNums, sizeof(int)*(nTypeNum+1), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(synapseNums), sizeof(int)*(sTypeNum+1)));
	checkCudaErrors(hipMemcpy(synapseNums, pCpuNet->synapseNums, sizeof(int)*(sTypeNum+1), hipMemcpyHostToDevice));

	//TODO support multitype N and S
	void **pNs = (void**)malloc(sizeof(void*)*nTypeNum);
	void **pSs = (void**)malloc(sizeof(void*)*sTypeNum);

	for (int i=0; i<nTypeNum; i++) {
		void *pNTmp = createType[pCpuNet->nTypes[i]]();
		memcpy(pNTmp, pCpuNet->pNeurons[i], getSize[pCpuNet->nTypes[i]]());
		cudaAllocType[pCpuNet->nTypes[i]](pCpuNet->pNeurons[i], pNTmp);
		void *pNGpu;
		checkCudaErrors(hipMalloc((void**)&(pNGpu), getSize[pCpuNet->nTypes[i]]()));
		checkCudaErrors(hipMemcpy(pNGpu, pNTmp, getSize[pCpuNet->nTypes[i]](), hipMemcpyHostToDevice));
		free(pNTmp);
		pNs[i] = pNGpu;
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pSTmp = createType[pCpuNet->sTypes[i]]();
		memcpy(pSTmp, pCpuNet->pSynapses[i], getSize[pCpuNet->sTypes[i]]());
		cudaAllocType[pCpuNet->sTypes[i]](pCpuNet->pSynapses[i], pSTmp);
		void *pSGpu;
		checkCudaErrors(hipMalloc((void**)&(pSGpu), getSize[pCpuNet->sTypes[i]]()));
		checkCudaErrors(hipMemcpy(pSGpu, pSTmp, getSize[pCpuNet->sTypes[i]](), hipMemcpyHostToDevice));
		free(pSTmp);
		pSs[i] = pSGpu;

	}

	//GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	//memcpy(pNTmp, pN, sizeof(GLIFNeurons));
	//GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	//memcpy(pSTmp, pS, sizeof(GExpSynapses));

	//GLIFNeurons *pNGpu;
	//GExpSynapses *pSGpu;

	//pN->allocGNeurons((GLIFNeurons*)pNTmp);
	//pS->allocGSynapses((GExpSynapses*)pSTmp);
	
	//checkCudaErrors(hipMalloc((void**)&(pNGpu), sizeof(GLIFNeurons)));
	//checkCudaErrors(hipMemcpy(pNGpu, pNTmp, sizeof(GLIFNeurons), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(pSGpu), sizeof(GExpSynapses)));
	//checkCudaErrors(hipMemcpy(pSGpu, pSTmp, sizeof(GExpSynapses), hipMemcpyHostToDevice));
	//free(pNTmp);
	//free(pSTmp);

	void **pNeurons, **pSynapses;
	checkCudaErrors(hipMalloc((void**)&(pNeurons), sizeof(void*)*nTypeNum));
	checkCudaErrors(hipMemcpy(pNeurons, pNs, sizeof(void*)*nTypeNum, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pSynapses), sizeof(void*)*sTypeNum));
	checkCudaErrors(hipMemcpy(pSynapses, pSs, sizeof(void*)*sTypeNum, hipMemcpyHostToDevice));
	free(pNs);
	free(pSs);

	tmpNet->pNeurons = pNeurons;
	tmpNet->pSynapses = pSynapses;
	tmpNet->nTypes = nTypes;
	tmpNet->sTypes = sTypes;
	tmpNet->neuronNums = neuronNums;
	tmpNet->synapseNums = synapseNums;

	return tmpNet;

	//GNetwork *pGpuNet;
	//checkCudaErrors(hipMalloc((void**)&(pGpuNet), sizeof(GNetwork)));
	//checkCudaErrors(hipMemcpy(pGpuNet, tmpNet, sizeof(GNetwork), hipMemcpyHostToDevice));
	//free(tmpNet);

	//return pGpuNet;
}


int freeGPUData(GNetwork *pGpuNet)
{
	GNetwork *pTmpNet = pGpuNet;
	//GNetwork *pTmpNet = (GNetwork*)malloc(sizeof(GNetwork));
	//checkCudaErrors(hipMemcpy(pTmpNet, pGpuNet, sizeof(GLIFNeurons), hipMemcpyDeviceToHost));

	int nTypeNum = pTmpNet->nTypeNum;
	int sTypeNum = pTmpNet->sTypeNum;

	void **pTmpNs = (void**)malloc(sizeof(void*) * nTypeNum);
	checkCudaErrors(hipMemcpy(pTmpNs, pTmpNet->pNeurons, sizeof(void*)*nTypeNum, hipMemcpyDeviceToHost));
	void **pTmpSs = (void**)malloc(sizeof(void*) * sTypeNum);
	checkCudaErrors(hipMemcpy(pTmpSs, pTmpNet->pSynapses, sizeof(void*)*sTypeNum, hipMemcpyDeviceToHost));

	Type * pTmpNT = (Type*)malloc(sizeof(Type)*nTypeNum);
	checkCudaErrors(hipMemcpy(pTmpNT, pTmpNet->nTypes, sizeof(Type)*nTypeNum, hipMemcpyDeviceToHost));
	Type * pTmpST = (Type*)malloc(sizeof(Type)*sTypeNum);
	checkCudaErrors(hipMemcpy(pTmpST, pTmpNet->sTypes, sizeof(Type)*sTypeNum, hipMemcpyDeviceToHost));

	for (int i=0; i<nTypeNum; i++) {
		void *pTmpN = createType[pTmpNT[i]]();
		checkCudaErrors(hipMemcpy(pTmpN, pTmpNs[i], getSize[pTmpNT[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpNT[i]](pTmpN);
		free(pTmpN);
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pTmpS = createType[pTmpST[i]]();
		checkCudaErrors(hipMemcpy(pTmpS, pTmpSs[i], getSize[pTmpST[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpST[i]](pTmpS);
		free(pTmpS);
	}

        //GLIFNeurons *pN = (GLIFNeurons*)pGpuNet->pNeurons;
	//GExpSynapses *pS = (GExpSynapses*)pGpuNet->pSynapses;

	//GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	//GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	//checkCudaErrors(hipMemcpy(pNTmp, pN, sizeof(GLIFNeurons), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(pSTmp, pS, sizeof(GExpSynapses), hipMemcpyDeviceToHost));

	//freeGNeurons(pNTmp);
	//freeGSynapses(pSTmp);


	checkCudaErrors(hipFree(pTmpNet->nTypes));
	checkCudaErrors(hipFree(pTmpNet->sTypes));
	free(pTmpNT);
	free(pTmpST);

	checkCudaErrors(hipFree(pTmpNet->neuronNums));
	checkCudaErrors(hipFree(pTmpNet->synapseNums));

	checkCudaErrors(hipFree(pTmpNet->pNeurons));
	checkCudaErrors(hipFree(pTmpNet->pSynapses));
	free(pTmpNs);
	free(pTmpSs);
	//checkCudaErrors(hipFree(pGpuNet));
	free(pTmpNet);

	return 0;
}
