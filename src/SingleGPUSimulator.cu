#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <stdio.h>

#include "utils/cuda/hip/hip_runtime_api.h"
#include "gpu_kernel.h"
#include "SingleGPUSimulator.h"

#define TEST

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

GNetwork* copyDataToGPU(GNetwork *);
int freeGPUData(GNetwork *);

int SingleGPUSimulator::run(real time)
{
	findCudaDevice(0, NULL);

	int sim_cycle = (int) (time/dt);

	reset();

	GNetwork *pCpuNet = network->buildNetwork();
	bool * c_n_fired = (bool*)malloc(sizeof(bool)*pCpuNet->neuronNum);

#ifdef TEST
	real * c_n_vm = (real*)malloc(sizeof(real)*pCpuNet->neuronNum);
	real * c_s_vm = (real*)malloc(sizeof(real)*pCpuNet->synapseNum);
#endif 

	FILE *logFile = fopen("SimGPU.log", "w+");
	if (logFile == NULL) {
		printf("ERROR: Open file SimGPU.log failed\n");
		return -1;
	}

#ifdef TEST
	FILE **fileN = (FILE**)malloc(sizeof(FILE*)*pCpuNet->neuronNum);
	char filename[128];
	for (int i=0; i<pCpuNet->neuronNum; i++) {
		sprintf(filename, "GNeuron_%d.log", i);
		fileN[i] = fopen(filename, "w+");
	}
	FILE **fileS = (FILE**)malloc(sizeof(FILE*)*pCpuNet->synapseNum);
	for (int i=0; i<pCpuNet->synapseNum; i++) {
		sprintf(filename, "GSynapse_%d.log", i);
		fileS[i] = fopen(filename, "w+");
	}
#endif

	GNetwork *c_pGpuNet = copyDataToGPU(pCpuNet);
	int *c_gTimeTable = NULL;
	bool *c_gFiredTable = NULL;
	bool *c_gSynapsesFiredTable = NULL;

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int totalNeuronNum = pCpuNet->neuronNums[nTypeNum+1];
	int totalSynapseNum = pCpuNet->synapseNums[sTypeNum+1];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = (int)(pCpuNet->MAX_DELAY/dt);
	printf("MAX_DELAY: %lf %lf %lf\n", network->maxDelay, pCpuNet->MAX_DELAY, dt);
	printf("MAX_DELAY: %u\n", MAX_DELAY);

	//GLIFNeurons *pN = (GLIFNeurons*)pCpuNet->pNeurons;
	//GExpSynapses *pS = (GExpSynapses*)pCpuNet->pSynapses;
	checkCudaErrors(hipMalloc((void**)&c_gTimeTable, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMemset(c_gTimeTable, 0, sizeof(int)*(MAX_DELAY+1)));
	//TODO: need to adapt the length
	checkCudaErrors(hipMalloc((void**)&c_gFiredTable, sizeof(bool)*((totalNeuronNum)*(MAX_DELAY+1))));
	checkCudaErrors(hipMemset(c_gFiredTable, 0, sizeof(bool)*((totalNeuronNum)*(MAX_DELAY+1))));
	checkCudaErrors(hipMalloc((void**)&c_gSynapsesFiredTable, sizeof(bool)*(totalSynapseNum)));
	checkCudaErrors(hipMemset(c_gSynapsesFiredTable, 0, sizeof(bool)*(totalSynapseNum)));

#ifdef TEST
	GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	checkCudaErrors(hipMemcpy(pNTmp, ((GLIFNeurons*)(c_pGpuNet->pNeurons)), sizeof(GLIFNeurons), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pSTmp, ((GExpSynapses*)(c_pGpuNet->pSynapses)), sizeof(GExpSynapses), hipMemcpyDeviceToHost));
	real *g_n_vm = pNTmp->p_vm;
	real *g_s_vm = pSTmp->p_I_syn;
	free(pNTmp);
	free(pSTmp);
#endif

	init_global<<<1, 1, 0>>>(MAX_DELAY, c_gTimeTable, MAX_DELAY+1, c_gFiredTable, pN->num, c_gSynapsesFiredTable, pS->num);

	printf("Start runing for %d cycles\n", sim_cycle);
	for (int time=0; time<sim_cycle; time++) {
		printf("\rCycle: %d", time);
		//update_lif_neuron<<<3, 2, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, c_pGpuNet->neuronNum, time);
		for (int i=0; i<nTypeNum; i++) {
			updateType[c_pGpuNet->nTypes[i]](c_pGpuNet->pNeurons[i], totalNeuronNum, simTime);
		}

		//update_pre_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, time);
		update_pre_synapse<<<1, 1, 0>>>(c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, time);
		//update_exp_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, c_pGpuNet->synapseNum, time);
		for (int i=0; i<sTypeNum; i++) {
			updateType[c_pGpuNet->sTypes[i]](c_pGpuNet->pSynapses[i], totalSynapseNum, simTime);
		}

		int currentIdx = time%(MAX_DELAY+1);
		checkCudaErrors(hipMemcpy(c_n_fired, c_gFiredTable + (currentIdx*pN->num), sizeof(bool)*pN->num, hipMemcpyDeviceToHost));

#ifdef TEST
		checkCudaErrors(hipMemcpy(c_n_vm, g_n_vm, sizeof(real)*pCpuNet->neuronNum,hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(c_s_vm, g_s_vm, sizeof(real)*pCpuNet->synapseNum,hipMemcpyDeviceToHost));
		for (int i=0; i<pCpuNet->neuronNum; i++) {
			fprintf(fileN[i], "%lf\n", c_n_vm[i]); 
		}
		for (int i=0; i<pCpuNet->synapseNum; i++) {
			fprintf(fileS[i], "%lf\n", c_s_vm[i]); 
		}
#endif

		int count = 0;
		for (int i=0; i<pCpuNet->neuronNums[nTypeNum+1]; i++) {
			if (c_n_fired[i]) {
				count++;
			}
		}
		if (count > 0) {
			fprintf(logFile, "Cycle %d: ", time);
			for (int i=0; i<pCpuNet->neuronNum; i++) {
				if (c_n_fired[i]) {
					fprintf(logFile, " %d_%d", pN->pID[i].groupId, pN->pID[i].id);
				}
			}
			fprintf(logFile, "\n");
			fflush(stdout);
		}
	}
	printf("\nFinish runing\n");

	checkCudaErrors(hipFree(c_gTimeTable));
	checkCudaErrors(hipFree(c_gFiredTable));
	checkCudaErrors(hipFree(c_gSynapsesFiredTable));
	freeGPUData(c_pGpuNet);

#ifdef TEST
	for (int i=0; i<pCpuNet->neuronNum; i++) {
		fflush(fileN[i]);
		fclose(fileN[i]);
	}
	for (int i=0; i<pCpuNet->synapseNum; i++) {
		fflush(fileS[i]);
		fclose(fileS[i]);
	}

	free(fileN);
	free(fileS);
#endif

	return 0;
}

GNetwork* copyDataToGPU(GNetwork *pCpuNet)
{
	GNetwork *tmpNet = (GNetwork*)malloc(sizeof(GNetwork));
	memcpy(tmpNet, pCpuNet, sizeof(GNetwork));

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;

	Type *nTypes, *sTypes;
	checkCudaErrors(hipMalloc((void**)&(nTypes), sizeof(Type)*nTypeNum));
	checkCudaErrors(hipMemcpy(nTypes, pCpuNet->nTypes, sizeof(Type)*nTypeNum, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(sTypes), sizeof(Type)*sTypeNum));
	checkCudaErrors(hipMemcpy(nTypes, pCpuNet->nsypes, sizeof(Type)*sTypeNum, hipMemcpyHostToDevice));

	int *neuronNums, *synapseNums;
	checkCudaErrors(hipMalloc((void**)&(neuronNums), sizeof(int)*(nTypeNum+1)));
	checkCudaErrors(hipMemcpy(neuronNums, pCpuNet->neuronNums, sizeof(int)*(nTypeNum+1), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(synapseNums), sizeof(int)*(sTypeNum+1)));
	checkCudaErrors(hipMemcpy(synapseNums, pCpuNet->synapseNums, sizeof(int)*(sTypeNum+1), hipMemcpyHostToDevice));

	//TODO support multitype N and S
	void **pNs = malloc(sizeof(void*)*nTypeNum);
	void **pSs = malloc(sizeof(void*)*sTypeNum);

	for (int i=0; i<nTypeNum; i++) {
		void *pNTmp = createType[pCpuNet->nTypes[i]]();
		memcpy(pNTmp, pCpuNet->pNeurons[i], getSize[pCpuNet->nTypes[i]]());
		cudaAllocType[pCpuNet->nTypes[i]](pCpuNet->pNeurons[i], pNTmp);
		void *pNGpu;
		checkCudaErrors(hipMalloc((void**)&(pNGpu), getSize[pCpuNet->nTypes[i]]()));
		checkCudaErrors(hipMemcpy(pNGpu, pNTmp, getSize[pCpuNet->nTypes[i]](), hipMemcpyHostToDevice));
		free(pNTmp);
		pNs[i] = pNGpu;
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pSTmp = createType[pCpuNet->sTypes[i]]();
		memcpy(pSTmp, pCpuNet->pSynapses[i], getSize[pCpuNet->sTypes[i]]());
		cudaAllocType[pCpuNet->sTypes[i]](pCpuNet->pSynapses[i], pSTmp);
		void *pSGpu;
		checkCudaErrors(hipMalloc((void**)&(pSGpu), getSize[pCpuNet->sTypes[i]]()));
		checkCudaErrors(hipMemcpy(pSGpu, pSTmp, getSize[pCpuNet->sTypes[i]](), hipMemcpyHostToDevice));
		free(pSTmp);
		pSs[i] = pSGpu;

	}

	//GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	//memcpy(pNTmp, pN, sizeof(GLIFNeurons));
	//GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	//memcpy(pSTmp, pS, sizeof(GExpSynapses));

	//GLIFNeurons *pNGpu;
	//GExpSynapses *pSGpu;

	//pN->allocGNeurons((GLIFNeurons*)pNTmp);
	//pS->allocGSynapses((GExpSynapses*)pSTmp);
	
	//checkCudaErrors(hipMalloc((void**)&(pNGpu), sizeof(GLIFNeurons)));
	//checkCudaErrors(hipMemcpy(pNGpu, pNTmp, sizeof(GLIFNeurons), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(pSGpu), sizeof(GExpSynapses)));
	//checkCudaErrors(hipMemcpy(pSGpu, pSTmp, sizeof(GExpSynapses), hipMemcpyHostToDevice));
	//free(pNTmp);
	//free(pSTmp);

	void **pNeurons, **pSynapses;
	checkCudaErrors(hipMalloc((void**)&(pNeurons), sizeof(void*)*nTypeNum));
	checkCudaErrors(hipMemcpy(pNeurons, pNs, sizeof(void*)*nTypeNum, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pSynapses), sizeof(void*)*sTypeNum));
	checkCudaErrors(hipMemcpy(pSynapses, pSs, sizeof(void*)*sTypeNum, hipMemcpyHostToDevice));
	free(pNs);
	free(pSs);

	tmpNet->pNeurons = pNeurons;
	tmpNet->pSynapses = pSynapses;
	tmpNet->nTypes = nTypes;
	tmpNet->sTypes = sTypes;
	tmpNet->neuronNums = neuronNums;
	tmpNet->synapseNums = synapseNums;

	return tmpNet;

	//GNetwork *pGpuNet;
	//checkCudaErrors(hipMalloc((void**)&(pGpuNet), sizeof(GNetwork)));
	//checkCudaErrors(hipMemcpy(pGpuNet, tmpNet, sizeof(GNetwork), hipMemcpyHostToDevice));
	//free(tmpNet);

	//return pGpuNet;
}


int freeGPUData(GNetwork *pGpuNet)
{
	GNetwork *pTmpNet = pGpuNet;
	//GNetwork *pTmpNet = (GNetwork*)malloc(sizeof(GNetwork));
	//checkCudaErrors(hipMemcpy(pTmpNet, pGpuNet, sizeof(GLIFNeurons), hipMemcpyDeviceToHost));

	int nTypeNum = pTmpNet->nTypeNum;
	int sTypeNum = pTmpNet->sTypeNum;

	void **pTmpNs = malloc(sizeof(void*) * nTypeNum);
	checkCudaErrors(hipMemcpy(pTmpNs, pTmpNet->pNeurons, sizeof(void*)*nTypeNum, hipMemcpyDeviceToHost));
	void **pTmpSs = malloc(sizeof(void*) * sTypeNum);
	checkCudaErrors(hipMemcpy(pTmpSs, pTmpNet->pSynapses, sizeof(void*)*sTypeNum, hipMemcpyDeviceToHost));

	Type * pTmpNT = malloc(sizeof(Type)*nTypeNum);
	checkCudaErrors(hipMemcpy(pTmpNTs, pTmpNet->nTypes, sizeof(void*)*nTypeNum, hipMemcpyDeviceToHost));
	Type * pTmpST = malloc(sizeof(Type)*sTypeNum);
	checkCudaErrors(hipMemcpy(pTmpSTs, pTmpNet->sTypes, sizeof(void*)*sTypeNum, hipMemcpyDeviceToHost));

	for (int i=0; i<nTypeNum; i++) {
		void *pTmpN = createType[pTmpNT[i]]();
		checkCudaErrors(hipMemcpy(pTmpN, pTmpNs[i], getSize[pTmpNT[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpNT[i]](pTmpN);
		free(pTmpN);
	}

	for (int i=0; i<sTypeNum; i++) {
		void *pTmpS = createType[pTmpST[i]]();
		checkCudaErrors(hipMemcpy(pTmpS, pTmpSs[i], getSize[pTmpST[i]](), hipMemcpyDeviceToHost));
		cudaFreeType[pTmpST[i]](pTmpS);
		free(pTmpS);
	}

        //GLIFNeurons *pN = (GLIFNeurons*)pGpuNet->pNeurons;
	//GExpSynapses *pS = (GExpSynapses*)pGpuNet->pSynapses;

	//GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	//GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	//checkCudaErrors(hipMemcpy(pNTmp, pN, sizeof(GLIFNeurons), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(pSTmp, pS, sizeof(GExpSynapses), hipMemcpyDeviceToHost));

	//freeGNeurons(pNTmp);
	//freeGSynapses(pSTmp);


	checkCudaErrors(hipFree(pTmpNet->nTypes));
	checkCudaErrors(hipFree(pTmpNet->sTypes));
	free(pTmpNT);
	free(pTmpST);

	checkCudaErrors(hipFree(pTmpNet->neuronNums));
	checkCudaErrors(hipFree(pTmpNet->synapseNums));

	checkCudaErrors(hipFree(pTmpNet->pNeurons));
	checkCudaErrors(hipFree(pTmpNet->pSynapses));
	free(pTmpNs);
	free(pTmpSs);
	//checkCudaErrors(hipFree(pGpuNet));
	free(pTmpNet);

	return 0;
}
