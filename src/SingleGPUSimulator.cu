#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <stdio.h>

#include "utils/cuda/hip/hip_runtime_api.h"
#include "gpu_kernel.h"
#include "SingleGPUSimulator.h"

SingleGPUSimulator::SingleGPUSimulator(Network *network, real dt)
	: SimulatorBase(network, dt)
{
}

SingleGPUSimulator::~SingleGPUSimulator()
{
}

PlainNetwork* copyDataToGPU(PlainNetwork *);
int freeGPUData(PlainNetwork *);

int SingleGPUSimulator::run(real time)
{
	findCudaDevice(0, NULL);

	int sim_cycle = (int) (time/dt);

	reset();

	PlainNetwork *pCpuNet = network->buildNetwrok();
	real * c_n_vm = (real*)malloc(sizeof(real)*pCpuNet->neuronNum);
	real * c_s_vm = (real*)malloc(sizeof(real)*pCpuNet->synapseNum);
	FILE **fileN = (FILE**)malloc(sizeof(FILE*)*pCpuNet->neuronNum);
	char filename[128];
	for (int i=0; i<pCpuNet->neuronNum; i++) {
		sprintf(filename, "GNeuron_%d.log", i);
		fileN[i] = fopen(filename, "w+");
	}
	FILE **fileS = (FILE**)malloc(sizeof(FILE*)*pCpuNet->synapseNum);
	for (int i=0; i<pCpuNet->synapseNum; i++) {
		sprintf(filename, "GSynapse_%d.log", i);
		fileS[i] = fopen(filename, "w+");
	}

	PlainNetwork *c_pGpuNet = copyDataToGPU(pCpuNet);
	unsigned int *c_gTimeTable = NULL;
	unsigned int *c_gFiredTable = NULL;
	bool *c_gSynapsesFiredTable = NULL;


	unsigned int MAX_DELAY = (unsigned int)(pCpuNet->MAX_DELAY/dt);
	printf("MAX_DELAY: %lf %lf %lf\n", network->maxDelay, pCpuNet->MAX_DELAY, dt);
	printf("MAX_DELAY: %u\n", MAX_DELAY);

	GLIFNeurons *pN = (GLIFNeurons*)pCpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pCpuNet->pSynapses;
	checkCudaErrors(hipMalloc((void**)&c_gTimeTable, sizeof(unsigned int)*(1000+MAX_DELAY+1)));
	checkCudaErrors(hipMemset(c_gTimeTable, 0, sizeof(unsigned int)*(1000+MAX_DELAY+1)));
	//TODO: need to adapt the length
	checkCudaErrors(hipMalloc((void**)&c_gFiredTable, sizeof(unsigned int)*((pN->num)*(MAX_DELAY+1)*1000)));
	checkCudaErrors(hipMemset(c_gFiredTable, 0, sizeof(unsigned int)*((pN->num)*(MAX_DELAY+1)*1000)));
	checkCudaErrors(hipMalloc((void**)&c_gSynapsesFiredTable, sizeof(bool)*(pS->num)));
	checkCudaErrors(hipMemset(c_gSynapsesFiredTable, 0, sizeof(bool)*(pS->num)));

	GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	checkCudaErrors(hipMemcpy(pNTmp, ((GLIFNeurons*)(c_pGpuNet->pNeurons)), sizeof(GLIFNeurons), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pSTmp, ((GExpSynapses*)(c_pGpuNet->pSynapses)), sizeof(GExpSynapses), hipMemcpyDeviceToHost));
	real *g_n_vm = pNTmp->p_vm;
	real *g_s_vm = pSTmp->p_I_syn;
	free(pNTmp);
	free(pSTmp);

	init_global<<<1, 1, 0>>>(MAX_DELAY, c_gTimeTable, 1000+MAX_DELAY+1, c_gFiredTable, (pN->num)*(MAX_DELAY+1)*1000, c_gSynapsesFiredTable, pS->num);

	printf("Start runing for %d cycles\n", sim_cycle);
	for (unsigned int time=0; time<sim_cycle; time++) {
		printf("\rCycle: %d", time);
		update_lif_neuron<<<1, 2, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, c_pGpuNet->neuronNum, time);
		update_pre_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, time);
		update_exp_synapse<<<1, 1, 0>>>((GLIFNeurons*)c_pGpuNet->pNeurons, (GExpSynapses*)c_pGpuNet->pSynapses, c_pGpuNet->synapseNum, time);

		checkCudaErrors(hipMemcpy(c_n_vm, g_n_vm, sizeof(real)*pCpuNet->neuronNum,hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(c_s_vm, g_s_vm, sizeof(real)*pCpuNet->synapseNum,hipMemcpyDeviceToHost));
		for (int i=0; i<pCpuNet->neuronNum; i++) {
			fprintf(fileN[i], "%lf\n", c_n_vm[i]); 
		}
		for (int i=0; i<pCpuNet->synapseNum; i++) {
			fprintf(fileS[i], "%lf\n", c_s_vm[i]); 
		}

		fflush(stdout);

	}
	printf("\nFinish runing\n");

	checkCudaErrors(hipFree(c_gTimeTable));
	checkCudaErrors(hipFree(c_gFiredTable));
	checkCudaErrors(hipFree(c_gSynapsesFiredTable));
	freeGPUData(c_pGpuNet);

	for (int i=0; i<pCpuNet->neuronNum; i++) {
		fflush(fileN[i]);
		fclose(fileN[i]);
	}
	for (int i=0; i<pCpuNet->synapseNum; i++) {
		fflush(fileS[i]);
		fclose(fileS[i]);
	}

	free(fileN);
	free(fileS);

	return 0;
}

PlainNetwork* copyDataToGPU(PlainNetwork *pCpuNet)
{
	PlainNetwork *tmpNet = (PlainNetwork*)malloc(sizeof(PlainNetwork));
	memcpy(tmpNet, pCpuNet, sizeof(PlainNetwork));

	GLIFNeurons *pN = (GLIFNeurons*)pCpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pCpuNet->pSynapses;

	GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	memcpy(pNTmp, pN, sizeof(GLIFNeurons));
	GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	memcpy(pSTmp, pS, sizeof(GExpSynapses));

	GLIFNeurons *pNGpu;
	GExpSynapses *pSGpu;

	pN->allocGNeurons((GLIFNeurons*)pNTmp);
	pS->allocGSynapses((GExpSynapses*)pSTmp);
	
	checkCudaErrors(hipMalloc((void**)&(pNGpu), sizeof(GLIFNeurons)));
	checkCudaErrors(hipMemcpy(pNGpu, pNTmp, sizeof(GLIFNeurons), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pSGpu), sizeof(GExpSynapses)));
	checkCudaErrors(hipMemcpy(pSGpu, pSTmp, sizeof(GExpSynapses), hipMemcpyHostToDevice));
	free(pNTmp);
	free(pSTmp);

	tmpNet->pNeurons = pNGpu;
	tmpNet->pSynapses = pSGpu;

	return tmpNet;
}


int freeGPUData(PlainNetwork *pGpuNet)
{
	GLIFNeurons *pN = (GLIFNeurons*)pGpuNet->pNeurons;
	GExpSynapses *pS = (GExpSynapses*)pGpuNet->pSynapses;

	GLIFNeurons *pNTmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons));
	GExpSynapses *pSTmp = (GExpSynapses*)malloc(sizeof(GExpSynapses));
	checkCudaErrors(hipMemcpy(pNTmp, pN, sizeof(GLIFNeurons), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pSTmp, pS, sizeof(GExpSynapses), hipMemcpyDeviceToHost));

	freeGNeurons(pNTmp);
	freeGSynapses(pSTmp);

	free(pNTmp);
	free(pSTmp);

	checkCudaErrors(hipFree(pN));
	checkCudaErrors(hipFree(pS));
	free(pGpuNet);

	return 0;
}
