
#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../gpu_utils/mem_op.h"
#include "GLIFENeurons.h"

int cudaAllocLIFE(void *pCpu, void *pGpu, int num)
{
	GLIFENeurons *pGpuNeurons = (GLIFENeurons*)pGpu;
	GLIFENeurons *p = (GLIFENeurons*)pCpu;
	pGpuNeurons->p_CI = copyToGPU<real>(p->p_CI, num);
	pGpuNeurons->p_vm = copyToGPU<real>(p->p_vm, num);
	pGpuNeurons->p_CE = copyToGPU<real>(p->p_CE, num);
	pGpuNeurons->p_refrac_step = copyToGPU<int>(p->p_refrac_step, num);
	pGpuNeurons->p_refrac_time = copyToGPU<int>(p->p_refrac_time, num);
	pGpuNeurons->p_i_tmp = copyToGPU<real>(p->p_i_tmp, num);
	pGpuNeurons->p_i_I = copyToGPU<real>(p->p_i_I, num);
	pGpuNeurons->p_i_E = copyToGPU<real>(p->p_i_E, num);
	pGpuNeurons->p_v_thresh = copyToGPU<real>(p->p_v_thresh, num);
	pGpuNeurons->p_v_reset = copyToGPU<real>(p->p_v_reset, num);
	pGpuNeurons->p_C2 = copyToGPU<real>(p->p_C2, num);
	pGpuNeurons->p_C1 = copyToGPU<real>(p->p_C1, num);
	return 0;
}

int cudaFreeLIFE(void *pCpu)
{
	GLIFENeurons *pGpuNeurons = (GLIFENeurons*)pGpu;
	gpuFreepGpuNeurons->p_CI);
	gpuFreepGpuNeurons->p_vm);
	gpuFreepGpuNeurons->p_CE);
	gpuFreepGpuNeurons->p_refrac_step);
	gpuFreepGpuNeurons->p_refrac_time);
	gpuFreepGpuNeurons->p_i_tmp);
	gpuFreepGpuNeurons->p_i_I);
	gpuFreepGpuNeurons->p_i_E);
	gpuFreepGpuNeurons->p_v_thresh);
	gpuFreepGpuNeurons->p_v_reset);
	gpuFreepGpuNeurons->p_C2);
	gpuFreepGpuNeurons->p_C1);
	return 0;
}

