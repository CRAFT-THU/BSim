
#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../gpu_utils/mem_op.h"
#include "GLIFENeurons.h"

int cudaAllocLIFE(void *pCpu, void *pGpu, int num)
{
	GLIFENeurons *pGpuNeurons = (GLIFENeurons*)pGpu;
	GLIFENeurons *p = (GLIFENeurons*)pCpu;
	pGpuNeurons->p_CI = copyToGPU<real>(p->p_CI, num);
	pGpuNeurons->p_vm = copyToGPU<real>(p->p_vm, num);
	pGpuNeurons->p_CE = copyToGPU<real>(p->p_CE, num);
	pGpuNeurons->p_refrac_step = copyToGPU<int>(p->p_refrac_step, num);
	pGpuNeurons->p_refrac_time = copyToGPU<int>(p->p_refrac_time, num);
	pGpuNeurons->p_i_tmp = copyToGPU<real>(p->p_i_tmp, num);
	pGpuNeurons->p_i_I = copyToGPU<real>(p->p_i_I, num);
	pGpuNeurons->p_i_E = copyToGPU<real>(p->p_i_E, num);
	pGpuNeurons->p_v_thresh = copyToGPU<real>(p->p_v_thresh, num);
	pGpuNeurons->p_v_reset = copyToGPU<real>(p->p_v_reset, num);
	pGpuNeurons->p_C2 = copyToGPU<real>(p->p_C2, num);
	pGpuNeurons->p_C1 = copyToGPU<real>(p->p_C1, num);
	return 0;
}

int cudaFreeLIFE(void *pCpu)
{
	GLIFENeurons *pGpuNeurons = (GLIFENeurons*)pGpu;
	gpuFree(pGpuNeurons->p_CI);
	gpuFree(pGpuNeurons->p_vm);
	gpuFree(pGpuNeurons->p_CE);
	gpuFree(pGpuNeurons->p_refrac_step);
	gpuFree(pGpuNeurons->p_refrac_time);
	gpuFree(pGpuNeurons->p_i_tmp);
	gpuFree(pGpuNeurons->p_i_I);
	gpuFree(pGpuNeurons->p_i_E);
	gpuFree(pGpuNeurons->p_v_thresh);
	gpuFree(pGpuNeurons->p_v_reset);
	gpuFree(pGpuNeurons->p_C2);
	gpuFree(pGpuNeurons->p_C1);
	return 0;
}

