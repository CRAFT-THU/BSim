/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "GTJNeurons.h"

int cudaAllocTJ(void *pCpu, void *pGpu, int num)
{
	GTJNeurons *pGpuNeurons = (GTJNeurons*)pGpu;
	GTJNeurons *p = (GTJNeurons*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_vm), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_vm, p->p_vm, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_reset), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_reset, p->p_v_reset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_thresh), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_thresh, p->p_v_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_cm, p->p_cm, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_cm, p->p_cm, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_refrac_step), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_refrac_step, p->p_refrac_step, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_refrac_time), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_refrac_time, p->p_refrac_time, sizeof(int)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeTJ(void *pGpu)
{
	GTJNeurons *pGpuNeurons = (GTJNeurons*)pGpu;

	checkCudaErrors(hipFree(pGpuNeurons->p_vm));
	checkCudaErrors(hipFree(pGpuNeurons->p_v_thresh));
	checkCudaErrors(hipFree(pGpuNeurons->p_v_reset));
	checkCudaErrors(hipFree(pGpuNeurons->p_i_tmp));
	checkCudaErrors(hipFree(pGpuNeurons->p_refrac_step));
	checkCudaErrors(hipFree(pGpuNeurons->p_refrac_time));

	return 0;
}
