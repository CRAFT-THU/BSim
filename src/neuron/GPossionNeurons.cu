/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../gpu_utils/gpu_func.h"
#include "../gpu_utils/gpu_kernel.h"
#include "GPossionNeurons.h"

int cudaAllocPossionNeurons(void *pCpu, void *pGpu, int num)
{
	GPossionNeurons *pGpuNeurons = (GPossionNeurons*)pGpu;
	GPossionNeurons *p = (GPossionNeurons*)pCpu;

	pGpuNeurons->p_rate = copyToGPU<real>(p->p_rate, num);
	pGpuNeurons->p_fire_cycle = copyToGPU<int>(p->p_fire_cycle, num);
	pGpuNeurons->p_end_cycle = copyToGPU<int>(p->p_end_cycle, num);
	pGpuNeurons->p_refrac_step = copyToGPU<int>(p->p_refrac_step, num);
	pGpuNeurons->p_state = gpuMalloc<hiprandState>(num);
	int block_num = (num+511)/512;
	curand_setup_kernel<<<block_num, 512>>>(pGpuNeurons->p_state, num);

	return 0;
}

int cudaFreePossionNeurons(void *pGpu)
{
	GPossionNeurons *pGpuNeurons = (GPossionNeurons*)pGpu;

	gpuFree(pGpuNeurons->p_rate);
	gpuFree(pGpuNeurons->p_fire_cycle);
	gpuFree(pGpuNeurons->p_end_cycle);
	gpuFree(pGpuNeurons->p_refrac_step);

	return 0;
}
