/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "../../gpu_utils/mem_op.h"
#include "GMaxNeurons.h"

int cudaAllocMax(void *pCpu, void *pGpu, int num)
{
	GMaxNeurons *pGpuNeurons = (GMaxNeurons*)pGpu;
	GMaxNeurons *p = (GMaxNeurons*)pCpu;

	pGpuNeurons->p_N = copyToGPU<int>(p->p_N, num);
	pGpuNeurons->p_count = gpuMalloc<int>(num);
	pGpuNeurons->p_record = gpuMalloc<int>(num*(p->max_N));

	return 0;
}

int cudaFreeMax(void *pGpu)
{
	GMaxNeurons *pGpuNeurons = (GMaxNeurons*)pGpu;

	gpuFree(pGpuNeurons->p_N);
	gpuFree(pGpuNeurons->p_count);
	gpuFree(pGpuNeurons->p_record);

	return 0;
}
