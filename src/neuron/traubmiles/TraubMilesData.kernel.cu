#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"
#include "../../net/Connection.h"

#include "TraubMilesData.h"

__global__ void update_traubmiles(Connection *connection, TraubMilesData *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
{
	int currentIdx = time % (connection->maxDelay+1);
	__shared__ int fireTableT[MAX_BLOCK_SIZE];
	__shared__ volatile int fireCNT;
	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x*gridDim.x) {
		int nid = idx;
		int gnid = offset + nid;
		int testLoc = 0;
		bool fired = false;

		real gNa = data->pGNa[nid];
		real ENa = data->pENa[nid];
		real gK = data->pGK[nid];
		real EK = data->pEK[nid];
		real gl = data->pGl[nid];
		real El = data->pEl[nid];
		real C = data->pC[nid];
		real v = data->pV[nid];
		real m = data->pM[nid];
		real h = data->pH[nid];
		real n = data->pN[nid];
		real decayE = data->pDecayE[nid];
		real decayI = data->pDecayI[nid];
		real EE = data->pEE[nid];
		real EI = data->pEI[nid];

		real mDt = DT/25.0;
		
		real i = currentE[gnid] * (EE-v);
		i += currentI[gnid] * (EI-v);

		const bool oldSpike = (v >= 0);

		for (int i=0; i<25; i++) {
           real iTmp = -(m*m*m*h*gNa*(V-ENa) + n*n*n*n*gK*(V-EK) + gl*(V-El)- i);
           real a = (v==-52.0)?1.28:(0.32*(-52.0-V)/(exp((-52.0-V)/4.0)-1.0));
           real b = (v==-25.0)?1.4:(0.28*(v+25.0)/(exp((v+25.0)/5.0)-1.0));

           m += (a*(1.0-m)-b*m)*mDt;
           a= 0.128*exp((-48.0-v)/18.0);
           b= 4.0 / (exp((-25.0-v)/5.0)+1.0);
           h += (a*(1.0-h)-b*h)*mDt;
		   a = (v == -50.0)?0.16:(0.032*(-50.0-v)/(exp((-50.0-v)/5.0)-1.0));
           b = 0.5*exp((-55.0-v)/40.0);
           n += (a*(1.0-n)-b*n)*mDt;
           v += iTmp/C*mDt;
		}

		// data->pGNa[nid] = gNa;
		// data->pENa[nid] = ENa;
		// data->pGK[nid] = gK;
		// data->pEK[nid] = EK;
		// data->pGl[nid] = gl;
		// data->pEl[nid] = El;
		// data->pC[nid] = C;
		data->pV[nid] = v;
		data->pM[nid] = m;
		data->pH[nid] = h;
		data->pN[nid] = n;
		data->pDecay[nid] = decay;
		data->pE[nid] = E;

		if (fired) {
			testLoc = atomicAdd((int*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fireTableT[testLoc] = gnid;
				fired = false;
			}
		}
		__syncthreads();

		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fireTableT, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			testLoc = atomicAdd((int*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fireTableT[testLoc] = gnid;
				fired = false;
			}
		}
		__syncthreads();

		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fireTableT, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fireTableT, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

	}
}


void cudaUpdateTraubMiles(Connection *connection, void *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time, BlockSize *pSize)
{
	update_traubmiles<<<pSize->gridSize, pSize->blockSize>>>(connection, (TraubMilesData *)data, currentE, currentI, firedTable, firedTableSizes, num, offset, time);
}


