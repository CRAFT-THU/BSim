#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "TraubMilesData.h"

void *cudaMallocTraubMiles()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(TraubMilesData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(TraubMilesData)*1));
	return ret;
}


void *cudaAllocTraubMiles(void *pCPU, int num)
{
	void *ret = cudaMallocTraubMiles();
	void *tmp = cudaAllocTraubMilesPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(TraubMilesData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}


void *cudaAllocTraubMilesPara(void *pCPU, int num)
{
	TraubMilesData *p = (TraubMilesData*)pCPU;
	TraubMilesData *ret = (TraubMilesData*)malloc(sizeof(TraubMilesData)*1);
	memset(ret, 0, sizeof(TraubMilesData)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pGNa), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pGNa, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pGNa, p->pGNa, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pENa), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pENa, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pENa, p->pENa, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pGK), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pGK, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pGK, p->pGK, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pEK), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pEK, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pEK, p->pEK, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pGl), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pGl, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pGl, p->pGl, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pEl), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pEl, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pEl, p->pEl, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC, p->pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV, p->pV, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pM), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pM, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pM, p->pM, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pH), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pH, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pH, p->pH, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pN), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pN, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pN, p->pN, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pDecay), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pDecay, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pDecay, p->pDecay, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pE), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pE, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pE, p->pE, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}


int cudaFetchTraubMiles(void *pCPU, void *pGPU, int num)
{
	TraubMilesData *pTmp = (TraubMilesData*)malloc(sizeof(TraubMilesData)*1);
	memset(pTmp, 0, sizeof(TraubMilesData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(TraubMilesData)*1, hipMemcpyDeviceToHost));

	cudaTraubMilesParaFromGPU(pCPU, pTmp, num);
	return 0;
}


int cudaTraubMilesParaToGPU(void *pCPU, void *pGPU, int num)
{
	TraubMilesData *pC = (TraubMilesData*)pCPU;
	TraubMilesData *pG = (TraubMilesData*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pGNa, pC->pGNa, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pENa, pC->pENa, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pGK, pC->pGK, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pEK, pC->pEK, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pGl, pC->pGl, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pEl, pC->pEl, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pC, pC->pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV, pC->pV, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pM, pC->pM, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pH, pC->pH, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pN, pC->pN, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pDecay, pC->pDecay, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pE, pC->pE, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}


int cudaTraubMilesParaFromGPU(void *pCPU, void *pGPU, int num)
{
	TraubMilesData *pC = (TraubMilesData*)pCPU;
	TraubMilesData *pG = (TraubMilesData*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pGNa, pG->pGNa, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pENa, pG->pENa, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pGK, pG->pGK, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pEK, pG->pEK, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pGl, pG->pGl, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pEl, pG->pEl, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC, pG->pC, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV, pG->pV, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pM, pG->pM, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pH, pG->pH, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pN, pG->pN, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pDecay, pG->pDecay, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pE, pG->pE, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}


int cudaFreeTraubMiles(void *pGPU)
{
	TraubMilesData *tmp = (TraubMilesData*)malloc(sizeof(TraubMilesData)*1);
	memset(tmp, 0, sizeof(TraubMilesData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(TraubMilesData)*1, hipMemcpyDeviceToHost));
	cudaFreeTraubMilesPara(tmp);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}


int cudaFreeTraubMilesPara(void *pGPU)
{
	TraubMilesData *p = (TraubMilesData*)pGPU;
	hipFree(p->pGNa);
	p->pGNa = NULL;
	hipFree(p->pENa);
	p->pENa = NULL;
	hipFree(p->pGK);
	p->pGK = NULL;
	hipFree(p->pEK);
	p->pEK = NULL;
	hipFree(p->pGl);
	p->pGl = NULL;
	hipFree(p->pEl);
	p->pEl = NULL;
	hipFree(p->pC);
	p->pC = NULL;
	hipFree(p->pV);
	p->pV = NULL;
	hipFree(p->pM);
	p->pM = NULL;
	hipFree(p->pH);
	p->pH = NULL;
	hipFree(p->pN);
	p->pN = NULL;
	hipFree(p->pDecay);
	p->pDecay = NULL;
	hipFree(p->pE);
	p->pE = NULL;

	return 0;
}


