#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

// #include "GConstantNeurons.h"
#include "constants.h"


__global__ void update_constant_neuron(GConstantNeurons *d_neurons, int num, int start_id, int time)
{
	int currentIdx = time % (MAX_DELAY+1);
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		fired = (time < d_neurons->p_end_cycle[idx]) && (((time - d_neurons->p_start_cycle[idx]) * d_neurons->p_fire_rate[idx]) > (d_neurons->p_fire_count[idx]));
		gFireCount[start_id + idx] += fired;

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = start_id + idx;
					d_neurons->p_fire_count[idx] = d_neurons->p_fire_count[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[currentIdx]), gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[currentIdx]), gFiredTableCap*currentIdx);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

int cudaUpdateConstant(void *data, int num, int start_id, int t, BlockSize *pSize)
{
	update_constant_neuron<<<pSize->gridSize, pSize->blockSize>>>((GConstantNeurons*)data, num, start_id, t);

	return 0;
}

