/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GConstantNeurons.h"

int cudaAllocConstant(void *pCpu, void *pGpu, int num)
{
	GConstantNeurons *pGpuNeurons = (GConstantNeurons*)pGpu;
	GConstantNeurons *p = (GConstantNeurons*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_fire_rate), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_fire_rate, p->p_fire_rate, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_start_cycle), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_start_cycle, p->p_start_cycle, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_end_cycle), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_end_cycle, p->p_end_cycle, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_fire_count), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_fire_count, p->p_fire_count, sizeof(int)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeConstant(void *pGpu)
{
	GConstantNeurons *pGpuNeurons = (GConstantNeurons*)pGpu;

	checkCudaErrors(hipFree(pGpuNeurons->p_fire_rate));
	checkCudaErrors(hipFree(pGpuNeurons->p_start_cycle));
	checkCudaErrors(hipFree(pGpuNeurons->p_end_cycle));
	checkCudaErrors(hipFree(pGpuNeurons->p_fire_count));

	return 0;
}
