/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GConstantNeurons.h"

int cudaAllocConstant(void *pCpu, void *pGpu, int num)
{
	GConstantNeurons *pGpuNeurons = (GConstantNeurons*)pGpu;
	GConstantNeurons *p = (GConstantNeurons*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_fire_rate), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_fire_rate, p->p_fire_rate, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_fire_count), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_fire_count, p->p_fire_count, sizeof(int)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeConstant(void *pGpu)
{
	GConstantNeurons *pGpuNeurons = (GConstantNeurons*)pGpu;

	checkCudaErrors(hipFree(pGpuNeurons->p_fire_rate));
	checkCudaErrors(hipFree(pGpuNeurons->p_fire_count));

	return 0;
}
