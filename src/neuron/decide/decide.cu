#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

#include "GDecideNeurons.h"

__global__ void update_decide_neuron(GDecideNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	const int PERIOD = 10;

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		int gnid = idx + start_id;


		fired = (PERIOD * d_neurons->p_fire_rate[idx]) > (d_neurons->p_fire_count[idx]);
		gFireCount[gnid] += fired;

		d_neurons->p_tmp_rate[idx] += gNeuronInput[gnid] + gNeuronInput_I[gnid];

		if (gCurrentCycle % PERIOD == 0) {
			d_neurons->p_fire_rate[idx] = 0.5 *d_neurons->p_fire_rate[idx] + d_neurons->p_tmp_rate[idx] * 0.05;
			d_neurons->p_fire_count[idx] = 0;
		}

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = start_id + idx;
					d_neurons->p_fire_count[idx] = d_neurons->p_fire_count[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;
		__syncthreads();
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

int cudaUpdateDecide(void *data, int num, int start_id, BlockSize *pSize)
{
	update_decide_neuron<<<pSize->gridSize, pSize->blockSize>>>((GDecideNeurons*)data, num, start_id);

	return 0;
}

