/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "GConstantNeurons.h"

int cudaAllocConstantNeurons(void *pCpu, void *pGpu)
{
	GConstantNeurons *pGpuNeurons = (GConstantNeurons*)pGpu;
	GConstantNeurons *p = (GConstantNeurons*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_fire_rate), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_fire_rate, p->p_fire_rate, sizeof(real)*p->num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_fire_count), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_fire_count, p->p_fire_count, sizeof(int)*p->num, hipMemcpyHostToDevice));

	NEURON_CONNECT_PARA_ALLOC(pGpuNeurons, p)

	return 0;
}

int cudaFreeConstantNeurons(void *pGpu)
{
	GConstantNeurons *pGpuNeurons = (GConstantNeurons*)pGpu;

	checkCudaErrors(hipFree(pGpuNeurons->p_fire_rate));
	checkCudaErrors(hipFree(pGpuNeurons->p_fire_count));

	NEURON_CONNECT_PARA_FREE(pGpuNeurons)

	return 0;
}
