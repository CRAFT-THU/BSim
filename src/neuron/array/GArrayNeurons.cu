/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "../../gpu_utils/mem_op.h"
#include "GArrayNeurons.h"

int cudaAllocArray(void *pCpu, void *pGpu, int num)
{
	GArrayNeurons *pGpuNeurons = (GArrayNeurons*)pGpu;
	GArrayNeurons *p = (GArrayNeurons*)pCpu;

	pGpuNeurons->p_start = copyToGPU<int>(p->p_start, num);
	pGpuNeurons->p_end = copyToGPU<int>(p->p_end, num);;
	pGpuNeurons->p_fire_time = copyToGPU<int>(p->p_fire_time, p->p_end[num-1]);

	return 0;
}

int hipFreeArray(void *pGpu)
{
	GArrayNeurons *pGpuNeurons = (GArrayNeurons*)pGpu;
	//TODO delete fire_time arrays

	gpuFree(pGpuNeurons->p_start);
	gpuFree(pGpuNeurons->p_end);
	gpuFree(pGpuNeurons->p_fire_time);

	return 0;
}
