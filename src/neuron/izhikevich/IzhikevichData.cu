#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "IzhikevichData.h"

void *cudaMallocIzhikevich()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(IzhikevichData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(IzhikevichData)*1));
	return ret;
}

void *cudaAllocIzhikevich(void *pCPU, int num)
{
	void *ret = cudaMallocIzhikevich();
	void *tmp = cudaAllocIzhikevichPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(IzhikevichData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocIzhikevichPara(void *pCPU, int num)
{
	IzhikevichData *p = (IzhikevichData*)pCPU;
	IzhikevichData *ret = (IzhikevichData*)malloc(sizeof(IzhikevichData)*1);
	memset(ret, 0, sizeof(IzhikevichData)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pV), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV, p->pV, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pU), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pU, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pU, p->pU, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pA), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pA, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pA, p->pA, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pB), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pB, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pB, p->pB, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC, p->pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pD), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pD, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pD, p->pD, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaFetchIzhikevich(void *pCPU, void *pGPU, int num)
{
	IzhikevichData *pTmp = (IzhikevichData*)malloc(sizeof(IzhikevichData)*1);
	memset(pTmp, 0, sizeof(IzhikevichData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(IzhikevichData)*1, hipMemcpyDeviceToHost));

	cudaIzhikevichParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaIzhikevichParaToGPU(void *pCPU, void *pGPU, int num)
{
	IzhikevichData *pC = (IzhikevichData*)pCPU;
	IzhikevichData *pG = (IzhikevichData*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pV, pC->pV, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pU, pC->pU, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pA, pC->pA, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pB, pC->pB, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pC, pC->pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pD, pC->pD, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaIzhikevichParaFromGPU(void *pCPU, void *pGPU, int num)
{
	IzhikevichData *pC = (IzhikevichData*)pCPU;
	IzhikevichData *pG = (IzhikevichData*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pV, pG->pV, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pU, pG->pU, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pA, pG->pA, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pB, pG->pB, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC, pG->pC, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pD, pG->pD, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeIzhikevich(void *pGPU)
{
	IzhikevichData *tmp = (IzhikevichData*)malloc(sizeof(IzhikevichData)*1);
	memset(tmp, 0, sizeof(IzhikevichData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(IzhikevichData)*1, hipMemcpyDeviceToHost));
	cudaFreeIzhikevichPara(tmp);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeIzhikevichPara(void *pGPU)
{
	IzhikevichData *p = (IzhikevichData*)pGPU;
	hipFree(p->pV);
	p->pV = NULL;
	hipFree(p->pU);
	p->pU = NULL;
	hipFree(p->pA);
	p->pA = NULL;
	hipFree(p->pB);
	p->pB = NULL;
	hipFree(p->pC);
	p->pC = NULL;
	hipFree(p->pD);
	p->pD = NULL;

	return 0;
}

