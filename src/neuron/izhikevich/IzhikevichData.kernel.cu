#include "hip/hip_runtime.h"

#include "IzhikevichData.h"

#include "../../gpu_utils/runtime.h"
#include "../../net/Connection.h"


__global__ void update_izhikevich_neuron(Connection *connection, IzhikevichData *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
{
	int currentIdx = time % (connection->maxDelay+1);
	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile int fire_cnt;
	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		int testLoc = 0;

		int nid = idx;
		int gnid = offset + nid; 

		real u = data->pU[nid];
		real v = data->pV[nid];
		real a = data->pA[nid];
		real b = data->pB[nid];
		real c = data->pC[nid];
		real d = data->pD[nid];

		const bool oldSpike = (v >= 29.99f);

		if (v >= 30.0f) {
			v = c;
			u += d;
		}

		const real I = currentE[gnid] + currentI[gnid];

		v += 0.5f * (0.04f * v * v + 5.0f * v + 140.0f -u + I) * DT;
		v += 0.5f * (0.04f * v * v + 5.0f * v + 140.0f -u + I) * DT;
	
		u += a*(b*v - u) * DT;

		if (v > 30.0f) {
			v = 30.0f;
		}

		bool fired = (v >= 29.99f) && (!oldSpike);

		data->pU[nid] = u;
		data->pV[nid] = v;
		data->pA[nid] = a;
		data->pB[nid] = b;
		data->pC[nid] = c;
		data->pD[nid] = d;

		if (fired) {
			testLoc = atomicAdd((int*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fire_table_t[testLoc] = gnid;
				fired = false;
			}
		} else {
			gXInput[gnid] += I;
		}

		currentE[gnid] = 0;
		currentI[gnid] = 0;

		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		__syncthreads();

		if (fired) {
			testLoc = atomicAdd((int*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fire_table_t[testLoc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

	}
	//__syncthreads();
	//if (threadIdx.x == 0 && blockIdx.x == 0) {
	//	gActiveTableSize = 0;
	//}
}

void cudaUpdateIzhikevich(Connection *conn, void *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time, BlockSize *pSize)
{
	update_izhikevich_neuron<<<pSize->gridSize, pSize->blockSize>>>(conn, (IzhikevichData *)data, currentE, currentI, firedTable, firedTableSizes, num, offset, time);
}
