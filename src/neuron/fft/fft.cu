#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

#include "GFFTNeurons.h"

__global__ void update_fft_neuron(GFFTNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		real res = 0;
		int gnid = idx + start_id;

		if (idx % 2 == 0) {
			res = d_neurons->p_res[idx/2].x;
		} else {
			res = d_neurons->p_res[idx/2].y;
		}

		fired = res > (d_neurons->p_fire_count[idx] + 0.5);

		gFireCount[gnid] += fired;

		int input = gNeuronInput[gnid] + gNeuronInput_I[gnid];
		gXInput[gnid] += input;

		if (idx % 2 == 0) {
			d_neurons->p_input[idx/2].x  += input;
		} else {
			d_neurons->p_input[idx/2].y  += input;
		}

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = start_id + idx;
				d_neurons->p_fire_count[idx] = d_neurons->p_fire_count[idx] + 1;
				fired = false;
			}
		} 
		
		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = start_id + idx;
				d_neurons->p_fire_count[idx] = d_neurons->p_fire_count[idx] + 1;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;

		__syncthreads();
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &(gFiredTableSizes[gCurrentIdx]), gFiredTableCap*gCurrentIdx);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

int cudaUpdateFFT(void *data, int num, int start_id, BlockSize *pSize)
{
	GFFTNeurons * p = (GFFTNeurons*)data;
	GFFTNeurons tmp;
	hipMemcpy(&tmp, p, sizeof(GFFTNeurons), hipMemcpyDeviceToHost);

	hipfftHandle plan;
	hipfftPlan1d(&plan, num/2, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, tmp.p_input, tmp.p_res, HIPFFT_FORWARD);

	update_fft_neuron<<<pSize->gridSize, pSize->blockSize>>>((GFFTNeurons*)data, num, start_id);

	return 0;
}

