#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GLIFNeurons.h"

void *cudaMallocLIF()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(GLIFNeurons)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(GLIFNeurons)*1));
	return ret;
}

void *cudaAllocLIF(void *pCPU, int num)
{
	void *ret = cudaMallocLIF();
	void *tmp = cudaAllocLIFPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(GLIFNeurons)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocLIFPara(void *pCPU, int num)
{
	GLIFNeurons *p = (GLIFNeurons*)pCPU;
	GLIFNeurons *ret = (GLIFNeurons*)malloc(sizeof(GLIFNeurons)*1);
	memset(ret, 0, sizeof(GLIFNeurons)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pRefracStep), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracStep, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracStep, p->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pRefracTime), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracTime, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracTime, p->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pV_m), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_m, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_m, p->pV_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pCi), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pCi, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pCi, p->pCi, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pCe), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pCe, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pCe, p->pCe, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_i, p->pC_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_e, p->pC_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_tmp, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_tmp, p->pV_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pI_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pI_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pI_i, p->pI_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pI_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pI_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pI_e, p->pI_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_i, p->pV_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_e, p->pV_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_thresh), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_thresh, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_thresh, p->pV_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_reset), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_reset, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_reset, p->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC_m), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_m, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_m, p->pC_m, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaLIFParaToGPU(void *pCPU, void *pGPU, int num)
{
	GLIFNeurons *pC = (GLIFNeurons*)pCPU;
	GLIFNeurons *pG = (GLIFNeurons*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pRefracStep, pC->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pRefracTime, pC->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG->pV_m, pC->pV_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pCi, pC->pCi, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pCe, pC->pCe, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pC_i, pC->pC_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pC_e, pC->pC_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_tmp, pC->pV_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pI_i, pC->pI_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pI_e, pC->pI_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_i, pC->pV_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_e, pC->pV_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_thresh, pC->pV_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_reset, pC->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pC_m, pC->pC_m, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaLIFParaFromGPU(void *pCPU, void *pGPU, int num)
{
	GLIFNeurons *pC = (GLIFNeurons*)pCPU;
	GLIFNeurons *pG = (GLIFNeurons*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pRefracStep, pG->pRefracStep, sizeof(int)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pRefracTime, pG->pRefracTime, sizeof(int)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pV_m, pG->pV_m, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pCi, pG->pCi, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pCe, pG->pCe, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_i, pG->pC_i, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_e, pG->pC_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_tmp, pG->pV_tmp, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pI_i, pG->pI_i, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pI_e, pG->pI_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_i, pG->pV_i, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_e, pG->pV_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_thresh, pG->pV_thresh, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_reset, pG->pV_reset, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_m, pG->pC_m, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeLIF(void *pGPU)
{
	GLIFNeurons *tmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons)*1);
	memset(tmp, 0, sizeof(GLIFNeurons)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(GLIFNeurons)*1, hipMemcpyDeviceToHost));
	cudaFreeLIFPara(pGPU);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeLIFPara(void *pGPU)
{
	GLIFNeurons *p = (GLIFNeurons*)pGPU;
	hipFree(p->pRefracStep);
	p->pRefracStep = NULL;
	hipFree(p->pRefracTime);
	p->pRefracTime = NULL;

	hipFree(p->pV_m);
	p->pV_m = NULL;
	hipFree(p->pCi);
	p->pCi = NULL;
	hipFree(p->pCe);
	p->pCe = NULL;
	hipFree(p->pC_i);
	p->pC_i = NULL;
	hipFree(p->pC_e);
	p->pC_e = NULL;
	hipFree(p->pV_tmp);
	p->pV_tmp = NULL;
	hipFree(p->pI_i);
	p->pI_i = NULL;
	hipFree(p->pI_e);
	p->pI_e = NULL;
	hipFree(p->pV_i);
	p->pV_i = NULL;
	hipFree(p->pV_e);
	p->pV_e = NULL;
	hipFree(p->pV_thresh);
	p->pV_thresh = NULL;
	hipFree(p->pV_reset);
	p->pV_reset = NULL;
	hipFree(p->pC_m);
	p->pC_m = NULL;

	return 0;
}

