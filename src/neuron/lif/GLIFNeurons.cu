#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GLIFNeurons.h"

void *cudaMallocLIF()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(GLIFNeurons)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(GLIFNeurons)*1));
	return ret;
}

void *cudaAllocLIF(void *pCPU, int num)
{
	void *ret = cudaMallocLIF();
	void *tmp = cudaAllocLIFPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(GLIFNeurons)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocLIFPara(void *pCPU, int num)
{
	GLIFNeurons *p = (GLIFNeurons*)pCPU;
	GLIFNeurons *ret = (GLIFNeurons*)malloc(sizeof(GLIFNeurons)*1);
	memset(ret, 0, sizeof(GLIFNeurons)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pRefracStep), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracStep, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracStep, p->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pRefracTime), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracTime, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracTime, p->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pV_m), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_m, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_m, p->pV_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pCi), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pCi, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pCi, p->pCi, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pCe), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pCe, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pCe, p->pCe, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_i, p->pC_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_e, p->pC_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_tmp, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_tmp, p->pV_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pI_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pI_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pI_i, p->pI_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pI_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pI_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pI_e, p->pI_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_i, p->pV_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_e, p->pV_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_thresh), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_thresh, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_thresh, p->pV_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_reset), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_reset, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_reset, p->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC_m), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_m, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_m, p->pC_m, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaLIFParaToGPU(void *pCPU, void *pGPU, int num)
{
	GLIFNeurons *pC = (GLIFNeurons*)pCPU;
	GLIFNeurons *pG = (GLIFNeurons*)pGPU;

	checkCudaErrors(hipMemcpy(pG->prefracStep, pC->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->prefracTime, pC->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG->pv_m, pC->pV_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pci, pC->pCi, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pce, pC->pCe, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pc_i, pC->pC_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pc_e, pC->pC_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pv_tmp, pC->pV_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pi_i, pC->pI_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pi_e, pC->pI_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pv_i, pC->pV_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pv_e, pC->pV_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pv_thresh, pC->pV_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pv_reset, pC->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pc_m, pC->pC_m, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaLIFParaFromGPU(void *pCPU, void *pGPU, int num)
{
	GLIFNeurons *pC = (GLIFNeurons*)pCPU;
	GLIFNeurons *pG = (GLIFNeurons*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pRefracStep, pG->prefracStep, sizeof(int)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pRefracTime, pG->prefracTime, sizeof(int)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pV_m, pG->pv_m, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pCi, pG->pci, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pCe, pG->pce, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_i, pG->pc_i, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_e, pG->pc_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_tmp, pG->pv_tmp, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pI_i, pG->pi_i, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pI_e, pG->pi_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_i, pG->pv_i, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_e, pG->pv_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_thresh, pG->pv_thresh, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_reset, pG->pv_reset, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_m, pG->pc_m, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeLIF(void *pGPU)
{
	GLIFNeurons *tmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons)*1);
	memset(tmp, 0, sizeof(GLIFNeurons)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(GLIFNeurons)*1, hipMemcpyDeviceToHost));
	cudaFreeLIFPara(pGPU);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeLIFPara(void *pGPU)
{
	GLIFNeurons *p = (GLIFNeurons*)pGPU;
	hipFree(p->pRefracStep);
	p->pRefracStep = NULL;
	hipFree(p->pRefracTime);
	p->pRefracTime = NULL;

	hipFree(p->pV_m);
	p->pV_m = NULL;
	hipFree(p->pCi);
	p->pCi = NULL;
	hipFree(p->pCe);
	p->pCe = NULL;
	hipFree(p->pC_i);
	p->pC_i = NULL;
	hipFree(p->pC_e);
	p->pC_e = NULL;
	hipFree(p->pV_tmp);
	p->pV_tmp = NULL;
	hipFree(p->pI_i);
	p->pI_i = NULL;
	hipFree(p->pI_e);
	p->pI_e = NULL;
	hipFree(p->pV_i);
	p->pV_i = NULL;
	hipFree(p->pV_e);
	p->pV_e = NULL;
	hipFree(p->pV_thresh);
	p->pV_thresh = NULL;
	hipFree(p->pV_reset);
	p->pV_reset = NULL;
	hipFree(p->pC_m);
	p->pC_m = NULL;

	return 0;
}

