
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "../../gpu_utils/mem_op.h"
#include "GLIFNeurons.h"

int cudaAllocLIF(void *pCpu, void *pGpu, int num)
{
	GLIFNeurons *pGpuNeurons = (GLIFNeurons*)pGpu;
	GLIFNeurons *p = (GLIFNeurons*)pCpu;
	pGpuNeurons->p_vm = copyToGPU<real>(p->p_vm, num);
	pGpuNeurons->p_CI = copyToGPU<real>(p->p_CI, num);
	pGpuNeurons->p_CE = copyToGPU<real>(p->p_CE, num);
	pGpuNeurons->p_C_I = copyToGPU<real>(p->p_C_I, num);
	pGpuNeurons->p_C_E = copyToGPU<real>(p->p_C_E, num);
	pGpuNeurons->p_v_tmp = copyToGPU<real>(p->p_v_tmp, num);
	pGpuNeurons->p_i_I = copyToGPU<real>(p->p_i_I, num);
	pGpuNeurons->p_i_E = copyToGPU<real>(p->p_i_E, num);
	pGpuNeurons->p_v_thresh = copyToGPU<real>(p->p_v_thresh, num);
	pGpuNeurons->p_v_reset = copyToGPU<real>(p->p_v_reset, num);
	pGpuNeurons->p_Cm = copyToGPU<real>(p->p_Cm, num);
	pGpuNeurons->p_refrac_step = copyToGPU<int>(p->p_refrac_step, num);
	pGpuNeurons->p_refrac_time = copyToGPU<int>(p->p_refrac_time, num);
	//pGpuNeurons->p_start_E = copyToGPU<int>(p->p_start_E, num);
	//pGpuNeurons->p_start_I = copyToGPU<int>(p->p_start_I, num);
	//pGpuNeurons->p_end = copyToGPU<int>(p->p_end, num);
	return 0;
}

int cudaFetchLIF(void *pGpu, void *pCpu, int num)
{
	GLIFNeurons *pGpuNeurons = copyFromGPU(static_cast<GLIFNeurons*>(pGpu), 1);
	GLIFNeurons *p = (GLIFNeurons*)pCpu;
	copyFromGPU<real>(p->p_vm, pGpuNeurons->p_vm, num);
	copyFromGPU<real>(p->p_i_E, pGpuNeurons->p_i_E, num);
	copyFromGPU<real>(p->p_i_I, pGpuNeurons->p_i_I, num);
	copyFromGPU<int>(p->p_refrac_step, pGpuNeurons->p_refrac_step, num);
	free(pGpuNeurons);

	return 0;
}

int cudaFreeLIF(void *pGpu)
{
	GLIFNeurons *pGpuNeurons = (GLIFNeurons*)pGpu;
	gpuFree(pGpuNeurons->p_vm);
	gpuFree(pGpuNeurons->p_CI);
	gpuFree(pGpuNeurons->p_CE);
	gpuFree(pGpuNeurons->p_C_I);
	gpuFree(pGpuNeurons->p_C_E);
	gpuFree(pGpuNeurons->p_v_tmp);
	gpuFree(pGpuNeurons->p_i_I);
	gpuFree(pGpuNeurons->p_i_E);
	gpuFree(pGpuNeurons->p_v_thresh);
	gpuFree(pGpuNeurons->p_v_reset);
	gpuFree(pGpuNeurons->p_Cm);
	gpuFree(pGpuNeurons->p_refrac_step);
	gpuFree(pGpuNeurons->p_refrac_time);
	//gpuFree(pGpuNeurons->p_start_E);
	//gpuFree(pGpuNeurons->p_start_I);
	//gpuFree(pGpuNeurons->p_end);
	return 0;
}

