#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GLIFNeurons.h"

void *cudaAllocLIF(void *pCPU, int num)
{
	void *ret = NULL;
	GLIFNeurons *p = (GLIFNeurons*)pCPU;
	GLIFNeurons * tmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons)*1);
	memset(tmp, 0, sizeof(GLIFNeurons)*1);

	checkCudaErrors(hipMalloc((void**)&(tmp->pI_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pI_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pI_i, p->pI_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pV_reset), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pV_reset, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pV_reset, p->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pCe), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pCe, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pCe, p->pCe, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pC_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pC_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pC_e, p->pC_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pC_m), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pC_m, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pC_m, p->pC_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pI_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pI_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pI_e, p->pI_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pV_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pV_tmp, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pV_tmp, p->pV_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pCi), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pCi, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pCi, p->pCi, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pV_thresh), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pV_thresh, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pV_thresh, p->pV_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pV_m), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pV_m, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pV_m, p->pV_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pV_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pV_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pV_i, p->pV_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pV_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pV_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pV_e, p->pV_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pC_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pC_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pC_i, p->pC_i, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(tmp->pRefracTime), sizeof(int)*num));
	checkCudaErrors(hipMemset(tmp->pRefracTime, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(tmp->pRefracTime, p->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pRefracStep), sizeof(int)*num));
	checkCudaErrors(hipMemset(tmp->pRefracStep, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(tmp->pRefracStep, p->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(GLIFNeurons)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(GLIFNeurons)*1));
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(GLIFNeurons)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

int cudaLIFToGPU(void *pCPU, void *pGPU, int num)
{
	GLIFNeurons *pC = (GLIFNeurons*)pCPU;
	GLIFNeurons *pG = (GLIFNeurons*)pGPU;

	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG, pC, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(int)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeLIF(void *pGPU)
{
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

