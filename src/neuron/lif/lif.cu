#include "hip/hip_runtime.h"

#include "GLIFNeurons.h"

#include "../../gpu_utils/runtime.h"

// #include "GLIFNeurons.h"
#include "lif.h"


__global__ void find_lif_neuron(GLIFNeurons *data, real * currentE, real * currentI, int num, int offset)
{
	__shared__ int active_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile int active_cnt;

	if (threadIdx.x == 0) {
		active_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		//bool actived = false;
		int test_loc = 0;
		bool actived = data->p_refrac_step[idx] <= 0;
		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		} else {
			currentE[offset + idx] = 0;
			currentI[offset + idx] = 0;
			data->p_refrac_step[idx] = data->p_refrac_step[idx] - 1;
		}
		__syncthreads();

		if (active_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(active_table_t, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		}
		__syncthreads();

		if (active_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(active_table_t, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (active_cnt > 0) {
			commit2globalTable(active_table_t, active_cnt, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
	}
}

__global__ void update_lif_neuron(GLIFNeurons *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
{
	int currentIdx = time % (MAX_DELAY+1);
	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile int fire_cnt;
	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int nid = gActiveTable[idx];
		int gnid = offset + nid; 

		//real I = sqrtf(data->p_CE[nid]) * data->p_i_E[nid] + sqrtf(data->p_CI[nid]) * data->p_i_I[nid] + data->p_i_tmp[nid];

		//real I = currentE[gnid] + data->p_i_tmp[nid];
		//data->p_vm[nid] = data->p_vm[nid] * data->p_C1[nid] + data->p_C2[nid] * I;

		data->p_vm[nid] = data->p_Cm[nid] * data->p_vm[nid] + data->p_v_tmp[nid] + data->p_i_E[nid] * data->p_C_E[nid] + data->p_i_I[nid] * data->p_C_I[nid];

		//data->p_i_syn[nid] = 0;

		data->p_i_E[nid] *= data->p_CE[nid];
		data->p_i_I[nid] *= data->p_CI[nid];

		fired = data->p_vm[nid] >= data->p_v_thresh[nid];

		gFireCount[gnid] += fired;

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}

			data->p_refrac_step[nid] = data->p_refrac_time[nid] - 1;
			data->p_vm[nid] = data->p_v_reset[nid];
		} else {
			gXInput[gnid] += currentE[gnid] + currentI[gnid];
			data->p_i_E[nid] += currentE[gnid];
			data->p_i_I[nid] += currentI[gnid];
		}

		currentE[gnid] = 0;
		currentI[gnid] = 0;

		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		__syncthreads();

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

	}
	//__syncthreads();
	//if (threadIdx.x == 0 && blockIdx.x == 0) {
	//	gActiveTableSize = 0;
	//}
}

__global__ void update_all_lif_neuron(GLIFNeurons *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
// __global__ void update_all_lif_neuron(GLIFNeurons *data, int num, int offset, int time)
{
	int currentIdx = time % (MAX_DELAY+1);
	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}

	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int nid = idx;
		int gnid = offset + idx; 
		bool actived = data->p_refrac_step[idx] <= 0;

		if (actived) {
			//real I = sqrtf(data->p_CE[nid]) * data->p_i_E[nid] + sqrtf(data->p_CI[nid]) * data->p_i_I[nid] + data->p_i_tmp[nid];

			//real I = currentE[gnid] + data->p_i_tmp[nid];
			//data->p_vm[nid] = data->p_vm[nid] * data->p_C1[nid] + data->p_C2[nid] * I;
			//data->p_i_syn[nid] = 0;

			data->p_vm[nid] = data->p_Cm[nid] * data->p_vm[nid] + data->p_v_tmp[nid] + data->p_i_E[nid] * data->p_C_E[nid] + data->p_i_I[nid] * data->p_C_I[nid];

			gXInput[gnid] += currentE[gnid] + currentI[gnid];

			data->p_i_E[nid] *= data->p_CE[nid];
			data->p_i_I[nid] *= data->p_CI[nid];

			fired = data->p_vm[nid] >= data->p_v_thresh[nid];

			gFireCount[gnid] += fired;

			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAX_BLOCK_SIZE) {
					fire_table_t[test_loc] = gnid;
					fired = false;
				}

				data->p_refrac_step[nid] = data->p_refrac_time[nid] - 1;
				data->p_vm[nid] = data->p_v_reset[nid];
			} else {
				data->p_i_E[nid] += currentE[gnid];
				data->p_i_I[nid] += currentI[gnid];
			}

			__syncthreads();
			if (fire_cnt >= MAX_BLOCK_SIZE) {
				commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}

			__syncthreads();

			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAX_BLOCK_SIZE) {
					fire_table_t[test_loc] = gnid;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAX_BLOCK_SIZE) {
				commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();

			if (fire_cnt > 0) {
				commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
		} else {
			data->p_refrac_step[idx] = data->p_refrac_step[idx] - 1;
		}
		currentE[gnid] = 0;
		currentI[gnid] = 0;
	}
	__syncthreads();
}

__global__ void update_dense_lif_neuron(GLIFNeurons *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
{
	//__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	//__shared__ volatile int fire_cnt;

	//if (threadIdx.x == 0) {
	//	fire_cnt = 0;
	//}
	//__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int currentIdx = time % (MAX_DELAY+1);
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		//bool fired = false;
		//int test_loc = 0;

		int nid = idx;
		int gnid = offset + idx; 
		bool actived = data->p_refrac_step[idx] <= 0;

		if (actived) {
			data->p_vm[nid] = data->p_Cm[nid] * data->p_vm[nid] + data->p_v_tmp[nid] + data->p_i_E[nid] * data->p_C_E[nid] + data->p_i_I[nid] * data->p_C_I[nid];

			data->p_i_E[nid] *= data->p_CE[nid];
			data->p_i_I[nid] *= data->p_CI[nid];

			bool fired = data->p_vm[nid] >= data->p_v_thresh[nid];

			firedTable[gFiredTableCap*currentIdx + gnid] = fired;

			gFireCount[gnid] += fired;

			if (fired) {
				data->p_refrac_step[nid] = data->p_refrac_time[nid] - 1;
				data->p_vm[nid] = data->p_v_reset[nid];

			} else {
				gXInput[gnid] += currentE[gnid] + currentI[gnid];
				data->p_i_E[nid] += currentE[gnid];
				data->p_i_I[nid] += currentI[gnid];
				//real input = 0, input_I = 0;
				//for (int i=data->p_start_E[nid]; i<data->p_start_I[nid]; i++) {
				//	input += currentE[i];
				//}
				//for (int i=data->p_start_I[nid]; i<data->p_end[nid]; i++) {
				//	input_I += currentE[i];
				//}
				//data->p_i_E[nid] += input;
				//data->p_i_I[nid] += input_I;
				//gXInput[gnid] += input + input_I;
			}

		} else {
			data->p_refrac_step[idx] = data->p_refrac_step[idx] - 1;
			firedTable[gFiredTableCap*currentIdx + gnid] = 0;
		}
		currentE[gnid] = 0;
		currentI[gnid] = 0;
	}
	__syncthreads();
}

void cudaUpdateLIF(void *data, void * unUsed, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time, BlockSize *pSize)
{
	find_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, currentE, currentI, num, offset);
	update_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, currentE, currentI, firedTable, firedTableSizes, num, offset, time);
	//update_dense_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, num, offset);

}
