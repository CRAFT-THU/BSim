#include "hip/hip_runtime.h"

#include "GLIFNeurons.h"

#include "../../gpu_utils/runtime.h"
#include "../../net/Connection.h"

// #include "GLIFNeurons.h"
#include "lif.h"


__global__ void find_lif_neuron(GLIFNeurons *data, real * currentE, real * currentI, int num, int offset)
{
	__shared__ int tActiveTable[MAX_BLOCK_SIZE];
	__shared__ volatile int activeCnt;

	if (threadIdx.x == 0) {
		activeCnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		//bool actived = false;
		int testLoc = 0;
		bool actived = data->pRefracStep[idx] <= 0;
		if (actived) {
			testLoc = atomicAdd((int*)&activeCnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				tActiveTable[testLoc] = idx;
				actived = false;
			}
		} else {
			currentE[offset + idx] = 0;
			currentI[offset + idx] = 0;
			data->pRefracStep[idx] = data->pRefracStep[idx] - 1;
		}
		__syncthreads();

		if (activeCnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(tActiveTable, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				activeCnt = 0;
			}
		}
		__syncthreads();

		if (actived) {
			testLoc = atomicAdd((int*)&activeCnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				tActiveTable[testLoc] = idx;
				actived = false;
			}
		}
		__syncthreads();

		if (activeCnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(tActiveTable, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				activeCnt = 0;
			}
		}
		__syncthreads();

		if (activeCnt > 0) {
			commit2globalTable(tActiveTable, activeCnt, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				activeCnt = 0;
			}
		}
		__syncthreads();
	}
}

__global__ void update_lif_neuron(Connection *connection, GLIFNeurons *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
{
	int currentIdx = time % (connection->maxDelay-connection->minDelay+1);
	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile int fire_cnt;
	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int testLoc = 0;

		int nid = gActiveTable[idx];
		int gnid = offset + nid; 

		//real I = sqrtf(data->pCe[nid]) * data->pI_e[nid] + sqrtf(data->pCi[nid]) * data->pI_i[nid] + data->p_i_tmp[nid];

		//real I = currentE[gnid] + data->p_i_tmp[nid];
		//data->pV_m[nid] = data->pV_m[nid] * data->p_C1[nid] + data->p_C2[nid] * I;

		data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];

		//data->p_i_syn[nid] = 0;

		data->pI_e[nid] *= data->pCe[nid];
		data->pI_i[nid] *= data->pCi[nid];

		fired = data->pV_m[nid] >= data->pV_thresh[nid];

		gFireCount[gnid] += fired;

		if (fired) {
			testLoc = atomicAdd((int*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fire_table_t[testLoc] = gnid;
				fired = false;
			}

			data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
			data->pV_m[nid] = data->pV_reset[nid];
		} else {
			gXInput[gnid] += currentE[gnid] + currentI[gnid];
			data->pI_e[nid] += currentE[gnid];
			data->pI_i[nid] += currentI[gnid];
		}

		currentE[gnid] = 0;
		currentI[gnid] = 0;

		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		__syncthreads();

		if (fired) {
			testLoc = atomicAdd((int*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fire_table_t[testLoc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

	}
	//__syncthreads();
	//if (threadIdx.x == 0 && blockIdx.x == 0) {
	//	gActiveTableSize = 0;
	//}
}

__global__ void update_all_lif_neuron(Connection *connection, GLIFNeurons *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
// __global__ void update_all_lif_neuron(GLIFNeurons *data, int num, int offset, int time)
{
	int currentIdx = time % (connection->maxDelay - connection->minDelay + 1);
	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}

	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int testLoc = 0;

		int nid = idx;
		int gnid = offset + idx; 
		bool actived = data->pRefracStep[idx] <= 0;

		if (actived) {
			//real I = sqrtf(data->pCe[nid]) * data->pI_e[nid] + sqrtf(data->pCi[nid]) * data->pI_i[nid] + data->p_i_tmp[nid];

			//real I = currentE[gnid] + data->p_i_tmp[nid];
			//data->pV_m[nid] = data->pV_m[nid] * data->p_C1[nid] + data->p_C2[nid] * I;
			//data->p_i_syn[nid] = 0;

			data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];

			gXInput[gnid] += currentE[gnid] + currentI[gnid];

			data->pI_e[nid] *= data->pCe[nid];
			data->pI_i[nid] *= data->pCi[nid];

			fired = data->pV_m[nid] >= data->pV_thresh[nid];

			gFireCount[gnid] += fired;

			if (fired) {
				testLoc = atomicAdd((int*)&fire_cnt, 1);
				if (testLoc < MAX_BLOCK_SIZE) {
					fire_table_t[testLoc] = gnid;
					fired = false;
				}

				data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
				data->pV_m[nid] = data->pV_reset[nid];
			} else {
				data->pI_e[nid] += currentE[gnid];
				data->pI_i[nid] += currentI[gnid];
			}

			__syncthreads();
			if (fire_cnt >= MAX_BLOCK_SIZE) {
				commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}

			__syncthreads();

			if (fired) {
				testLoc = atomicAdd((int*)&fire_cnt, 1);
				if (testLoc < MAX_BLOCK_SIZE) {
					fire_table_t[testLoc] = gnid;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAX_BLOCK_SIZE) {
				commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();

			if (fire_cnt > 0) {
				commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
		} else {
			data->pRefracStep[idx] = data->pRefracStep[idx] - 1;
		}
		currentE[gnid] = 0;
		currentI[gnid] = 0;
	}
	__syncthreads();
}

__global__ void update_dense_lif_neuron(Connection *connection, GLIFNeurons *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
{
	//__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	//__shared__ volatile int fire_cnt;

	//if (threadIdx.x == 0) {
	//	fire_cnt = 0;
	//}
	//__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int currentIdx = time % (connection->maxDelay-connection->minDelay+1);
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		//bool fired = false;
		//int testLoc = 0;

		int nid = idx;
		int gnid = offset + idx; 
		bool actived = data->pRefracStep[idx] <= 0;

		if (actived) {
			data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];

			data->pI_e[nid] *= data->pCe[nid];
			data->pI_i[nid] *= data->pCi[nid];

			bool fired = data->pV_m[nid] >= data->pV_thresh[nid];

			firedTable[gFiredTableCap*currentIdx + gnid] = fired;

			gFireCount[gnid] += fired;

			if (fired) {
				data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
				data->pV_m[nid] = data->pV_reset[nid];

			} else {
				gXInput[gnid] += currentE[gnid] + currentI[gnid];
				data->pI_e[nid] += currentE[gnid];
				data->pI_i[nid] += currentI[gnid];
				//real input = 0, input_I = 0;
				//for (int i=data->p_start_E[nid]; i<data->p_start_I[nid]; i++) {
				//	input += currentE[i];
				//}
				//for (int i=data->p_start_I[nid]; i<data->p_end[nid]; i++) {
				//	input_I += currentE[i];
				//}
				//data->pI_e[nid] += input;
				//data->pI_i[nid] += input_I;
				//gXInput[gnid] += input + input_I;
			}

		} else {
			data->pRefracStep[idx] = data->pRefracStep[idx] - 1;
			firedTable[gFiredTableCap*currentIdx + gnid] = 0;
		}
		currentE[gnid] = 0;
		currentI[gnid] = 0;
	}
	__syncthreads();
}

void cudaUpdateLIF(Connection *conn, void *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time, BlockSize *pSize)
{
	find_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, currentE, currentI, num, offset);
	update_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>(conn, (GLIFNeurons*)data, currentE, currentI, firedTable, firedTableSizes, num, offset, time);
	//update_dense_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, num, offset);

}
