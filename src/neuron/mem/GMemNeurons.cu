/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GMemNeurons.h"

int cudaAllocMem(void *pCpu, void *pGpu, int num)
{
	GMemNeurons *pGpuNeurons = (GMemNeurons*)pGpu;
	GMemNeurons *p = (GMemNeurons*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_fire_rate), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_fire_rate, p->p_fire_rate, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_fire_count), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_fire_count, p->p_fire_count, sizeof(int)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeMem(void *pGpu)
{
	GMemNeurons *pGpuNeurons = (GMemNeurons*)pGpu;

	checkCudaErrors(hipFree(pGpuNeurons->p_fire_rate));
	checkCudaErrors(hipFree(pGpuNeurons->p_fire_count));

	return 0;
}
