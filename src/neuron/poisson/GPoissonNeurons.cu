/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../../gpu_utils/mem_op.h"
#include "../../gpu_utils/gpu_func.h"
#include "../../gpu_utils/runtime.h"
#include "GPoissonNeurons.h"

int cudaAllocPoisson(void *pCpu, void *pGpu, int num)
{
	GPoissonNeurons *pGpuNeurons = (GPoissonNeurons*)pGpu;
	GPoissonNeurons *p = (GPoissonNeurons*)pCpu;

	pGpuNeurons->p_rate = copyToGPU<real>(p->p_rate, num);
	pGpuNeurons->p_fire_cycle = copyToGPU<int>(p->p_fire_cycle, num);
	pGpuNeurons->p_end_cycle = copyToGPU<int>(p->p_end_cycle, num);
	pGpuNeurons->p_refrac_step = copyToGPU<int>(p->p_refrac_step, num);
	pGpuNeurons->p_state = gpuMalloc<hiprandState>(num);
	int block_num = (num+511)/512;
	curand_setup_kernel<<<block_num, 512>>>(pGpuNeurons->p_state, num);

	return 0;
}

int cudaFreePoisson(void *pGpu)
{
	GPoissonNeurons *pGpuNeurons = (GPoissonNeurons*)pGpu;

	gpuFree(pGpuNeurons->p_rate);
	gpuFree(pGpuNeurons->p_fire_cycle);
	gpuFree(pGpuNeurons->p_end_cycle);
	gpuFree(pGpuNeurons->p_refrac_step);
	gpuFree(pGpuNeurons->p_state);

	return 0;
}
