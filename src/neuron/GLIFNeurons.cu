/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "GLIFNeurons.h"

int cudaAllocLIF(void *pCpu, void *pGpu, int num)
{
	GLIFNeurons *pGpuNeurons = (GLIFNeurons*)pGpu;
	GLIFNeurons *p = (GLIFNeurons*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_reset), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_reset, p->p_v_reset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_thresh), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_thresh, p->p_v_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_syn, p->p_i_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_vm), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_vm, p->p_vm, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_C1, p->p_C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_C2), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_C2, p->p_C2, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_tmp, p->p_i_tmp, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_refrac_step), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_refrac_step, p->p_refrac_step, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_refrac_time), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_refrac_time, p->p_refrac_time, sizeof(int)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeLIF(void *pGpu)
{
	GLIFNeurons *pGpuNeurons = (GLIFNeurons*)pGpu;

	checkCudaErrors(hipFree(pGpuNeurons->p_v_reset));
	checkCudaErrors(hipFree(pGpuNeurons->p_v_thresh));
	checkCudaErrors(hipFree(pGpuNeurons->p_i_syn));
	checkCudaErrors(hipFree(pGpuNeurons->p_vm));
	checkCudaErrors(hipFree(pGpuNeurons->p_C1));
	checkCudaErrors(hipFree(pGpuNeurons->p_C2));
	checkCudaErrors(hipFree(pGpuNeurons->p_i_tmp));
	checkCudaErrors(hipFree(pGpuNeurons->p_refrac_step));
	checkCudaErrors(hipFree(pGpuNeurons->p_refrac_time));

	return 0;
}
