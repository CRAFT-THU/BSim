/* This program is writen by qp09.
 * usually just for fun.
 * Sat March 12 2016
 */

#include "Neuron.h"
#include "gpu_kernel.h"
#include "gpu_func.h"


void* (*createType[])() = { createLIFNeurons, /*createNengoNeurons, createInputNeurons, createPossionNeurons, createProbeNeurons,*/ createAlphaSynapses, createExpSynapses/*, createLowpassSynapses*/ };

size_t (*getSize[])() = { getLIFSize, /*getNengoSize, getInputSize, getPossionSize, getProbeSize,*/ getAlphaSize, getExpSize/*, getLowpassSize*/ };

int (*allocType[])(void *, int) = { allocLIFNeurons, /*allocNengoNeurons, allocInputNeurons, allocPossionNeurons, allocProbeNeurons,*/ allocAlphaSynapses, allocExpSynapses/*, allocLowpassSynapses*/ };

int (*allocConnect[])(void *, int *, int *, int *, int) = { allocLIFConnects, /*allocNengoConnects, allocInputConnects, allocPossionConnects, allocProbeConnects,*/ allocAlphaConnects, allocExpConnects/*, allocLowpassConnects*/ };

int (*cudaAllocType[])(void *, void *) = { cudaAllocLIFNeurons, /*cudaAllocNengoNeurons, cudaAllocInputNeurons, cudaAllocPossionNeurons, cudaAllocProbeNeurons,*/ cudaAllocAlphaSynapses, cudaAllocExpSynapses/*, cudaAllocLowpassSynapses*/ };

int (*cudaFreeType[])(void *) = { cudaFreeLIFNeurons, /*cudaFreeNengoNeurons, cudaFreeInputNeurons, cudaFreePossionNeurons, cudaFreeProbeNeurons,*/ cudaFreeAlphaSynapses, cudaFreeExpSynapses/*, cudaFreeLowpassSynapses*/ };

int updateLIFNeuron(void *data, int num, int simTime, BlockSize *pSize)
{
	update_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFNeurons*)data, num, simTime);

	return 0;
}

int updateAlphaSynapses(void *data, int num, int simTime, BlockSize *pSize)
{
	update_alpha_synapse<<<pSize->gridSize, pSize->blockSize>>>((GAlphaSynapses*)data, num, simTime);

	return 0;
}

int updateExpSynapses(void *data, int num, int simTime, BlockSize *pSize)
{
	update_exp_synapse<<<pSize->gridSize, pSize->blockSize>>>((GExpSynapses*)data, num, simTime);

	return 0;
}

int (*updateType[])(void *, int, int, BlockSize*) = { updateLIFNeuron, updateAlphaSynapses, updateExpSynapses };

BlockSize * getBlockSize(int nSize, int sSize)
{
	BlockSize *ret = (BlockSize*)malloc(sizeof(BlockSize)*TypeSize);
	hipOccupancyMaxPotentialBlockSize(&(ret[LIF].minGridSize), &(ret[LIF].blockSize), update_lif_neuron, 0, nSize); 
	ret[LIF].gridSize = (nSize + (ret[LIF].blockSize) - 1) / (ret[LIF].blockSize);
	hipOccupancyMaxPotentialBlockSize(&(ret[Exp].minGridSize), &(ret[Exp].blockSize), update_exp_synapse, 0, sSize); 
	ret[Exp].gridSize = (sSize + (ret[Exp].blockSize) - 1) / (ret[Exp].blockSize);
	hipOccupancyMaxPotentialBlockSize(&(ret[Alpha].minGridSize), &(ret[Alpha].blockSize), update_alpha_synapse, 0, sSize); 
	ret[Alpha].gridSize = (sSize + (ret[Alpha].blockSize) - 1) / (ret[Alpha].blockSize);

	return ret;
}
