
#include "utils/cuda/hip/hip_runtime_api.h"
#include "GBasicSynapses.h"

int cudaAllocBasicSynapses(void *pCpu, void *pGpu)
{
	GBasicSynapses *pGpuSynapses = (GBasicSynapses*)pGpu;
	GBasicSynapses *p = (GBasicSynapses*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_weight), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_weight, p->p_weight, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay_steps), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay_steps, p->p_delay_steps, sizeof(int)*p->num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pDst), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pDst, p->pDst, sizeof(int)*p->num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeBasicSynapses(void *pGpu)
{
	GBasicSynapses *pGpuSynapses = (GBasicSynapses*)pGpu;

	checkCudaErrors(hipFree(pGpuSynapses->p_weight));
	checkCudaErrors(hipFree(pGpuSynapses->p_delay_steps));

	checkCudaErrors(hipFree(pGpuSynapses->pDst));

	return 0;
}
