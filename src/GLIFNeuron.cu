/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */

#include <stdlib.h>

#include "./utils/cuda/hip/hip_runtime_api.h"
#include "GLIFNeuron.h"

int GLIFNeurons::allocNeurons(unsigned int N)
{
	num = N;
	pID = (ID*)malloc(N*sizeof(ID));
	pType = (NeuronType*)malloc(N*sizeof(NeuronType));
	p_v_init = (real*)malloc(N*sizeof(real));
	p_v_rest = (real*)malloc(N*sizeof(real));
	p_v_reset = (real*)malloc(N*sizeof(real));
	p_cm = (real*)malloc(N*sizeof(real));
	p_tau_m = (real*)malloc(N*sizeof(real));
	p_tau_refrac = (real*)malloc(N*sizeof(real));
	p_tau_syn_E = (real*)malloc(N*sizeof(real));
	p_tau_syn_I = (real*)malloc(N*sizeof(real));
	p_v_thresh = (real*)malloc(N*sizeof(real));
	p_i_offset = (real*)malloc(N*sizeof(real));
	p_i_syn = (real*)malloc(N*sizeof(real));
	p_vm = (real*)malloc(N*sizeof(real));
	p__dt = (real*)malloc(N*sizeof(real));
	p_C1 = (real*)malloc(N*sizeof(real));
	p_C2 = (real*)malloc(N*sizeof(real));
	p_i_tmp = (real*)malloc(N*sizeof(real));
	p_refrac_step = (int*)malloc(N*sizeof(int));
	pSynapsesNum = (unsigned int*)malloc(N*sizeof(unsigned int));
	pSynapsesLoc = (unsigned int*)malloc(N*sizeof(unsigned int));
	
	return 0;
}

int GLIFNeurons::allocConnects(unsigned int S)
{
	synapsesNum = S;
	pSynapsesIdx = (unsigned int*)malloc(S*sizeof(unsigned int));
	return 0;
}

int GLIFNeurons::allocGNeurons(GLIFNeurons * pGpuNeurons)
{
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pID), sizeof(ID)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pID, pID, sizeof(ID)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pType), sizeof(NeuronType)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pType, pType, sizeof(NeuronType)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_init), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_init, p_v_init, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_rest), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_rest, p_v_rest, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_reset), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_reset, p_v_init, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_cm), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_cm, p_cm, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_m), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_m, p_tau_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_refrac), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_refrac, p_tau_refrac, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_syn_E), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_syn_E, p_tau_syn_E, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_syn_I), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_syn_I, p_tau_syn_E, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_thresh), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_thresh, p_v_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_offset), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_offset, p_i_offset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_syn, p_i_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_vm), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_vm, p_vm, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p__dt), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p__dt, p__dt, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_C1, p_C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_C2), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_C2, p_C2, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_tmp, p_i_tmp, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_refrac_step), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_refrac_step, p_refrac_step, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pSynapsesNum), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pSynapsesNum, pSynapsesNum, sizeof(unsigned int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pSynapsesLoc), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pSynapsesLoc, pSynapsesLoc, sizeof(unsigned int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pSynapsesIdx), sizeof(unsigned int)*synapsesNum));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pSynapsesIdx, pSynapsesIdx, sizeof(unsigned int)*synapsesNum, hipMemcpyHostToDevice));

	return 0;
}

