/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */

#include "utils/cuda/hip/hip_runtime_api.h"
#include "GAlphaSynapses.h"

int GAlphaSynapses::allocGSynapses(GAlphaSynapses *pGpuSynapses)
{
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pID), sizeof(ID)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pID, pID, sizeof(ID)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pType), sizeof(NeuronType)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pType, pType, sizeof(NeuronType)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_weight), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_weight, p_weight, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay, p_delay, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C1, p_C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C2), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C2, p_C2, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C1, p__C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C2), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C2, p__C2, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_tau_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_tau_syn, p_tau_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_syn, p_I_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_tmp, p_I_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__dt), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__dt, p__dt, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pSrc), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pSrc, pSrc, sizeof(unsigned int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pDst), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pDst, pDst, sizeof(unsigned int)*num, hipMemcpyHostToDevice));

	return 0;
}

int freeGSynapses(GAlphaSynapses *pGpuSynapses)
{
	checkCudaErrors(hipFree(pGpuSynapses->pID));

	checkCudaErrors(hipFree(pGpuSynapses->pType));

	checkCudaErrors(hipFree(pGpuSynapses->p_weight));
	checkCudaErrors(hipFree(pGpuSynapses->p_delay));
	checkCudaErrors(hipFree(pGpuSynapses->p_C1));
	checkCudaErrors(hipFree(pGpuSynapses->p_C2));
	checkCudaErrors(hipFree(pGpuSynapses->p__C1));
	checkCudaErrors(hipFree(pGpuSynapses->p__C2));
	checkCudaErrors(hipFree(pGpuSynapses->p_tau_syn));
	checkCudaErrors(hipFree(pGpuSynapses->p_I_syn));
	checkCudaErrors(hipFree(pGpuSynapses->p_I_tmp));
	checkCudaErrors(hipFree(pGpuSynapses->p__dt));

	checkCudaErrors(hipFree(pGpuSynapses->pSrc));
	checkCudaErrors(hipFree(pGpuSynapses->pDst));

	return 0;
}
