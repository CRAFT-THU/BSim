#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <mpi.h>
#include <sys/time.h>
#include <stdio.h>

#include "utils/cuda/hip/hip_runtime_api.h"
#include "gpu_utils.h"
#include "gpu_func.h"
#include "gpu_kernel.h"
#include "MultiGPUSimulator.h"

#define TEST

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : SimulatorBase(network, dt)
{
	rank = -1;
	size = -1;
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

GNetwork* getGlobalData(GNetwork *, int, int);
GNetwork* splitNetwork(GNetwork *, GNetwork *, int, int);

int MultiGPUSimulator::init(int argc, char**argv)
{
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &size);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	return size;
}

int MultiGPUSimulator::run(real time)
{
	findCudaDevice(0, NULL);

	int sim_cycle = round(time/dt);

	reset();

	GNetwork *pCpuNet = NULL;
	GNetwork *pAllNet = NULL;
	FILE *logFile = NULL;
	FILE *dataFile = NULL;
	if (rank == 0) {
		logFile = fopen("MGSim.log", "w+");
		if (logFile == NULL) {
			printf("ERROR: Open file MGSim.log failed\n");
			return -1;
		}
		dataFile = fopen("MGSim.data", "w+");
		if (dataFile == NULL) {
			printf("ERROR: Open file MGSim.log failed\n");
			return -1;
		}

		pAllNet = network->buildNetwork();
	}

	pCpuNet = getGlobalData(pAllNet, size, rank);
	MPI_Bcast(pCpuNet, sizeof(GNetwork), MPI_BYTE, 0, MPI_COMM_WORLD);
	MPI_Bcast(pCpuNet->nTypes, sizeof(Type)*pCpuNet->nTypeNum, MPI_BYTE, 0, MPI_COMM_WORLD);
	MPI_Bcast(pCpuNet->sTypes, sizeof(Type)*pCpuNet->sTypeNum, MPI_BYTE, 0, MPI_COMM_WORLD);
	MPI_Bcast(pCpuNet->gNeuronNums, sizeof(int)*(pCpuNet->nTypeNum+1), MPI_BYTE, 0, MPI_COMM_WORLD);
	MPI_Bcast(pCpuNet->gSynapseNums, sizeof(int)*(pCpuNet->sTypeNum+1), MPI_BYTE, 0, MPI_COMM_WORLD);
	splitNetwork(pCpuNet, pAllNet, size, rank);


	GNetwork *c_pGpuNet = copyDataToGPU(pCpuNet);
	GNetwork *pGpuNet;
	void **c_pNeurons;
	void **c_pSynapses;
	int *c_gTimeTable = NULL;
	int *c_gFiredTable = NULL;
	bool *c_gSynapsesFiredTable = NULL;
	real *c_gNeuronInput = NULL;

	int nTypeNum = pCpuNet->nTypeNum;
	int sTypeNum = pCpuNet->sTypeNum;
	int thisNeuronNum = pCpuNet->neuronNums[nTypeNum];
	int thisSynapseNum = pCpuNet->synapseNums[sTypeNum];
	int totalNeuronNum = pCpuNet->gNeuronNums[nTypeNum];
	int totalSynapseNum = pCpuNet->gSynapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", thisNeuronNum, thisSynapseNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);

	int MAX_DELAY = (int)(pCpuNet->MAX_DELAY/dt);
	printf("MAX_DELAY: %lf %lf %lf\n", network->maxDelay, pCpuNet->MAX_DELAY, dt);
	printf("MAX_DELAY: %u\n", MAX_DELAY);

	int * c_n_fired = (int*)malloc(sizeof(int)*((totalNeuronNum)));
	bool * c_s_fired = (bool*)malloc(sizeof(bool)*((totalSynapseNum)));
	real * c_n_input = (real*)malloc(sizeof(real)*((totalNeuronNum)));

	checkCudaErrors(hipMalloc((void**)&(pGpuNet), sizeof(GNetwork)));
	checkCudaErrors(hipMemcpy(pGpuNet, c_pGpuNet, sizeof(GNetwork), hipMemcpyHostToDevice));

	c_pNeurons = (void**)malloc(sizeof(void*)*pCpuNet->nTypeNum);
	checkCudaErrors(hipMemcpy(c_pNeurons, c_pGpuNet->pNeurons, sizeof(void*)*(pCpuNet->nTypeNum), hipMemcpyDeviceToHost));
	c_pSynapses = (void**)malloc(sizeof(void*)*pCpuNet->sTypeNum);
	checkCudaErrors(hipMemcpy(c_pSynapses, c_pGpuNet->pSynapses, sizeof(void*)*(pCpuNet->sTypeNum), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMalloc((void**)&c_gTimeTable, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMemset(c_gTimeTable, 0, sizeof(int)*(MAX_DELAY+1)));
	checkCudaErrors(hipMalloc((void**)&c_gFiredTable, sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1))));
	checkCudaErrors(hipMemset(c_gFiredTable, 0, sizeof(int)*((totalNeuronNum)*(MAX_DELAY+1))));
	checkCudaErrors(hipMalloc((void**)&c_gSynapsesFiredTable, sizeof(bool)*(totalSynapseNum)));
	checkCudaErrors(hipMemset(c_gSynapsesFiredTable, 0, sizeof(bool)*(totalSynapseNum)));
	checkCudaErrors(hipMalloc((void**)&c_gNeuronInput, sizeof(real)*(totalNeuronNum)));
	checkCudaErrors(hipMemset(c_gNeuronInput, 0, sizeof(real)*(totalNeuronNum)));

	init_global<<<1, 1, 0>>>(MAX_DELAY, c_gTimeTable, c_gNeuronInput, c_gFiredTable, totalNeuronNum, c_gSynapsesFiredTable, totalSynapseNum, pGpuNet);

	BlockSize *updateSize = getBlockSize(thisNeuronNum, thisSynapseNum);
	BlockSize preSize = { 0, 0, 0};
	BlockSize postSize = { 0, 0, 0};
	hipOccupancyMaxPotentialBlockSize(&(preSize.minGridSize), &(preSize.blockSize), update_lif_neuron, 0, totalNeuronNum); 
	preSize.gridSize = (totalNeuronNum + (preSize.blockSize) - 1) / (preSize.blockSize);
	hipOccupancyMaxPotentialBlockSize(&(postSize.minGridSize), &(postSize.blockSize), update_lif_neuron, 0, totalSynapseNum); 
	postSize.gridSize = (totalSynapseNum + (postSize.blockSize) - 1) / (postSize.blockSize);

	vector<int> firedInfo;
	printf("Start runing for %d cycles\n", sim_cycle);
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<sim_cycle; time++) {
		printf("\rCycle: %d", time);

		checkCudaErrors(hipMemcpy(c_n_input, c_gNeuronInput, sizeof(real)*(totalNeuronNum), hipMemcpyDeviceToHost));
		MPI_Allreduce(MPI_IN_PLACE, c_n_input, totalNeuronNum, MPI_CREAL, MPI_SUM, MPI_COMM_WORLD);
		checkCudaErrors(hipMemcpy(c_gNeuronInput, c_n_input, sizeof(real)*(totalNeuronNum), hipMemcpyHostToDevice));

		for (int i=0; i<nTypeNum; i++) {
			updateType[pCpuNet->nTypes[i]](c_pNeurons[i], pCpuNet->neuronNums[i+1]-pCpuNet->neuronNums[i], time, &updateSize[pCpuNet->nTypes[i]]);
		}

		update_pre_synapse<<<preSize.gridSize, preSize.blockSize>>>(pGpuNet, time);

		checkCudaErrors(hipMemcpy(c_s_fired, c_gSynapsesFiredTable, sizeof(bool)*(totalSynapseNum), hipMemcpyDeviceToHost));
		MPI_Allreduce(MPI_IN_PLACE, c_s_fired, sizeof(bool)*totalSynapseNum, MPI_BYTE, MPI_SUM, MPI_COMM_WORLD);
		checkCudaErrors(hipMemcpy(c_gSynapsesFiredTable, c_s_fired, sizeof(bool)*(totalSynapseNum), hipMemcpyHostToDevice));

		for (int i=0; i<sTypeNum; i++) {
			updateType[pCpuNet->sTypes[i]](c_pSynapses[i], pCpuNet->synapseNums[i+1]-pCpuNet->synapseNums[i], time, &updateSize[pCpuNet->nTypes[i]]);
		}

		update_post_synapse<<<postSize.gridSize, postSize.blockSize>>>(pGpuNet, time);

		int currentIdx = time%(MAX_DELAY+1);
		checkCudaErrors(hipMemcpy(c_n_fired, c_gFiredTable + totalNeuronNum*currentIdx, sizeof(int)*(totalNeuronNum), hipMemcpyDeviceToHost));
		MPI_Allreduce(MPI_IN_PLACE, c_n_fired, totalNeuronNum, MPI_CREAL, MPI_SUM, MPI_COMM_WORLD);

		//int count = 0;
		fprintf(dataFile, "%d", time);
		for (int i=0; i<pCpuNet->neuronNums[nTypeNum]; i++) {
			fprintf(dataFile, ", %lf", c_n_input[i]);
		}
		fprintf(dataFile, "\n");

		fprintf(logFile, "Cycle %d: ", time);
		firedInfo.clear();
		for (int i=0; i<pCpuNet->neuronNums[nTypeNum]; i++) {
			if (c_n_fired[i] > 0) {
				firedInfo.push_back(i);
			}
		}

		if (rank == 0) {
			int size = firedInfo.size();
			if (size > 0) {
				fprintf(logFile, "%d_%d", network->idx2nid[firedInfo[0]].groupId, network->idx2nid[firedInfo[0]].id);
				for (int i=1; i<size; i++) {
					fprintf(logFile, ", %d_%d", network->idx2nid[firedInfo[i]].groupId, network->idx2nid[firedInfo[i]].id);
				}
			}
			firedInfo.clear();
			for (int i=0; i<pCpuNet->synapseNums[sTypeNum]; i++) {
				if (c_s_fired[i]) {
					firedInfo.push_back(i);
				}
			}
			int size2 = firedInfo.size();
			if (size2 > 0) {
				if (size > 0) {
					printf(", ");
				}
				fprintf(logFile, "%d_%d", network->idx2sid[firedInfo[0]].groupId, network->idx2sid[firedInfo[0]].id);
				for (int i=1; i<size2; i++) {
					fprintf(logFile, ", %d_%d", network->idx2sid[firedInfo[i]].groupId, network->idx2sid[firedInfo[i]].id);
				}
			}
			fprintf(logFile, "\n");
		}
	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("\nSimulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	if (rank == 0) {
		fclose(logFile);
		fclose(dataFile);
	}

	checkCudaErrors(hipFree(c_gTimeTable));
	checkCudaErrors(hipFree(c_gFiredTable));
	checkCudaErrors(hipFree(c_gSynapsesFiredTable));

	freeGPUData(c_pGpuNet);
	checkCudaErrors(hipFree(pGpuNet));

	return 0;
}

GNetwork* getGlobalData(GNetwork *network, int rankSize, int rank)
{
	int nTypeNum = network->nTypeNum;
	int sTypeNum = network->sTypeNum;

	GNetwork *ret = (GNetwork *)malloc(sizeof(GNetwork));
	memcpy(ret, network, sizeof(GNetwork));

	ret->pNeurons = (void**)malloc(sizeof(void*)*nTypeNum);
	ret->pSynapses = (void**)malloc(sizeof(void*)*sTypeNum);
	ret->nOffsets = (int*)malloc(sizeof(int)*(nTypeNum));
	ret->sOffsets = (int*)malloc(sizeof(int)*(sTypeNum));
	ret->neuronNums = (int*)malloc(sizeof(int)*(nTypeNum + 1));
	ret->synapseNums = (int*)malloc(sizeof(int)*(sTypeNum + 1));
	ret->neuronNums[0] = 0;
	ret->synapseNums[0] = 0;

	ret->nTypes = (Type*)malloc(sizeof(Type)*nTypeNum);
	ret->sTypes = (Type*)malloc(sizeof(Type)*sTypeNum);
	ret->gNeuronNums = (int*)malloc(sizeof(int)*(nTypeNum + 1));
	ret->gSynapseNums = (int*)malloc(sizeof(int)*(sTypeNum + 1));

	if (rank == 0) {
		memcpy(ret->nTypes, network->nTypes, sizeof(Type)*nTypeNum);
		memcpy(ret->sTypes, network->sTypes, sizeof(Type)*sTypeNum);
		memcpy(ret->gNeuronNums, network->gNeuronNums, sizeof(int)*(nTypeNum+1));
		memcpy(ret->gSynapseNums, network->gSynapseNums, sizeof(int)*(sTypeNum+1));
	}

	for (int i=0; i<nTypeNum; i++) {
		int num_i = network->gNeuronNums[i+1] - network->gNeuronNums[i];
		int size = num_i/rankSize;
		int range = num_i%size;

		int size_tt = 0;
		if (range > 0) {
			size_tt = size+1;
		} else {
			size_tt = size;
		}

		network->neuronNums[i+1] = network->neuronNums[i] + size_tt;
	}
	for (int i=0; i<sTypeNum; i++) {
		int num_i = network->gSynapseNums[i+1] - network->gSynapseNums[i];
		int size = num_i/rankSize;
		int range = num_i%size;

		int size_tt = 0;
		if (range > 0) {
			size_tt = size+1;
		} else {
			size_tt = size;
		}

		network->synapseNums[i+1] = network->synapseNums[i] + size_tt;
	}


	return ret;
}

GNetwork* splitNetwork(GNetwork *network, GNetwork *allNet, int rankSize, int rank)
{
	int nTypeNum = network->nTypeNum;
	int sTypeNum = network->sTypeNum;
	int totalNeuronNum = network->gNeuronNums[nTypeNum];
	int totalSynapseNum = network->gSynapseNums[sTypeNum];
	printf("NeuronTypeNum: %d, SynapseTypeNum: %d\n", nTypeNum, sTypeNum);
	printf("NeuronNum: %d, SynapseNum: %d\n", totalNeuronNum, totalSynapseNum);
	printf("MAX_DELAY: %lf\n", network->MAX_DELAY);

	if (rank == 0) {
		for (int i=0; i<nTypeNum; i++) {
			int size = network->neuronNums[i+1] - network->neuronNums[i];

			network->nOffsets[i] = 0;
			//Copy neurons
			//copyNeurons[i](network, allNet, i, size);
		}
		for (int i=0; i<sTypeNum; i++) {
			int size = network->neuronNums[i+1] - network->neuronNums[i];

			network->sOffsets[i] = 0;
			//Copy neurons
			//copySynapse[i](network, allNet, i, size);
		}
	} else {
		for (int i=0; i<nTypeNum; i++) {
			int num_i = network->gNeuronNums[i+1] - network->gNeuronNums[i];
			int size = num_i/rankSize;
			int range = num_i%size;

			if (i < range) {
				network->nOffsets[i] = (size+1)*range;
			} else {
				network->nOffsets[i] = (size+1)*range + size*(i-range);
			}
			//Copy neurons
			//mpiSendNeurons(network, allNet, i, size);
		}
		for (int i=0; i<sTypeNum; i++) {
			int num_i = network->gNeuronNums[i+1] - network->gNeuronNums[i];
			int size = num_i/rankSize;
			int range = num_i%size;

			if (i < range) {
				network->nOffsets[i] = (size+1)*range;
			} else {
				network->nOffsets[i] = (size+1)*range + size*(i-range);
			}
			//Copy neurons
			//mpiSendSynapse(network, allNet, i, size);
		}
	}

	return network;
}
