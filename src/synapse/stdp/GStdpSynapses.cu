
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "../../gpu_utils/mem_op.h"
#include "GStaticSynapses.h"

int cudaAllocStatic(void *pCpu, void *pGpu, int num)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	GStaticSynapses *p = (GStaticSynapses*)pCpu;
	pGpuSynapses->p_weight = copyToGPU<real>(p->p_weight, num);
	pGpuSynapses->p_apre = copyToGPU<real>(p->p_apre, num);
	pGpuSynapses->p_apost = copyToGPU<real>(p->p_apost, num);
	pGpuSynapses->p_d_apre = copyToGPU<real>(p->p_d_apre, num);
	pGpuSynapses->p_d_apost = copyToGPU<real>(p->p_d_apost, num);
	pGpuSynapses->p_tau_apre = copyToGPU<real>(p->p_tau_apre, num);
	pGpuSynapses->p_tau_apost = copyToGPU<real>(p->p_tau_apost, num);
	//pGpuSynapses->p_delay = copyToGPU<int>(p->p_delay, num);
	pGpuSynapses->p_src = copyToGPU<int>(p->p_src, num);
	pGpuSynapses->p_dst = copyToGPU<int>(p->p_dst, num);
	return 0;
}

int cudaFreeStatic(void *pGpu)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	gpuFree(pGpuSynapses->p_weight);
	gpuFree(pGpuSynapses->p_apre);
	gpuFree(pGpuSynapses->p_apost);
	gpuFree(pGpuSynapses->p_d_apre);
	gpuFree(pGpuSynapses->p_d_apost);
	gpuFree(pGpuSynapses->p_tau_apre);
	gpuFree(pGpuSynapses->p_tau_apost);
	//gpuFree(pGpuSynapses->p_delay);
	gpuFree(pGpuSynapses->p_src);
	gpuFree(pGpuSynapses->p_dst);
	return 0;
}

