#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

#include "STDPData.h"


__global__ void update_stdp_hit(Connection *connection, STDPData *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int time)
{
	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int delayLength = connection->maxDelay - connection->minDelay + 1;
	for (int delta_t = 0; delta_t<delayLength; delta_t++) {
		int block_idx = blockIdx.x;
		int timeIdx = (time+delayLength-delta_t)%(connection->maxDelay+1);
		int firedSize = firedTableSizes[timeIdx];
		int numPerBlock = (firedSize - 1)/gridDim.x + 1;
		int blockNumMinusOne = (firedSize - 1) / numPerBlock;

		int firedSizeBlock = 0;
		if (block_idx == blockNumMinusOne) {
			firedSizeBlock = firedSize - block_idx * numPerBlock;
		} else if (block_idx < blockNumMinusOne) {
			firedSizeBlock = numPerBlock;
		} else {
			firedSizeBlock = 0;
		}

		for (int idx = 0; idx < firedSizeBlock; idx++) {
			int nid = firedTable[timeIdx*gFiredTableCap + (block_idx)*numPerBlock + idx];
			int startLoc = connection->pDelayStart[delta_t + nid * delayLength];
			int synapseNum = connection->pDelayNum[delta_t + nid * delayLength];
			if (threadIdx.x == 0) {
				gLayerInput[nid]++;
			}
			for (int j=threadIdx.x; j<synapseNum; j += blockDim.x) {
				//int sid = connection->pSynapsesIdx[j+startLoc];
				int sid = j+startLoc;
				real weight = data->pWeight[sid];
				if (weight >= 0) {
					atomicAdd(&(currentE[data->pDst[sid]]), weight);
				} else {
					atomicAdd(&(currentI[data->pDst[sid]]), weight);
				}

				data->pAPre[sid] *= exp((data->pLastUpdate[sid] - time) / (data->pTauPre[sid]));
				data->pAPost[sid] *= exp((data->pLastUpdate[sid] - time) / (data->pTauPost[sid]));

				data->pAPre[sid] += data->pDPre[sid];
				data->pWeight[sid] = _clip(weight + data->pAPost[sid], G_MIN, G_MAX);
				data->pLastUpdate[sid] = time;
			}
		}
		__syncthreads();
	}
}

__global__ void learn_stdp_synapse_post(Connection *connection, STDPData * data, int *firedTable, int *firedTableSizes, int num, int start_id, int time)
{
	int block_idx = blockIdx.x;
	int timeIdx = time%(connection->maxDelay+1);
	int firedSize = firedTableSizes[timeIdx];
	int numPerBlock = (firedSize - 1) / gridDim.x + 1;
	int blockNumMinusOne = (firedSize - 1) / numPerBlock;
	int firedSizeBlock = 0;
	if (block_idx == blockNumMinusOne) {
		firedSizeBlock = firedSize - block_idx * numPerBlock;
	} else if (block_idx < blockNumMinusOne) {
		firedSizeBlock = numPerBlock;
	} else {
		firedSizeBlock = 0;
	}

	for (int idx = 0; idx < firedSizeBlock; idx++) {
		int nid = firedTable[timeIdx * gFiredTableCap + (block_idx) * numPerBlock + idx];
		int startLoc = connection->pDelayStartRev[nid];
		int synapseNum = connection->pDelayNumRev[nid];
		for (int j=threadIdx.x; j<synapseNum; j+=blockDim.x) {
			int sid = connection->pSidMapRev[j+startLoc];
			data->pAPre[sid] *= exp((data->pLastUpdate[sid] - time) / (data->pTauPre[sid]));
			data->pAPost[sid] *= exp((data->pLastUpdate[sid] - time) / (data->pTauPost[sid]));
			data->pAPost[sid] += data->pDPost[sid];
			data->pWeight[sid] = _clip(data->pWeight[sid] + data->pAPre[sid], G_MIN, G_MAX);
			data->pLastUpdate[sid] = time;
		}
	}
	__syncthreads();
}

void cudaUpdateSTDP(Connection * connection, void *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int time, BlockSize *pSize)
{
	//update_stdp_hit<<<pSize->gridSize, pSize->blockSize>>>((STDPData*)data, num, start_id);
	//reset_active_synapse<<<1, 1>>>();
	update_stdp_hit<<<pSize->gridSize, pSize->blockSize>>>((Connection *)connection,  (STDPData *)data, currentE, currentI, firedTable, firedTableSizes, num, start_id, time);
	learn_stdp_synapse_post<<<pSize->gridSize, pSize->blockSize>>>((Connection *)connection,  (STDPData *)data, firedTable, firedTableSizes, num, start_id, time);

}

