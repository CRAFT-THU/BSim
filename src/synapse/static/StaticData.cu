#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "StaticData.h"

void *cudaMallocStatic()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(StaticData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(StaticData)*1));
	return ret;
}

void *cudaAllocStatic(void *pCPU, int num)
{
	void *ret = cudaMallocStatic();
	void *tmp = cudaAllocStaticPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(StaticData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocStaticPara(void *pCPU, int num)
{
	StaticData *p = (StaticData*)pCPU;
	StaticData *ret = (StaticData*)malloc(sizeof(StaticData)*1);
	memset(ret, 0, sizeof(StaticData)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pDst), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pDst, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pDst, p->pDst, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pWeight), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pWeight, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pWeight, p->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaFetchStatic(void *pCPU, void *pGPU, int num)
{
	StaticData *pTmp = (StaticData*)malloc(sizeof(StaticData)*1);
	memset(pTmp, 0, sizeof(StaticData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(StaticData)*1, hipMemcpyDeviceToHost));

	cudaStaticParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaStaticParaToGPU(void *pCPU, void *pGPU, int num)
{
	StaticData *pC = (StaticData*)pCPU;
	StaticData *pG = (StaticData*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pDst, pC->pDst, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG->pWeight, pC->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaStaticParaFromGPU(void *pCPU, void *pGPU, int num)
{
	StaticData *pC = (StaticData*)pCPU;
	StaticData *pG = (StaticData*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pDst, pG->pDst, sizeof(int)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pWeight, pG->pWeight, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeStatic(void *pGPU)
{
	StaticData *tmp = (StaticData*)malloc(sizeof(StaticData)*1);
	memset(tmp, 0, sizeof(StaticData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(StaticData)*1, hipMemcpyDeviceToHost));
	cudaFreeStaticPara(tmp);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeStaticPara(void *pGPU)
{
	StaticData *p = (StaticData*)pGPU;
	hipFree(p->pDst);
	p->pDst = NULL;

	hipFree(p->pWeight);
	p->pWeight = NULL;

	return 0;
}

