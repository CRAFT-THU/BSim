#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GStaticSynapses.h"

void *cudaMallocStatic()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(GStaticSynapses)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(GStaticSynapses)*1));
	return ret;
}

void *cudaAllocStatic(void *pCPU, int num)
{
	void *ret = cudaMallocStatic();
	void *tmp = cudaAllocStaticPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(GStaticSynapses)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocStaticPara(void *pCPU, int num)
{
	GStaticSynapses *p = (GStaticSynapses*)pCPU;
	GStaticSynapses *ret = (GStaticSynapses*)malloc(sizeof(GStaticSynapses)*1);
	memset(ret, 0, sizeof(GStaticSynapses)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pDst), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pDst, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pDst, p->pDst, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pWeight), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pWeight, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pWeight, p->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaFetchStatic(void *pCPU, void *pGPU, int num)
{
	GStaticSynapses *pTmp = (GStaticSynapses*)malloc(sizeof(GStaticSynapses)*1);
	memset(pTmp, 0, sizeof(GStaticSynapses)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(GStaticSynapses)*1, hipMemcpyDeviceToHost));

	cudaStaticParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaStaticParaToGPU(void *pCPU, void *pGPU, int num)
{
	GStaticSynapses *pC = (GStaticSynapses*)pCPU;
	GStaticSynapses *pG = (GStaticSynapses*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pDst, pC->pDst, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG->pWeight, pC->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaStaticParaFromGPU(void *pCPU, void *pGPU, int num)
{
	GStaticSynapses *pC = (GStaticSynapses*)pCPU;
	GStaticSynapses *pG = (GStaticSynapses*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pDst, pG->pDst, sizeof(int)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pWeight, pG->pWeight, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeStatic(void *pGPU)
{
	GStaticSynapses *tmp = (GStaticSynapses*)malloc(sizeof(GStaticSynapses)*1);
	memset(tmp, 0, sizeof(GStaticSynapses)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(GStaticSynapses)*1, hipMemcpyDeviceToHost));
	cudaFreeStaticPara(tmp);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeStaticPara(void *pGPU)
{
	GStaticSynapses *p = (GStaticSynapses*)pGPU;
	hipFree(p->pDst);
	p->pDst = NULL;

	hipFree(p->pWeight);
	p->pWeight = NULL;

	return 0;
}

