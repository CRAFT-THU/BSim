#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GStaticSynapses.h"

void *cudaAllocStatic(void *pCPU, int num)
{
	void *ret = NULL;
	GStaticSynapses *p = (GStaticSynapses*)pCPU;
	GStaticSynapses *tmp = (GStaticSynapses*)malloc(sizeof(GStaticSynapses)*1);
	memset(tmp, 0, sizeof(GStaticSynapses)*1);

	checkCudaErrors(hipMalloc((void**)&(tmp->pDst), sizeof(int)*num));
	checkCudaErrors(hipMemset(tmp->pDst, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(tmp->pDst, p->pDst, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(tmp->pWeight), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pWeight, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pWeight, p->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(GStaticSynapses)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(GStaticSynapses)*1));
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(GStaticSynapses)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

int cudaStaticToGPU(void *pCPU, void *pGPU, int num)
{
	GStaticSynapses *pC = (GStaticSynapses*)pCPU;
	GStaticSynapses *pG = (GStaticSynapses*)pGPU;

	checkCudaErrors(hipMemcpy(pG, pC, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeStatic(void *pGPU)
{
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

