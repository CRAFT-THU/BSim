#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime_info.h"

// #include "GStaticSynapses.h"
#include "static.h"


__global__ void update_dense_static_hit(Connection *connection, GStaticSynapses *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int time)
{
	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int delayLength = connection->maxDelay - connection->minDelay + 1;
	for (int delta_t = 0; delta_t<delayLength; delta_t++) {
		int block_idx = blockIdx.x;
		int time_idx = (time - connection->minDelay - delta_t)%(delayLength);
		int firedSize = firedTableSizes[time_idx];
		int num_per_block = (firedSize - 1)/gridDim.x + 1;
		int block_nums_minus_1 = (firedSize - 1) / num_per_block;

		int fired_size_block = 0;
		if (block_idx == block_nums_minus_1) {
			fired_size_block = firedSize - block_idx * num_per_block;
		} else if (block_idx < block_nums_minus_1) {
			fired_size_block = num_per_block;
		} else {
			fired_size_block = 0;
		}

		for (int idx = 0; idx < fired_size_block; idx++) {
			int nid = firedTable[time_idx*gFiredTableCap + (block_idx)*num_per_block + idx];
			int start_loc = connection->pDelayStart[delta_t + nid * delayLength];
			int synapseNum = connection->pDelayNum[delta_t + nid * delayLength];
			if (threadIdx.x == 0) {
				gLayerInput[nid]++;
			}
			for (int j=threadIdx.x; j<synapseNum; j += blockDim.x) {
				//int sid = connection->pSynapsesIdx[j+start_loc];
				int sid = j+start_loc;
				real weight = data->pWeight[sid];
				if (weight >= 0) {
					atomicAdd(&(currentE[data->pDst[sid]]), weight);
				} else {
					atomicAdd(&(currentI[data->pDst[sid]]), weight);
				}
			}
		}
		__syncthreads();
	}
}

void cudaUpdateStatic(void * connection, void *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int time, BlockSize *pSize)
{
	//update_static_hit<<<pSize->gridSize, pSize->blockSize>>>((GStaticSynapses*)data, num, start_id);
	//reset_active_synapse<<<1, 1>>>();
	update_dense_static_hit<<<pSize->gridSize, pSize->blockSize>>>((Connection *)connection,  (GStaticSynapses *)data, currentE, currentI, firedTable, firedTableSizes, num, start_id, time);

}

