#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime_info.h"

#include "GStatic.h"

__global__ void update_dense_static_hit(GStaticSynapses *d_synapses, int num, int start_id)
{
#define FAST_TEST 2
#if  FAST_TEST == 1
	__shared__ int fire_neuron_id[MAXBLOCKSIZE];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
		int block_idx = blockIdx.x;
		int time_idx = (gCurrentIdx+MAX_DELAY-delta_t)%(MAX_DELAY+1);
		int firedSize = gFiredTableSizes[time_idx];
		int block_nums_minus_1 = (firedSize - 1 + blockDim.x) / blockDim.x - 1;
		int grid_nums = (firedSize - 1 + blockDim.x*gridDim.x)/(blockDim.x * gridDim.x);
		int oid = tid;
		for (int idx = 0; idx < grid_nums; idx++) {
			if (oid < firedSize) {
				fire_neuron_id[threadIdx.x] = gFiredTable[time_idx*gFiredTableCap + oid];
			} else {
				fire_neuron_id[threadIdx.x] = -1;
			}
			oid += blockDim.x * gridDim.x;
			__syncthreads();

			int size = 0;
			if (block_idx == block_nums_minus_1) {
				size = firedSize - block_idx * blockDim.x;
			} else if (block_idx < block_nums_minus_1) {
				size = blockDim.x;
			} else {
				size = 0;
			}

			for (int i=0; i<size; i++) {
				int nid = fire_neuron_id[i];
				int start_loc = gConnection->delayStart[delta_t + nid * MAX_DELAY];
				int synapseNum = gConnection->delayNum[delta_t + nid * MAX_DELAY];
				gLayerInput[nid]++;
				for (int j=threadIdx.x; j<synapseNum; j += blockDim.x) {
					//int sid = gConnection->pSynapsesIdx[j+start_loc];
					int sid = j+start_loc;
					real weight = d_synapses->p_weight[sid];
					if (weight >= 0) {
						atomicAdd(&(gNeuronInput[d_synapses->p_dst[sid]]), weight);
					} else {
						atomicAdd(&(gNeuronInput_I[d_synapses->p_dst[sid]]), weight);
					}
				}
			}
			block_idx += gridDim.x;
			__syncthreads();
		}
		__syncthreads();
	}
#elif FAST_TEST == 2
	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
		int block_idx = blockIdx.x;
		int time_idx = (gCurrentIdx+MAX_DELAY-delta_t)%(MAX_DELAY+1);
		int firedSize = gFiredTableSizes[time_idx];
		int num_per_block = (firedSize - 1)/gridDim.x + 1;
		int block_nums_minus_1 = (firedSize - 1) / num_per_block;

		int fired_size_block = 0;
		if (block_idx == block_nums_minus_1) {
			fired_size_block = firedSize - block_idx * num_per_block;
		} else if (block_idx < block_nums_minus_1) {
			fired_size_block = num_per_block;
		} else {
			fired_size_block = 0;
		}

		for (int idx = 0; idx < fired_size_block; idx++) {
			int nid = gFiredTable[time_idx*gFiredTableCap + (block_idx)*num_per_block + idx];
			int start_loc = gConnection->delayStart[delta_t + nid * MAX_DELAY];
			int synapseNum = gConnection->delayNum[delta_t + nid * MAX_DELAY];
			if (threadIdx.x == 0) {
				gLayerInput[nid]++;
			}
			for (int j=threadIdx.x; j<synapseNum; j += blockDim.x) {
				//int sid = gConnection->pSynapsesIdx[j+start_loc];
				int sid = j+start_loc;
				real weight = d_synapses->p_weight[sid];
				if (weight >= 0) {
					atomicAdd(&(gNeuronInput[d_synapses->p_dst[sid]]), weight);
				} else {
					atomicAdd(&(gNeuronInput_I[d_synapses->p_dst[sid]]), weight);
				}
			}
		}
		__syncthreads();
	}
#else
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
		int time_idx = (gCurrentIdx+MAX_DELAY-delta_t)%(MAX_DELAY+1);
		int firedSize = gFiredTableSizes[time_idx];
		for (int idx = tid; idx < firedSize; idx += blockDim.x*gridDim.x) {
			int nid = gFiredTable[time_idx*gFiredTableCap + idx];
			int start_loc = gConnection->delayStart[delta_t + nid * MAX_DELAY];
			int synapseNum = gConnection->delayNum[delta_t + nid * MAX_DELAY];
			gLayerInput[nid]++;
			for (int i=0; i<synapseNum; i++) {
				//int sid = gConnection->pSynapsesIdx[i+start_loc];
				int sid = i+start_loc;
				real weight = d_synapses->p_weight[sid];
				if (weight >= 0) {
					atomicAdd(&(gNeuronInput[d_synapses->p_dst[sid]]), weight);
				} else {
					atomicAdd(&(gNeuronInput_I[d_synapses->p_dst[sid]]), weight);
				}
			}
		}
	}
#endif
}
