/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "GAlphaSynapses.h"

int cudaAllocAlphaSynapses(void *pCpu, void *pGpu)
{
	GAlphaSynapses *pGpuSynapses = (GAlphaSynapses*)pGpu;
	GAlphaSynapses *p = (GAlphaSynapses*)pCpu;

	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pID), sizeof(ID)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->pID, p->pID, sizeof(ID)*p->num, hipMemcpyHostToDevice));

	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pType), sizeof(Type)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->pType, p->pType, sizeof(Type)*p->num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_weight), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_weight, p->p_weight, sizeof(real)*p->num, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay), sizeof(real)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay, p->p_delay, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay_steps), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay_steps, p->p_delay_steps, sizeof(int)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C1), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C1, p->p_C1, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C2), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C2, p->p_C2, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C1), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C1, p->p__C1, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C2), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C2, p->p__C2, sizeof(real)*p->num, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_tau_syn), sizeof(real)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->p_tau_syn, p->p_tau_syn, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_syn), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_syn, p->p_I_syn, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_tmp), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_tmp, p->p_I_tmp, sizeof(real)*p->num, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__dt), sizeof(real)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->p__dt, p->p__dt, sizeof(real)*p->num, hipMemcpyHostToDevice));

	//checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pSrc), sizeof(int)*p->num));
	//checkCudaErrors(hipMemcpy(pGpuSynapses->pSrc, pSrc, sizeof(int)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pDst), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pDst, p->pDst, sizeof(int)*p->num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeAlphaSynapses(void *pGpu)
{
	GAlphaSynapses *pGpuSynapses = (GAlphaSynapses*)pGpu;
	//checkCudaErrors(hipFree(pGpuSynapses->pID));

	//checkCudaErrors(hipFree(pGpuSynapses->pType));

	checkCudaErrors(hipFree(pGpuSynapses->p_weight));
	//checkCudaErrors(hipFree(pGpuSynapses->p_delay));
	checkCudaErrors(hipFree(pGpuSynapses->p_delay_steps));
	checkCudaErrors(hipFree(pGpuSynapses->p_C1));
	checkCudaErrors(hipFree(pGpuSynapses->p_C2));
	checkCudaErrors(hipFree(pGpuSynapses->p__C1));
	checkCudaErrors(hipFree(pGpuSynapses->p__C2));
	//checkCudaErrors(hipFree(pGpuSynapses->p_tau_syn));
	checkCudaErrors(hipFree(pGpuSynapses->p_I_syn));
	checkCudaErrors(hipFree(pGpuSynapses->p_I_tmp));
	//checkCudaErrors(hipFree(pGpuSynapses->p__dt));

	//checkCudaErrors(hipFree(pGpuSynapses->pSrc));
	checkCudaErrors(hipFree(pGpuSynapses->pDst));

	return 0;
}
