/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "GExpSynapses.h"

int cudaAllocExp(void *pCpu, void *pGpu, int num)
{
	GExpSynapses *pGpuSynapses = (GExpSynapses*)pGpu;
	GExpSynapses *p = (GExpSynapses*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_weight), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_weight, p->p_weight, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay_steps), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay_steps, p->p_delay_steps, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C1, p->p_C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C1, p->p__C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_syn, p->p_I_syn, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pDst), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pDst, p->pDst, sizeof(int)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeExp(void *pGpu)
{
	GExpSynapses *pGpuSynapses = (GExpSynapses*)pGpu;

	checkCudaErrors(hipFree(pGpuSynapses->p_weight));
	checkCudaErrors(hipFree(pGpuSynapses->p_delay_steps));
	checkCudaErrors(hipFree(pGpuSynapses->p_C1));
	checkCudaErrors(hipFree(pGpuSynapses->p__C1));
	checkCudaErrors(hipFree(pGpuSynapses->p_I_syn));

	checkCudaErrors(hipFree(pGpuSynapses->pDst));

	return 0;
}
