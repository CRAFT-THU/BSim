
#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../gpu_utils/mem_op.h"
#include "GStaticSynapses.h"

int cudaAllocStatic(void *pCpu, void *pGpu, int num)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	GStaticSynapses *p = (GStaticSynapses*)pCpu;
	pGpuSynapses->p_weight = copyToGPU<real>(p->p_weight, num);
	return 0;
}

int cudaFreeStatic(void *pCpu)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	gpuFree(pGpuSynapses->p_weight);
	return 0;
}

