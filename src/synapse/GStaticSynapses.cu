
#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../gpu_utils/mem_op.h"
#include "GStaticSynapses.h"

int cudaAllocStatic(void *pCpu, void *pGpu, int num)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	GStaticSynapses *p = (GStaticSynapses*)pCpu;
	pGpuSynapses->p_weight = copyToGPU<real>(p->p_weight, num);
	pGpuSynapses->pDst = copyToGPU<int>(p->pDst, num);
	return 0;
}

int cudaFreeStatic(void *pGpu)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	gpuFree(pGpuSynapses->p_weight);
	gpuFree(pGpuSynapses->pDst);
	return 0;
}

