
#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../gpu_utils/mem_op.h"
#include "GStaticSynapses.h"

int cudaAllocStatic(void *pCpu, void *pGpu, int num)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	GStaticSynapses *p = (GStaticSynapses*)pCpu;
	pGpuSynapses->p_weight = copyToGPU<real>(p->p_weight, num);
	pGpuSynapses->p_dst = copyToGPU<int>(p->p_dst, num);
	return 0;
}

int cudaFreeStatic(void *pGpu)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	gpuFree(pGpuSynapses->p_weight);
	gpuFree(pGpuSynapses->p_dst);
	return 0;
}

