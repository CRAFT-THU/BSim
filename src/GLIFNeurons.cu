/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "./utils/cuda/hip/hip_runtime_api.h"
#include "GLIFNeurons.h"

int GLIFNeurons::allocGNeurons(GLIFNeurons * pGpuNeurons)
{
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pID), sizeof(ID)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pID, pID, sizeof(ID)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pType), sizeof(NeuronType)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pType, pType, sizeof(NeuronType)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_init), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_init, p_v_init, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_rest), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_rest, p_v_rest, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_reset), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_reset, p_v_init, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_cm), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_cm, p_cm, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_m), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_m, p_tau_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_refrac), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_refrac, p_tau_refrac, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_syn_E), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_syn_E, p_tau_syn_E, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_syn_I), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_syn_I, p_tau_syn_E, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_thresh), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_thresh, p_v_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_offset), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_offset, p_i_offset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_syn, p_i_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_vm), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_vm, p_vm, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p__dt), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p__dt, p__dt, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_C1, p_C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_C2), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_C2, p_C2, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_tmp, p_i_tmp, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_refrac_step), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_refrac_step, p_refrac_step, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pSynapsesNum), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pSynapsesNum, pSynapsesNum, sizeof(unsigned int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pSynapsesLoc), sizeof(unsigned int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pSynapsesLoc, pSynapsesLoc, sizeof(unsigned int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pSynapsesIdx), sizeof(unsigned int)*synapsesNum));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pSynapsesIdx, pSynapsesIdx, sizeof(unsigned int)*synapsesNum, hipMemcpyHostToDevice));

	return 0;
}

int freeGNeurons(GLIFNeurons * pGpuNeurons)
{
	checkCudaErrors(hipFree(pGpuNeurons->pID));
	checkCudaErrors(hipFree(pGpuNeurons->pType));

	checkCudaErrors(hipFree(pGpuNeurons->p_v_init));
	checkCudaErrors(hipFree(pGpuNeurons->p_v_rest));
	checkCudaErrors(hipFree(pGpuNeurons->p_v_reset));
	checkCudaErrors(hipFree(pGpuNeurons->p_cm));
	checkCudaErrors(hipFree(pGpuNeurons->p_tau_m));
	checkCudaErrors(hipFree(pGpuNeurons->p_tau_refrac));
	checkCudaErrors(hipFree(pGpuNeurons->p_tau_syn_E));
	checkCudaErrors(hipFree(pGpuNeurons->p_tau_syn_I));
	checkCudaErrors(hipFree(pGpuNeurons->p_v_thresh));
	checkCudaErrors(hipFree(pGpuNeurons->p_i_offset));
	checkCudaErrors(hipFree(pGpuNeurons->p_i_syn));
	checkCudaErrors(hipFree(pGpuNeurons->p_vm));
	checkCudaErrors(hipFree(pGpuNeurons->p__dt));
	checkCudaErrors(hipFree(pGpuNeurons->p_C1));
	checkCudaErrors(hipFree(pGpuNeurons->p_C2));
	checkCudaErrors(hipFree(pGpuNeurons->p_i_tmp));

	checkCudaErrors(hipFree(pGpuNeurons->p_refrac_step));

	checkCudaErrors(hipFree(pGpuNeurons->pSynapsesNum));
	checkCudaErrors(hipFree(pGpuNeurons->pSynapsesLoc));
	checkCudaErrors(hipFree(pGpuNeurons->pSynapsesIdx));

	return 0;
}
