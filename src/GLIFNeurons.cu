/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "./utils/cuda/hip/hip_runtime_api.h"
#include "GLIFNeurons.h"

int cudaAllocLIFNeurons(void *pCpu, void *pGpu)
{
	GLIFNeurons *pGpuNeurons = (GLIFNeurons*)pGpu;
	GLIFNeurons *p = (GLIFNeurons*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pID), sizeof(ID)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pID, p->pID, sizeof(ID)*p->num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pType), sizeof(Type)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pType, p->pType, sizeof(Type)*p->num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_init), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_init, p->p_v_init, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_rest), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_rest, p->p_v_rest, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_reset), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_reset, p->p_v_init, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_cm), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_cm, p->p_cm, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_m), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_m, p->p_tau_m, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_refrac), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_refrac, p->p_tau_refrac, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_syn_E), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_syn_E, p->p_tau_syn_E, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_tau_syn_I), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_tau_syn_I, p->p_tau_syn_E, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_v_thresh), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_v_thresh, p->p_v_thresh, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_offset), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_offset, p->p_i_offset, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_syn), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_syn, p->p_i_syn, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_vm), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_vm, p->p_vm, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p__dt), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p__dt, p->p__dt, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_C1), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_C1, p->p_C1, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_C2), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_C2, p->p_C2, sizeof(real)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_i_tmp), sizeof(real)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_i_tmp, p->p_i_tmp, sizeof(real)*p->num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_refrac_step), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_refrac_step, p->p_refrac_step, sizeof(int)*p->num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pSynapsesNum), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pSynapsesNum, p->pSynapsesNum, sizeof(int)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pSynapsesLoc), sizeof(int)*p->num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pSynapsesLoc, p->pSynapsesLoc, sizeof(int)*p->num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->pSynapsesIdx), sizeof(int)*p->synapsesNum));
	checkCudaErrors(hipMemcpy(pGpuNeurons->pSynapsesIdx, p->pSynapsesIdx, sizeof(int)*p->synapsesNum, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeLIFNeurons(void *pGpu)
{
	GLIFNeurons *pGpuNeurons = (GLIFNeurons*)pGpu;

	checkCudaErrors(hipFree(pGpuNeurons->pID));
	checkCudaErrors(hipFree(pGpuNeurons->pType));

	checkCudaErrors(hipFree(pGpuNeurons->p_v_init));
	checkCudaErrors(hipFree(pGpuNeurons->p_v_rest));
	checkCudaErrors(hipFree(pGpuNeurons->p_v_reset));
	checkCudaErrors(hipFree(pGpuNeurons->p_cm));
	checkCudaErrors(hipFree(pGpuNeurons->p_tau_m));
	checkCudaErrors(hipFree(pGpuNeurons->p_tau_refrac));
	checkCudaErrors(hipFree(pGpuNeurons->p_tau_syn_E));
	checkCudaErrors(hipFree(pGpuNeurons->p_tau_syn_I));
	checkCudaErrors(hipFree(pGpuNeurons->p_v_thresh));
	checkCudaErrors(hipFree(pGpuNeurons->p_i_offset));
	checkCudaErrors(hipFree(pGpuNeurons->p_i_syn));
	checkCudaErrors(hipFree(pGpuNeurons->p_vm));
	checkCudaErrors(hipFree(pGpuNeurons->p__dt));
	checkCudaErrors(hipFree(pGpuNeurons->p_C1));
	checkCudaErrors(hipFree(pGpuNeurons->p_C2));
	checkCudaErrors(hipFree(pGpuNeurons->p_i_tmp));

	checkCudaErrors(hipFree(pGpuNeurons->p_refrac_step));

	checkCudaErrors(hipFree(pGpuNeurons->pSynapsesNum));
	checkCudaErrors(hipFree(pGpuNeurons->pSynapsesLoc));
	checkCudaErrors(hipFree(pGpuNeurons->pSynapsesIdx));

	return 0;
}
