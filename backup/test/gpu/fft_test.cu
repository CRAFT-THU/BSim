/* This program is writen by qp09.
 * usually just for fun.
 * Tue December 15 2015
 */

#include <hipfft/hipfft.h>
#include "../../include/BSim.h"

using namespace std;

int main(int argc, char **argv)
{
	bool load = false;

	if ((argc > 1) && (0==strcmp(argv[1], "load"))) {
		load = true;
	}

	const int N = 10;
	Network c;

	Population<Constant_spikes> *pn0 = c.createPopulation(0, N, Constant_spikes(ConstantNeuron(0.05, 0, 0.101), 1.0, 0.101));
	for (int i = 0; i < pn0->getNum(); i++) {
		Constant_spikes * n = static_cast<Constant_spikes*>(pn0->getNeuron(i));
		n->setRate((N/2-i)*0.04 + 0.1);
	}

	Population<FFTCompute> *pn1 = c.createPopulation(1, N, FFTCompute(FFTNeuron(), 1.0, 1.0));

	real * weight0 = NULL;
	real * delay = NULL;

	if (load) { 
		//printf("LOAD DATA...\n");
		weight0 = loadArray("weight0.csv", N);
		delay = loadArray("delay.csv", N);
		//printf("LOAD DATA FINISHED\n");
	} else {
		//printf("GENERATE DATA...\n");
		//real * array = getConstArray(weight_value, num);
		weight0 = getConstArray((real)1.0, N);
		//weight2 = getRandomArray((real)20e-3, N*N);
		delay = getConstArray((real)1e-3, N);
		//printf("GENERATE DATA FINISHED\n");
	}

	//Network.connect(population1, population2, weight_array, delay_array, Exec or Inhi array, num)
	c.connectOne2One(pn0, pn1, weight0, delay, NULL, N);

	//STSim st(&c, 1.0e-3);
	ArrayInfo zero_array = {0, NULL};
	FireInfo fire_info;
	fire_info["count"] = zero_array;
	//fire_info["X"] = zero_array;
	//fire_info["Y"] = zero_array;

	SGSim sg(&c, 1.0e-3);


	sg.run(0.11, fire_info);
	
	int *rate = (int*) fire_info["count"].data;
	//int *X = (int*) fire_info["X"].data;
	//real *Y = (real*) fire_info["Y"].data;

	printf("Input:\t");
	for (int i=0; i<N/2; i++) {
		printf("%d+%di ", rate[i*2], rate[i*2+1]);
	}
	printf("\n");

	//printf("Input2:\t");
	//for (int i=0; i<N; i++) {
	//	printf("%d ", X[i]);
	//}
	//printf("\n");

	//printf("Input3:\t");
	//for (int i=0; i<N; i++) {
	//	printf("%f ", Y[N + i]);
	//}
	//printf("\n");

	printf("Result:\t");
	for (int i=0; i<N/2; i++) {
		printf("%d+%di ", rate[N + i*2], rate[N + i*2 + 1]);
	}
	printf("\n");

	hipfftComplex *data_dev = NULL;
	hipfftComplex *data_Host = (hipfftComplex*)malloc(N/2*sizeof(hipfftComplex));

	for (int i =0; i < N/2; i++)
	{
		data_Host[i].x = rate[i*2];
		data_Host[i].y = rate[i*2+1];
	}

	hipMalloc((void**)&data_dev, sizeof(hipfftComplex)*N/2);
	hipMemcpy(data_dev, data_Host, N/2 * sizeof(hipfftComplex), hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan1d(&plan, N/2, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, data_dev, data_dev, HIPFFT_FORWARD);
	hipMemcpy(data_Host, data_dev, N/2 * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	printf("Check:\t");
	for (int i=0; i<N/2; i++) {
		printf("%f+%fi ", data_Host[i].x, data_Host[i].y);
	}
	printf("\n");


	//if (!load) {
	//	printf("SAVE DATA...\n");
	//	saveArray("weight0.csv", weight0, N);
	//	saveArray("delay.csv", delay, N);
	//	printf("SAVE DATA FINISHED\n");
	//}
	
	return 0;
} 
