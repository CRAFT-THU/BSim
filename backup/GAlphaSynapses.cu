/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */

#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "GAlphaSynapses.h"

int cudaAllocAlphaSynapses(void *pCpu, void *pGpu, int num)
{
	GAlphaSynapses *pGpuSynapses = (GAlphaSynapses*)pGpu;
	GAlphaSynapses *p = (GAlphaSynapses*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_weight), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_weight, p->p_weight, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_delay_steps), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_delay_steps, p->p_delay_steps, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C1, p->p_C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_C2), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_C2, p->p_C2, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C1), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C1, p->p__C1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p__C2), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p__C2, p->p__C2, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_syn), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_syn, p->p_I_syn, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->p_I_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->p_I_tmp, p->p_I_tmp, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuSynapses->pDst), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuSynapses->pDst, p->pDst, sizeof(int)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeAlphaSynapses(void *pGpu)
{
	GAlphaSynapses *pGpuSynapses = (GAlphaSynapses*)pGpu;

	checkCudaErrors(hipFree(pGpuSynapses->p_weight));
	checkCudaErrors(hipFree(pGpuSynapses->p_delay_steps));
	checkCudaErrors(hipFree(pGpuSynapses->p_C1));
	checkCudaErrors(hipFree(pGpuSynapses->p_C2));
	checkCudaErrors(hipFree(pGpuSynapses->p__C1));
	checkCudaErrors(hipFree(pGpuSynapses->p__C2));
	checkCudaErrors(hipFree(pGpuSynapses->p_I_syn));
	checkCudaErrors(hipFree(pGpuSynapses->p_I_tmp));

	checkCudaErrors(hipFree(pGpuSynapses->pDst));

	return 0;
}
