#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"
#include "../../net/Connection.h"

#include "IzhikevichData.h"

__global__ void update_izhikevich(Connection *connection, IzhikevichData *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
{
	int currentIdx = time % (connection->maxDelay+1);
	__shared__ int fireTableT[MAX_BLOCK_SIZE];
	__shared__ volatile int fire_cnt;
	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		int nid = idx;
		int gnid = offset + nid;
		int testLoc = 0;

		real v = data->pV[nid];
		real u = data->pU[nid];
		real a = data->pA[nid];
		real b = data->pB[nid];
		real c = data->pC[nid];
		real d = data->pD[nid];

		const bool oldSpike = (v >= 29.99f);

		if (v >= 30.0f) {
			v = c;
			u += d;
		}

		const real I = currentE[gnid] + currentI[gnid];

		v += 0.5f * (0.04f * v * v + 5.0f * v + 140.0f -u + I) * DT;
		v += 0.5f * (0.04f * v * v + 5.0f * v + 140.0f -u + I) * DT;
	
		u += a*(b*v - u) * DT;

		if (v > 30.0f) {
			v = 30.0f;
		}

		bool fired = (v >= 29.99f) && (!oldSpike);

		data->pV[nid] = v;
		data->pU[nid] = u;
		data->pA[nid] = a;
		data->pB[nid] = b;
		data->pC[nid] = c;
		data->pD[nid] = d;

		if (fired) {
			testLoc = atomicAdd((int*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fireTableT[testLoc] = gnid;
				fired = false;
			}
		} else {
			gXInput[gnid] += I;
		}

		currentE[gnid] = 0;
		currentI[gnid] = 0;

		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fireTableT, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			testLoc = atomicAdd((int*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fireTableT[testLoc] = gnid;
				fired = false;
			}
		}
		__syncthreads();

		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fireTableT, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fireTableT, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
	}
	//__syncthreads();
	//if (threadIdx.x == 0 && blockIdx.x == 0) {
	//	gActiveTableSize = 0;
	//}
}

void cudaUpdateIzhikevich(Connection *connection, void *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time, BlockSize *pSize)
{
	update_izhikevich<<<pSize->gridSize, pSize->blockSize>>>(connection, (IzhikevichData *)data, currentE, currentI, firedTable, firedTableSizes, num, offset, time);
}
