
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "../../gpu_utils/mem_op.h"
#include "GStaticSynapses.h"

int cudaAllocStatic(void *pCpu, void *pGpu, int num)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	GStaticSynapses *p = (GStaticSynapses*)pCpu;
	pGpuSynapses->p_weight = copyToGPU<real>(p->p_weight, num);
	//pGpuSynapses->p_delay = copyToGPU<int>(p->p_delay, num);
	//pGpuSynapses->p_src = copyToGPU<int>(p->p_src, num);
	pGpuSynapses->p_dst = copyToGPU<int>(p->p_dst, num);
	return 0;
}

int cudaFreeStatic(void *pGpu)
{
	GStaticSynapses *pGpuSynapses = (GStaticSynapses*)pGpu;
	gpuFree(pGpuSynapses->p_weight);
	//gpuFree(pGpuSynapses->p_delay);
	//gpuFree(pGpuSynapses->p_src);
	gpuFree(pGpuSynapses->p_dst);
	return 0;
}

