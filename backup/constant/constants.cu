#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

// #include "GConstantNeurons.h"
#include "constants.h"


__global__ void update_constant_neuron(GConstantNeurons *d_neurons, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int time)
// __global__ void update_constant_neuron(GConstantNeurons *d_neurons, int num, int start_id, int time)
{
	int currentIdx = time % (MAX_DELAY+1);
	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		fired = (time < d_neurons->p_end_cycle[idx]) && (((time - d_neurons->p_start_cycle[idx]) * d_neurons->p_fire_rate[idx]) > (d_neurons->p_fire_count[idx]));
		gFireCount[start_id + idx] += fired;

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAX_BLOCK_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					d_neurons->p_fire_count[idx] = d_neurons->p_fire_count[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAX_BLOCK_SIZE) {
				commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &(firedTableSizes[currentIdx]), gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, firedTable, &(firedTableSizes[currentIdx]), gFiredTableCap*currentIdx);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

void cudaUpdateConstant(void *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int t, BlockSize *pSize)
{
	update_constant_neuron<<<pSize->gridSize, pSize->blockSize>>>((GConstantNeurons*)data, currentE, currentI, firedTable, firedTableSizes, num, start_id, t);
}

