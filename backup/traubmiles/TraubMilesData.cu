#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "TraubMilesData.h"

void *cudaMallocTraubMiles()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(TraubMilesData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(TraubMilesData)*1));
	return ret;
}


void *cudaAllocTraubMiles(void *pCPU, int num)
{
	void *ret = cudaMallocTraubMiles();
	void *tmp = cudaAllocTraubMilesPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(TraubMilesData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}


void *cudaAllocTraubMilesPara(void *pCPU, int num)
{
	TraubMilesData *p = (TraubMilesData*)pCPU;
	TraubMilesData *ret = (TraubMilesData*)malloc(sizeof(TraubMilesData)*1);
	memset(ret, 0, sizeof(TraubMilesData)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pGNa), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pGNa, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pGNa, p->pGNa, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pENa), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pENa, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pENa, p->pENa, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pGK), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pGK, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pGK, p->pGK, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pEK), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pEK, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pEK, p->pEK, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pGl), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pGl, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pGl, p->pGl, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pEl), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pEl, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pEl, p->pEl, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC, p->pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV, p->pV, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pM), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pM, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pM, p->pM, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pH), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pH, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pH, p->pH, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pN), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pN, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pN, p->pN, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pDecayE), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pDecayE, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pDecayE, p->pDecayE, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pDecayI), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pDecayI, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pDecayI, p->pDecayI, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pEE), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pEE, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pEE, p->pEE, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pEI), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pEI, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pEI, p->pEI, sizeof(real)*num, hipMemcpyHostToDevice));


	return ret;
}


int cudaFetchTraubMiles(void *pCPU, void *pGPU, int num)
{
	TraubMilesData *pTmp = (TraubMilesData*)malloc(sizeof(TraubMilesData)*1);
	memset(pTmp, 0, sizeof(TraubMilesData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(TraubMilesData)*1, hipMemcpyDeviceToHost));

	cudaTraubMilesParaFromGPU(pCPU, pTmp, num);
	return 0;
}


int cudaTraubMilesParaToGPU(void *pCPU, void *pGPU, int num)
{
	TraubMilesData *pC = (TraubMilesData*)pCPU;
	TraubMilesData *pG = (TraubMilesData*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pGNa, pC->pGNa, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pENa, pC->pENa, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pGK, pC->pGK, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pEK, pC->pEK, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pGl, pC->pGl, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pEl, pC->pEl, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pC, pC->pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV, pC->pV, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pM, pC->pM, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pH, pC->pH, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pN, pC->pN, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pDecayE, pC->pDecayE, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pDecayI, pC->pDecayI, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pEE, pC->pEE, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pEI, pC->pEI, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}


int cudaTraubMilesParaFromGPU(void *pCPU, void *pGPU, int num)
{
	TraubMilesData *pC = (TraubMilesData*)pCPU;
	TraubMilesData *pG = (TraubMilesData*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pGNa, pG->pGNa, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pENa, pG->pENa, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pGK, pG->pGK, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pEK, pG->pEK, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pGl, pG->pGl, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pEl, pG->pEl, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC, pG->pC, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV, pG->pV, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pM, pG->pM, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pH, pG->pH, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pN, pG->pN, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pDecayE, pG->pDecayE, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pDecayI, pG->pDecayI, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pEE, pG->pEE, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pEI, pG->pEI, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}


int cudaFreeTraubMiles(void *pGPU)
{
	TraubMilesData *tmp = (TraubMilesData*)malloc(sizeof(TraubMilesData)*1);
	memset(tmp, 0, sizeof(TraubMilesData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(TraubMilesData)*1, hipMemcpyDeviceToHost));
	cudaFreeTraubMilesPara(tmp);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}


int cudaFreeTraubMilesPara(void *pGPU)
{
	TraubMilesData *p = (TraubMilesData*)pGPU;
	hipFree(p->pGNa);
	p->pGNa = NULL;
	hipFree(p->pENa);
	p->pENa = NULL;
	hipFree(p->pGK);
	p->pGK = NULL;
	hipFree(p->pEK);
	p->pEK = NULL;
	hipFree(p->pGl);
	p->pGl = NULL;
	hipFree(p->pEl);
	p->pEl = NULL;
	hipFree(p->pC);
	p->pC = NULL;
	hipFree(p->pV);
	p->pV = NULL;
	hipFree(p->pM);
	p->pM = NULL;
	hipFree(p->pH);
	p->pH = NULL;
	hipFree(p->pN);
	p->pN = NULL;
	hipFree(p->pDecayE);
	p->pDecayE = NULL;
	hipFree(p->pDecayI);
	p->pDecayI = NULL;
	hipFree(p->pEE);
	p->pEE = NULL;
	hipFree(p->pEI);
	p->pEI = NULL;

	return 0;
}


