#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "STDPData.h"

void *cudaMallocSTDP()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(STDPData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(STDPData)*1));
	return ret;
}

void *cudaAllocSTDP(void *pCPU, int num)
{
	void *ret = cudaMallocSTDP();
	void *tmp = cudaAllocSTDPPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(STDPData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocSTDPPara(void *pCPU, int num)
{
	STDPData *p = (STDPData*)pCPU;
	STDPData *ret = (STDPData*)malloc(sizeof(STDPData)*1);
	memset(ret, 0, sizeof(STDPData)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pDst), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pDst, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pDst, p->pDst, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pLastUpdate), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pLastUpdate, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pLastUpdate, p->pLastUpdate, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pWeight), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pWeight, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pWeight, p->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pAPre), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pAPre, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pAPre, p->pAPre, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pAPost), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pAPost, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pAPost, p->pAPost, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pDPre), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pDPre, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pDPre, p->pDPre, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pDPost), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pDPost, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pDPost, p->pDPost, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pTauPre), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pTauPre, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pTauPre, p->pTauPre, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pTauPost), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pTauPost, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pTauPost, p->pTauPost, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaFetchSTDP(void *pCPU, void *pGPU, int num)
{
	STDPData *pTmp = (STDPData*)malloc(sizeof(STDPData)*1);
	memset(pTmp, 0, sizeof(STDPData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(STDPData)*1, hipMemcpyDeviceToHost));

	cudaSTDPParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaSTDPParaToGPU(void *pCPU, void *pGPU, int num)
{
	STDPData *pC = (STDPData*)pCPU;
	STDPData *pG = (STDPData*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pDst, pC->pDst, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pLastUpdate, pC->pLastUpdate, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG->pWeight, pC->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pAPre, pC->pAPre, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pAPost, pC->pAPost, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pDPre, pC->pDPre, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pDPost, pC->pDPost, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pTauPre, pC->pTauPre, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pTauPost, pC->pTauPost, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaSTDPParaFromGPU(void *pCPU, void *pGPU, int num)
{
	STDPData *pC = (STDPData*)pCPU;
	STDPData *pG = (STDPData*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pDst, pG->pDst, sizeof(int)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pLastUpdate, pG->pLastUpdate, sizeof(int)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pWeight, pG->pWeight, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pAPre, pG->pAPre, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pAPost, pG->pAPost, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pDPre, pG->pDPre, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pDPost, pG->pDPost, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pTauPre, pG->pTauPre, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pTauPost, pG->pTauPost, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeSTDP(void *pGPU)
{
	STDPData *tmp = (STDPData*)malloc(sizeof(STDPData)*1);
	memset(tmp, 0, sizeof(STDPData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(STDPData)*1, hipMemcpyDeviceToHost));
	cudaFreeSTDPPara(tmp);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeSTDPPara(void *pGPU)
{
	STDPData *p = (STDPData*)pGPU;
	hipFree(p->pDst);
	p->pDst = NULL;
	hipFree(p->pLastUpdate);
	p->pLastUpdate = NULL;

	hipFree(p->pWeight);
	p->pWeight = NULL;
	hipFree(p->pAPre);
	p->pAPre = NULL;
	hipFree(p->pAPost);
	p->pAPost = NULL;
	hipFree(p->pDPre);
	p->pDPre = NULL;
	hipFree(p->pDPost);
	p->pDPost = NULL;
	hipFree(p->pTauPre);
	p->pTauPre = NULL;
	hipFree(p->pTauPost);
	p->pTauPost = NULL;

	return 0;
}

