#include "hip/hip_runtime.h"
#include "runtime.h"
#include "stdp_synapse.h"


__global__ void update_stdp_synapse_gpu(Stdp_synapse *data, int num, int start_id, int t)
{
	for (int delta_t=MIN_DELAY; delta_t<=MAX_DELAY; delta_t++) {
		int block_idx = blockIdx.x;
		int delay_idx = (t + MAX_DELAY + 1 - delta_t) % ( MAX_DELAY + 1);
		int fired_size = g_fired_table_sizes[delay_idx];
		int num_per_block = (fired_size - 1) / gridDim.x + 1;
		int block_nums_minus_1 = (fired_size - 1) / num_per_block;
		int fired_size_block = 0;
		if (block_idx == block_nums_minus_1) {
		fired_size_block = fired_size - block_idx * num_per_block;
		} else if (block_idx < block_nums_minus_1) {
		fired_size_block = num_per_block;
		} else {
		fired_size_block = 0;
		}
		for (int idx = 0; idx < fired_size_block; idx++) {
			int nid = g_fired_table[delay_idx * FIRED_TABLE_SIZE + (block_idx) * num_per_block + idx];
			int start_loc = g_connection_stdp_synapse->delay_start[delta_t - MIN_DELAY + nid * (MAX_DELAY - MIN_DELAY + 1)];
			int synapse_num = g_connection_stdp_synapse->delay_num[delta_t - MIN_DELAY + nid * (MAX_DELAY - MIN_DELAY + 1)];
			for (int j=threadIdx.x; j < synapse_num; j += blockDim.x) {
				int sid = j+start_loc;
				double weight = data->p_weight[sid];
				if (weight >= 0) {
					atomicAdd(&(g_i_exec[data->p_dst[sid]]), weight);
				} else {
					atomicAdd(&(g_i_inh[data->p_dst[sid]]), weight);
				}

				data->p_apre[sid] *= exp((data->p_last_update[sid] - t) / (data->p_tau_pre[sid]));
				data->p_apost[sid] *= exp((data->p_last_update[sid] - t) / (data->p_tau_post[sid]));

				data->p_apre[sid] += data->p_d_apre[sid];
				data->p_weight[sid] = _clip(weight + data->p_apost[sid], G_MIN, G_MAX);
				data->p_last_update[sid] = t;
			}
		}
		__syncthreads();
	}
}

__global__ void learn_stdp_synapse_post(Stdp_synapse * data, int num, int start_id, int t)
{
	int block_idx = blockIdx.x;
	int delay_idx = t%(MAX_DELAY+1);
	int fired_size = g_fired_table_sizes[delay_idx];
	int num_per_block = (fired_size - 1) / gridDim.x + 1;
	int block_nums_minus_1 = (fired_size - 1) / num_per_block;
	int fired_size_block = 0;
	if (block_idx == block_nums_minus_1) {
		fired_size_block = fired_size - block_idx * num_per_block;
	} else if (block_idx < block_nums_minus_1) {
		fired_size_block = num_per_block;
	} else {
		fired_size_block = 0;
	}

	for (int idx = 0; idx < fired_size_block; idx++) {
		int nid = g_fired_table[delay_idx * FIRED_TABLE_SIZE + (block_idx) * num_per_block + idx];
		int start_loc = g_connection_stdp_synapse->rev_delay_start[nid];
		int synapse_num = g_connection_stdp_synapse->rev_delay_num[nid];
		for (int j=threadIdx.x; j<synapse_num; j+=blockDim.x) {
			int sid = g_connection_stdp_synapse->rev_map2sid[j+start_loc];
			data->p_apre[sid] *= exp((data->p_last_update[sid] - t) / (data->p_tau_pre[sid]));
			data->p_apost[sid] *= exp((data->p_last_update[sid] - t) / (data->p_tau_post[sid]));
			data->p_apost[sid] += data->p_d_apost[sid];
			data->p_weight[sid] = _clip(data->p_weight[sid] + data->p_apre[sid], G_MIN, G_MAX);
			data->p_last_update[sid] = t;
		}
	}
	__syncthreads();
}

void update_stdp_synapse(Stdp_synapse *data, int num, int start_id, int t)
{
	update_stdp_synapse_gpu<<<STDP_SYNAPSE_GRID_SIZE, STDP_SYNAPSE_BLOCK_SIZE>>>((Stdp_synapse*)data, num, start_id, t);
	learn_stdp_synapse_post<<<STDP_SYNAPSE_GRID_SIZE, STDP_SYNAPSE_BLOCK_SIZE>>>((Stdp_synapse*)data, num, start_id, t);
}

