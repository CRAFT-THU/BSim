/* This program is writen by qp09.
 * usually just for fun.
 * Sat March 12 2016
 */



int addCrossNeurons(int *ids, int num, int time)
{
	add_cross_neuron<<<(num+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(ids, num, time);
	return 0;
}


int cudaDeliverNeurons(int *idx2index, int *crossnode_index2idx, int *global_cross_data, int *fired_n_num, int node_num, int neuron_num, int time)
{
	deliver_neurons<<<(neuron_num + MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(idx2index, crossnode_index2idx, global_cross_data, fired_n_num, node_num, time);

	return 0;
}

BlockSize * getBlockSize(int nSize, int sSize)
{
	BlockSize *ret = (BlockSize*)malloc(sizeof(BlockSize)*TYPESIZE);
	memset(ret, 0, sizeof(BlockSize)*TYPESIZE);

	hipOccupancyMaxPotentialBlockSize(&(ret[Array].minGridSize), &(ret[Array].blockSize), update_array_neuron, 0, nSize); 
	ret[Array].gridSize = (upzero_else_set_one(nSize) + (ret[Array].blockSize) - 1) / (ret[Array].blockSize);

	hipOccupancyMaxPotentialBlockSize(&(ret[LIF].minGridSize), &(ret[LIF].blockSize), update_lif_neuron, 0, nSize); 
	ret[LIF].gridSize = (upzero_else_set_one(nSize) + (ret[LIF].blockSize) - 1) / (ret[LIF].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Constant].minGridSize), &(ret[Constant].blockSize), update_constant_neuron, 0, nSize); 
	// ret[Constant].gridSize = (upzero_else_set_one(nSize) + (ret[Constant].blockSize) - 1) / (ret[Constant].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Poisson].minGridSize), &(ret[Poisson].blockSize), update_poisson_neuron, 0, nSize); 
	// ret[Poisson].gridSize = (upzero_else_set_one(nSize) + (ret[Poisson].blockSize) - 1) / (ret[Poisson].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Decide].minGridSize), &(ret[Decide].blockSize), update_max_neuron, 0, nSize); 
	// ret[Decide].gridSize = (upzero_else_set_one(nSize) + (ret[Decide].blockSize) - 1) / (ret[Decide].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[FFT].minGridSize), &(ret[FFT].blockSize), update_fft_neuron, 0, nSize); 
	// ret[FFT].gridSize = (upzero_else_set_one(nSize) + (ret[FFT].blockSize) - 1) / (ret[FFT].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Mem].minGridSize), &(ret[Mem].blockSize), update_mem_neuron, 0, nSize); 
	// ret[Mem].gridSize = (upzero_else_set_one(nSize) + (ret[Mem].blockSize) - 1) / (ret[Mem].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Max].minGridSize), &(ret[Max].blockSize), update_max_neuron, 0, nSize); 
	// ret[Max].gridSize = (upzero_else_set_one(nSize) + (ret[Max].blockSize) - 1) / (ret[Max].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[TJ].minGridSize), &(ret[TJ].blockSize), update_tj_neuron, 0, nSize); 
	// ret[TJ].gridSize = (upzero_else_set_one(nSize) + (ret[TJ].blockSize) - 1) / (ret[TJ].blockSize);

	// hipOccupancyMaxPotentialBlockSize(&(ret[Static].minGridSize), &(ret[Static].blockSize), update_static_hit, 0, sSize); 
	ret[Static].blockSize = 128;
	ret[Static].gridSize = (upzero_else_set_one(nSize) + (ret[Static].blockSize) - 1) / (ret[Static].blockSize);

	return ret;
}
