#include "hip/hip_runtime.h"

#include "GLIFENeurons.h"

#include "../../gpu_utils/runtime.h"

// #include "GLIFENeurons.h"
#include "lif.h"


__global__ void find_life_neuron(GLIFENeurons *d_neurons, int num, int start_id)
{
	__shared__ int active_table_t[MAXBLOCKSIZE];
	__shared__ volatile int active_cnt;

	if (threadIdx.x == 0) {
		active_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		//bool actived = false;
		int test_loc = 0;
		bool actived = d_neurons->p_refrac_step[idx] <= 0;
		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			gNeuronInput_I[start_id + idx] = 0;
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		__syncthreads();

		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		}
		__syncthreads();

		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (active_cnt > 0) {
			commit2globalTable(active_table_t, active_cnt, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
	}
}

__global__ void update_life_neuron(GLIFENeurons *d_neurons, int num, int start_id, int time)
{
	int currentIdx = time % (MAX_DELAY+1);
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;
	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int nid = gActiveTable[idx];
		int gnid = start_id + nid; 

		//real I = sqrtf(d_neurons->p_CE[nid]) * d_neurons->p_i_E[nid] + sqrtf(d_neurons->p_CI[nid]) * d_neurons->p_i_I[nid] + d_neurons->p_i_tmp[nid];

		//real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
		//d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;

		d_neurons->p_vm[nid] = d_neurons->p_Cm[nid] * d_neurons->p_vm[nid] + d_neurons->p_v_tmp[nid] + d_neurons->p_i_E[nid] * d_neurons->p_C_E[nid] + d_neurons->p_i_I[nid] * d_neurons->p_C_I[nid];

		//d_neurons->p_i_syn[nid] = 0;

		d_neurons->p_i_E[nid] *= d_neurons->p_CE[nid];
		d_neurons->p_i_I[nid] *= d_neurons->p_CI[nid];

		fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

		gFireCount[gnid] += fired;

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}

			d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
			d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
		} else {
			gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];
			d_neurons->p_i_E[nid] += gNeuronInput[gnid];
			d_neurons->p_i_I[nid] += gNeuronInput_I[gnid];
		}

		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;

		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		__syncthreads();

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[currentIdx], gFiredTableCap*currentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

	}
	//__syncthreads();
	//if (threadIdx.x == 0 && blockIdx.x == 0) {
	//	gActiveTableSize = 0;
	//}
}

__global__ void update_all_life_neuron(GLIFENeurons *d_neurons, int num, int start_id, int time)
{
	int currentIdx = time % (MAX_DELAY+1);
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}

	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int nid = idx;
		int gnid = start_id + idx; 
		bool actived = d_neurons->p_refrac_step[idx] <= 0;

		if (actived) {
			//real I = sqrtf(d_neurons->p_CE[nid]) * d_neurons->p_i_E[nid] + sqrtf(d_neurons->p_CI[nid]) * d_neurons->p_i_I[nid] + d_neurons->p_i_tmp[nid];

			//real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
			//d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
			//d_neurons->p_i_syn[nid] = 0;

			d_neurons->p_vm[nid] = d_neurons->p_Cm[nid] * d_neurons->p_vm[nid] + d_neurons->p_v_tmp[nid] + d_neurons->p_i_E[nid] * d_neurons->p_C_E[nid] + d_neurons->p_i_I[nid] * d_neurons->p_C_I[nid];

			gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];

			d_neurons->p_i_E[nid] *= d_neurons->p_CE[nid];
			d_neurons->p_i_I[nid] *= d_neurons->p_CI[nid];

			fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

			gFireCount[gnid] += fired;

			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = gnid;
					fired = false;
				}

				d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
				d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
			} else {
				d_neurons->p_i_E[nid] += gNeuronInput[gnid];
				d_neurons->p_i_I[nid] += gNeuronInput_I[gnid];
			}

			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[currentIdx], gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}

			__syncthreads();

			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = gnid;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[currentIdx], gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();

			if (fire_cnt > 0) {
				commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[currentIdx], gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
		} else {
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;
	}
	__syncthreads();
}

__global__ void update_dense_life_neuron(GLIFENeurons *d_neurons, int num, int start_id, int time)
{
	//__shared__ int fire_table_t[MAXBLOCKSIZE];
	//__shared__ volatile int fire_cnt;

	//if (threadIdx.x == 0) {
	//	fire_cnt = 0;
	//}
	//__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int currentIdx = time % (MAX_DELAY+1);
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		//bool fired = false;
		//int test_loc = 0;

		int nid = idx;
		int gnid = start_id + idx; 
		bool actived = d_neurons->p_refrac_step[idx] <= 0;

		if (actived) {
			d_neurons->p_vm[nid] = d_neurons->p_Cm[nid] * d_neurons->p_vm[nid] + d_neurons->p_v_tmp[nid] + d_neurons->p_i_E[nid] * d_neurons->p_C_E[nid] + d_neurons->p_i_I[nid] * d_neurons->p_C_I[nid];

			d_neurons->p_i_E[nid] *= d_neurons->p_CE[nid];
			d_neurons->p_i_I[nid] *= d_neurons->p_CI[nid];

			bool fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

			gFiredTable[gFiredTableCap*currentIdx + gnid] = fired;

			gFireCount[gnid] += fired;

			if (fired) {
				d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
				d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];

			} else {
				gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];
				d_neurons->p_i_E[nid] += gNeuronInput[gnid];
				d_neurons->p_i_I[nid] += gNeuronInput_I[gnid];
				//real input = 0, input_I = 0;
				//for (int i=d_neurons->p_start_E[nid]; i<d_neurons->p_start_I[nid]; i++) {
				//	input += gNeuronInput[i];
				//}
				//for (int i=d_neurons->p_start_I[nid]; i<d_neurons->p_end[nid]; i++) {
				//	input_I += gNeuronInput[i];
				//}
				//d_neurons->p_i_E[nid] += input;
				//d_neurons->p_i_I[nid] += input_I;
				//gXInput[gnid] += input + input_I;
			}

		} else {
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
			gFiredTable[gFiredTableCap*currentIdx + gnid] = 0;
		}
		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;
	}
	__syncthreads();
}

int cudaUpdateLIFE(void *data, int num, int start_id, int time, BlockSize *pSize)
{
	find_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFENeurons*)data, num, start_id);
	update_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFENeurons*)data, num, start_id, time);
	//update_dense_life_neuron<<<pSize->gridSize, pSize->blockSize>>>((GLIFENeurons*)data, num, start_id);

	return 0;
}
