/* This program is writen by qp09.
 * usually just for fun.
 * Mon December 14 2015
 */


#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GFFTNeurons.h"

int cudaAllocFFT(void *pCpu, void *pGpu, int num)
{
	GFFTNeurons *pGpuNeurons = (GFFTNeurons*)pGpu;
	GFFTNeurons *p = (GFFTNeurons*)pCpu;

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_input), sizeof(hipfftComplex)*num/2));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_input, p->p_input, sizeof(hipfftComplex)*num/2, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_res), sizeof(hipfftComplex)*num/2));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_res, p->p_res, sizeof(hipfftComplex)*num/2, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGpuNeurons->p_fire_count), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(pGpuNeurons->p_fire_count, p->p_fire_count, sizeof(int)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaFreeFFT(void *pGpu)
{
	GFFTNeurons *pGpuNeurons = (GFFTNeurons*)pGpu;

	checkCudaErrors(hipFree(pGpuNeurons->p_input));
	checkCudaErrors(hipFree(pGpuNeurons->p_res));
	checkCudaErrors(hipFree(pGpuNeurons->p_fire_count));

	return 0;
}
