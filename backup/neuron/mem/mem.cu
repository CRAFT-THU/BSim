#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

// #include "GMemNeurons.h"
#include "mem.h"


__global__ void update_mem_neuron(GMemNeurons *d_neurons, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int time)
{
	int currentIdx = time % (MAX_DELAY+1);
	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		int gnid = idx + start_id;


		//fired = (time * d_neurons->p_fire_rate[idx]) > (d_neurons->p_fire_count[idx]);
		d_neurons->p_fire_rate[idx] = d_neurons->p_fire_rate[idx] + currentE[gnid] + currentI[gnid];
		fired = d_neurons->p_fire_rate[idx] > d_neurons->p_fire_count[idx];
		gFireCount[gnid] += fired;

		//d_neurons->p_fire_rate[idx] = (d_neurons->p_fire_rate[idx] * (time) + currentE[gnid] + currentI[gnid])/(time + 1);

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAX_BLOCK_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					d_neurons->p_fire_count[idx] = d_neurons->p_fire_count[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAX_BLOCK_SIZE) {
				commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &(firedTableSizes[currentIdx]), gFiredTableCap*currentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}

		gXInput[gnid] += currentE[gnid] + currentI[gnid];

		currentE[gnid] = 0;
		currentI[gnid] = 0;
		__syncthreads();
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, firedTable, &(firedTableSizes[currentIdx]), gFiredTableCap*currentIdx);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

void cudaUpdateMem(void *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int time, BlockSize *pSize)
{
	update_mem_neuron<<<pSize->gridSize, pSize->blockSize>>>((GMemNeurons*)data, currentE, currentI, firedTable, firedTableSizes, num, start_id, time);
}

