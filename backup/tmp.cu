#include "hip/hip_runtime.h"

__global__ void find_life_neuron(GLIFENeurons *d_neurons, int num, int start_id)
{
	__shared__ int active_table_t[MAXBLOCKSIZE];
	__shared__ volatile int active_cnt;

	if (threadIdx.x == 0) {
		active_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool actived = false;
		int test_loc = 0;

		actived = d_neurons->p_refrac_step[idx] <= 0;

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			gNeuronInput_I[start_id + idx] = 0;
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		__syncthreads();

		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		}
		__syncthreads();
		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
	}
	
	if (active_cnt > 0) {
		commit2globalTable(active_table_t, active_cnt, gActiveTable, &gActiveTableSize, 0);
		if (threadIdx.x == 0) {
			active_cnt = 0;
		}
	}
	__syncthreads();

}

__device__ void reset_life_neuron(GLIFENeurons *d_neurons, int *shared_buf, volatile int size, int start_id) 
{
	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		int nid = shared_buf[idx] - start_id;
		d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
		d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
	}
}

__global__ void update_life_neuron(GLIFENeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int nid = gActiveTable[idx];
		int gnid = start_id + nid; 
		d_neurons->p_i_E[nid] *= d_neurons->p_CE[nid]
		d_neurons->p_i_I[nid] *= d_neurons->p_CI[nid]

		real I = d_neurons->p_i_E[nid] + d_neurons->p_i_I[nid] + d_neurons->p_i_tmp[nid];
 
		//real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
		d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
		//d_neurons->p_i_syn[nid] = 0;
		gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];

		fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}

			d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
			d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
		} else {
			d_neurons->p_i_E[nid] += (sqrtf(d_neurons->p_CE[nid]) * gNeuronInput[gnid])
			d_neurons->p_i_I[nid] += (sqrtf(d_neurons->p_CI[nid]) * gNeuronInput_I[gnid])
		}

		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		__syncthreads();

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;
	}
	__syncthreads();
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		gActiveTableSize = 0;
	}
}

__global__ void update_all_life_neuron(GLIFNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		int gnid = start_id + idx; 
		bool actived = d_neurons->p_refrac_step[idx] <= 0;
		if (actived) {
			d_neurons->p_i_E[nid] *= d_neurons->p_CE[nid]
				d_neurons->p_i_I[nid] *= d_neurons->p_CI[nid]

				real I = d_neurons->p_i_E[nid] + d_neurons->p_i_I[nid] + d_neurons->p_i_tmp[nid];

			//real I = gNeuronInput[gnid] + d_neurons->p_i_tmp[nid];
			d_neurons->p_vm[nid] = d_neurons->p_vm[nid] * d_neurons->p_C1[nid] + d_neurons->p_C2[nid] * I;
			//d_neurons->p_i_syn[nid] = 0;
			gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];

			fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];

			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = gnid;
					fired = false;
				}

				d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
				d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
			} else {
				d_neurons->p_i_E[nid] += (sqrtf(d_neurons->p_CE[nid]) * gNeuronInput[gnid])
					d_neurons->p_i_I[nid] += (sqrtf(d_neurons->p_CI[nid]) * gNeuronInput_I[gnid])
			}

			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}

			__syncthreads();

			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAXBLOCKSIZE) {
					fire_table_t[test_loc] = gnid;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAXBLOCKSIZE) {
				commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();

			if (fire_cnt > 0) {
				commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
		} else {
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;
	}
	__syncthreads();
}
