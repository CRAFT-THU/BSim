#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

// #include "GArrayNeurons.h"
#include "array.h"


__global__ void update_array_neuron(GArrayNeurons *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int time)
{
	int currentIdx = time % (MAX_DELAY+1);
	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile unsigned int fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		bool fired = false;
		int test_loc = 0;

		fired = (data->p_start[idx] < data->p_end[idx]) &&  (time >= data->p_fire_time[data->p_start[idx]]);
		gFireCount[start_id + idx] += fired;

		for (int i=0; i<2; i++) {
			if (fired) {
				test_loc = atomicAdd((int*)&fire_cnt, 1);
				if (test_loc < MAX_BLOCK_SIZE) {
					fire_table_t[test_loc] = start_id + idx;
					data->p_start[idx] = data->p_start[idx] + 1;
					fired = false;
				}
			}
			__syncthreads();
			if (fire_cnt >= MAX_BLOCK_SIZE) {
				commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &(firedTableSizes[currentIdx]), gFiredTableCap*currentIdx);
				//advance_array_neuron(data, fire_table_t, MAX_BLOCK_SIZE, start_id);
				if (threadIdx.x == 0) {
					fire_cnt = 0;
				}
			}
			__syncthreads();
		}
	}
	__syncthreads();

	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, firedTable, &(firedTableSizes[currentIdx]), gFiredTableCap*currentIdx);
		//advance_array_neuron(data, fire_table_t, fire_cnt, start_id);
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
}

void cudaUpdateArray(void *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int start_id, int time, BlockSize *pSize)
{
	update_array_neuron<<<pSize->gridSize, pSize->blockSize>>>((GArrayNeurons*)data, currentE, currentI, firedTable, firedTableSizes, num, start_id, time);
}

