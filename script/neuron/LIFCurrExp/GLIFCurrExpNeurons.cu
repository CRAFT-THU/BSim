
#include "../third_party/cuda/hip/hip_runtime_api.h"
#include "../gpu_utils/mem_op.h"
#include "GLIFCurrExpNeurons.h"

int cudaAllocLIFCurrExp(void *pCpu, void *pGpu, int num)
{
	GLIFCurrExpNeurons *pGpuNeurons = (GLIFCurrExpNeurons*)pGpu;
	GLIFCurrExpNeurons *p = (GLIFCurrExpNeurons*)pCpu;
	pGpuNeurons->p_i_offset = copyToGPU<real>(p->p_i_offset, num);
	pGpuNeurons->p_i_injection = copyToGPU<real>(p->p_i_injection, num);
	pGpuNeurons->p_cm = copyToGPU<real>(p->p_cm, num);
	pGpuNeurons->p_tau_m = copyToGPU<real>(p->p_tau_m, num);
	pGpuNeurons->p_tau_e = copyToGPU<real>(p->p_tau_e, num);
	pGpuNeurons->p_tau_i = copyToGPU<real>(p->p_tau_i, num);
	pGpuNeurons->p_vm = copyToGPU<real>(p->p_vm, num);
	pGpuNeurons->p_v_thresh = copyToGPU<real>(p->p_v_thresh, num);
	pGpuNeurons->p_v_reset = copyToGPU<real>(p->p_v_reset, num);
	pGpuNeurons->p_v_reset = copyToGPU<real>(p->p_v_reset, num);
	pGpuNeurons->p_i_E = copyToGPU<real>(p->p_i_E, num);
	pGpuNeurons->p_i_I = copyToGPU<real>(p->p_i_I, num);
	pGpuNeurons->p_refrac_time = copyToGPU<int>(p->p_refrac_time, num);
	pGpuNeurons->p_refrac_step = copyToGPU<int>(p->p_refrac_step, num);
	return 0;
}

int cudaFreeLIFCurrExp(void *pGpu)
{
	GLIFCurrExpNeurons *pGpuNeurons = (GLIFCurrExpNeurons*)pGpu;
	gpuFree(pGpuNeurons->p_i_offset);
	gpuFree(pGpuNeurons->p_i_injection);
	gpuFree(pGpuNeurons->p_cm);
	gpuFree(pGpuNeurons->p_tau_m);
	gpuFree(pGpuNeurons->p_tau_e);
	gpuFree(pGpuNeurons->p_tau_i);
	gpuFree(pGpuNeurons->p_vm);
	gpuFree(pGpuNeurons->p_v_thresh);
	gpuFree(pGpuNeurons->p_v_reset);
	gpuFree(pGpuNeurons->p_v_reset);
	gpuFree(pGpuNeurons->p_i_E);
	gpuFree(pGpuNeurons->p_i_I);
	gpuFree(pGpuNeurons->p_refrac_time);
	gpuFree(pGpuNeurons->p_refrac_step);
	return 0;
}

