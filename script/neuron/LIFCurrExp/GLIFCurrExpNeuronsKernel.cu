#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

#include "GLIFCurrExpNeuronsKernel.h"

__global void find_lifcurrexp_neurons(GLIFCurrExpNeurons *d_neurons, int num, int start_id)
{
	__shared__ int active_table_t[MAXBLOCKSIZE];
	__shared__ volatile int active_cnt;

	if (threadIdx.x == 0) {
		active_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		int test_loc = 0;
		
		bool actived = d_neurons->p_refrac_step[idx] <= 0;
		
		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			gNeuronInput_I[start_id + idx] = 0;
			d_neurons->p_refrac_step[idx] = d_neurons->p_refrac_step[idx] - 1;
		}
		__syncthreads();
		
		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
		
		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		}
		__syncthreads();
		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
		if (active_cnt > 0) {
			commit2globalTable(active_table_t, active_cnt, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
	}
}


__global void update_lifcurrexp_neurons(GLIFCurrExpNeurons *d_neurons, int num, int start_id)
{
	__shared__ int fire_table_t[MAXBLOCKSIZE];
	__shared__ volatile int fire_cnt;
	
	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	__syncthreads();
	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		
		int nid = gActiveTable[idx];
		int gnid = start_id + nid; 
		
		d_neurons->p_vm[nid] = d_neurons->p_Cm[nid] * d_neurons->p_vm[nid] + d_neurons->p_v_tmp[nid] + d_neurons->p_i_E[nid] * d_neurons->p_C_E[nid] + d_neurons->p_i_I[nid] * d_neurons->p_C_I[nid];
		
		d_neurons->p_i_E[nid] *= d_neurons->p_CE[nid];
		d_neurons->p_i_I[nid] *= d_neurons->p_CI[nid];
		
		fired = d_neurons->p_vm[nid] >= d_neurons->p_v_thresh[nid];
		
		gFireCount[gnid] += fired;
		
		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
			
			d_neurons->p_refrac_step[nid] = d_neurons->p_refrac_time[nid] - 1;
			d_neurons->p_vm[nid] = d_neurons->p_v_reset[nid];
		} else {
			gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];
			d_neurons->p_i_E[nid] += gNeuronInput[gnid];
			d_neurons->p_i_I[nid] += gNeuronInput_I[gnid];
		}
		
		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;
		__syncthreads();
		
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
		
		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
		
		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		
	}
}
