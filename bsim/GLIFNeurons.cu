#include <stdlib.h>
#include <string.h>
#include "../../third_party/cuda/hip/hip_runtime_api.h"
#include "GLIFNeurons.h"

void *cudaAllocLIF(void *pCPU, int num)
{
	void *ret = NULL;
	GLIFNeurons *p = (GLIFNeurons*)pCPU;
	GLIFNeurons * tmp = (GLIFNeurons*)malloc(sizeof(GLIFNeurons)*1);
	memset(tmp, 0, sizeof(GLIFNeurons)*1);

	checkCudaErrors(hipMalloc((void**)&(tmp->pRefracStep), sizeof(int)*num));
	checkCudaErrors(hipMemset(tmp->pRefracStep, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(tmp->pRefracStep, p->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pRefracTime), sizeof(int)*num));
	checkCudaErrors(hipMemset(tmp->pRefracTime, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(tmp->pRefracTime, p->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(tmp->pVm), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pVm, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pVm, p->pVm, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pCi), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pCi, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pCi, p->pCi, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pCe), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pCe, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pCe, p->pCe, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pC_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pC_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pC_i, p->pC_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pC_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pC_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pC_e, p->pC_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pVtmp), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pVtmp, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pVtmp, p->pVtmp, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pIi), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pIi, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pIi, p->pIi, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pIe), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pIe, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pIe, p->pIe, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pVthresh), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pVthresh, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pVthresh, p->pVthresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pVreset), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pVreset, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pVreset, p->pVreset, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(tmp->pCm), sizeof(real)*num));
	checkCudaErrors(hipMemset(tmp->pCm, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(tmp->pCm, p->pCm, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(GLIFNeurons)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(GLIFNeurons)*1));
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(GLIFNeurons)*1, hipMemcpyHostToDevice));
	free(tmp);
	{} = NULL;
	return ret;
}

void *cudaLIFToGPU(void *pCPU, void *pGPU, int num)
{
	GLIFNeurons *pC = (GLIFNeurons*)pCPU;
	LIFData *pG = (LIFData*)pGPU;

	checkCudaErrors(hipMemcpy(pG, pC, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG, pC, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

void *cudaFreeLIF(void *pGPU)
{
	hipFree(pGPU);
	{} = NULL;
	return 0;
}

