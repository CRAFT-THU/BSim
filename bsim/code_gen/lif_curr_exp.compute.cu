#include "hip/hip_runtime.h"

#include "runtime.h"
#include "lif_curr_exp.h"


__global__ void find_lif_curr_exp_gpu(Lif_curr_exp *data, int num, int start_id)
{
	__shared__ int active_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile int active_cnt;

	if (threadIdx.x == 0 && blockIdx.x == 0) {
		active_cnt = 0;
		g_active_lif_curr_exp_table_size = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		int test_loc = 0;
		bool actived = data->p_refract_step[idx] <= 0;
		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		} else {
			g_i_exec[start_id + idx] = 0;
			g_i_inh[start_id + idx] = 0;
			data->p_refract_time[idx] = data->p_refract_time[idx] - 1;
		}
		__syncthreads();

		if (active_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(active_table_t, MAX_BLOCK_SIZE, g_active_lif_curr_exp_table, &g_active_lif_curr_exp_table_size, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		}
		__syncthreads();

		if (active_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(active_table_t, MAX_BLOCK_SIZE, g_active_lif_curr_exp_table, &g_active_lif_curr_exp_table_size, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (active_cnt > 0) {
			commit2globalTable(active_table_t, active_cnt, g_active_lif_curr_exp_table, &g_active_lif_curr_exp_table_size, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
	}
}

__global__ void update_lif_curr_exp_gpu(Lif_curr_exp *data, int num, int start_id, int t)
{
	int delay_idx = t % (MAX_DELAY+1);
	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile int fire_cnt;
	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		g_fired_table_sizes[delay_idx] = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < g_active_lif_curr_exp_table_size; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		int nid = g_active_lif_curr_exp_table[idx];
		int gnid = start_id + nid; 

		data->p_v[nid] = data->p_C[nid] * data->p_v[nid] + data->p_v_tmp[nid] + data->p_i_exec[nid] * data->p_C_exec[nid] + data->p_i_inh[nid] * data->p_C_inh[nid];
		data->p_i_exec[nid] *= data->p_Cexec[nid];
		data->p_i_inh[nid] *= data->p_Cinh[nid];

		fired = data->p_v[nid] >= data->p_v_threshold[nid];
		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
			data->p_refract_step[nid] = data->p_refract_time[nid] - 1;
			data->p_v[nid] = data->p_v_reset[nid];
		} else {
			data->p_i_exec[nid] += g_i_exec[gnid];
			data->p_i_inh[nid] += g_i_inh[gnid];
		}

		g_i_exec[gnid] = 0;
		g_i_inh[gnid] = 0;
		__syncthreads();

		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, g_fired_table, &g_fired_table_sizes[delay_idx], FIRED_TABLE_SIZE*delay_idx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, g_fired_table, &g_fired_table_sizes[delay_idx], FIRED_TABLE_SIZE*delay_idx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
		
		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, g_fired_table, &g_fired_table_sizes[delay_idx], FIRED_TABLE_SIZE*delay_idx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		
	}
}

void update_lif_curr_exp(Lif_curr_exp *data, int num, int start_id, int t)
{
	find_lif_curr_exp_gpu<<<LIF_CURR_EXP_GRID_SIZE, LIF_CURR_EXP_BLOCK_SIZE>>>((Lif_curr_exp*)data, num, start_id);
	update_lif_curr_exp_gpu<<<LIF_CURR_EXP_GRID_SIZE, LIF_CURR_EXP_BLOCK_SIZE>>>((Lif_curr_exp*)data, num, start_id, t);
}

