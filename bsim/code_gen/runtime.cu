#include <stdio.h>
#include "../c_code/hip/hip_runtime_api.h"
#include "runtime.h"


__device__ int * g_fired_table;
__device__ int * g_fired_table_sizes;
__device__ int * g_active_lif_curr_exp_table;
__device__ int g_active_lif_curr_exp_table_size;
__device__ CConnection * g_connection_static;
__device__ double * i_exec;
__device__ double * g_i_exec;
__device__ double * g_i_inh;
__device__ double * i_inh;


void **init_runtime(CConnection ** connections)
{
	int zero = 0;
	int *p_int = NULL;
	double *p_double = NULL;

	void **ret = static_cast<void**>(malloc(sizeof(void*) * 2));

	checkCudaErrors(hipMalloc((void**)&(p_int), sizeof(int)*2));
	checkCudaErrors(hipMemset(p_int, 0, sizeof(int)*2));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_fired_table_sizes), &p_int, sizeof(p_int)));
	ret[0] = static_cast<void*>(p_int);
	checkCudaErrors(hipMalloc((void**)&(p_int), sizeof(int)*60));
	checkCudaErrors(hipMemset(p_int, 0, sizeof(int)*60));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_fired_table), &p_int, sizeof(p_int)));
	ret[1] = static_cast<void*>(p_int);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_active_lif_curr_exp_table_size), &zero, sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&(p_int), sizeof(int)*30));
	checkCudaErrors(hipMemset(p_int, 0, sizeof(int)*30));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_active_lif_curr_exp_table), &p_int, sizeof(p_int)));


	checkCudaErrors(hipMalloc((void**)&(p_double), sizeof(double)*30));
	checkCudaErrors(hipMemset(p_double, 0, sizeof(double)*30));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(i_exec), &p_double, sizeof(p_double)));
	checkCudaErrors(hipMalloc((void**)&(p_double), sizeof(double)*30));
	checkCudaErrors(hipMemset(p_double, 0, sizeof(double)*30));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_i_exec), &p_double, sizeof(p_double)));
	checkCudaErrors(hipMalloc((void**)&(p_double), sizeof(double)*30));
	checkCudaErrors(hipMemset(p_double, 0, sizeof(double)*30));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_i_inh), &p_double, sizeof(p_double)));
	checkCudaErrors(hipMalloc((void**)&(p_double), sizeof(double)*30));
	checkCudaErrors(hipMemset(p_double, 0, sizeof(double)*30));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(i_inh), &p_double, sizeof(p_double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_connection_static), &(connections[0]), sizeof(CConnection*)));
	return ret;
}
