#include "hip/hip_runtime.h"
#include "runtime.h"
#include "static.h"


__device__ double _clip(double a, double min, double max)
{
	if (a < min) {
		return min;
	} else if (a > max) {
		return max;
	} else {
		return a;
	}
}
__global__ void update_static_gpu(Static *data, int num, int start_id, int t)
{
	for (int delta_t=MIN_DELAY; delta_t<=MAX_DELAY; delta_t++) {
		int block_idx = blockIdx.x;
		int delay_idx = (t + MAX_DELAY + 1 - delta_t) % ( MAX_DELAY + 1);
		int fired_size = g_fired_table_sizes[delay_idx];
		int num_per_block = (fired_size - 1) / gridDim.x + 1;
		int block_nums_minus_1 = (fired_size - 1) / num_per_block;
		int fired_size_block = 0;
		if (block_idx == block_nums_minus_1) {
		fired_size_block = fired_size - block_idx * num_per_block;
		} else if (block_idx < block_nums_minus_1) {
		fired_size_block = num_per_block;
		} else {
		fired_size_block = 0;
		}
		for (int idx = 0; idx < fired_size_block; idx++) {
			int nid = g_fired_table[delay_idx * FIRED_TABLE_SIZE + (block_idx) * num_per_block + idx];
			int start_loc = g_connection_static->delay_start[delta_t - MIN_DELAY + nid * (MAX_DELAY - MIN_DELAY + 1)];
			int synapse_num = g_connection_static->delay_num[delta_t - MIN_DELAY + nid * (MAX_DELAY - MIN_DELAY + 1)];
			for (int j=threadIdx.x; j < synapse_num; j += blockDim.x) {
				int sid = j+start_loc;
				double weight = data->p_weight[sid];
				if (weight >= 0) {
					atomicAdd(&(g_i_exec[data->p_dst[sid]]), weight);
				} else {
					atomicAdd(&(g_i_inh[data->p_dst[sid]]), weight);
				}

			}
		}
		__syncthreads();
	}
}

void update_static(Static *data, int num, int start_id, int t)
{
	update_static_gpu<<<STATIC_GRID_SIZE, STATIC_BLOCK_SIZE>>>((Static*)data, num, start_id, t);
}

