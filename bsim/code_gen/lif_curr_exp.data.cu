

#include <stdlib.h>

#include "../c_code/hip/hip_runtime_api.h"
#include "lif_curr_exp.h"


Lif_curr_exp * to_gpu_lif_curr_exp(Lif_curr_exp *cpu, int num)
{
	Lif_curr_exp * gpu = static_cast<Lif_curr_exp*>(malloc(sizeof(Lif_curr_exp)*1));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_refract_time), sizeof(int)*num));
	checkCudaErrors(hipMemset(gpu->p_refract_time, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_refract_time, cpu->p_refract_time, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_refract_step), sizeof(int)*num));
	checkCudaErrors(hipMemset(gpu->p_refract_step, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_refract_step, cpu->p_refract_step, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_Cinh), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_Cinh, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_Cinh, cpu->p_Cinh, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_v_tmp), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_v_tmp, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_v_tmp, cpu->p_v_tmp, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_v_reset), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_v_reset, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_v_reset, cpu->p_v_reset, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_Cexec), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_Cexec, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_Cexec, cpu->p_Cexec, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_C_inh), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_C_inh, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_C_inh, cpu->p_C_inh, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_C), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_C, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_C, cpu->p_C, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_C_exec), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_C_exec, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_C_exec, cpu->p_C_exec, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_v_threshold), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_v_threshold, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_v_threshold, cpu->p_v_threshold, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_i_exec), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_i_exec, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_i_exec, cpu->p_i_exec, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_v), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_v, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_v, cpu->p_v, sizeof(double)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_i_inh), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_i_inh, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_i_inh, cpu->p_i_inh, sizeof(double)*num, hipMemcpyHostToDevice));
	Lif_curr_exp * ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(Lif_curr_exp)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(Lif_curr_exp)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(Lif_curr_exp)*1, hipMemcpyHostToDevice));
	return ret;
}

Lif_curr_exp * from_gpu_lif_curr_exp(Lif_curr_exp *gpu, int num)
{
	Lif_curr_exp * cpu = static_cast<Lif_curr_exp*>(malloc(sizeof(Lif_curr_exp)*1));
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(Lif_curr_exp)*1, hipMemcpyDeviceToHost));
	return cpu;
}

