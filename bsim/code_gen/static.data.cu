

#include <stdlib.h>

#include "../c_code/hip/hip_runtime_api.h"
#include "static.h"


Static * to_gpu_static(Static *cpu, int num)
{
	Static * gpu = static_cast<Static*>(malloc(sizeof(Static)*1));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_dst), sizeof(int)*num));
	checkCudaErrors(hipMemset(gpu->p_dst, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_dst, cpu->p_dst, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_weight), sizeof(double)*num));
	checkCudaErrors(hipMemset(gpu->p_weight, 0, sizeof(double)*num));
	checkCudaErrors(hipMemcpy(gpu->p_weight, cpu->p_weight, sizeof(double)*num, hipMemcpyHostToDevice));
	Static * ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(Static)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(Static)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(Static)*1, hipMemcpyHostToDevice));
	return ret;
}

Static * from_gpu_static(Static *gpu, int num)
{
	Static * cpu = static_cast<Static*>(malloc(sizeof(Static)*1));
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(Static)*1, hipMemcpyDeviceToHost));
	return cpu;
}

