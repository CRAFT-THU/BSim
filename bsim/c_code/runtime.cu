#include "hip/hip_runtime_api.h"
#include "runtime.h"


void init_runtime(CConnection ** connections)
{
	int zero = 0;
	int *p_int = NULL;
	float *p_float = NULL;

	checkCudaErrors(hipMalloc((void**)&(p_int), sizeof(int)*24));
	checkCudaErrors(hipMemset(p_int, 0, sizeof(int)*24));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_fired_table), &p_int, sizeof(p_int)));
	checkCudaErrors(hipMalloc((void**)&(p_int), sizeof(int)*3));
	checkCudaErrors(hipMemset(p_int, 0, sizeof(int)*3));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_fired_table_sizes), &p_int, sizeof(p_int)));

	checkCudaErrors(hipMalloc((void**)&(p_int), sizeof(int)*8));
	checkCudaErrors(hipMemset(p_int, 0, sizeof(int)*8));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_active_lif_curr_exp_table), &p_int, sizeof(p_int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_active_lif_curr_exp_table_size), &zero, sizeof(int)));


	checkCudaErrors(hipMalloc((void**)&(p_float), sizeof(float)*8));
	checkCudaErrors(hipMemset(p_float, 0, sizeof(float)*8));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_i_exec), &p_float, sizeof(p_float)));
	checkCudaErrors(hipMalloc((void**)&(p_float), sizeof(float)*8));
	checkCudaErrors(hipMemset(p_float, 0, sizeof(float)*8));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_i_inh), &p_float, sizeof(p_float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_connection_stdp_synapse), &(connections[0]), sizeof(CConnection*)));
}
