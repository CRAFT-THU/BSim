

#include <stdlib.h>

#include "hip/hip_runtime_api.h"
#include "stdp_synapse.h"


Stdp_synapse * to_gpu_stdp_synapse(Stdp_synapse *cpu, int num)
{
	Stdp_synapse * gpu = static_cast<Stdp_synapse*>(malloc(sizeof(Stdp_synapse)*1));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_last_update), sizeof(int)*num));
	checkCudaErrors(hipMemset(gpu->p_last_update, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_last_update, cpu->p_last_update, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_delay), sizeof(int)*num));
	checkCudaErrors(hipMemset(gpu->p_delay, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_delay, cpu->p_delay, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_dst), sizeof(int)*num));
	checkCudaErrors(hipMemset(gpu->p_dst, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_dst, cpu->p_dst, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_d_apost), sizeof(float)*num));
	checkCudaErrors(hipMemset(gpu->p_d_apost, 0, sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_d_apost, cpu->p_d_apost, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_d_apre), sizeof(float)*num));
	checkCudaErrors(hipMemset(gpu->p_d_apre, 0, sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_d_apre, cpu->p_d_apre, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_tau_pre), sizeof(float)*num));
	checkCudaErrors(hipMemset(gpu->p_tau_pre, 0, sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_tau_pre, cpu->p_tau_pre, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_tau_post), sizeof(float)*num));
	checkCudaErrors(hipMemset(gpu->p_tau_post, 0, sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_tau_post, cpu->p_tau_post, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_apre), sizeof(float)*num));
	checkCudaErrors(hipMemset(gpu->p_apre, 0, sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_apre, cpu->p_apre, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_weight), sizeof(float)*num));
	checkCudaErrors(hipMemset(gpu->p_weight, 0, sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_weight, cpu->p_weight, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_apost), sizeof(float)*num));
	checkCudaErrors(hipMemset(gpu->p_apost, 0, sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_apost, cpu->p_apost, sizeof(float)*num, hipMemcpyHostToDevice));
	Stdp_synapse * ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(Stdp_synapse)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(Stdp_synapse)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(Stdp_synapse)*1, hipMemcpyHostToDevice));
	return ret;
}

Stdp_synapse * from_gpu_stdp_synapse(Stdp_synapse *gpu, int num)
{
	Stdp_synapse * cpu = static_cast<Stdp_synapse*>(malloc(sizeof(Stdp_synapse)*1));
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(Stdp_synapse)*1, hipMemcpyDeviceToHost));
	return cpu;
}

