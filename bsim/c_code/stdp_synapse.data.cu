

#include <stdlib.h>

#include "hip/hip_runtime_api.h"
#include "stdp_synapse.h"


Stdp_synapse * to_gpu_stdp_synapse(Stdp_synapse *cpu, int num)
{
	Stdp_synapse * gpu = static_cast<Stdp_synapse*>(malloc(sizeof(Stdp_synapse)*1));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_last_update), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_last_update, cpu->p_last_update, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_delay), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_delay, cpu->p_delay, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_dst), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_dst, cpu->p_dst, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_tau_post), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_tau_post, cpu->p_tau_post, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_d_apost), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_d_apost, cpu->p_d_apost, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_tau_pre), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_tau_pre, cpu->p_tau_pre, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_d_apre), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_d_apre, cpu->p_d_apre, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_weight), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_weight, cpu->p_weight, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_apre), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_apre, cpu->p_apre, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_apost), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_apost, cpu->p_apost, sizeof(float)*num, hipMemcpyHostToDevice));
	Stdp_synapse * ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(Stdp_synapse)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(Stdp_synapse)*1, hipMemcpyHostToDevice));
	return ret;
}

Stdp_synapse * from_gpu_stdp_synapse(Stdp_synapse *gpu, int num)
{
	Stdp_synapse * cpu = static_cast<Stdp_synapse*>(malloc(sizeof(Stdp_synapse)*1));
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(Stdp_synapse)*1, hipMemcpyDeviceToHost));
	return cpu;
}

