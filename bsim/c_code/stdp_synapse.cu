#include "hip/hip_runtime.h"
#include "runtime.h"
#include "connection.h"

__global__ void find_stdp_synapse_gpu(Stdp_synapse *data, int num, int start_id);;
__global__ void update_stdp_synapse_gpu(Stdp_synapse *data, int num, int start_id);;

extern "C" {
	void update_stdp_synapse(Stdp_synapse *data, int num, int start_id, int t)
	{
		update_stdp_synapse_gpu<<<GRIDSIZE, BLOCKSIZE>>>((Stdp_synapse*)data, num, start_id, t);
		learn_stdp_synapse_post<<<GRIDSIZE, BLOCKSIZE>>>((Stdp_synapse*)data, num, start_id, t);
	}
}

__device__ void _clip(float a, float min, float max)
{
	if (a < min) {
		return min;
	} else if (a > max) {
		return max;
	} else {
		return a;
	}
}
__global__ void update_stdp_synapse_gpu(Stdp_synapse *data, int num, int start_id, int t)
{
	for (int delta_t=MIN_DELAY; delta_t<=MAX_DELAY; delta_t++) {
		int block_idx = blockIdx.x;
		int time_idx = (t + MAX_DELAY - delta_t) % ( MAX_DELAY + 1);
		int firedSize = gFiredTableSizes[time_idx];
		int num_per_block = (firedSize - 1) / gridDim.x + 1;
		int block_nums_minus_1 = (firedSize - 1) / num_per_block;
		int fired_size_block = 0;
		if (block_idx == block_nums_minus_1) {
		fired_size_block = firedSize - block_idx * num_per_block;
		} else if (block_idx < block_nums_minus_1) {
		fired_size_block = num_per_block;
		} else {
		fired_size_block = 0;
		}
		for (int idx = 0; idx < fired_size_block; idx++) {
			int nid = gFiredTable[time_idx * gFiredTableCap + (block_idx) * num_per_block + idx];
			int start_loc = connection->delayStart[delta_t + nid * MAX_DELAY];
			int synapseNum = connection->delayNum[delta_t + nid * MAX_DELAY];
			if (threadIdx.x == 0) {
				gLayerInput[nid]++;
			}
			for (int j=threadIdx.x; j < synapseNum; j += blockDim.x) {
				int sid = j+start_loc;
				real weight = d_synapses->p_weight[sid];
				if (weight >= 0) {
					atomicAdd(&(gNeuronInput[datas->p_dst[sid]]), weight);
				} else {
					atomicAdd(&(gNeuronInput_I[datas->p_dst[sid]]), weight);
				}

				data->p_apre[sid] *= exp((data->p_last_update[sid] - t) / (data->p_tau_pre[sid]));
				data->p_apost[sid] *= exp((data->p_last_update[sid] - t) / (data->p_tau_post[sid]));

				data->p_pre[sid] += data->p_d_apre[sid];
				data->p_weight[sid] = _clip(weight + data->p_apost[sid], GMIN, GMAX);
				data->p_last_update[sid] = gCurrentCycle;
			}
		}
		__syncthreads();
	}
}

__global__ void update_stdp_synapse_gpu(Stdp_synapse * data, int num, int start_id, int t)
{
	int block_idx = blockIdx.x;
	int time_idx = t%(MAX_DELAY+1);
	int firedSize = gFiredTableSizes[time_idx];
	int num_per_block = (firedSize - 1) / gridDim.x + 1;
	int block_nums_minus_1 = (firedSize - 1) / num_per_block;
	int fired_size_block = 0;
	if (block_idx == block_nums_minus_1) {
	fired_size_block = firedSize - block_idx * num_per_block;
	} else if (block_idx < block_nums_minus_1) {
	fired_size_block = num_per_block;
	} else {
		fired_size_block = 0;
	}

	for (int idx = 0; idx < fired_size_block; idx++) {
		int nid = gFiredTable[time_idx * gFiredTableCap + (block_idx) * num_per_block + idx];
		int start_loc = gConnectionStdp_synapse->rev_delayStart[nid];
		int synapseNum = gConnectionStdp_synapse->rev_delayNum[nid];
		for (int j=threadIdx.x; j < synapseNum; j += blockDim.x) {
			int sid = connection->rev_map2sid[j+start_loc];
			data->p_apre[sid] *= exp((data->p_last_update[sid] - t) / (data->p_tau_pre[sid]));
			data->p_apost[sid] *= exp((data->p_last_update[sid] - t) / (data->p_tau_post[sid]));
			data->p_apost[sid] += data->p_d_apost[sid];
			data->p_weight[sid] = _clip(weight + data->p_pre[sid], GMIN, GMAX);
			data->p_last_update[sid] = gCurrentCycle;
		}
	}
	__syncthreads();
}

