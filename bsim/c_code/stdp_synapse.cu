#include "hip/hip_runtime.h"
extern "C" {
	void update_stdp_synapse(Stdp_synapse *data, int num, int start_id, BlockSize *size)
	{
		update_stdp_synapse_gpu<<<size=>gridSize, size->blockSize>>>((Stdp_synapse*)data, num, start_id);
		learn_stdp_synapse_post<<<size=>gridSize, size->blockSize>>>((Stdp_synapse*)data, num, start_id);
	}
}

__global void update_stdp_synapse_gpu(stdp_synapse *data, int num, int start_id)
{
	for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
		int block_idx = blockIdx.x;
		int time_idx = (gCurrentIdx + MAX_DELAY - delta_t) % ( MAX_DELAY + 1);
		int firedSize = gFiredTableSizes[time_idx];
		int num_per_block = (firedSize - 1) / gridDim.x + 1;
		int block_nums_minus_1 = (firedSize - 1) / num_per_block;
		int fired_size_block = 0;
		if (block_idx == block_nums_minus_1) {
		fired_size_block = firedSize - block_idx * num_per_block;
		} else if (block_idx < block_nums_minus_1) {
		fired_size_block = num_per_block;
		} else {
		fired_size_block = 0;
		}
		for (int idx = 0; idx < fired_size_block; idx++) {
			int nid = gFiredTable[time_idx * gFiredTableCap + (block_idx) * num_per_block + idx];
			int start_loc = gConnection->rev_delayStart[delta_t + nid * MAX_DELAY];
			int synapseNum = gConnection->rev_delayNum[delta_t + nid * MAX_DELAY];
			if (threadIdx.x == 0) {
				gLayerInput[nid]++;
			}
			for (int j=threadIdx.x; j < synapseNum; j += blockDim.x) {
				int sid = j+start_loc;
				real weight = d_synapses->p_weight[sid];
				if (weight >= 0) {
					atomicAdd(&(gNeuronInput[d_synapses->p_dst[sid]]), weight);
				} else {
					atomicAdd(&(gNeuronInput_I[d_synapses->p_dst[sid]]), weight);
				}

				d_synapse->p_apre[sid] *= exp((d_synapse->p_last_update[sid] - t) / (d_synapse->p_tau_pre[sid]));
				d_synapse->p_apost[sid] *= exp((d_synapse->p_last_update[sid] - t) / (d_synapse->p_tau_post[sid]));

				d_synapse->p_pre[sid] += d_synapse->p_d_apre[sid];
				d_synapse->p_weight[sid] = _clip(weight + d_synapse->p_apost[sid], gMin, gMax);
				d_synapse->p_last_update[sid] = gCurrentCycle;
			}
		}
		__syncthreads();
	}
}

__global void update_stdp_synapse_gpu(stdp_synapse *data, int num, int start_id)
{
	for (int delta_t = 0; delta_t<MAX_DELAY; delta_t++) {
		int block_idx = blockIdx.x;
		int time_idx = (gCurrentIdx + MAX_DELAY - delta_t) % (MAX_DELAY + 1);
		int firedSize = gFiredTableSizes[time_idx];
		int num_per_block = (firedSize - 1) / gridDim.x + 1;
		int block_nums_minus_1 = (firedSize - 1) / num_per_block;

		int fired_size_block = 0;
		if (block_idx == block_nums_minus_1) {
		fired_size_block = firedSize - block_idx * num_per_block;
		} else if (block_idx < block_nums_minus_1) {
		fired_size_block = num_per_block;
		} else {
		fired_size_block = 0;
		}

		for (int idx = 0; idx < fired_size_block; idx++) {
			int nid = gFiredTable[time_idx * gFiredTableCap + (block_idx) * num_per_block + idx];
			int start_loc = gConnection->rev_delayStart[delta_t + nid * MAX_DELAY];
			int synapseNum = gConnection->rev_delayNum[delta_t + nid * MAX_DELAY];
			if (threadIdx.x == 0) {
			gLayerInput[nid]++;
			}
			for (int j=threadIdx.x; j < synapseNum; j += blockDim.x) {
			int sid = gConnection->rev_map2sid[j+start_loc];

			d_synapse->p_apre[sid] *= exp((d_synapse->p_last_update[sid] - t) / (d_synapse->p_tau_pre[sid]));
			d_synapse->p_apost[sid] *= exp((d_synapse->p_last_update[sid] - t) / (d_synapse->p_tau_post[sid]));
			d_synapse->p_apost[sid] += d_synapse->p_d_apost[sid];
			d_synapse->p_weight[sid] = _clip(weight + d_synapse->p_pre[sid], gMin, gMax);
			d_synapse->p_last_update[sid] = gCurrentCycle;
			}
		}
		__syncthreads();
	}
}

