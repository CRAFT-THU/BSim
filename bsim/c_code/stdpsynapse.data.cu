

#include <stdlib.h>

#include "hip/hip_runtime_api.h"
#include "stdpsynapse.h"


Stdpsynapse * to_gpu_stdpsynapse(Stdpsynapse *cpu, int num)
{
	Stdpsynapse * gpu = static_cast<Stdpsynapse*>(malloc(sizeof(Stdpsynapse)*1));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_last_update), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_last_update, cpu->p_last_update, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_dst), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_dst, cpu->p_dst, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_delay), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_delay, cpu->p_delay, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_tau_pre), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_tau_pre, cpu->p_tau_pre, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_tau_post), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_tau_post, cpu->p_tau_post, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_d_apre), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_d_apre, cpu->p_d_apre, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_d_apost), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_d_apost, cpu->p_d_apost, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_apre), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_apre, cpu->p_apre, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_weight), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_weight, cpu->p_weight, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_apost), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_apost, cpu->p_apost, sizeof(float)*num, hipMemcpyHostToDevice));
	Stdpsynapse * ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(Stdpsynapse)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(Stdpsynapse)*1, hipMemcpyHostToDevice));
	return ret;
}

Stdpsynapse * from_gpu_stdpsynapse(Stdpsynapse *gpu, int num)
{
	Stdpsynapse * cpu = static_cast<Stdpsynapse*>(malloc(sizeof(Stdpsynapse)*1));
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(Stdpsynapse)*1, hipMemcpyDeviceToHost));
	return cpu;
}

