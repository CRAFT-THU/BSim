#include "hip/hip_runtime.h"

#include "runtime.h"
#include "lif_curr_exp.h"


__global__ void find_lif_curr_exp_gpu(Lif_curr_exp *data, int num, int start_id)
{
	__shared__ int active_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile int active_cnt;

	if (threadIdx.x == 0 && blockIdx.x == 0) {
		active_cnt = 0;
		g_active_lif_curr_exp_table_size = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		int test_loc = 0;
		bool actived = data->p_refract_step[idx] <= 0;
		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		} else {
			g_i_exec[start_id + idx] = 0;
			g_i_inh[start_id + idx] = 0;
			data->p_refract_time[idx] = data->p_refract_time[idx] - 1;
		}
		__syncthreads();

		if (active_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(active_table_t, MAX_BLOCK_SIZE, g_active_lif_curr_exp_table, &g_active_lif_curr_exp_table_size, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		}
		__syncthreads();

		if (active_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(active_table_t, MAX_BLOCK_SIZE, g_active_lif_curr_exp_table, &g_active_lif_curr_exp_table_size, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (active_cnt > 0) {
			commit2globalTable(active_table_t, MAX_BLOCK_SIZE, g_active_lif_curr_exp_table, &g_active_lif_curr_exp_table_size, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
	}
}

__global__ void update_lif_curr_exp_gpu(Lif_curr_exp *data, int num, int start_id, int t)
{
	int delay_idx = t % (MAX_DELAY+1)
	if (threadIdx.x == 0 && blockIdx.x==0) {
		g_fired_table_sizes[delay_idx] = 0;
	}
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < g_active_lif_curr_exp_table_size; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		int nid = g_active_lif_curr_exptable[idx];
		int gnid = start_id + nid; 

		data->p_vm[nid] = data->p_Cm[nid] * data->p_vm[nid] + data->p_v_tmp[nid] + data->p_i_E[nid] * data->p_C_E[nid] + data->p_i_I[nid] * data->p_C_I[nid];
		data->p_i_E[nid] *= data->p_CE[nid];
		data->p_i_I[nid] *= data->p_CI[nid];

		fired = data->p_vm[nid] >= data->p_v_thresh[nid];
		gFireCount[gnid] += fired;
		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
			data->p_refract_time[nid] = data->p_refrac_time[nid] - 1;
			data->p_vm[nid] = data->p_v_reset[nid];
		} else {
			data->p_i_E[nid] += g_i_exec[gnid];
			data->p_i_I[nid] += g_i_inh[gnid];
		}

		g_i_exec[gnid] = 0;
		g_i_inh[gnid] = 0;
		__syncthreads();

		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, g_fired_table, &g_fired_table_sizes[delay_idx], g_fired_tableCap*delay_idx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAX_BLOCK_SIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, g_fired_table, &g_fired_tableSizes[delay_idx], g_fired_tableCap*delay_idx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
		
		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, g_fired_table, &g_fired_tableSizes[delay_idx], g_fired_tableCap*delay_idx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		
	}
}

void update_lif_curr_exp(Lif_curr_exp *data, int num, int start_id, int t)
{
	find_lif_curr_exp_gpu<<<LIF_CURR_EXP_GRID_SIZE, LIF_CURR_EXP_BLOCK_SIZE>>>((Lif_curr_exp*)data, num, start_id);
	update_lif_curr_exp_gpu<<<LIF_CURR_EXP_GRID_SIZE, LIF_CURR_EXP_BLOCK_SIZE>>>((Lif_curr_exp*)data, num, start_id);
}

