#include "hip/hip_runtime.h"

#include "runtime.h"
#include "LIF_curr_exp.h"


void update_lif_curr_exp(Lif_curr_exp *data, int num, int start_id, int t)
{
	find_lif_curr_exp_gpu<<<LIF_CURR_EXP_GRID_SIZE, LIF_CURR_EXP_BLOCK_SIZE>>>((Lif_curr_exp*)data, num, start_id);
	update_lif_curr_exp_gpu<<<LIF_CURR_EXP_GRID_SIZE, LIF_CURR_EXP_BLOCK_SIZE>>>((Lif_curr_exp*)data, num, start_id);
}

__global void find_lif_curr_exp_gpu(LIF_curr_exp *data, int num, int start_id)
{
	__shared__ int active_table_t[MAXBLOCKSIZE];
	__shared__ volatile int active_cnt;

	if (threadIdx.x == 0) {
		active_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
		int test_loc = 0;
		bool actived = data->p_refrac_step[idx] <= 0;
		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		} else {
			gNeuronInput[start_id + idx] = 0;
			gNeuronInput_I[start_id + idx] = 0;
			data->p_refrac_step[idx] = data->p_refrac_step[idx] - 1;
		}
		__syncthreads();

		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (actived) {
			test_loc = atomicAdd((int*)&active_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				active_table_t[test_loc] = idx;
				actived = false;
			}
		}
		__syncthreads();

		if (active_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(active_table_t, MAXBLOCKSIZE, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();

		if (active_cnt > 0) {
			commit2globalTable(active_table_t, active_cnt, gActiveTable, &gActiveTableSize, 0);
			if (threadIdx.x == 0) {
				active_cnt = 0;
			}
		}
		__syncthreads();
	}
}

__global void update_lif_curr_exp_gpu(LIF_curr_exp *data, int num, int start_id)
{
	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		int test_loc = 0;
		int nid = gActiveTable[idx];
		int gnid = start_id + nid; 

		data->p_vm[nid] = data->p_Cm[nid] * data->p_vm[nid] + data->p_v_tmp[nid] + data->p_i_E[nid] * data->p_C_E[nid] + data->p_i_I[nid] * data->p_C_I[nid];
		data->p_i_E[nid] *= data->p_CE[nid];
		data->p_i_I[nid] *= data->p_CI[nid];

		fired = data->p_vm[nid] >= data->p_v_thresh[nid];
		gFireCount[gnid] += fired;
		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
			data->p_refrac_step[nid] = data->p_refrac_time[nid] - 1;
			data->p_vm[nid] = data->p_v_reset[nid];
		} else {
			gXInput[gnid] += gNeuronInput[gnid] + gNeuronInput_I[gnid];
			data->p_i_E[nid] += gNeuronInput[gnid];
			data->p_i_I[nid] += gNeuronInput_I[gnid];
		}

		gNeuronInput[gnid] = 0;
		gNeuronInput_I[gnid] = 0;
		__syncthreads();

		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

		if (fired) {
			test_loc = atomicAdd((int*)&fire_cnt, 1);
			if (test_loc < MAXBLOCKSIZE) {
				fire_table_t[test_loc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAXBLOCKSIZE) {
			commit2globalTable(fire_table_t, MAXBLOCKSIZE, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();
		
		if (fire_cnt > 0) {
			commit2globalTable(fire_table_t, fire_cnt, gFiredTable, &gFiredTableSizes[gCurrentIdx], gFiredTableCap*gCurrentIdx);
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		
	}
}

