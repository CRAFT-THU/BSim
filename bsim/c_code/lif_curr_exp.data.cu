

#include <stdlib.h>

#include "hip/hip_runtime_api.h"
#include "lif_curr_exp.h"


Lif_curr_exp * to_gpu_lif_curr_exp(Lif_curr_exp *cpu, int num)
{
	Lif_curr_exp * gpu = static_cast<Lif_curr_exp*>(malloc(sizeof(Lif_curr_exp)*1));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_refract_step), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_refract_step, cpu->p_refract_step, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_v), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_v, cpu->p_v, sizeof(float)*num, hipMemcpyHostToDevice));
	Lif_curr_exp * ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(Lif_curr_exp)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(Lif_curr_exp)*1, hipMemcpyHostToDevice));
	return ret;
}

