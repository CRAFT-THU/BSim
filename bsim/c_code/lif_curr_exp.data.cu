

#include <stdlib.h>

#include "hip/hip_runtime_api.h"
#include "lif_curr_exp.h"


Lif_curr_exp * to_gpu_lif_curr_exp(Lif_curr_exp *cpu, int num)
{
	Lif_curr_exp * gpu = static_cast<Lif_curr_exp*>(malloc(sizeof(Lif_curr_exp)*1));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_refract_time), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_refract_time, cpu->p_refract_time, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_refract_step), sizeof(int)*num));
	checkCudaErrors(hipMemcpy(gpu->p_refract_step, cpu->p_refract_step, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_C_exec), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_C_exec, cpu->p_C_exec, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_C_inh), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_C_inh, cpu->p_C_inh, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_v_tmp), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_v_tmp, cpu->p_v_tmp, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_Cm), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_Cm, cpu->p_Cm, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_Cinh), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_Cinh, cpu->p_Cinh, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_v_threshold), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_v_threshold, cpu->p_v_threshold, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_Cexec), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_Cexec, cpu->p_Cexec, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_v_reset), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_v_reset, cpu->p_v_reset, sizeof(float)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->p_v), sizeof(float)*num));
	checkCudaErrors(hipMemcpy(gpu->p_v, cpu->p_v, sizeof(float)*num, hipMemcpyHostToDevice));
	Lif_curr_exp * ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(Lif_curr_exp)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(Lif_curr_exp)*1, hipMemcpyHostToDevice));
	return ret;
}

Lif_curr_exp * from_gpu_lif_curr_exp(Lif_curr_exp *gpu, int num)
{
	Lif_curr_exp * cpu = static_cast<Lif_curr_exp*>(malloc(sizeof(Lif_curr_exp)*1));
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(Lif_curr_exp)*1, hipMemcpyDeviceToHost));
	return cpu;
}

