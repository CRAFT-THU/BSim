#include "hip/hip_runtime.h"

#include "hiprand/hiprand_kernel.h"
#include "hiprand.h"
#include "hip/hip_runtime_api.h"
#include "runtime.h"

__device__ int commit2globalTable(int *shared_buf, volatile unsigned int size, int *global_buf, int * global_size, int offset)
{
	__shared__ volatile unsigned int start_loc;
	if (threadIdx.x == 0) {
		start_loc = atomicAdd(global_size, (int)size);
	}
	__syncthreads();

	for (int idx=threadIdx.x; idx<size; idx+=blockDim.x) {
		global_buf[offset + start_loc + idx] = shared_buf[idx];
	}

	return 0;
}

__global__ void curand_setup_kernel(hiprandState *state, int num)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < num) {
		hiprand_init(1234, id, 0, &state[id]);
	}
}


__global__ void add_cross_neuron(int *ids, int num, int t)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int delay_idx = t % (MAX_DELAY+1);
	if (tid < num) {
		g_fired_table[FIRED_TABLE_SIZE*delay_idx + g_fired_table_sizes[delay_idx] + tid] = ids[tid];
	}
	__syncthreads();

	if (tid == 0) {
		g_fired_table_sizes[delay_idx] += num;
	}
}

__global__ void deliver_neurons(int *idx2index, int *crossnode_index2idx, int *global_cross_data, int *fired_n_num, int node_num, int t)
{
	__shared__ int cross_neuron_id[MAX_BLOCK_SIZE];
	__shared__ volatile int cross_cnt;

	if (threadIdx.x == 0) {
		cross_cnt = 0;
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int delay_idx = t % (MAX_DELAY+1);
	int fired_size = g_fired_table_sizes[delay_idx];
	for (int node = 0; node < node_num; node++) {
		for (int idx = tid; idx < fired_size; idx += blockDim.x * gridDim.x) {
			int nid = g_fired_table[FIRED_TABLE_SIZE*delay_idx+ idx];
			int tmp = idx2index[nid];
			if (tmp >= 0) {
				int map_nid = crossnode_index2idx[tmp*node_num + node];
				if (map_nid >= 0) {
					int test_loc = atomicAdd((int*)&cross_cnt, 1);
					if (test_loc < MAX_BLOCK_SIZE) {
						cross_neuron_id[test_loc] = map_nid;
					}
				}
			}
			__syncthreads();

			if (cross_cnt > 0) {
				commit2globalTable(cross_neuron_id, cross_cnt, global_cross_data, &fired_n_num[node], FIRED_TABLE_SIZE*node);
				if (threadIdx.x == 0) {
					cross_cnt = 0;
				}
			}
			__syncthreads();
		}
		__syncthreads();
	}
}