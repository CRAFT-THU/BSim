

#include <stdlib.h>

#include "hip/hip_runtime_api.h"
#include "connection.h"


CConnection * to_gpu_connection(CConnection *cpu)
{
	CConnection * gpu = (CConnection*)malloc(sizeof(CConnection));
	gpu->n_length = cpu->n_length;
	gpu->s_length = cpu->s_length;
	checkCudaErrors(hipMalloc((void**)&(gpu->delay_start), sizeof(int)*cpu->n_length));
	checkCudaErrors(hipMemcpy(gpu->delay_start, cpu->delay_start, sizeof(int)*cpu->n_length, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->delay_num), sizeof(int)*cpu->n_length));
	checkCudaErrors(hipMemcpy(gpu->delay_num, cpu->delay_num, sizeof(int)*cpu->n_length, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->rev_delay_start), sizeof(int)*cpu->n_length));
	checkCudaErrors(hipMemcpy(gpu->rev_delay_start, cpu->rev_delay_start, sizeof(int)*cpu->n_length, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->rev_delay_num), sizeof(int)*cpu->n_length));
	checkCudaErrors(hipMemcpy(gpu->rev_delay_num, cpu->rev_delay_num, sizeof(int)*cpu->n_length, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->rev_map2sid), sizeof(int)*cpu->s_length));
	checkCudaErrors(hipMemcpy(gpu->rev_map2sid, cpu->rev_map2sid, sizeof(int)*cpu->s_length, hipMemcpyHostToDevice));
	CConnection * ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(CConnection)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(CConnection)*1, hipMemcpyHostToDevice));
	return ret;
}

CConnection * from_gpu_connection(CConnection *gpu)
{
	CConnection * ret = static_cast<CConnection*>(malloc(sizeof(CConnection)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(CConnection)*1, hipMemcpyDeviceToHost));
	return ret;
}

