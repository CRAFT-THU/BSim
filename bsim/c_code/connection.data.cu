

#include <stdlib.h>

#include "hip/hip_runtime_api.h"
#include "connection.h"


CConnection * to_gpu_connection(CConnection *cpu)
{
	CConnection * gpu = (CConnection*)malloc(sizeof(CConnection));
	gpu->n_len = cpu->n_len;
	gpu->r_n_len = cpu->r_n_len;
	gpu->s_len = cpu->s_len;
	checkCudaErrors(hipMalloc((void**)&(gpu->delay_start), sizeof(int)*cpu->n_len));
	checkCudaErrors(hipMemset(gpu->delay_start, 0, sizeof(int)*cpu->n_len));
	checkCudaErrors(hipMemcpy(gpu->delay_start, cpu->delay_start, sizeof(int)*cpu->n_len, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->delay_num), sizeof(int)*cpu->n_len));
	checkCudaErrors(hipMemset(gpu->delay_num, 0, sizeof(int)*cpu->n_len));
	checkCudaErrors(hipMemcpy(gpu->delay_num, cpu->delay_num, sizeof(int)*cpu->n_len, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->rev_delay_start), sizeof(int)*cpu->r_n_len));
	checkCudaErrors(hipMemset(gpu->rev_delay_start, 0, sizeof(int)*cpu->r_n_len));
	checkCudaErrors(hipMemcpy(gpu->rev_delay_start, cpu->rev_delay_start, sizeof(int)*cpu->r_n_len, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->rev_delay_num), sizeof(int)*cpu->r_n_len));
	checkCudaErrors(hipMemset(gpu->rev_delay_num, 0, sizeof(int)*cpu->r_n_len));
	checkCudaErrors(hipMemcpy(gpu->rev_delay_num, cpu->rev_delay_num, sizeof(int)*cpu->r_n_len, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(gpu->rev_map2sid), sizeof(int)*cpu->s_len));
	checkCudaErrors(hipMemset(gpu->rev_map2sid, 0, sizeof(int)*cpu->s_len));
	checkCudaErrors(hipMemcpy(gpu->rev_map2sid, cpu->rev_map2sid, sizeof(int)*cpu->s_len, hipMemcpyHostToDevice));
	CConnection * ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(CConnection)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(CConnection)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(CConnection)*1, hipMemcpyHostToDevice));
	return ret;
}

CConnection * from_gpu_connection(CConnection *gpu)
{
	CConnection * ret = static_cast<CConnection*>(malloc(sizeof(CConnection)*1));
	checkCudaErrors(hipMemcpy(ret, gpu, sizeof(CConnection)*1, hipMemcpyDeviceToHost));
	return ret;
}

