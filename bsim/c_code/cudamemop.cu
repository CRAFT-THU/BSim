

#include "hip/hip_runtime_api.h"
#include "cudamemop.h"


int* gpu_malloc_int(int size)
{
	int * gpu = NULL;
	checkCudaErrors(hipMalloc((void**)&(gpu), sizeof(int)*size));
	return gpu;
}

void cpu2gpu_int(int *cpu, int *gpu, int size)
{
	checkCudaErrors(hipMemcpy(gpu, cpu, sizeof(int)*size, hipMemcpyHostToDevice));
}

void gpu2cpu_int(int *gpu, int *cpu, int size)
{
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(int)*size, hipMemcpyDeviceToHost));
}

int* to_gpu_int(int *cpu, int size)
{
	int * gpu = NULL;
	checkCudaErrors(hipMalloc((void**)&(gpu), sizeof(int)*size));
	checkCudaErrors(hipMemcpy(gpu, cpu, sizeof(int)*size, hipMemcpyHostToDevice));
	return gpu;
}

int* from_gpu_int(int *gpu, int size)
{
	int * cpu = static_cast<int*>(malloc(sizeof(int)*size));
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(int)*size, hipMemcpyDeviceToHost));
	return cpu;
}

float* gpu_malloc_float(int size)
{
	float * gpu = NULL;
	checkCudaErrors(hipMalloc((void**)&(gpu), sizeof(float)*size));
	return gpu;
}

void cpu2gpu_float(float *cpu, float *gpu, int size)
{
	checkCudaErrors(hipMemcpy(gpu, cpu, sizeof(float)*size, hipMemcpyHostToDevice));
}

void gpu2cpu_float(float *gpu, float *cpu, int size)
{
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(float)*size, hipMemcpyDeviceToHost));
}

float* to_gpu_float(float *cpu, int size)
{
	float * gpu = NULL;
	checkCudaErrors(hipMalloc((void**)&(gpu), sizeof(float)*size));
	checkCudaErrors(hipMemcpy(gpu, cpu, sizeof(float)*size, hipMemcpyHostToDevice));
	return gpu;
}

float* from_gpu_float(float *gpu, int size)
{
	float * cpu = static_cast<float*>(malloc(sizeof(float)*size));
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(float)*size, hipMemcpyDeviceToHost));
	return cpu;
}

double* gpu_malloc_double(int size)
{
	double * gpu = NULL;
	checkCudaErrors(hipMalloc((void**)&(gpu), sizeof(double)*size));
	return gpu;
}

void cpu2gpu_double(double *cpu, double *gpu, int size)
{
	checkCudaErrors(hipMemcpy(gpu, cpu, sizeof(double)*size, hipMemcpyHostToDevice));
}

void gpu2cpu_double(double *gpu, double *cpu, int size)
{
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(double)*size, hipMemcpyDeviceToHost));
}

double* to_gpu_double(double *cpu, int size)
{
	double * gpu = NULL;
	checkCudaErrors(hipMalloc((void**)&(gpu), sizeof(double)*size));
	checkCudaErrors(hipMemcpy(gpu, cpu, sizeof(double)*size, hipMemcpyHostToDevice));
	return gpu;
}

double* from_gpu_double(double *gpu, int size)
{
	double * cpu = static_cast<double*>(malloc(sizeof(double)*size));
	checkCudaErrors(hipMemcpy(cpu, gpu, sizeof(double)*size, hipMemcpyDeviceToHost));
	return cpu;
}


